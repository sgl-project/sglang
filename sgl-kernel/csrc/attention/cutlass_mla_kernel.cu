#include "hip/hip_runtime.h"
/*
Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
Copyright 2025 SGLang Team. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <cutlass/cutlass.h>
#include <cutlass/kernel_hardware_info.h>
#include <torch/all.h>

#include <cute/tensor.hpp>
#include <device/sm100_mla.hpp>
#include <kernel/sm100_mla_tile_scheduler.hpp>

#if defined CUDA_VERSION && CUDA_VERSION >= 12040

#define CUTLASS_CHECK(status)                                                       \
  {                                                                                 \
    cutlass::Status error = status;                                                 \
    TORCH_CHECK(error == cutlass::Status::kSuccess, cutlassGetStatusString(error)); \
  }

using namespace cute;
using namespace cutlass::fmha::kernel;

template <bool v>
struct IsPersistent {
  static const bool value = v;
};

template <typename T, typename PersistenceOption = IsPersistent<true>>
struct MlaSm100 {
  using Element = T;
  using ElementAcc = float;
  using ElementOut = T;

  using TileShape = Shape<_128, _128, Shape<_512, _64>>;
  using TileShapeH = cute::tuple_element_t<0, TileShape>;
  using TileShapeD = cute::tuple_element_t<2, TileShape>;

  // H K (D_latent D_rope) B
  using ProblemShape = cute::tuple<TileShapeH, int, TileShapeD, int>;

  using StrideQ = cute::tuple<int64_t, _1, int64_t>;  // H D B
  using StrideK = cute::tuple<int64_t, _1, int64_t>;  // K D B
  using StrideO = StrideK;                            // H D B
  using StrideLSE = cute::tuple<_1, int>;             // H B

  using TileScheduler =
      std::conditional_t<PersistenceOption::value, Sm100MlaPersistentTileScheduler, Sm100MlaIndividualTileScheduler>;

  using FmhaKernel = cutlass::fmha::kernel::Sm100FmhaMlaKernelTmaWarpspecialized<
      TileShape,
      Element,
      ElementAcc,
      ElementOut,
      ElementAcc,
      TileScheduler,
      /*kIsCpAsync=*/true>;
  using Fmha = cutlass::fmha::device::MLA<FmhaKernel>;
};

template <typename T>
typename T::Fmha::Arguments args_from_options(
    at::Tensor const& out,
    at::Tensor const& q_nope_and_q_pe,
    at::Tensor const& kv_c_and_k_pe_cache,
    at::Tensor const& seq_lens,
    at::Tensor const& page_table) {
  cutlass::KernelHardwareInfo hw_info;
  hw_info.device_id = q_nope_and_q_pe.device().index();
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  int batches = q_nope_and_q_pe.sizes()[0];
  int page_count_per_seq = page_table.sizes()[1];
  int page_count_total = kv_c_and_k_pe_cache.sizes()[0];
  int page_size = kv_c_and_k_pe_cache.sizes()[1];
  int max_seq_len = page_size * page_count_per_seq;
  using TileShapeH = typename T::TileShapeH;
  using TileShapeD = typename T::TileShapeD;
  auto problem_shape = cute::make_tuple(TileShapeH{}, max_seq_len, TileShapeD{}, batches);

  auto [H, K, D, B] = problem_shape;
  auto [D_latent, D_rope] = D;

  // the scale is based on the non-absorbed sizes, change as appropriate
  // we can't determine this parameter from the info we have, it's an input
  int D_non_latent = 128;
  float scale = 1.0 / sqrt(1.0 * (D_non_latent + D_rope));

  using StrideQ = typename T::StrideQ;
  using StrideK = typename T::StrideK;
  using StrideO = typename T::StrideO;
  using StrideLSE = typename T::StrideLSE;

  StrideQ stride_Q = cute::make_tuple(
      static_cast<int64_t>(0 + D_latent + D_rope), _1{}, static_cast<int64_t>(H * (0 + D_latent + D_rope)));
  StrideK stride_C = cute::make_tuple(
      static_cast<int64_t>(0 + D_latent + D_rope), _1{}, static_cast<int64_t>(page_size * (D_latent + D_rope)));
  StrideLSE stride_PT = cute::make_stride(_1{}, page_count_per_seq);
  StrideLSE stride_LSE = cute::make_tuple(_1{}, 0 + H);
  StrideO stride_O = cute::make_tuple(static_cast<int64_t>(0 + D_latent), _1{}, static_cast<int64_t>(0 + H * D_latent));

  using Element = typename T::Element;
  using ElementOut = typename T::ElementOut;
  using ElementAcc = typename T::ElementAcc;
  auto Q_ptr = static_cast<Element*>(q_nope_and_q_pe.data_ptr());
  auto C_ptr = static_cast<Element*>(kv_c_and_k_pe_cache.data_ptr());
  typename T::Fmha::Arguments arguments{
      problem_shape,
      {scale,
       Q_ptr,
       stride_Q,
       Q_ptr + D_latent,
       stride_Q,
       C_ptr,
       stride_C,
       C_ptr + D_latent,
       stride_C,
       static_cast<int*>(seq_lens.data_ptr()),
       static_cast<int*>(page_table.data_ptr()),
       stride_PT,
       page_count_total,
       page_size},
      {static_cast<ElementOut*>(out.data_ptr()), stride_O, static_cast<ElementAcc*>(nullptr), stride_LSE},
      hw_info,
      -1,       // split_kv
      nullptr,  // is_var_split_kv
  };
  // TODO(kaixih@nvidia): When split_kv=-1 and is_var_split_kv=false, we compute
  // split_kv automatically based on batch size and sequence length to balance
  // workload across available SMs. Consider using var_split_kv for manual
  // control if needed.
  T::Fmha::set_split_kv(arguments);
  return arguments;
}

template <typename Element>
void runMla(
    at::Tensor const& out,
    at::Tensor const& q_nope_and_q_pe,
    at::Tensor const& kv_c_and_k_pe_cache,
    at::Tensor const& seq_lens,
    at::Tensor const& page_table,
    at::Tensor const& workspace,
    hipStream_t stream) {
  using MlaSm100Type = MlaSm100<Element>;
  typename MlaSm100Type::Fmha fmha;
  auto arguments = args_from_options<MlaSm100Type>(out, q_nope_and_q_pe, kv_c_and_k_pe_cache, seq_lens, page_table);

  CUTLASS_CHECK(fmha.can_implement(arguments));

  CUTLASS_CHECK(fmha.initialize(arguments, workspace.data_ptr(), stream));

  CUTLASS_CHECK(fmha.run(arguments, workspace.data_ptr(), stream));
}

void cutlass_mla_decode(
    torch::Tensor const& out,
    torch::Tensor const& q_nope_and_q_pe,
    torch::Tensor const& kv_c_and_k_pe_cache,
    torch::Tensor const& seq_lens,
    torch::Tensor const& page_table,
    torch::Tensor const& workspace) {
  auto in_dtype = q_nope_and_q_pe.dtype();
  at::cuda::CUDAGuard device_guard{(char)q_nope_and_q_pe.get_device()};
  const hipStream_t stream = at::cuda::getCurrentCUDAStream(q_nope_and_q_pe.get_device());
  if (in_dtype == at::ScalarType::Half) {
    runMla<cutlass::half_t>(out, q_nope_and_q_pe, kv_c_and_k_pe_cache, seq_lens, page_table, workspace, stream);
  } else if (in_dtype == at::ScalarType::BFloat16) {
    runMla<cutlass::bfloat16_t>(out, q_nope_and_q_pe, kv_c_and_k_pe_cache, seq_lens, page_table, workspace, stream);
  } else if (in_dtype == at::ScalarType::Float8_e4m3fn) {
    runMla<cutlass::float_e4m3_t>(out, q_nope_and_q_pe, kv_c_and_k_pe_cache, seq_lens, page_table, workspace, stream);
  } else {
    TORCH_CHECK(false, "Unsupported input data type of MLA");
  }
}

int64_t cutlass_mla_get_workspace_size(int64_t max_seq_len, int64_t num_batches, int64_t sm_count) {
  // Workspace size depends on ElementAcc and ElementLSE (same as ElementAcc)
  // which are float, so Element type here doesn't matter.
  using MlaSm100Type = MlaSm100<cutlass::half_t>;

  // Get split kv. Requires problem shape and sm_count only.
  typename MlaSm100Type::Fmha::Arguments arguments;
  using TileShapeH = typename MlaSm100Type::TileShapeH;
  using TileShapeD = typename MlaSm100Type::TileShapeD;
  arguments.problem_shape =
      cute::make_tuple(TileShapeH{}, static_cast<int>(max_seq_len), TileShapeD{}, static_cast<int>(num_batches));
  // Assumes device 0 when getting sm_count.
  arguments.hw_info.sm_count =
      sm_count <= 0 ? cutlass::KernelHardwareInfo::query_device_multiprocessor_count(/*device_id=*/0) : sm_count;
  MlaSm100Type::Fmha::set_split_kv(arguments);

  return MlaSm100Type::Fmha::get_workspace_size(arguments);
}

#endif
