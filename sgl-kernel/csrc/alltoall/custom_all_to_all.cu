#include "hip/hip_runtime.h"
/* Copyright 2025 SGLang Team. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

// reference:
// https://github.com/NVIDIA/TensorRT-LLM/blob/release/0.14/cpp/tensorrt_llm/kernels/customAllReduceKernels.cu
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <ATen/cuda/Exceptions.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <cassert>
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <tuple>

#include "allreduce/custom_all_reduce.cuh"
#include "utils.h"

using namespace sglang;

constexpr int MAX_RANKS_PER_NODE = 8;
constexpr int MAX_ALL_TO_ALL_BLOCKS = kMaxAll2AllBlocks;
constexpr int MAX_ALL_TO_ALL_WARPS = 16;
// Fake pointer type, must match fptr_t type in ops.h.
// We use this type alias to indicate when pointers are passed in as int64_t.
using fptr_t = int64_t;
static_assert(sizeof(void*) == sizeof(fptr_t));
int g_sm_count = 0;

namespace {
inline int divUp(int a, int b) {
  return (a + b - 1) / b;
}

struct All2AllParams {
  int64_t elts_size;
  int64_t rank;
  int64_t ranks_per_node;
  RankData* peer_comm_buffer_ptrs;
  void* local_input_buffer_ptr;
  int64_t* plan_meta_ptr;

  int64_t output_elts_total;
  int64_t input_elts_total;

  int64_t input_stride0;
  int64_t input_stride1;
  int64_t input_dim1;

  RankSignals sg;
  Signal* self_sg;
};

struct All2AllPlanParams {
  int64_t elts_size;
  int64_t rank;
  int64_t ranks_per_node;
  int64_t* output_split_sizes;
  int64_t* input_split_sizes;
  int64_t* output_split_offsets;
  int64_t* input_split_offsets;
  int64_t* plan_meta_ptr;

  int64_t output_elts_total;
  int64_t input_elts_total;

  int64_t chunk_size;
  int64_t output_stride0;
  int64_t output_stride1;
  int64_t output_dim1;

  int64_t blocks_per_grid;
  int64_t threads_per_block;

  RankSignals sg;
  Signal* self_sg;
};

struct __align__(16) AllToAllCommMeta {
  int32_t output_elts_offset[MAX_RANKS_PER_NODE];
  int32_t output_elts_length[MAX_RANKS_PER_NODE];
  int32_t output_dim1;
  int32_t output_stride0;
  int32_t output_stride1;
};

struct __align__(16) AllToAllPeerMeta {
  int32_t output_elts_offset;
  int32_t output_dim1;
  int32_t output_stride0;
  int32_t output_stride1;
};

struct __align__(16) AllToAllPlanMeta {
  int32_t local_input_elts_offset[MAX_RANKS_PER_NODE];
  int32_t local_input_elts_length[MAX_RANKS_PER_NODE];

  AllToAllPeerMeta peer_meta[MAX_RANKS_PER_NODE];
  int32_t chunk_size;
  int32_t input_split_elts_total;
  int32_t total_opt_warps_count;
  int32_t warps_beg[MAX_RANKS_PER_NODE];
  int8_t warp_peer_gpu[8];  // real size total_opt_warps_count
};

inline int getSmCount() {
  if (g_sm_count <= 0) {
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, 0);
    g_sm_count = prop.multiProcessorCount;
  }
  return g_sm_count;
}

template <int ngpus>
static __global__ void __launch_bounds__(32, 1) all2AllPlanKernel(All2AllPlanParams params) {
  int const bidx = blockIdx.x;
  int const tidx = threadIdx.x;
  const int chunk_size = params.chunk_size;
  auto input_split_sizes = params.input_split_sizes;
  auto output_split_sizes = params.output_split_sizes;

  int64_t* split_offset_size[ngpus];
#pragma unroll
  for (int i = 0; i < ngpus; i++) {
    split_offset_size[i] = get_tmp_buf<int64_t>(params.sg.signals[i]);
  }
  auto& cur_meta = *reinterpret_cast<struct AllToAllCommMeta*>(split_offset_size[params.rank]);

  auto& plan_meta = *reinterpret_cast<struct AllToAllPlanMeta*>(params.plan_meta_ptr);
  if (bidx == 0 && tidx == 0) {
    plan_meta.chunk_size = params.chunk_size;

    int32_t cur_offset = 0;
    int32_t cur_total_len = 0;
    int32_t offset;
    int32_t len;
    // Set the length and offset of each device's output copied from current device
    for (int i = 0; i < ngpus; i++) {
      len = output_split_sizes[i];
      if (params.output_split_offsets != nullptr) {
        offset = params.output_split_offsets[i];
        cur_offset = std::max(cur_offset, offset + len);
      } else {
        offset = cur_offset;
        cur_offset += output_split_sizes[i];
      }
      if (cur_offset * chunk_size > params.output_elts_total) {
        assert(false & "invalid output_split_sizes");
        return;
      }
      cur_meta.output_elts_offset[i] = offset * chunk_size;
      cur_meta.output_elts_length[i] = len * chunk_size;
      cur_total_len += len;
    }
    cur_meta.output_dim1 = params.output_dim1;
    cur_meta.output_stride0 = params.output_stride0;
    cur_meta.output_stride1 = params.output_stride1;

    cur_offset = 0;
    cur_total_len = 0;
    // Calculate the length and offset of the current device's input copied to each device's output
    for (int i = 0; i < ngpus; i++) {
      len = input_split_sizes[i];
      if (params.input_split_offsets != nullptr) {
        offset = params.input_split_offsets[i];
        cur_offset = std::max(cur_offset, offset + len);
      } else {
        offset = cur_offset;
        cur_offset += input_split_sizes[i];
      }
      if (cur_offset * chunk_size > params.input_elts_total) {
        assert(false & "invalid input_split_sizes");
        return;
      }
      plan_meta.local_input_elts_offset[i] = offset * chunk_size;
      plan_meta.local_input_elts_length[i] = len * chunk_size;
      cur_total_len += len;
    }
    plan_meta.input_split_elts_total = cur_total_len * chunk_size;
  }
  multi_gpu_barrier<ngpus, true>(params.sg, params.self_sg, params.rank);

  // Get the length and offset of each target device's output
  if (bidx == 0 && tidx < ngpus) {
    int peer_rank = tidx;
    // get peer output offset and length from current rank
    auto& peer_meta = *reinterpret_cast<struct AllToAllCommMeta*>(split_offset_size[peer_rank]);
    // offset in peer output buffer
    int64_t peer_elts_offset = peer_meta.output_elts_offset[params.rank];
    // element count copy to peer output buffer
    int64_t peer_elts_cnt = peer_meta.output_elts_length[params.rank];
    if (peer_elts_cnt != input_split_sizes[peer_rank] * chunk_size) {
      assert(false & "input_split_sizes mismatch peer output_split_sizes");
      multi_gpu_barrier<ngpus, false>(params.sg, params.self_sg, params.rank);
      return;
    }
    plan_meta.peer_meta[peer_rank].output_elts_offset = peer_elts_offset;
    plan_meta.peer_meta[peer_rank].output_dim1 = peer_meta.output_dim1;
    plan_meta.peer_meta[peer_rank].output_stride0 = peer_meta.output_stride0;
    plan_meta.peer_meta[peer_rank].output_stride1 = peer_meta.output_stride1;
  }
  // Calculate the number of warps used to copy to each target device
  int total_opt_warps_count = 0;
  const int elts_once = sizeof(int4) / params.elts_size * WARP_SIZE;
  for (int i = 0; i < ngpus; i++) {
    const int peer_rank = (i + params.rank) % ngpus;
    const int opt_warps_count = (input_split_sizes[peer_rank] * chunk_size + elts_once - 1) / elts_once;
    for (int k = bidx * blockDim.x + tidx; k < opt_warps_count; k += gridDim.x * blockDim.x) {
      plan_meta.warp_peer_gpu[k + total_opt_warps_count] = peer_rank;
    }
    if (bidx == 0 && tidx == 0) {
      plan_meta.warps_beg[peer_rank] = total_opt_warps_count;
    }
    total_opt_warps_count += opt_warps_count;
  }
  if (bidx == 0 && tidx == 0) {
    plan_meta.total_opt_warps_count = total_opt_warps_count;
  }
  multi_gpu_barrier<ngpus, false>(params.sg, params.self_sg, params.rank);
}

__device__ __forceinline__ int32_t
real_offset(int32_t offset, int32_t dim1, int32_t chunk_size, int32_t stride0, int32_t stride1) {
  auto dim01 = offset / chunk_size;
  return (dim01 / dim1) * stride0 + (dim01 % dim1) * stride1 + offset % chunk_size;
}

template <typename T, int ngpus>
static __global__ void __launch_bounds__(MAX_ALL_TO_ALL_WARPS* WARP_SIZE, 1) all2AllKernel(All2AllParams params) {
  int const bidx = blockIdx.x;
  int const tidx = threadIdx.x;
  int const lane_id = tidx % WARP_SIZE;
  // The source pointers. Distributed round-robin for the different warps.
  auto peer_comm_buffer_ptrs = params.peer_comm_buffer_ptrs->ptrs;
  // Start and end offsets of the thread
  const auto& plan_meta = *reinterpret_cast<struct AllToAllPlanMeta*>(params.plan_meta_ptr);
  int32_t input_split_elts_total = plan_meta.input_split_elts_total;
  if (input_split_elts_total > params.input_elts_total) {
    assert(false & "invalid input_split_sizes");
    return;
  }

  constexpr int VEC_SIZE = 16 / sizeof(T);  // every copy element cnt

  const int32_t chunk_size = plan_meta.chunk_size;
  const int32_t cur_dim1 = params.input_dim1;
  const int32_t cur_stride0 = params.input_stride0;
  const int32_t cur_stride1 = params.input_stride1;
  const T* __restrict__ const local_input = reinterpret_cast<const T*>(params.local_input_buffer_ptr);

  int warp_idx = (bidx * blockDim.x + tidx) / WARP_SIZE;
  const int warp_stride = gridDim.x * blockDim.x / WARP_SIZE;
  const int total_opt_warps_count = plan_meta.total_opt_warps_count;
  for (; warp_idx < total_opt_warps_count; warp_idx += warp_stride) {
    int peer_rank = plan_meta.warp_peer_gpu[warp_idx];
    auto warp_beg = plan_meta.warps_beg[peer_rank];
    const int32_t local_offset = plan_meta.local_input_elts_offset[peer_rank];

    auto& peer_meta = plan_meta.peer_meta[peer_rank];
    const int32_t peer_offset = peer_meta.output_elts_offset;
    const int32_t peer_dim1 = peer_meta.output_dim1;
    const int32_t peer_stride0 = peer_meta.output_stride0;
    const int32_t peer_stride1 = peer_meta.output_stride1;

    T* __restrict__ const peer_output = reinterpret_cast<T*>(peer_comm_buffer_ptrs[peer_rank]);

    int32_t chunk_start = ((warp_idx - warp_beg) * WARP_SIZE + lane_id) * VEC_SIZE;

    auto output_offset = real_offset(chunk_start + peer_offset, peer_dim1, chunk_size, peer_stride0, peer_stride1);
    auto input_offset = real_offset(chunk_start + local_offset, cur_dim1, chunk_size, cur_stride0, cur_stride1);
    *reinterpret_cast<int4*>(peer_output + output_offset) = *reinterpret_cast<int4 const*>(local_input + input_offset);
  }
  multi_gpu_barrier<ngpus, false>(params.sg, params.self_sg, params.rank);
}

std::tuple<int, int> all2AllkernelLaunchConfig(int64_t output_elts_total, int64_t input_elts_total, int64_t elts_size) {
  int chunk_size = sizeof(int4) * WARP_SIZE / elts_size;
  int elts_total = input_elts_total;
  int total_splits = std::max(1, divUp(elts_total, chunk_size));
  int sm_count = getSmCount();
  int blocks_per_grid = std::min(sm_count, MAX_ALL_TO_ALL_BLOCKS);
  int splits_per_block = divUp(total_splits, blocks_per_grid);

  int num_warps = std::min(MAX_ALL_TO_ALL_WARPS, splits_per_block);
  int threads_per_block = num_warps * WARP_SIZE;
  return std::make_tuple(blocks_per_grid, threads_per_block);
}

template <typename T>
void invokeAll2AllKernel(All2AllParams& params, hipStream_t stream) {
  auto [blocks_per_grid, threads_per_block] =
      all2AllkernelLaunchConfig(params.output_elts_total, params.input_elts_total, params.elts_size);

#define A2A_KERNEL(ngpus) all2AllKernel<T, ngpus><<<blocks_per_grid, threads_per_block, 0, stream>>>(params)
  switch (params.ranks_per_node) {
    case 2:
      A2A_KERNEL(2);
      break;
    case 4:
      A2A_KERNEL(4);
      break;
    case 6:
      A2A_KERNEL(6);
      break;
    case 8:
      A2A_KERNEL(8);
      break;
    default:
      throw std::runtime_error("invalid world size " + std::to_string(params.ranks_per_node));
  }
#undef A2A_KERNEL

  CHECK_CUDA_SUCCESS(hipGetLastError());
}

void trtCustomAll2All(All2AllParams& params, hipStream_t stream) {
  switch (params.elts_size) {
    case 2:
      invokeAll2AllKernel<half>(params, stream);
      break;
    case 4:
      invokeAll2AllKernel<float>(params, stream);
      break;
    default:
      throw std::runtime_error("Unsupported data element size " + std::to_string(params.elts_size));
  }
}

void invokeAll2AllPlanKernel(All2AllPlanParams& params, hipStream_t stream) {
  switch (params.ranks_per_node) {
    case 2:
      all2AllPlanKernel<2><<<32, 32, 0, stream>>>(params);
      break;
    case 4:
      all2AllPlanKernel<4><<<32, 32, 0, stream>>>(params);
      break;
    case 6:
      all2AllPlanKernel<6><<<32, 32, 0, stream>>>(params);
      break;
    case 8:
      all2AllPlanKernel<8><<<32, 32, 0, stream>>>(params);
      break;
    default:
      throw std::runtime_error("invalid world size " + std::to_string(params.ranks_per_node));
  }
  CHECK_CUDA_SUCCESS(hipGetLastError());
}
}  // namespace

void all_to_all(fptr_t _fa, torch::Tensor& out, torch::Tensor& inp, torch::Tensor& plan_meta, fptr_t _reg_buffer) {
  auto fa = reinterpret_cast<CustomAllreduce*>(_fa);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(inp));
  auto stream = c10::cuda::getCurrentCUDAStream().stream();

  if (inp.dim() != 3 || out.dim() != 3) {
    throw std::runtime_error(
        "custom all_to_all currently requires input or output dim count to be 3, but got input dim count " +
        std::to_string(inp.dim()) + " and output dim count of " + std::to_string(out.dim()) + ".");
    return;
  }

  TORCH_CHECK_EQ(inp.scalar_type(), out.scalar_type());
  All2AllParams params;
  auto reg_buffer = reinterpret_cast<void*>(_reg_buffer);
  auto it = fa->buffers_.find(reg_buffer);
  if (it == fa->buffers_.end()) throw std::runtime_error("output buffer address is not registered!");
  params.peer_comm_buffer_ptrs = it->second;

  params.ranks_per_node = fa->world_size_;
  params.rank = fa->rank_;
  params.sg = fa->sg_;
  params.self_sg = fa->self_sg_;

  params.local_input_buffer_ptr = inp.data_ptr();
  params.plan_meta_ptr = reinterpret_cast<int64_t*>(plan_meta.data_ptr());

  params.input_elts_total = inp.numel();
  params.output_elts_total = out.numel();
  params.elts_size = inp.element_size();
  params.input_stride0 = inp.stride(0);
  params.input_stride1 = inp.stride(1);
  params.input_dim1 = inp.size(1);

  trtCustomAll2All(params, stream);
  if (out.numel() != 0 && out.data_ptr() != reg_buffer) {
    auto output_size = out.numel() * out.element_size();
    AT_CUDA_CHECK(hipMemcpyAsync(out.data_ptr(), reg_buffer, output_size, hipMemcpyDeviceToDevice, stream));
  }
}

void all_to_all_plan(
    fptr_t _fa,
    torch::Tensor& out,
    torch::Tensor& inp,
    torch::Tensor& output_split_sizes,
    torch::Tensor& input_split_sizes,
    int64_t chunk_size,
    torch::Tensor& output_split_offsets,
    torch::Tensor& input_split_offsets,
    torch::Tensor& plan_meta) {
  auto fa = reinterpret_cast<CustomAllreduce*>(_fa);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(inp));
  auto stream = c10::cuda::getCurrentCUDAStream().stream();

  if (plan_meta.numel() * plan_meta.element_size() < sizeof(AllToAllPlanMeta)) {
    assert(false && "Invalid plan meta size");
    throw std::runtime_error(
        "custom all_to_all: invalid plan meta size, requires >= " + std::to_string(sizeof(AllToAllPlanMeta)) +
        ", but got " + std::to_string(plan_meta.numel() * plan_meta.element_size()) + ".");
    return;
  }
  if (inp.dim() != 3 || out.dim() != 3) {
    throw std::runtime_error(
        "custom all_to_all currently requires input or output dim count to be 3, but got input dim count " +
        std::to_string(inp.dim()) + " and output dim count of " + std::to_string(out.dim()) + ".");
    return;
  }

  All2AllPlanParams params;
  params.ranks_per_node = fa->world_size_;
  params.rank = fa->rank_;
  params.sg = fa->sg_;
  params.self_sg = fa->self_sg_;

  params.input_elts_total = inp.numel();
  params.output_elts_total = out.numel();

  params.output_split_sizes = reinterpret_cast<int64_t*>(output_split_sizes.data_ptr());
  params.input_split_sizes = reinterpret_cast<int64_t*>(input_split_sizes.data_ptr());
  if (output_split_offsets.numel() == 0) {
    params.output_split_offsets = nullptr;
  } else {
    params.output_split_offsets = reinterpret_cast<int64_t*>(output_split_offsets.data_ptr());
  }
  if (input_split_offsets.numel() == 0) {
    params.input_split_offsets = nullptr;
  } else {
    params.input_split_offsets = reinterpret_cast<int64_t*>(input_split_offsets.data_ptr());
  }

  params.plan_meta_ptr = reinterpret_cast<int64_t*>(plan_meta.data_ptr());

  params.chunk_size = chunk_size;
  params.output_stride0 = out.stride(0);
  params.output_stride1 = out.stride(1);
  params.output_dim1 = out.size(1);

  params.elts_size = inp.element_size();
  int d = sizeof(int4) * WARP_SIZE / params.elts_size;
  if (params.input_elts_total % d != 0) {
    throw std::runtime_error(
        "custom all_to_all currently requires input length to be multiple "
        "of " +
        std::to_string(d));
    return;
  }
  if (params.input_elts_total % d != 0) {
    throw std::runtime_error(
        "custom all_to_all currently requires output length to be multiple "
        "of " +
        std::to_string(d));
    return;
  }
  auto [blocks_per_grid, threads_per_block] =
      all2AllkernelLaunchConfig(params.output_elts_total, params.input_elts_total, params.elts_size);
  params.blocks_per_grid = blocks_per_grid;
  params.threads_per_block = threads_per_block;
  invokeAll2AllPlanKernel(params, stream);
}
