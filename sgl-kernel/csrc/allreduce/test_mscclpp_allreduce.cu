#include "hip/hip_runtime.h"
/*
 * this file is used to test mscclpp_allreduce.cu using mpirun
 * this file is adapted from https://github.com/flashinfer-ai/flashinfer/blob/v0.2.5/src/test_sum_all_reduce.cu
usage:
cd PATH-TO-THIS-FILE
export MPI_HOME=/usr/local/mpi
# export MPI_HOME=/opt/hpcx/ompi/
export MSCCLPP_HOME=/workspace/test/mscclpp
nvcc -O2 -arch=native -std=c++17 test_mscclpp_allreduce.cu \
  -o test_mscclpp_allreduce -D_GLIBCXX_USE_CXX11_ABI=0 \
  -I${MSCCLPP_HOME}/include -L${MSCCLPP_HOME}/build -lmscclpp \
  -lnccl -I${MPI_HOME}/include -L${MPI_HOME}/lib -lmpi

/opt/hpcx/ompi/bin/
mpirun --allow-run-as-root -H 127.0.0.1:8 -np 8 \
  --map-by ppr:8:node \
  --mca btl_openib_warn_no_device_params_found 0 \
  --mca btl_tcp_if_include bond0 \
  --allow-run-as-root -np 8 \
  -x NCCL_RUNTIME_CONNECT=0 -x NCCL_IB_GID_INDEX=3 -x NCCL_DEBUG=WARN \
  -x LD_PRELOAD=${MSCCLPP_HOME}/build/libmscclpp.so ./test_mscclpp_allreduce
 */
#include <mpi.h>
#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#ifndef CHECK_CUDA_SUCCESS
#define CHECK_CUDA_SUCCESS(cmd)                                                             \
  do {                                                                                      \
    hipError_t e = cmd;                                                                    \
    if (e != hipSuccess) {                                                                 \
      printf("Failed: Cuda error %s:%d '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
      exit(EXIT_FAILURE);                                                                   \
    }                                                                                       \
  } while (0)
#endif

#include <cstdint>

#include "mscclpp_allreduce.cuh"

template <typename T>
bool isclose(T a, T b, float rtol = 1e-5, float atol = 1e-8) {
  return fabs(a - b) <= (atol + rtol * fabs(b));
}

int main(int argc, char* argv[]) {
  // init mpi
  MPI_Init(&argc, &argv);
  printf("MPI Initialized.\n");
  int nranks, rank;

  // get work size and rank id
  MPI_Comm_size(MPI_COMM_WORLD, &nranks);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  hipSetDevice(rank);
  printf("nranks: %d, rank: %d\n", nranks, rank);

  // init host and device buffers
  using T = float;
  using ReduceT = float;
  const size_t num_elems = 2 * 1024 * 1024;
  std::vector<T> host_buf(num_elems);
  for (uint32_t i = 0; i < num_elems; ++i) {
    host_buf[i] = T(i + rank);
  }
  thrust::device_vector<T> device_buf(host_buf);
  const size_t buf_size_in_bytes = num_elems * sizeof(T);
  std::vector<T> host_result_buf(num_elems);
  thrust::device_vector<T> device_result_buf(host_result_buf);

  std::vector<T> host_scratch_buf(num_elems * 8);
  for (uint32_t i = 0; i < num_elems; ++i) {
    host_scratch_buf[i] = 1;
  }
  thrust::device_vector<T> device_scratch_buf(host_scratch_buf);
  std::vector<T> host_put_buf(num_elems);
  thrust::device_vector<T> device_put_buf(host_put_buf);

  mscclpp::UniqueId unique_id;
  if (rank == 0) unique_id = mscclpp::TcpBootstrap::createUniqueId();
  MPI_Bcast(&unique_id, sizeof(unique_id), MPI_BYTE, 0, MPI_COMM_WORLD);

  std::vector<int64_t> rank_to_node(nranks);
  std::vector<int64_t> rank_to_ib(nranks);
  for (int i = 0; i < nranks; i++) {
    rank_to_node[i] = i / 8;
    rank_to_ib[i] = i % 8;
  }

  hipStream_t s;
  CHECK_CUDA_SUCCESS(hipStreamCreate(&s));
  CHECK_CUDA_SUCCESS(hipStreamSynchronize(s));
  if (nranks == 8) {
    auto context = std::make_shared<sglang::Msccl1NodeLLcontext>(
        unique_id,
        rank,
        nranks,
        thrust::raw_pointer_cast(device_scratch_buf.data()),
        buf_size_in_bytes * 8,
        rank_to_node,
        rank_to_ib);
    printf("rank: %d, Msccl1NodeLLcontext setup.\n", rank);
    MPI_Barrier(MPI_COMM_WORLD);
    context->allreduce<T>(
        s,
        thrust::raw_pointer_cast(device_buf.data()),
        thrust::raw_pointer_cast(device_result_buf.data()),
        device_buf.size());
  } else if (nranks == 16) {
    // TODO: this branch is untested since there is something wrong with mpirun in my test machince
    auto context = std::make_shared<sglang::Msccl2NodeLLcontext>(
        unique_id,
        rank,
        nranks,
        thrust::raw_pointer_cast(device_scratch_buf.data()),
        buf_size_in_bytes * 8,
        thrust::raw_pointer_cast(device_put_buf.data()),
        buf_size_in_bytes,
        rank_to_node,
        rank_to_ib);
    printf("rank: %d, Msccl2NodeLLcontext setup.\n", rank);
    MPI_Barrier(MPI_COMM_WORLD);
    context->allreduce<T>(
        s,
        thrust::raw_pointer_cast(device_buf.data()),
        thrust::raw_pointer_cast(device_result_buf.data()),
        device_buf.size());
  }

  // check result correctness
  thrust::host_vector<T> host_buf_result = device_result_buf;
  size_t num_results_error_atol_1e_3_rtol_1e_3 = 0;
  bool nan_detected = false;

  for (uint32_t i = 0; i < num_elems; ++i) {
    T expected = T(i * nranks + (nranks - 1) * nranks / 2);
    if (std::isnan(float(host_buf_result[i]))) {
      nan_detected = true;
    }
    if (!isclose(float(host_buf_result[i]), float(expected), 1e-3, 1e-3)) {
      num_results_error_atol_1e_3_rtol_1e_3++;
    }
  }
  float result_accuracy = 1. - float(num_results_error_atol_1e_3_rtol_1e_3) / float(num_elems);

  printf("rank: %d, nan_detected: %d accuracy: %f\n", rank, nan_detected, result_accuracy);

  CHECK_CUDA_SUCCESS(hipStreamDestroy(s));
  MPI_Finalize();
  return 0;
}
