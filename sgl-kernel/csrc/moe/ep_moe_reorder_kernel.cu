#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <THC/THCAtomics.cuh>
#include <flashinfer/vec_dtypes.cuh>

#include "utils.h"

template <typename scalar_t>
__global__ void ep_pre_reorder_cuda_kernel(
    const scalar_t* __restrict__ input_ptr,
    scalar_t* __restrict__ gateup_input_ptr,
    const int* __restrict__ src2dst_ptr,
    const int* __restrict__ topk_ids_ptr,
    const float* __restrict__ a1_scales_ptr,
    int start_expert_id,
    int end_expert_id,
    int topk,
    int hidden_size,
    bool use_per_token_if_dynamic) {
  int token_idx = blockIdx.x;
  int tid = threadIdx.x;

  const scalar_t* src_ptr = input_ptr + int64_t(token_idx) * hidden_size;
  const int* token_src2dst = src2dst_ptr + token_idx * topk;
  const int* token_topk_ids = topk_ids_ptr + token_idx * topk;

  float scale = 1.0f;

  if (a1_scales_ptr != nullptr and use_per_token_if_dynamic) {
    scale = 1.0f / a1_scales_ptr[token_idx];
  }

  for (int k = 0; k < topk; ++k) {
    int expert_id = token_topk_ids[k];
    if (expert_id < start_expert_id || expert_id > end_expert_id) continue;

    if (a1_scales_ptr != nullptr) {
      if (!use_per_token_if_dynamic) {
        scale = 1.0f / a1_scales_ptr[expert_id - start_expert_id];
      }
    }

    int dst_idx = token_src2dst[k];
    scalar_t* dst_ptr = gateup_input_ptr + int64_t(dst_idx) * hidden_size;

    constexpr uint32_t vec_size = 16 / sizeof(scalar_t);
    using vec_t = flashinfer::vec_t<scalar_t, vec_size>;

    int vec_elements = (hidden_size / vec_size) * vec_size;
    for (int idx = tid; idx < hidden_size / vec_size; idx += blockDim.x) {
      vec_t input_vec, output_vec;
      input_vec.cast_load(src_ptr + idx * vec_size);
#pragma unroll
      for (uint32_t i = 0; i < vec_size; ++i) {
        float val = static_cast<float>(input_vec[i]);
        output_vec[i] = static_cast<scalar_t>(val * scale);
      }
      output_vec.cast_store(dst_ptr + idx * vec_size);
    }

    for (int idx = vec_elements + tid; idx < hidden_size; idx += blockDim.x) {
      float val = static_cast<float>(src_ptr[idx]);
      dst_ptr[idx] = static_cast<scalar_t>(val * scale);
    }
  }
}

template <typename scalar_t>
__global__ void ep_post_reorder_cuda_kernel(
    const scalar_t* __restrict__ down_output_ptr,
    scalar_t* __restrict__ output_ptr,
    const int* __restrict__ src2dst_ptr,
    const int* __restrict__ topk_ids_ptr,
    const scalar_t* __restrict__ topk_weights_ptr,
    int start_expert_id,
    int end_expert_id,
    int topk,
    int hidden_size) {
  const int token_idx = blockIdx.x;
  const int tid = threadIdx.x;

  const int* token_src2dst = src2dst_ptr + token_idx * topk;
  const int* token_topk_ids = topk_ids_ptr + token_idx * topk;
  const scalar_t* token_topk_weights = topk_weights_ptr + token_idx * topk;

  scalar_t* dst_ptr = output_ptr + static_cast<int64_t>(token_idx) * hidden_size;

  constexpr uint32_t vec_size = 16 / sizeof(scalar_t);
  using vec_t = flashinfer::vec_t<scalar_t, vec_size>;

  const int vec_iters = hidden_size / vec_size;
  for (int idx = tid; idx < vec_iters; idx += blockDim.x) {
    float acc[vec_size] = {0};

    for (int k = 0; k < topk; ++k) {
      const int expert_id = token_topk_ids[k];
      if (expert_id < start_expert_id || expert_id > end_expert_id) continue;
      const int src_row = token_src2dst[k];
      const scalar_t* src_ptr = down_output_ptr + static_cast<int64_t>(src_row) * hidden_size;
      const float weight = static_cast<float>(token_topk_weights[k]);

      vec_t src_vec;
      src_vec.cast_load(src_ptr + idx * vec_size);

#pragma unroll
      for (uint32_t i = 0; i < vec_size; ++i) {
        acc[i] += static_cast<float>(src_vec[i]) * weight;
      }
    }
    vec_t out_vec;
#pragma unroll
    for (uint32_t i = 0; i < vec_size; ++i)
      out_vec[i] = static_cast<scalar_t>(acc[i]);

    out_vec.cast_store(dst_ptr + idx * vec_size);
  }
}

void ep_moe_pre_reorder(
    torch::Tensor input,
    torch::Tensor gateup_input,
    torch::Tensor src2dst,
    torch::Tensor topk_ids,
    torch::Tensor a1_scales,
    int64_t start_expert_id,
    int64_t end_expert_id,
    int64_t topk,
    bool use_per_token_if_dynamic) {
  const int total_blocks = input.size(0);
  const int block_size = 512;
  dim3 grid(total_blocks);
  dim3 block(block_size);
  int hidden_size = input.size(1);

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), scalar_t, [&] {
    ep_pre_reorder_cuda_kernel<scalar_t><<<grid, block>>>(
        static_cast<scalar_t*>(input.data_ptr()),
        static_cast<scalar_t*>(gateup_input.data_ptr()),
        src2dst.data_ptr<int>(),
        topk_ids.data_ptr<int>(),
        a1_scales.defined() ? a1_scales.data_ptr<float>() : nullptr,
        start_expert_id,
        end_expert_id,
        topk,
        hidden_size,
        use_per_token_if_dynamic);
    return true;
  });
}

void ep_moe_post_reorder(
    torch::Tensor down_output,
    torch::Tensor output,
    torch::Tensor src2dst,
    torch::Tensor topk_ids,
    torch::Tensor topk_weights,
    int64_t start_expert_id,
    int64_t end_expert_id,
    int64_t topk) {
  const int total_tokens = output.size(0);
  const int block_size = 512;
  dim3 grid(total_tokens);
  dim3 block(block_size);
  const int hidden_size = output.size(1);

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(down_output.scalar_type(), scalar_t, [&] {
    ep_post_reorder_cuda_kernel<scalar_t><<<grid, block>>>(
        static_cast<scalar_t*>(down_output.data_ptr()),
        static_cast<scalar_t*>(output.data_ptr()),
        src2dst.data_ptr<int>(),
        topk_ids.data_ptr<int>(),
        static_cast<scalar_t*>(topk_weights.data_ptr()),
        static_cast<int>(start_expert_id),
        static_cast<int>(end_expert_id),
        static_cast<int>(topk),
        hidden_size);
    return true;
  });
}
