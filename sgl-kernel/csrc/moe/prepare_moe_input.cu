#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAGuard.h>
#include <cudaTypedefs.h>
#include <torch/all.h>

#include <iostream>

#include "cutlass/array.h"

constexpr uint64_t THREADS_PER_EXPERT = 512;

__global__ void compute_problem_sizes(
    const int* __restrict__ topk_ids,
    int32_t* problem_sizes1,
    int32_t* problem_sizes2,
    int32_t* atomic_buffer,
    const int64_t topk_length,
    const int64_t n,
    const int64_t k) {
  int expert_id = blockIdx.x;

  int occurrences = 0;
  for (int i = threadIdx.x; i < topk_length; i += THREADS_PER_EXPERT) {
    occurrences += (topk_ids[i] == expert_id);
  }
  atomicAdd(&atomic_buffer[expert_id], occurrences);
  __syncthreads();

  if (threadIdx.x == 0) {
    int final_occurrences = atomic_buffer[expert_id];
    problem_sizes1[expert_id * 3] = final_occurrences;
    problem_sizes1[expert_id * 3 + 1] = static_cast<int32_t>(2 * n);
    problem_sizes1[expert_id * 3 + 2] = static_cast<int32_t>(k);
    problem_sizes2[expert_id * 3] = final_occurrences;
    problem_sizes2[expert_id * 3 + 1] = static_cast<int32_t>(k);
    problem_sizes2[expert_id * 3 + 2] = static_cast<int32_t>(n);
  }
}

__global__ void compute_expert_offsets(
    const int32_t* __restrict__ problem_sizes1,
    int32_t* expert_offsets,
    int32_t* atomic_buffer,
    const int64_t num_experts) {
  int32_t tot_offset = 0;
  expert_offsets[0] = 0;
  for (int i = 0; i < num_experts; ++i) {
    atomic_buffer[i] = tot_offset;
    tot_offset += problem_sizes1[i * 3];
    expert_offsets[i + 1] = tot_offset;
  }
}

__global__ void compute_expert_blockscale_offsets(
    const int32_t* __restrict__ problem_sizes1,
    int32_t* expert_offsets,
    int32_t* blockscale_offsets,
    int32_t* atomic_buffer,
    const int64_t num_experts) {
  int32_t tot_offset = 0;
  int32_t tot_rounded_offset = 0;
  expert_offsets[0] = 0;
  blockscale_offsets[0] = 0;
  for (int i = 0; i < num_experts; ++i) {
    atomic_buffer[i] = tot_offset;
    int num_tokens = problem_sizes1[i * 3];
    int rounded_num_tokens = (num_tokens + (128 - 1)) / 128 * 128;
    tot_offset += num_tokens;
    tot_rounded_offset += rounded_num_tokens;
    expert_offsets[i + 1] = tot_offset;
    blockscale_offsets[i + 1] = tot_rounded_offset;
  }
}

__global__ void compute_arg_sorts(
    const int32_t* __restrict__ topk_ids,
    int32_t* input_permutation,
    int32_t* output_permutation,
    int32_t* atomic_buffer,
    const int64_t topk_length,
    const int64_t topk) {
  int expert_id = blockIdx.x;

  for (int i = threadIdx.x; i < topk_length; i += THREADS_PER_EXPERT) {
    if (topk_ids[i] == expert_id) {
      int start = atomicAdd(&atomic_buffer[expert_id], 1);
      input_permutation[start] = i / topk;
      output_permutation[i] = start;
    }
  }
}

void get_moe_prepare_input_caller(
    const torch::Tensor& topk_ids,
    torch::Tensor& expert_offsets,
    const std::optional<torch::Tensor>& blockscale_offsets,
    torch::Tensor& problem_sizes1,
    torch::Tensor& problem_sizes2,
    torch::Tensor& input_permutation,
    torch::Tensor& output_permutation,
    const int64_t num_experts,
    const int64_t n,
    const int64_t k) {
  auto stream = at::cuda::getCurrentCUDAStream(topk_ids.device().index());
  auto options_int32 = torch::TensorOptions().dtype(torch::kInt32).device(topk_ids.device());
  torch::Tensor atomic_buffer = torch::zeros(num_experts, options_int32);

  uint32_t num_threads = static_cast<uint32_t>(min(THREADS_PER_EXPERT, topk_ids.numel()));
  uint32_t num_blocks = static_cast<uint32_t>(num_experts);

  compute_problem_sizes<<<num_blocks, num_threads, 0, stream>>>(
      static_cast<const int32_t*>(topk_ids.data_ptr()),
      static_cast<int32_t*>(problem_sizes1.data_ptr()),
      static_cast<int32_t*>(problem_sizes2.data_ptr()),
      static_cast<int32_t*>(atomic_buffer.data_ptr()),
      topk_ids.numel(),
      n,
      k);
  if (blockscale_offsets.has_value()) {
    compute_expert_blockscale_offsets<<<1, 1, 0, stream>>>(
        static_cast<const int32_t*>(problem_sizes1.data_ptr()),
        static_cast<int32_t*>(expert_offsets.data_ptr()),
        static_cast<int32_t*>(blockscale_offsets.value().data_ptr()),
        static_cast<int32_t*>(atomic_buffer.data_ptr()),
        num_experts);
  } else {
    compute_expert_offsets<<<1, 1, 0, stream>>>(
        static_cast<const int32_t*>(problem_sizes1.data_ptr()),
        static_cast<int32_t*>(expert_offsets.data_ptr()),
        static_cast<int32_t*>(atomic_buffer.data_ptr()),
        num_experts);
  }
  compute_arg_sorts<<<num_blocks, num_threads, 0, stream>>>(
      static_cast<const int32_t*>(topk_ids.data_ptr()),
      static_cast<int32_t*>(input_permutation.data_ptr()),
      static_cast<int32_t*>(output_permutation.data_ptr()),
      static_cast<int32_t*>(atomic_buffer.data_ptr()),
      topk_ids.numel(),
      topk_ids.size(1));
}

void prepare_moe_input(
    const torch::Tensor& topk_ids,
    torch::Tensor& expert_offsets,
    const std::optional<torch::Tensor>& blockscale_offsets,
    torch::Tensor& problem_sizes1,
    torch::Tensor& problem_sizes2,
    torch::Tensor& input_permutation,
    torch::Tensor& output_permutation,
    const int64_t num_experts,
    const int64_t n,
    const int64_t k) {
  TORCH_CHECK(topk_ids.dtype() == torch::kInt32);
  get_moe_prepare_input_caller(
      topk_ids,
      expert_offsets,
      blockscale_offsets,
      problem_sizes1,
      problem_sizes2,
      input_permutation,
      output_permutation,
      num_experts,
      n,
      k);
  return;
}

template <typename T>
__global__ void shuffleRowsKernel(
    const T* input,
    const int32_t* dst2src_map,
    T* output,
    int64_t num_src_rows,
    int64_t num_dst_rows,
    int64_t num_cols) {
  int64_t dest_row_idx = blockIdx.x;
  int64_t const source_row_idx = dst2src_map[dest_row_idx];

  if (blockIdx.x < num_dst_rows) {
    // Load 128-bits per thread
    constexpr uint64_t ELEM_PER_THREAD = 128 / sizeof(T) / 8;
    using DataElem = cutlass::Array<T, ELEM_PER_THREAD>;

    // Duplicate and permute rows
    auto const* source_row_ptr = reinterpret_cast<DataElem const*>(input + source_row_idx * num_cols);
    auto* dest_row_ptr = reinterpret_cast<DataElem*>(output + dest_row_idx * num_cols);

    auto const start_offset = threadIdx.x;
    auto const stride = blockDim.x;
    auto const num_elems_in_col = num_cols / ELEM_PER_THREAD;

    for (auto elem_index = start_offset; elem_index < num_elems_in_col; elem_index += stride) {
      dest_row_ptr[elem_index] = source_row_ptr[elem_index];
    }
  }
}

#define DECLARE_SHUFFLE_ROWS(T)      \
  __global__ void shuffleRowsKernel( \
      const T* input,                \
      const int32_t* dst2src_map,    \
      T* output,                     \
      int64_t num_src_rows,          \
      int64_t num_dest_rows,         \
      int64_t num_cols);

DECLARE_SHUFFLE_ROWS(float);
DECLARE_SHUFFLE_ROWS(half);
DECLARE_SHUFFLE_ROWS(__hip_bfloat16);
DECLARE_SHUFFLE_ROWS(__hip_fp8_e4m3_fnuz);
DECLARE_SHUFFLE_ROWS(uint8_t);

#define SHUFFLE_ROWS(T)                                    \
  shuffleRowsKernel<T><<<blocks, threads, 0, stream>>>(    \
      reinterpret_cast<const T*>(input),                   \
      static_cast<const int32_t*>(dst2src_map.data_ptr()), \
      reinterpret_cast<T*>(output),                        \
      num_src_rows,                                        \
      num_dst_rows,                                        \
      num_cols)

#define DTYPE_DISPATCH_CASE(T, CUDA_T) \
  case T:                              \
    SHUFFLE_ROWS(CUDA_T);              \
    break;

void shuffle_rows_caller(
    const torch::Tensor& input_tensor, const torch::Tensor& dst2src_map, torch::Tensor& output_tensor) {
  TORCH_CHECK(
      input_tensor.scalar_type() == output_tensor.scalar_type(),
      "Input and output tensors must have the same data type");
  auto stream = at::cuda::getCurrentCUDAStream().stream();
  uint32_t blocks = static_cast<uint32_t>(output_tensor.size(0));
  uint32_t threads = 256;
  int64_t num_dst_rows = output_tensor.size(0);
  int64_t num_src_rows = input_tensor.size(0);
  int64_t num_cols = input_tensor.size(1);
  const void* input = input_tensor.data_ptr();
  void* output = output_tensor.data_ptr();
  switch (input_tensor.scalar_type()) {
    DTYPE_DISPATCH_CASE(torch::kFloat16, half);
    DTYPE_DISPATCH_CASE(torch::kBFloat16, __hip_bfloat16);
    DTYPE_DISPATCH_CASE(torch::kFloat32, float);
    DTYPE_DISPATCH_CASE(torch::kFloat8_e4m3fn, __hip_fp8_e4m3_fnuz);
    DTYPE_DISPATCH_CASE(torch::kUInt8, uint8_t);
    default:
      TORCH_CHECK(false, "[moe replicate input] data type dispatch fail!");
  }
  return;
}

void shuffle_rows(const torch::Tensor& input_tensor, const torch::Tensor& dst2src_map, torch::Tensor& output_tensor) {
  shuffle_rows_caller(input_tensor, dst2src_map, output_tensor);
  return;
}

template <typename scalar_t>
__global__ void apply_shuffle_mul_sum_kernel(
    const scalar_t* __restrict__ input_tensor,  // [m * topk, row_stride]
    scalar_t* __restrict__ output_tensor,       // [m, row_stride]
    const int32_t* __restrict__ permutation,    // [m * topk]
    int m,
    int topk,
    int row_stride,
    const scalar_t* __restrict__ factors)  // [m * topk] or nullptr
{
  int i = blockIdx.x;   // [0, m * topk)
  int d = threadIdx.x;  // [0, row_stride)

  if (i >= m || d >= row_stride) return;

  scalar_t sum_val = 0.0;

  for (int j = 0; j < topk; ++j) {
    int index_2d = i * topk + j;
    int src_row = permutation[index_2d];
    if (src_row >= m) continue;

    scalar_t val = input_tensor[src_row * row_stride + d];

    scalar_t factor = 1.0;
    if (factors != nullptr) {
      factor = factors[index_2d];
    }

    sum_val += factor * val;
  }

  output_tensor[i * row_stride + d] = sum_val;
}

void get_apply_shuffle_mul_sum_caller(
    const torch::Tensor& input_tensor,                // [m * topk, row_stride], bf16/f16
    torch::Tensor& output_tensor,                     // [m, row_stride], bf16/f16
    const torch::Tensor& permutation,                 // [m * topk], int32
    const std::optional<torch::Tensor>& factors_opt)  // optional [m * topk], bf16/f16
{
  TORCH_CHECK(input_tensor.dim() == 2, "input_tensor must be 2D [m * topk, row_stride]");
  TORCH_CHECK(output_tensor.dim() == 2, "output_tensor must be 2D [m, row_stride]");
  TORCH_CHECK(permutation.dim() == 1, "permutation must be 1D [m * topk]");

  int m = output_tensor.size(0);
  int topk = int(permutation.size(0) / m);
  int row_stride = output_tensor.size(1);

  TORCH_CHECK(permutation.size(0) == m * topk, "permutation size must match m * topk");

  dim3 block(std::min(256, row_stride));
  dim3 grid(m);  // blockIdx.x = j, blockIdx.y = i
  auto stream = at::cuda::getCurrentCUDAStream(input_tensor.device().index());

  const int32_t* perm_ptr = permutation.data_ptr<int32_t>();

  void* factors_ptr = nullptr;
  if (factors_opt.has_value()) {
    TORCH_CHECK(factors_opt->dtype() == output_tensor.dtype(), "Factors must match output dtype");
    TORCH_CHECK(factors_opt->numel() == m * topk, "Factors must have shape [m * topk]");
    factors_ptr = factors_opt->data_ptr();
  }

  if (output_tensor.scalar_type() == at::ScalarType::Half) {
    const at::Half* factor_data = static_cast<const at::Half*>(factors_ptr);
    apply_shuffle_mul_sum_kernel<at::Half><<<grid, block, 0, stream>>>(
        input_tensor.data_ptr<at::Half>(),
        output_tensor.data_ptr<at::Half>(),
        perm_ptr,
        m,
        topk,
        row_stride,
        static_cast<const at::Half*>(factors_ptr));
  } else if (output_tensor.scalar_type() == at::ScalarType::BFloat16) {
    const c10::BFloat16* factor_data = static_cast<const c10::BFloat16*>(factors_ptr);
    apply_shuffle_mul_sum_kernel<c10::BFloat16><<<grid, block, 0, stream>>>(
        input_tensor.data_ptr<c10::BFloat16>(),
        output_tensor.data_ptr<c10::BFloat16>(),
        perm_ptr,
        m,
        topk,
        row_stride,
        static_cast<const c10::BFloat16*>(factors_ptr));
  } else {
    TORCH_CHECK(false, "Unsupported output dtype for cast+mul kernel: ", output_tensor.scalar_type());
  }
}

/**
 * @brief Applies a permutation-based shuffle, element-wise multiplication, and reduction over the second dimension.
 *
 * This function performs the equivalent of the following PyTorch expression:
 *
 *     (c2[c_map].view(m, topk, k) * topk_weights.view(m, topk, 1).to(out_dtype)).sum(dim=1)
 *
 * Specifically:
 * - `input` is shuffled using the `permutation` tensor.
 * - The shuffled tensor is reshaped and multiplied element-wise with `factors` (e.g., top-k weights).
 * - The result is summed along dimension 1 (the top-k dimension), and stored in `output`.
 *
 * @param input        Input tensor of shape (m * topk, k), representing c2.
 * @param output       Output tensor of shape (m, k), where the final reduced results are stored.
 * @param permutation  Index tensor (e.g., c_map) that maps positions in `input` to shuffled layout.
 * @param factors      Optional scaling factors (e.g., top-k weights), shape (m * topk) or (m, topk).
 */
void apply_shuffle_mul_sum(
    const torch::Tensor& input,
    torch::Tensor& output,
    const torch::Tensor& permutation,
    const std::optional<torch::Tensor>& factors) {
  get_apply_shuffle_mul_sum_caller(input, output, permutation, factors);
}
