#include "hip/hip_runtime.h"
/* Copyright 2025 SGLang Team. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <THC/THCAtomics.cuh>

#include "utils.h"

#define WARP_SIZE 32

#define VEC_SIZE 4
using Vec = int4;

template <typename scalar_t>
__global__ void count_and_sort_expert_tokens_kernel(
    const scalar_t* __restrict__ topk_ids,
    int32_t* __restrict__ sorted_token_ids,
    int32_t* __restrict__ cumsum_buffer,
    size_t numel) {
  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;

  for (size_t i = tid; i < numel; i += stride) {
    int32_t expert_id = topk_ids[i];
    int32_t rank_post_pad = atomicAdd(&cumsum_buffer[expert_id], 1);
    sorted_token_ids[rank_post_pad] = i;
  }
}

template <typename scalar_t>
__global__ void moe_align_block_size_kernel(
    const scalar_t* __restrict__ topk_ids,
    int32_t* __restrict__ sorted_token_ids,
    int32_t* __restrict__ expert_ids,
    int32_t* __restrict__ total_tokens_post_pad,
    int32_t num_experts,
    int32_t block_size,
    size_t numel,
    int32_t* __restrict__ cumsum,
    bool pad_sorted_token_ids,
    const int32_t scan_size) {
  extern __shared__ int32_t smem[];
  int32_t* shared_counts = smem;                  // [num_experts]
  int32_t* prefix = shared_counts + num_experts;  // [num_experts + 1]
  int32_t* scan_buf = prefix + num_experts + 1;   // [scan_size]
  __shared__ int32_t s_total_tokens_post_pad;

  const size_t tid = threadIdx.x;
  const size_t stride = blockDim.x;

  if (tid < num_experts) {
    shared_counts[tid] = 0;
  }

  __syncthreads();

  for (size_t i = tid; i < numel; i += stride) {
    int expert_id = topk_ids[i];
    atomicAdd(&shared_counts[expert_id], 1);
  }

  __syncthreads();

  int32_t padded_count = 0;
  if (tid < num_experts) {
    int32_t count = shared_counts[tid];
    padded_count = (count + block_size - 1) / block_size * block_size;
    scan_buf[tid] = padded_count;
  }

  if (tid >= num_experts && tid < scan_size) {
    scan_buf[tid] = 0;
  }

  __syncthreads();

  // Blelloch scan
  int offset = 1;
#pragma unroll
  for (int d = scan_size >> 1; d > 0; d >>= 1) {
    if (tid < d) {
      int ai = offset * (2 * tid + 1) - 1;
      int bi = offset * (2 * tid + 2) - 1;
      scan_buf[bi] += scan_buf[ai];
    }
    offset <<= 1;
    __syncthreads();
  }

  // down-sweep
  if (tid == 0) {
    prefix[num_experts] = scan_buf[scan_size - 1];
    scan_buf[scan_size - 1] = 0;
  }
  __syncthreads();

#pragma unroll
  for (int d = 1; d < scan_size; d <<= 1) {
    offset >>= 1;
    if (tid < d) {
      int ai = offset * (2 * tid + 1) - 1;
      int bi = offset * (2 * tid + 2) - 1;
      if (bi < scan_size) {
        int temp = scan_buf[ai];
        scan_buf[ai] = scan_buf[bi];
        scan_buf[bi] += temp;
      }
    }
    __syncthreads();
  }

  if (tid < num_experts) {
    prefix[tid] = scan_buf[tid];
  }

  if (tid == 0) {
    s_total_tokens_post_pad = prefix[num_experts];
    *total_tokens_post_pad = s_total_tokens_post_pad;
  }

  __syncthreads();

  if (tid <= num_experts) {
    cumsum[tid] = prefix[tid];
  }

  // fill expert_ids
  const int32_t num_blocks = s_total_tokens_post_pad / block_size;
  for (int32_t i = tid; i < num_blocks; i += stride) {
    int32_t block_start = i * block_size;
    int left = 0, right = num_experts;
    while (left < right) {
      int mid = (left + right) >> 1;
      if (prefix[mid] <= block_start) {
        left = mid + 1;
      } else {
        right = mid;
      }
    }
    expert_ids[i] = left - 1;
  }

  if (pad_sorted_token_ids) {
    Vec fill_vec;
    fill_vec.x = fill_vec.y = fill_vec.z = fill_vec.w = numel;
    int32_t total_vecs = (s_total_tokens_post_pad + VEC_SIZE - 1) / VEC_SIZE;
    Vec* out_ptr = reinterpret_cast<Vec*>(sorted_token_ids);
    for (int32_t i = tid; i < total_vecs; i += stride) {
      out_ptr[i] = fill_vec;
    }
  }
}

template <typename scalar_t>
__global__ void moe_align_block_size_small_batch_expert_kernel(
    const scalar_t* __restrict__ topk_ids,
    int32_t* __restrict__ sorted_token_ids,
    int32_t* __restrict__ expert_ids,
    int32_t* __restrict__ total_tokens_post_pad,
    int32_t num_experts,
    int32_t block_size,
    size_t numel,
    bool pad_sorted_token_ids) {
  const size_t tid = threadIdx.x;
  const size_t stride = blockDim.x;

  extern __shared__ int32_t shared_mem[];
  int32_t* cumsum = shared_mem;
  int32_t* tokens_cnts = (int32_t*)(shared_mem + num_experts + 1);

  for (int i = 0; i < num_experts; ++i) {
    tokens_cnts[(threadIdx.x + 1) * num_experts + i] = 0;
  }

  for (size_t i = tid; i < numel; i += stride) {
    ++tokens_cnts[(threadIdx.x + 1) * num_experts + topk_ids[i]];
  }

  __syncthreads();

  if (threadIdx.x < num_experts) {
    tokens_cnts[threadIdx.x] = 0;
    for (int i = 1; i <= blockDim.x; ++i) {
      tokens_cnts[i * num_experts + threadIdx.x] += tokens_cnts[(i - 1) * num_experts + threadIdx.x];
    }
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    cumsum[0] = 0;
    for (int i = 1; i <= num_experts; ++i) {
      cumsum[i] = cumsum[i - 1] + CEILDIV(tokens_cnts[blockDim.x * num_experts + i - 1], block_size) * block_size;
    }
    *total_tokens_post_pad = static_cast<int32_t>(cumsum[num_experts]);
  }

  __syncthreads();

  if (threadIdx.x < num_experts) {
    for (int i = cumsum[threadIdx.x]; i < cumsum[threadIdx.x + 1]; i += block_size) {
      expert_ids[i / block_size] = threadIdx.x;
    }
  }

  if (pad_sorted_token_ids) {
    Vec fill_vec;
    fill_vec.x = fill_vec.y = fill_vec.z = fill_vec.w = numel;
    int32_t total_vecs = (*total_tokens_post_pad + VEC_SIZE - 1) / VEC_SIZE;
    Vec* out_ptr = reinterpret_cast<Vec*>(sorted_token_ids);
    for (int32_t i = tid; i < total_vecs; i += stride) {
      out_ptr[i] = fill_vec;
    }
  }

  __syncthreads();

  for (size_t i = tid; i < numel; i += stride) {
    int32_t expert_id = topk_ids[i];
    int32_t rank_post_pad = tokens_cnts[threadIdx.x * num_experts + expert_id] + cumsum[expert_id];
    sorted_token_ids[rank_post_pad] = i;
    ++tokens_cnts[threadIdx.x * num_experts + expert_id];
  }
}

void moe_align_block_size(
    torch::Tensor topk_ids,
    int64_t num_experts,
    int64_t block_size,
    torch::Tensor sorted_token_ids,
    torch::Tensor experts_ids,
    torch::Tensor num_tokens_post_pad,
    torch::Tensor token_cnts_buffer,
    torch::Tensor cumsum_buffer,
    bool pad_sorted_token_ids) {
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int64_t padded_num_experts = ((num_experts + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;

  int experts_per_warp = WARP_SIZE;
  int threads = 1024;

  threads = ((threads + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;

  DISPATCH_INTEGRAL_TYPES(topk_ids.scalar_type(), "moe_align_block_size_kernel", [&] {
    bool small_batch_expert_mode = (topk_ids.numel() < 1024) && (num_experts <= 64);

    if (small_batch_expert_mode) {
      const int32_t threads = max((int32_t)num_experts, WARP_SIZE);
      const int32_t shared_mem_size = ((threads + 1) * num_experts + (num_experts + 1)) * sizeof(int32_t);

      auto small_batch_expert_kernel = moe_align_block_size_small_batch_expert_kernel<scalar_t>;
      small_batch_expert_kernel<<<1, threads, shared_mem_size, stream>>>(
          topk_ids.data_ptr<scalar_t>(),
          sorted_token_ids.data_ptr<int32_t>(),
          experts_ids.data_ptr<int32_t>(),
          num_tokens_post_pad.data_ptr<int32_t>(),
          num_experts,
          block_size,
          topk_ids.numel(),
          pad_sorted_token_ids);
    } else {
      auto align_kernel = moe_align_block_size_kernel<scalar_t>;

      const size_t scan_size = next_pow2(num_experts);
      const size_t shared_mem_size = (num_experts + (num_experts + 1) + scan_size) * sizeof(int32_t);

      align_kernel<<<1, threads, shared_mem_size, stream>>>(
          topk_ids.data_ptr<scalar_t>(),
          sorted_token_ids.data_ptr<int32_t>(),
          experts_ids.data_ptr<int32_t>(),
          num_tokens_post_pad.data_ptr<int32_t>(),
          num_experts,
          block_size,
          topk_ids.numel(),
          cumsum_buffer.data_ptr<int32_t>(),
          pad_sorted_token_ids,
          scan_size);

      const int block_threads = std::min(256, (int)threads);
      const int num_blocks = (topk_ids.numel() + block_threads - 1) / block_threads;
      const int max_blocks = 65535;
      const int actual_blocks = std::min(num_blocks, max_blocks);

      auto sort_kernel = count_and_sort_expert_tokens_kernel<scalar_t>;
      sort_kernel<<<actual_blocks, block_threads, 0, stream>>>(
          topk_ids.data_ptr<scalar_t>(),
          sorted_token_ids.data_ptr<int32_t>(),
          cumsum_buffer.data_ptr<int32_t>(),
          topk_ids.numel());
    }
  });
}
