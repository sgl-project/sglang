#include "hip/hip_runtime.h"
/* Copyright 2025 SGLang Team. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <THC/THCAtomics.cuh>

#include "utils.h"

#define WARP_SIZE 32

template <typename scalar_t>
__global__ void count_and_sort_expert_tokens_kernel(
    const scalar_t* __restrict__ topk_ids,
    int32_t* __restrict__ sorted_token_ids,
    int32_t* __restrict__ cumsum_buffer,
    size_t numel) {
  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;

  for (size_t i = tid; i < numel; i += stride) {
    int32_t expert_id = topk_ids[i];
    int32_t rank_post_pad = atomicAdd(&cumsum_buffer[expert_id], 1);
    sorted_token_ids[rank_post_pad] = i;
  }
}

template <typename scalar_t>
__global__ void moe_align_block_size_kernel(
    const scalar_t* __restrict__ topk_ids,
    int32_t* __restrict__ sorted_token_ids,
    int32_t* __restrict__ expert_ids,
    int32_t* __restrict__ total_tokens_post_pad,
    int32_t num_experts,
    int32_t padded_num_experts,
    int32_t experts_per_warp,
    int32_t block_size,
    size_t numel,
    int32_t* __restrict__ cumsum) {
  extern __shared__ int32_t shared_counts[];

  const int warp_id = threadIdx.x / WARP_SIZE;
  const int my_expert_start = warp_id * experts_per_warp;

  for (int i = 0; i < experts_per_warp; ++i) {
    if (my_expert_start + i < padded_num_experts) {
      shared_counts[warp_id * experts_per_warp + i] = 0;
    }
  }

  __syncthreads();

  const size_t tokens_per_thread = CEILDIV(numel, blockDim.x);
  const size_t start_idx = threadIdx.x * tokens_per_thread;

  for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
    int expert_id = topk_ids[i];
    int warp_idx = expert_id / experts_per_warp;
    int expert_offset = expert_id % experts_per_warp;
    atomicAdd(&shared_counts[warp_idx * experts_per_warp + expert_offset], 1);
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    cumsum[0] = 0;
    for (int i = 1; i <= num_experts; ++i) {
      int expert_count = 0;
      int warp_idx = (i - 1) / experts_per_warp;
      int expert_offset = (i - 1) % experts_per_warp;
      expert_count = shared_counts[warp_idx * experts_per_warp + expert_offset];

      cumsum[i] = cumsum[i - 1] + CEILDIV(expert_count, block_size) * block_size;
    }
    *total_tokens_post_pad = cumsum[num_experts];
  }

  __syncthreads();

  if (threadIdx.x < num_experts) {
    for (int i = cumsum[threadIdx.x]; i < cumsum[threadIdx.x + 1]; i += block_size) {
      expert_ids[i / block_size] = threadIdx.x;
    }
  }
}

void moe_align_block_size(
    torch::Tensor topk_ids,
    int64_t num_experts,
    int64_t block_size,
    torch::Tensor sorted_token_ids,
    torch::Tensor experts_ids,
    torch::Tensor num_tokens_post_pad,
    torch::Tensor token_cnts_buffer,
    torch::Tensor cumsum_buffer) {
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int64_t padded_num_experts = ((num_experts + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;

  int experts_per_warp;
  int threads;

  if (num_experts <= 8) {
    experts_per_warp = 8;
    threads = 256;
  } else if (num_experts <= 16) {
    experts_per_warp = 16;
    threads = 512;
  } else {
    experts_per_warp = WARP_SIZE;
    threads = 1024;
  }

  threads = ((threads + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;

  DISPATCH_INTEGRAL_TYPES(topk_ids.scalar_type(), "moe_align_block_size_kernel", [&] {
    auto align_kernel = moe_align_block_size_kernel<scalar_t>;

    size_t num_warps = CEILDIV(padded_num_experts, experts_per_warp);
    size_t shared_mem_size = num_warps * experts_per_warp * sizeof(int32_t);

    align_kernel<<<1, threads, shared_mem_size, stream>>>(
        topk_ids.data_ptr<scalar_t>(),
        sorted_token_ids.data_ptr<int32_t>(),
        experts_ids.data_ptr<int32_t>(),
        num_tokens_post_pad.data_ptr<int32_t>(),
        num_experts,
        padded_num_experts,
        experts_per_warp,
        block_size,
        topk_ids.numel(),
        cumsum_buffer.data_ptr<int32_t>());

    const int block_threads = std::min(256, (int)threads);
    const int num_blocks = (topk_ids.numel() + block_threads - 1) / block_threads;
    const int max_blocks = 65535;
    const int actual_blocks = std::min(num_blocks, max_blocks);

    auto sort_kernel = count_and_sort_expert_tokens_kernel<scalar_t>;
    sort_kernel<<<actual_blocks, block_threads, 0, stream>>>(
        topk_ids.data_ptr<scalar_t>(),
        sorted_token_ids.data_ptr<int32_t>(),
        cumsum_buffer.data_ptr<int32_t>(),
        topk_ids.numel());
  });
}
