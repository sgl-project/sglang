#include "hip/hip_runtime.h"
#pragma once

#include <c10/cuda/CUDAStream.h>
#include <hip/hip_runtime.h>
#include <torch/all.h>

#include "cutlass/bfloat16.h"
#include "cutlass/float8.h"

template <
    typename ElementAB,
    typename ElementC,
    typename ElementAccumulator,
    typename LayoutSFA,
    typename LayoutSFB,
    typename ScaleConfig>
__global__ void get_group_gemm_starts(
    int32_t* expert_offsets,
    ElementAB** a_offsets,
    ElementAB** b_offsets,
    ElementC** out_offsets,
    ElementAccumulator** a_scales_offsets,
    ElementAccumulator** b_scales_offsets,
    ElementAB* a_base_as_int,
    ElementAB* b_base_as_int,
    ElementC* out_base_as_int,
    ElementAccumulator* a_scales_base_as_int,
    ElementAccumulator* b_scales_base_as_int,
    LayoutSFA* layout_sfa_base_as_int,
    LayoutSFB* layout_sfb_base_as_int,
    int* problem_sizes,
    int* problem_sizes_transpose,
    bool transpose = false) {
  int expert_id = threadIdx.x;

  if (expert_id >= gridDim.x * blockDim.x) {
    return;
  }

  int m = problem_sizes[expert_id * 3];
  int n = problem_sizes[expert_id * 3 + 1];
  int k = problem_sizes[expert_id * 3 + 2];
  if (transpose) {
    problem_sizes_transpose[expert_id * 3] = n;
    problem_sizes_transpose[expert_id * 3 + 1] = m;
    problem_sizes_transpose[expert_id * 3 + 2] = k;
  }

  int32_t expert_offset = expert_offsets[expert_id];
  int a_stride = 0;
  int b_stride = 0;
  int a_scale_stride = 0;
  int b_scale_stride = 0;
  if (!transpose) {
    a_stride = expert_offset * k;
    b_stride = expert_id * k * n;
    a_scale_stride = expert_offset * k / 128;
    b_scale_stride = expert_id * k * n / 128 / 128;
  } else {
    a_stride = expert_id * k * n;
    b_stride = expert_offset * k;
    a_scale_stride = expert_id * k * n / 128 / 128;
    b_scale_stride = expert_offset * k / 128;
  }
  a_offsets[expert_id] = a_base_as_int + a_stride;
  b_offsets[expert_id] = b_base_as_int + b_stride;
  out_offsets[expert_id] = out_base_as_int + expert_offset * n;
  a_scales_offsets[expert_id] = a_scales_base_as_int + a_scale_stride;
  b_scales_offsets[expert_id] = b_scales_base_as_int + b_scale_stride;

  LayoutSFA* layout_sfa_ptr = layout_sfa_base_as_int + expert_id;
  LayoutSFB* layout_sfb_ptr = layout_sfb_base_as_int + expert_id;

  if (!transpose) {
    *layout_sfa_ptr = ScaleConfig::tile_atom_to_shape_SFA(cute::make_shape(m, n, k, 1));
    *layout_sfb_ptr = ScaleConfig::tile_atom_to_shape_SFB(cute::make_shape(m, n, k, 1));
  } else {
    *layout_sfa_ptr = ScaleConfig::tile_atom_to_shape_SFA(cute::make_shape(n, m, k, 1));
    *layout_sfb_ptr = ScaleConfig::tile_atom_to_shape_SFB(cute::make_shape(n, m, k, 1));
  }
}

#define __CALL_GET_STARTS_KERNEL(TENSOR_C_TYPE, C_TYPE, LayoutSFA, LayoutSFB, ScaleConfig)         \
  else if (out_tensors.dtype() == TENSOR_C_TYPE) {                                                 \
    get_group_gemm_starts<cutlass::float_e4m3_t, C_TYPE, float, LayoutSFA, LayoutSFB, ScaleConfig> \
        <<<1, num_experts, 0, stream>>>(                                                           \
            static_cast<int32_t*>(expert_offsets.data_ptr()),                                      \
            static_cast<cutlass::float_e4m3_t**>(a_ptrs.data_ptr()),                               \
            static_cast<cutlass::float_e4m3_t**>(b_ptrs.data_ptr()),                               \
            static_cast<C_TYPE**>(out_ptrs.data_ptr()),                                            \
            static_cast<float**>(a_scales_ptrs.data_ptr()),                                        \
            static_cast<float**>(b_scales_ptrs.data_ptr()),                                        \
            static_cast<cutlass::float_e4m3_t*>(a_tensors.data_ptr()),                             \
            static_cast<cutlass::float_e4m3_t*>(b_tensors.data_ptr()),                             \
            static_cast<C_TYPE*>(out_tensors.data_ptr()),                                          \
            static_cast<float*>(a_scales.data_ptr()),                                              \
            static_cast<float*>(b_scales.data_ptr()),                                              \
            reinterpret_cast<LayoutSFA*>(layout_sfa.data_ptr()),                                   \
            reinterpret_cast<LayoutSFB*>(layout_sfb.data_ptr()),                                   \
            static_cast<int*>(problem_sizes.data_ptr()),                                           \
            static_cast<int*>(problem_sizes_transpose.data_ptr()),                                 \
            transpose);                                                                            \
  }

namespace {
template <typename LayoutSFA, typename LayoutSFB, typename ScaleConfig>
void run_get_group_gemm_starts(
    torch::Tensor const& expert_offsets,
    torch::Tensor& a_ptrs,
    torch::Tensor& b_ptrs,
    torch::Tensor& out_ptrs,
    torch::Tensor& a_scales_ptrs,
    torch::Tensor& b_scales_ptrs,
    torch::Tensor const& a_tensors,
    torch::Tensor const& b_tensors,
    torch::Tensor& out_tensors,
    torch::Tensor const& a_scales,
    torch::Tensor const& b_scales,
    torch::Tensor const& layout_sfa,
    torch::Tensor const& layout_sfb,
    torch::Tensor const& problem_sizes,
    torch::Tensor& problem_sizes_transpose,
    bool transpose = false) {
  TORCH_CHECK(a_tensors.dtype() == torch::kFloat8_e4m3fn);
  TORCH_CHECK(b_tensors.dtype() == torch::kFloat8_e4m3fn);
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(out_tensors.size(1) % 128 == 0 or out_tensors.size(0) % 128 == 0);
  TORCH_CHECK(a_tensors.size(1) % 128 == 0 or a_tensors.size(0) % 128 == 0);

  int num_experts = (int)expert_offsets.size(0);
  auto stream = at::cuda::getCurrentCUDAStream(a_tensors.device().index());

  if (false) {
  }
  __CALL_GET_STARTS_KERNEL(torch::kBFloat16, cutlass::bfloat16_t, LayoutSFA, LayoutSFB, ScaleConfig)
  __CALL_GET_STARTS_KERNEL(torch::kFloat16, half, LayoutSFA, LayoutSFB, ScaleConfig)
  else {
    TORCH_CHECK(false, "Invalid output type (must be float16 or bfloat16)");
  }
}
}  // namespace
