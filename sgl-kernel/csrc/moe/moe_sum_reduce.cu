#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <hip/hip_runtime.h>
#include <torch/all.h>

#include <iostream>
#include <type_traits>

#include "cutlass/array.h"
#include "utils.h"

template <typename T>
__device__ __forceinline__ float to_float(T x) {
  return static_cast<float>(x);
}

template <>
__device__ __forceinline__ float to_float<half>(half x) {
  return __half2float(x);
}

template <typename T>
__device__ __forceinline__ T from_float(float x) {
  return static_cast<T>(x);
}

template <>
__device__ __forceinline__ half from_float<half>(float x) {
  return __float2half_rn(x);
}

template <typename T>
__device__ __forceinline__ T ldg_cg(const T* p) {
  return __ldg(p);
}

union Pack16B {
  uint4 v;
  __hip_bfloat16 u16[8];
};

template <int WARPS_PER_BLOCK>
__global__ void moe_sum_reduce_warp_per_token_vec_kernel(
    const at::BFloat16* __restrict__ x,
    at::BFloat16* __restrict__ y,
    const int64_t token_num,
    const int64_t hidden_dim,
    const int64_t topk_num,
    const int64_t stride_token,      // in elements
    const int64_t stride_topk,       // in elements
    const int64_t out_stride_token,  // in elements
    const float scale) {
  constexpr int VEC = 16;
  constexpr int PACKS = VEC / 8;

  const int warp_id = threadIdx.x / 32;
  const int lane = threadIdx.x % 32;
  const int64_t t = (int64_t)blockIdx.y * WARPS_PER_BLOCK + warp_id;
  if (t >= token_num) return;

  const int64_t n_chunks = hidden_dim / VEC;

  for (int64_t chunk = (int64_t)blockIdx.x * 32 + lane; chunk < n_chunks; chunk += (int64_t)gridDim.x * 32) {
    const int64_t d = chunk * VEC;
    const int64_t base = t * stride_token + d;

    float acc[VEC];
#pragma unroll
    for (int i = 0; i < VEC; ++i)
      acc[i] = 0.f;

#pragma unroll
    for (int k = 0; k < topk_num; ++k) {
#pragma unroll
      for (int p = 0; p < PACKS; ++p) {
        const int64_t offset = base + (int64_t)k * stride_topk + p * 8;
        Pack16B pack = {ldg_cg(reinterpret_cast<const uint4*>(x + offset))};

#pragma unroll
        for (int i = 0; i < 8; ++i) {
          acc[p * 8 + i] += __bfloat162float(pack.u16[i]);
        }
      }
    }

#pragma unroll
    for (int i = 0; i < VEC; ++i)
      acc[i] *= scale;

#pragma unroll
    for (int p = 0; p < PACKS; ++p) {
      Pack16B outp;
#pragma unroll
      for (int i = 0; i < 8; ++i) {
        outp.u16[i] = __float2bfloat16_rn(acc[p * 8 + i]);
      }
      const int64_t dst = t * out_stride_token + d + p * 8;
      *reinterpret_cast<uint4*>(y + dst) = outp.v;
    }
  }
}

template <typename scalar_t, int TOPK, int WARPS_PER_BLOCK>
__global__ void moe_sum_reduce_kernel_warp_token_topk(
    const scalar_t* __restrict__ x,
    scalar_t* __restrict__ y,
    const int64_t token_num,
    const int64_t hidden_dim,
    const int64_t stride_token,
    const int64_t stride_topk,
    const int64_t out_stride_token,
    const float scale) {
  const int warp_id = threadIdx.x / 32;
  const int lane = threadIdx.x % 32;
  const int64_t t = (int64_t)blockIdx.y * WARPS_PER_BLOCK + warp_id;
  if (t >= token_num) return;

  for (int64_t d = (int64_t)blockIdx.x * 32 + lane; d < hidden_dim; d += (int64_t)gridDim.x * 32) {
    float acc = 0.f;
    const int64_t base = t * stride_token + d;

#pragma unroll
    for (int k = 0; k < TOPK; ++k) {
      acc += to_float<scalar_t>(ldg_cg(&x[base + (int64_t)k * stride_topk]));
    }
    acc *= scale;
    y[t * out_stride_token + d] = from_float<scalar_t>(acc);
  }
}

template <typename scalar_t, int TOPK>
__global__ void moe_sum_reduce_kernel(
    const scalar_t* __restrict__ x,
    scalar_t* __restrict__ y,
    const int64_t token_num,
    const int64_t hidden_dim,
    const int64_t stride_token,
    const int64_t stride_topk,
    const int64_t out_stride_token,
    const float scale) {
  for (int t = blockIdx.y; t < token_num; t += gridDim.y) {
    for (int d = blockIdx.x * blockDim.x + threadIdx.x; d < hidden_dim; d += blockDim.x * gridDim.x) {
      const int64_t base = t * stride_token + d;
      float acc = 0.f;

#pragma unroll
      for (int k = 0; k < TOPK; ++k) {
        acc += to_float<scalar_t>(x[base + (int64_t)k * stride_topk]);
      }

      acc *= scale;
      y[t * out_stride_token + d] = from_float<scalar_t>(acc);
    }
  }
}

void moe_sum_reduce(at::Tensor& input, at::Tensor& output, double routed_scaling_factor) {
  TORCH_CHECK(input.is_cuda(), "input must be CUDA tensor");
  TORCH_CHECK(output.is_cuda(), "output must be CUDA tensor");
  TORCH_CHECK(input.dim() == 3, "input must be a 3D tensor like [token_num, topk_num, hidden_dim]");
  TORCH_CHECK(output.dim() == 2, "output must be [token_num, hidden_dim]");
  TORCH_CHECK(input.size(0) == output.size(0), "token dim mismatch");
  TORCH_CHECK(input.size(2) == output.size(1), "hidden_dim mismatch");

  TORCH_CHECK(input.is_contiguous(), "expect input to be contiguous");
  TORCH_CHECK(output.is_contiguous(), "expect output to be contiguous");

  const int64_t token_num = input.size(0);
  const int64_t topk_num = input.size(1);
  const int64_t hidden_dim = input.size(2);

  const int64_t in_stride_token = input.stride(0);
  const int64_t in_stride_topk = input.stride(1);
  const int64_t out_stride_token = output.stride(0);

  const float scale = static_cast<float>(routed_scaling_factor);

  auto stream = at::cuda::getCurrentCUDAStream();

  const bool fast_bf16_vec_ok = (input.scalar_type() == at::kBFloat16) && (token_num > 256) && (hidden_dim % 8 == 0);

  // Fast path for bf16 vectorize
  if (fast_bf16_vec_ok) {
    constexpr int WARPS_PER_BLOCK = 8;
    constexpr int THREADS = WARPS_PER_BLOCK * 32;

    const int64_t n_chunks = hidden_dim / 8;
    int64_t grid_x = (n_chunks + 32 - 1) / 32;
    if (grid_x > 65535) grid_x = 65535;

    int64_t grid_y = (token_num + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;
    if (grid_y > 65535) grid_y = 65535;

    dim3 block(THREADS);
    dim3 grid(static_cast<unsigned>(grid_x), static_cast<unsigned>(grid_y));

    auto stream = at::cuda::getCurrentCUDAStream();

    moe_sum_reduce_warp_per_token_vec_kernel<WARPS_PER_BLOCK><<<grid, block, 0, stream>>>(
        reinterpret_cast<const at::BFloat16*>(input.data_ptr<at::BFloat16>()),
        reinterpret_cast<at::BFloat16*>(output.data_ptr<at::BFloat16>()),
        token_num,
        hidden_dim,
        topk_num,
        in_stride_token,
        in_stride_topk,
        out_stride_token,
        scale);

    TORCH_CHECK(hipGetLastError() == hipSuccess, "moe_sum_reduce CUDA kernel launch failed");
    return;
  }

  const bool per_token_use_one_warp = (token_num > 128);

  auto dispatch_topk = [&](auto&& launch_kernel) {
    switch (topk_num) {
      case 2:
        launch_kernel(std::integral_constant<int, 2>{});
        break;
      case 4:
        launch_kernel(std::integral_constant<int, 4>{});
        break;
      case 8:
        launch_kernel(std::integral_constant<int, 8>{});
        break;
      case 9:
        launch_kernel(std::integral_constant<int, 9>{});
        break;
      default:
        launch_kernel(std::integral_constant<int, -1>{});
        break;
    }
  };

  if (!per_token_use_one_warp) {
    // ---------- small-token ----------
    const int block_size = 256;
    int64_t grid_x = (hidden_dim + block_size - 1) / block_size;
    grid_x = grid_x > 65535 ? 65535 : grid_x;
    int64_t grid_y = token_num < 65535 ? token_num : 65535;

    dim3 block(block_size);
    dim3 grid(static_cast<unsigned>(grid_x), static_cast<unsigned>(grid_y));

    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::kHalf, at::kBFloat16, input.scalar_type(), "moe_sum_reduce_cuda_small_token", [&] {
          using scalar_t_ = scalar_t;

          auto lauch_small_token_kernel = [&](auto topk_c) {
            constexpr int TK = decltype(topk_c)::value;

            moe_sum_reduce_kernel<scalar_t_, TK><<<grid, block, 0, stream>>>(
                input.data_ptr<scalar_t_>(),
                output.data_ptr<scalar_t_>(),
                token_num,
                hidden_dim,
                in_stride_token,
                in_stride_topk,
                out_stride_token,
                scale);
          };
          dispatch_topk(lauch_small_token_kernel);
        });

  } else {
    // ---------- warp-token ----------
    constexpr int WARPS_PER_BLOCK = 4;
    constexpr int THREADS = WARPS_PER_BLOCK * 32;

    int64_t gx = (hidden_dim + 32 - 1) / 32;
    gx = gx > 65535 ? 65535 : gx;

    int64_t gy = (token_num + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;
    gy = gy > 65535 ? 65535 : gy;

    dim3 block(THREADS);
    dim3 grid(static_cast<unsigned>(gx), static_cast<unsigned>(gy));

    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::kHalf, at::kBFloat16, input.scalar_type(), "moe_sum_reduce_cuda_large_token", [&] {
          using scalar_t_ = scalar_t;

          auto launch_large_token_kernel = [&](auto topk_c) {
            constexpr int TK = decltype(topk_c)::value;

            moe_sum_reduce_kernel_warp_token_topk<scalar_t_, TK, WARPS_PER_BLOCK><<<grid, block, 0, stream>>>(
                input.data_ptr<scalar_t_>(),
                output.data_ptr<scalar_t_>(),
                token_num,
                hidden_dim,
                in_stride_token,
                in_stride_topk,
                out_stride_token,
                scale);
          };
          dispatch_topk(launch_large_token_kernel);
        });
  }
  TORCH_CHECK(hipGetLastError() == hipSuccess, "CUDA kernel launch failed");
}
