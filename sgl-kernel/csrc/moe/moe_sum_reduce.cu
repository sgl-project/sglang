#include "hip/hip_runtime.h"
#include <ATen/OpMathType.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <hip/hip_runtime.h>
#include <torch/all.h>

#include <iostream>
#include <type_traits>

#include "cutlass/array.h"
#include "utils.h"

template <typename T>
using opmath_t = at::opmath_type<T>;

template <typename T>
__device__ __forceinline__ opmath_t<T> to_acc(T x) {
  return static_cast<opmath_t<T>>(x);
}

template <typename T>
__device__ __forceinline__ T from_acc(opmath_t<T> x) {
  return static_cast<T>(x);
}

template <>
__device__ __forceinline__ opmath_t<at::Half> to_acc<at::Half>(at::Half x) {
  return __half2float(__nv_half(x));
}
template <>
__device__ __forceinline__ at::Half from_acc<at::Half>(opmath_t<at::Half> x) {
  return __float2half_rn(x);
}

template <>
__device__ __forceinline__ opmath_t<at::BFloat16> to_acc<at::BFloat16>(at::BFloat16 x) {
  return __bfloat162float(__hip_bfloat16(x));
}
template <>
__device__ __forceinline__ at::BFloat16 from_acc<at::BFloat16>(opmath_t<at::BFloat16> x) {
  return __float2bfloat16_rn(x);
}

template <typename T>
__device__ __forceinline__ T ldg_cg(const T* p) {
  return __ldg(p);
}

union Pack16B {
  uint4 v;
  __hip_bfloat16 u16[8];
};

template <int WARPS_PER_BLOCK>
__global__ void moe_sum_reduce_warp_per_token_vec_kernel(
    const at::BFloat16* __restrict__ x,
    at::BFloat16* __restrict__ y,
    const int64_t token_num,
    const int64_t hidden_dim,
    const int64_t topk_num,
    const int64_t stride_token,      // in elements
    const int64_t stride_topk,       // in elements
    const int64_t out_stride_token,  // in elements
    const float scale) {
  constexpr int VEC = 16;
  constexpr int PACKS = VEC / 8;

  const int warp_id = threadIdx.x / 32;
  const int lane = threadIdx.x % 32;
  const int64_t t = (int64_t)blockIdx.y * WARPS_PER_BLOCK + warp_id;
  if (t >= token_num) return;

  const int64_t n_chunks = hidden_dim / VEC;

  for (int64_t chunk = (int64_t)blockIdx.x * 32 + lane; chunk < n_chunks; chunk += (int64_t)gridDim.x * 32) {
    const int64_t d = chunk * VEC;
    const int64_t base = t * stride_token + d;

    float acc[VEC];
#pragma unroll
    for (int i = 0; i < VEC; ++i)
      acc[i] = 0.f;

#pragma unroll
    for (int k = 0; k < topk_num; ++k) {
#pragma unroll
      for (int p = 0; p < PACKS; ++p) {
        const int64_t offset = base + (int64_t)k * stride_topk + p * 8;
        Pack16B pack = {ldg_cg(reinterpret_cast<const uint4*>(x + offset))};

#pragma unroll
        for (int i = 0; i < 8; ++i) {
          acc[p * 8 + i] += __bfloat162float(pack.u16[i]);
        }
      }
    }

#pragma unroll
    for (int i = 0; i < VEC; ++i)
      acc[i] *= scale;

#pragma unroll
    for (int p = 0; p < PACKS; ++p) {
      Pack16B outp;
#pragma unroll
      for (int i = 0; i < 8; ++i) {
        outp.u16[i] = __float2bfloat16_rn(acc[p * 8 + i]);
      }
      const int64_t dst = t * out_stride_token + d + p * 8;
      *reinterpret_cast<uint4*>(y + dst) = outp.v;
    }
  }
}

template <typename scalar_t, int TOPK, int WARPS_PER_BLOCK>
__global__ void moe_sum_reduce_kernel_warp_token_topk(
    const scalar_t* __restrict__ x,
    scalar_t* __restrict__ y,
    const int64_t token_num,
    const int64_t hidden_dim,
    const int64_t stride_token,
    const int64_t stride_topk,
    const int64_t out_stride_token,
    const opmath_t<scalar_t> scale) {
  const int warp_id = threadIdx.x / 32;
  const int lane = threadIdx.x % 32;
  const int64_t t = (int64_t)blockIdx.y * WARPS_PER_BLOCK + warp_id;
  if (t >= token_num) return;

  for (int64_t d = (int64_t)blockIdx.x * 32 + lane; d < hidden_dim; d += (int64_t)gridDim.x * 32) {
    opmath_t<scalar_t> acc = opmath_t<scalar_t>(0);
    const int64_t base = t * stride_token + d;

#pragma unroll
    for (int k = 0; k < TOPK; ++k) {
      acc += to_acc<scalar_t>(x[base + (int64_t)k * stride_topk]);
    }
    acc *= scale;
    y[t * out_stride_token + d] = from_acc<scalar_t>(acc);
  }
}

template <typename scalar_t, int TOPK>
__global__ void moe_sum_reduce_kernel(
    const scalar_t* __restrict__ x,
    scalar_t* __restrict__ y,
    const int64_t token_num,
    const int64_t hidden_dim,
    const int64_t stride_token,
    const int64_t stride_topk,
    const int64_t out_stride_token,
    const opmath_t<scalar_t> scale) {
  for (int t = blockIdx.y; t < token_num; t += gridDim.y) {
    for (int d = blockIdx.x * blockDim.x + threadIdx.x; d < hidden_dim; d += blockDim.x * gridDim.x) {
      const int64_t base = t * stride_token + d;
      opmath_t<scalar_t> acc = opmath_t<scalar_t>(0);

#pragma unroll
      for (int k = 0; k < TOPK; ++k) {
        acc += to_acc<scalar_t>(x[base + (int64_t)k * stride_topk]);
      }

      acc *= scale;
      y[t * out_stride_token + d] = from_acc<scalar_t>(acc);
    }
  }
}

// -------------------- general-topk fallback kernels --------------------
// small-token
template <typename scalar_t>
__global__ void moe_sum_reduce_kernel_general(
    const scalar_t* __restrict__ x,
    scalar_t* __restrict__ y,
    const int64_t token_num,
    const int64_t hidden_dim,
    const int64_t stride_token,
    const int64_t stride_topk,
    const int64_t out_stride_token,
    const int topk_num,
    const opmath_t<scalar_t> scale) {
  for (int t = blockIdx.y; t < token_num; t += gridDim.y) {
    for (int d = blockIdx.x * blockDim.x + threadIdx.x; d < hidden_dim; d += blockDim.x * gridDim.x) {
      const int64_t base = t * stride_token + d;
      opmath_t<scalar_t> acc = opmath_t<scalar_t>(0);
#pragma unroll 1
      for (int k = 0; k < topk_num; ++k) {
        acc += to_acc<scalar_t>(x[base + (int64_t)k * stride_topk]);
      }
      acc *= scale;
      y[t * out_stride_token + d] = from_acc<scalar_t>(acc);
    }
  }
}

// warp-per-token
template <typename scalar_t, int WARPS_PER_BLOCK>
__global__ void moe_sum_reduce_kernel_warp_token_general(
    const scalar_t* __restrict__ x,
    scalar_t* __restrict__ y,
    const int64_t token_num,
    const int64_t hidden_dim,
    const int64_t stride_token,
    const int64_t stride_topk,
    const int64_t out_stride_token,
    const int topk_num,
    const opmath_t<scalar_t> scale) {
  const int warp_id = threadIdx.x / 32;
  const int lane = threadIdx.x % 32;
  const int64_t t = (int64_t)blockIdx.y * WARPS_PER_BLOCK + warp_id;
  if (t >= token_num) return;

  for (int64_t d = (int64_t)blockIdx.x * 32 + lane; d < hidden_dim; d += (int64_t)gridDim.x * 32) {
    opmath_t<scalar_t> acc = opmath_t<scalar_t>(0);
    const int64_t base = t * stride_token + d;
#pragma unroll 1
    for (int k = 0; k < topk_num; ++k) {
      acc += to_acc<scalar_t>(x[base + (int64_t)k * stride_topk]);
    }
    acc *= scale;
    y[t * out_stride_token + d] = from_acc<scalar_t>(acc);
  }
}

void moe_sum_reduce(at::Tensor& input, at::Tensor& output, double routed_scaling_factor) {
  TORCH_CHECK(input.is_cuda(), "input must be CUDA tensor");
  TORCH_CHECK(output.is_cuda(), "output must be CUDA tensor");
  TORCH_CHECK(input.dim() == 3, "input must be a 3D tensor like [token_num, topk_num, hidden_dim]");
  TORCH_CHECK(output.dim() == 2, "output must be [token_num, hidden_dim]");
  TORCH_CHECK(input.size(0) == output.size(0), "token dim mismatch");
  TORCH_CHECK(input.size(2) == output.size(1), "hidden_dim mismatch");

  TORCH_CHECK(input.is_contiguous(), "expect input to be contiguous");
  TORCH_CHECK(output.is_contiguous(), "expect output to be contiguous");

  const int64_t token_num = input.size(0);
  const int64_t topk_num = input.size(1);
  const int64_t hidden_dim = input.size(2);

  const int64_t in_stride_token = input.stride(0);
  const int64_t in_stride_topk = input.stride(1);
  const int64_t out_stride_token = output.stride(0);

  auto stream = at::cuda::getCurrentCUDAStream();

  const bool fast_bf16_vec_ok = (input.scalar_type() == at::kBFloat16) && (token_num > 256) && (hidden_dim % 8 == 0);

  // Fast path for bf16 vectorize
  if (fast_bf16_vec_ok) {
    constexpr int WARPS_PER_BLOCK = 8;
    constexpr int THREADS = WARPS_PER_BLOCK * 32;

    const int64_t n_chunks = hidden_dim / 8;
    int64_t grid_x = (n_chunks + 32 - 1) / 32;
    if (grid_x > 65535) grid_x = 65535;

    int64_t grid_y = (token_num + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;
    if (grid_y > 65535) grid_y = 65535;

    dim3 block(THREADS);
    dim3 grid(static_cast<unsigned>(grid_x), static_cast<unsigned>(grid_y));

    auto stream = at::cuda::getCurrentCUDAStream();

    const float scale = static_cast<float>(routed_scaling_factor);
    moe_sum_reduce_warp_per_token_vec_kernel<WARPS_PER_BLOCK><<<grid, block, 0, stream>>>(
        reinterpret_cast<const at::BFloat16*>(input.data_ptr<at::BFloat16>()),
        reinterpret_cast<at::BFloat16*>(output.data_ptr<at::BFloat16>()),
        token_num,
        hidden_dim,
        topk_num,
        in_stride_token,
        in_stride_topk,
        out_stride_token,
        scale);

    TORCH_CHECK(hipGetLastError() == hipSuccess, "moe_sum_reduce CUDA kernel (bf16 vec) launch failed");
    return;
  }

  const bool per_token_use_one_warp = (token_num > 128);

  if (!per_token_use_one_warp) {
    // ---------- small-token ----------
    const int block_size = 256;
    int64_t grid_x = (hidden_dim + block_size - 1) / block_size;
    grid_x = grid_x > 65535 ? 65535 : grid_x;
    int64_t grid_y = token_num < 65535 ? token_num : 65535;

    dim3 block(block_size);
    dim3 grid(static_cast<unsigned>(grid_x), static_cast<unsigned>(grid_y));

#define LAUNCH_SMALL_TOKEN_KERNEL(TOPK)                               \
  moe_sum_reduce_kernel<scalar_t_, TOPK><<<grid, block, 0, stream>>>( \
      input.data_ptr<scalar_t_>(),                                    \
      output.data_ptr<scalar_t_>(),                                   \
      token_num,                                                      \
      hidden_dim,                                                     \
      in_stride_token,                                                \
      in_stride_topk,                                                 \
      out_stride_token,                                               \
      scale);

    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::kHalf, at::kBFloat16, input.scalar_type(), "moe_sum_reduce_cuda_small_token", [&] {
          using scalar_t_ = scalar_t;
          using acc_t_ = opmath_t<scalar_t_>;
          const acc_t_ scale = static_cast<acc_t_>(routed_scaling_factor);

          switch (topk_num) {
            case 2:
              LAUNCH_SMALL_TOKEN_KERNEL(2);
              break;
            case 4:
              LAUNCH_SMALL_TOKEN_KERNEL(4);
              break;
            case 8:
              LAUNCH_SMALL_TOKEN_KERNEL(8);
              break;
            case 9:
              LAUNCH_SMALL_TOKEN_KERNEL(9);
              break;
            default:  // launch general kernel
              moe_sum_reduce_kernel_general<scalar_t_><<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t_>(),
                  output.data_ptr<scalar_t_>(),
                  token_num,
                  hidden_dim,
                  in_stride_token,
                  in_stride_topk,
                  out_stride_token,
                  static_cast<int>(topk_num),
                  scale);
          }
        });
#undef LAUNCH_SMALL_TOKEN_KERNEL

    TORCH_CHECK(hipGetLastError() == hipSuccess, "moe_sum_reduce CUDA kernel (small-token) launch failed");

  } else {
    // ---------- warp-per-token ----------
    constexpr int WARPS_PER_BLOCK = 4;
    constexpr int THREADS = WARPS_PER_BLOCK * 32;

    int64_t gx = (hidden_dim + 32 - 1) / 32;
    gx = gx > 65535 ? 65535 : gx;

    int64_t gy = (token_num + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;
    gy = gy > 65535 ? 65535 : gy;

    dim3 block(THREADS);
    dim3 grid(static_cast<unsigned>(gx), static_cast<unsigned>(gy));

#define LAUNCH_WARP_PER_TOKEN_KERNEL(TOPK)                                                             \
  moe_sum_reduce_kernel_warp_token_topk<scalar_t_, TOPK, WARPS_PER_BLOCK><<<grid, block, 0, stream>>>( \
      input.data_ptr<scalar_t_>(),                                                                     \
      output.data_ptr<scalar_t_>(),                                                                    \
      token_num,                                                                                       \
      hidden_dim,                                                                                      \
      in_stride_token,                                                                                 \
      in_stride_topk,                                                                                  \
      out_stride_token,                                                                                \
      scale);

    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::kHalf, at::kBFloat16, input.scalar_type(), "moe_sum_reduce_cuda_large_token", [&] {
          using scalar_t_ = scalar_t;
          using acc_t_ = opmath_t<scalar_t_>;
          const acc_t_ scale = static_cast<acc_t_>(routed_scaling_factor);

          switch (topk_num) {
            case 2:
              LAUNCH_WARP_PER_TOKEN_KERNEL(2);
              break;
            case 4:
              LAUNCH_WARP_PER_TOKEN_KERNEL(4);
              break;
            case 8:
              LAUNCH_WARP_PER_TOKEN_KERNEL(8);
              break;
            case 9:
              LAUNCH_WARP_PER_TOKEN_KERNEL(9);
              break;
            default:  // launch general kernel
              moe_sum_reduce_kernel_warp_token_general<scalar_t_, WARPS_PER_BLOCK><<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t_>(),
                  output.data_ptr<scalar_t_>(),
                  token_num,
                  hidden_dim,
                  in_stride_token,
                  in_stride_topk,
                  out_stride_token,
                  static_cast<int>(topk_num),
                  scale);
          }
        });
#undef LAUNCH_WARP_PER_TOKEN_KERNEL

    TORCH_CHECK(hipGetLastError() == hipSuccess, "moe_sum_reduce CUDA kernel (warp-token) launch failed");
  }
}
