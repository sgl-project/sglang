/* Copyright 2025 SGLang Team. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <ATen/cuda/HIPContext.h>
#include <cutlass/cutlass.h>
#include <cutlass/epilogue/thread/linear_combination.h>
#include <cutlass/epilogue/threadblock/epilogue_with_visitor.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/gemm/device/gemm_universal_adapter.h>
#include <cutlass/numeric_types.h>

#include <cute/atom/mma_atom.hpp>
#include <cute/tensor.hpp>
#include <cutlass/epilogue/collective/collective_builder.hpp>
#include <cutlass/gemm/collective/collective_builder.hpp>
#include <cutlass/gemm/kernel/gemm_universal.hpp>
#include <cutlass/util/packed_stride.hpp>

#include "cutlass_extensions/epilogue/epilogue_per_row_per_col_scale.h"
#include "cutlass_extensions/gemm/gemm_universal_base_compat.h"
#include "cutlass_extensions/gemm/gemm_with_epilogue_visitor.h"
#include "utils.h"

using namespace cute;

template <
    typename ElementOutput,
    typename ArchTag,
    typename ThreadblockShape,
    typename WarpShape,
    typename InstructionShape,
    int NumStages>
void cutlass_int8_scaled_mm(
    torch::Tensor& out,
    const torch::Tensor& mat_a,
    const torch::Tensor& mat_b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  using ElementAccumulator = int32_t;
  using ElementCompute = float;
  using ElementInputA = int8_t;
  using ElementInputB = int8_t;

  using OperatorClass = cutlass::arch::OpClassTensorOp;
  using ThreadblockSwizzle = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>;

  using DefaultGemmConf = cutlass::gemm::device::
      DefaultGemmConfiguration<OperatorClass, ArchTag, ElementInputA, ElementInputB, ElementOutput, ElementCompute>;
  using EpilogueOutputOp = typename DefaultGemmConf::EpilogueOutputOp;

  using GemmKernel_ = typename cutlass::gemm::kernel::DefaultGemm<
      ElementInputA,
      cutlass::layout::RowMajor,
      DefaultGemmConf::kAlignmentA,
      ElementInputB,
      cutlass::layout::ColumnMajor,
      DefaultGemmConf::kAlignmentB,
      ElementOutput,
      cutlass::layout::RowMajor,
      ElementAccumulator,
      OperatorClass,
      ArchTag,
      ThreadblockShape,
      WarpShape,
      InstructionShape,
      EpilogueOutputOp,
      ThreadblockSwizzle,
      NumStages,
      true,
      typename DefaultGemmConf::Operator>::GemmKernel;

  using AlphaColTileIterator = cutlass::epilogue::threadblock::PredicatedTileIterator<
      cutlass::epilogue::threadblock::OutputTileOptimalThreadMap<
          typename GemmKernel_::Epilogue::OutputTileIterator::ThreadMap::Shape,
          typename GemmKernel_::Epilogue::OutputTileIterator::ThreadMap::Count,
          GemmKernel_::Epilogue::OutputTileIterator::ThreadMap::kThreads,
          GemmKernel_::Epilogue::OutputTileIterator::kElementsPerAccess,
          cutlass::sizeof_bits<ElementOutput>::value>,
      ElementCompute>;

  using EpilogueVisitor = typename cutlass::epilogue::threadblock::EpilogueVisitorPerRowPerCol<
      ThreadblockShape,
      GemmKernel_::kThreadCount,
      AlphaColTileIterator,
      typename GemmKernel_::Epilogue::OutputTileIterator,
      ElementAccumulator,
      ElementCompute,
      EpilogueOutputOp>;

  using Epilogue = typename cutlass::epilogue::threadblock::
      EpilogueWithVisitorFromExistingEpilogue<EpilogueVisitor, typename GemmKernel_::Epilogue>::Epilogue;

  using GemmKernel =
      cutlass::gemm::kernel::GemmWithEpilogueVisitor<typename GemmKernel_::Mma, Epilogue, ThreadblockSwizzle>;

  using Gemm = cutlass::gemm::device::GemmUniversalBaseCompat<GemmKernel>;

  Gemm gemm_op;

  int m = mat_a.size(0);
  int k = mat_a.size(1);
  int n = mat_b.size(1);

  auto a_ptr = static_cast<ElementInputA*>(mat_a.data_ptr());
  auto b_ptr = static_cast<ElementInputB*>(mat_b.data_ptr());
  auto o_ptr = static_cast<ElementOutput*>(out.data_ptr());

  auto a_s_ptr = static_cast<ElementCompute*>(scales_a.data_ptr());
  auto b_s_ptr = static_cast<ElementCompute*>(scales_b.data_ptr());

  int64_t lda = mat_a.stride(0);
  int64_t ldb = mat_b.stride(1);
  int64_t ldd = out.stride(0);

  ElementOutput* bias_ptr = nullptr;
  int64_t ldc = 0;
  if (bias) {
    bias_ptr = static_cast<ElementOutput*>(bias->data_ptr());
  }

  typename EpilogueOutputOp::Params linearScalingParams;
  typename EpilogueVisitor::Arguments visitor_args{linearScalingParams};

  typename Gemm::Arguments args{
      {m, n, k}, {a_ptr, lda}, {b_ptr, ldb}, {b_s_ptr, 0}, {a_s_ptr, 0}, {bias_ptr, ldc}, {o_ptr, ldd}, visitor_args};

  auto workspace = torch::empty(
      gemm_op.get_workspace_size(args), torch::TensorOptions().dtype(torch::kUInt8).device(mat_a.device()));

  auto stream = at::cuda::getCurrentCUDAStream(mat_a.get_device());

  auto can_implement = gemm_op.can_implement(args);
  TORCH_CHECK(
      can_implement == cutlass::Status::kSuccess,
      "gemm cannot implement, error: ",
      cutlassGetStatusString(can_implement));

  auto status = gemm_op(args, workspace.data_ptr(), stream);
  TORCH_CHECK(status == cutlass::Status::kSuccess, "gemm executioin failed, error: ", cutlassGetStatusString(status));
}

template <typename ElementOutput, typename ArchTag, typename InstructionShape>
void sm75_dispatch_shape(
    torch::Tensor& out,
    const torch::Tensor& mat_a,
    const torch::Tensor& mat_b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  int m = mat_a.size(0);
  if (m <= 32) {
    cutlass_int8_scaled_mm<
        ElementOutput,
        ArchTag,
        cutlass::gemm::GemmShape<32, 128, 64>,
        cutlass::gemm::GemmShape<32, 64, 64>,
        InstructionShape,
        2>(out, mat_a, mat_b, scales_a, scales_b, bias);
  } else if (m <= 64) {
    cutlass_int8_scaled_mm<
        ElementOutput,
        ArchTag,
        cutlass::gemm::GemmShape<64, 128, 128>,
        cutlass::gemm::GemmShape<64, 64, 64>,
        InstructionShape,
        2>(out, mat_a, mat_b, scales_a, scales_b, bias);
  } else if (m <= 256) {
    cutlass_int8_scaled_mm<
        ElementOutput,
        ArchTag,
        cutlass::gemm::GemmShape<128, 128, 128>,
        cutlass::gemm::GemmShape<64, 64, 64>,
        InstructionShape,
        2>(out, mat_a, mat_b, scales_a, scales_b, bias);
  } else {
    cutlass_int8_scaled_mm<
        ElementOutput,
        ArchTag,
        cutlass::gemm::GemmShape<128, 128, 64>,
        cutlass::gemm::GemmShape<64, 64, 64>,
        InstructionShape,
        2>(out, mat_a, mat_b, scales_a, scales_b, bias);
  }
}

template <typename ElementOutput, typename ArchTag, typename InstructionShape>
void sm80_dispatch_shape(
    torch::Tensor& out,
    const torch::Tensor& mat_a,
    const torch::Tensor& mat_b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  int m = mat_a.size(0);
  int n = mat_b.size(1);
  if (m <= 16) {
    if (n <= 4096) {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<16, 64, 128>,
          cutlass::gemm::GemmShape<16, 64, 64>,
          InstructionShape,
          6>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<16, 64, 128>,
          cutlass::gemm::GemmShape<16, 64, 64>,
          InstructionShape,
          5>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
  } else if (m <= 32) {
    if (n <= 4096) {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<32, 64, 128>,
          cutlass::gemm::GemmShape<32, 64, 64>,
          InstructionShape,
          6>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<32, 64, 128>,
          cutlass::gemm::GemmShape<32, 64, 64>,
          InstructionShape,
          5>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
  } else if (m <= 64) {
    if (n <= 4096) {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<64, 64, 128>,
          cutlass::gemm::GemmShape<32, 64, 64>,
          InstructionShape,
          5>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<64, 128, 128>,
          cutlass::gemm::GemmShape<64, 64, 64>,
          InstructionShape,
          5>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
  } else if (m <= 128 && n < 8192) {
    cutlass_int8_scaled_mm<
        ElementOutput,
        ArchTag,
        cutlass::gemm::GemmShape<64, 128, 128>,
        cutlass::gemm::GemmShape<64, 64, 64>,
        InstructionShape,
        5>(out, mat_a, mat_b, scales_a, scales_b, bias);
  } else {
    cutlass_int8_scaled_mm<
        ElementOutput,
        ArchTag,
        cutlass::gemm::GemmShape<128, 128, 64>,
        cutlass::gemm::GemmShape<64, 64, 64>,
        InstructionShape,
        5>(out, mat_a, mat_b, scales_a, scales_b, bias);
  }
}

// Dispatch shape for sm89 (L40S, L20, RTX 4090), according to:
// https://github.com/vllm-project/vllm/blob/main/csrc/quantization/cutlass_w8a8/scaled_mm_c2x_sm89_int8_dispatch.cuh
template <typename ElementOutput, typename ArchTag, typename InstructionShape>
void sm89_dispatch_shape(
    torch::Tensor& out,
    const torch::Tensor& mat_a,
    const torch::Tensor& mat_b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  int m = mat_a.size(0);
  int n = mat_b.size(1);
  if (m <= 16) {
    if (n <= 8192) {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<16, 64, 128>,
          cutlass::gemm::GemmShape<16, 64, 64>,
          InstructionShape,
          5>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<16, 128, 128>,
          cutlass::gemm::GemmShape<16, 64, 64>,
          InstructionShape,
          4>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
  } else if (m <= 32) {
    if (n <= 8192) {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<32, 64, 128>,
          cutlass::gemm::GemmShape<16, 64, 64>,
          InstructionShape,
          5>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<32, 128, 128>,
          cutlass::gemm::GemmShape<32, 64, 64>,
          InstructionShape,
          4>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
  } else if (m <= 64) {
    if (n <= 8192) {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<64, 64, 128>,
          cutlass::gemm::GemmShape<32, 64, 64>,
          InstructionShape,
          5>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<64, 128, 128>,
          cutlass::gemm::GemmShape<64, 64, 64>,
          InstructionShape,
          3>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
  } else if (m <= 128) {
    if (n <= 8192) {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<64, 128, 128>,
          cutlass::gemm::GemmShape<32, 64, 64>,
          InstructionShape,
          3>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else if (n <= 16384) {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<128, 128, 64>,
          cutlass::gemm::GemmShape<64, 64, 64>,
          InstructionShape,
          5>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<64, 64, 128>,
          cutlass::gemm::GemmShape<32, 64, 64>,
          InstructionShape,
          5>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
  } else if (m <= 256) {
    if (n <= 4096) {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<64, 128, 128>,
          cutlass::gemm::GemmShape<64, 64, 64>,
          InstructionShape,
          3>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else if (n <= 8192) {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<128, 128, 64>,
          cutlass::gemm::GemmShape<64, 64, 64>,
          InstructionShape,
          5>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else if (n <= 16384) {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<256, 128, 64>,
          cutlass::gemm::GemmShape<64, 64, 64>,
          InstructionShape,
          3>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      cutlass_int8_scaled_mm<
          ElementOutput,
          ArchTag,
          cutlass::gemm::GemmShape<128, 128, 64>,
          cutlass::gemm::GemmShape<64, 64, 64>,
          InstructionShape,
          5>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
  } else {
    cutlass_int8_scaled_mm<
        ElementOutput,
        ArchTag,
        cutlass::gemm::GemmShape<32, 64, 128>,
        cutlass::gemm::GemmShape<16, 64, 64>,
        InstructionShape,
        5>(out, mat_a, mat_b, scales_a, scales_b, bias);
  }
}

template <
    typename ElementOutput,
    typename TileShape,
    typename ClusterShape,
    typename MainloopScheduleType,
    bool WithBias>
void cutlass_int8_scaled_mm_sm90(
    torch::Tensor& out,
    const torch::Tensor& mat_a,
    const torch::Tensor& mat_b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  using ArchTag = cutlass::arch::Sm90;

  using ElementAccumulator = int32_t;
  using ElementCompute = float;
  using ElementInputA = int8_t;
  using ElementInputB = int8_t;

  static constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementInputA>::value;
  static constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementInputB>::value;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementOutput>::value;
  static constexpr int AlignmentOutput = 128 / cutlass::sizeof_bits<ElementOutput>::value;

  using OperatorClass = cutlass::arch::OpClassTensorOp;

  using EpilogueScheduleType = cutlass::epilogue::TmaWarpSpecialized;
  using TileSchedulerType = cutlass::gemm::PersistentScheduler;

  using XScale = cutlass::epilogue::fusion::
      Sm90ColBroadcast<0, TileShape, ElementCompute, ElementCompute, Stride<Int<1>, Int<0>, Int<0>>>;

  using WScale = cutlass::epilogue::fusion::
      Sm90RowBroadcast<0, TileShape, ElementCompute, ElementCompute, Stride<Int<0>, Int<1>, Int<0>>>;

  using Bias = cutlass::epilogue::fusion::
      Sm90RowBroadcast<0, TileShape, ElementOutput, ElementOutput, Stride<Int<0>, Int<1>, Int<0>>>;

  using Accum = cutlass::epilogue::fusion::Sm90AccFetch;

  // Scale
  using Compute0 = cutlass::epilogue::fusion::
      Sm90Compute<cutlass::multiplies, ElementCompute, ElementCompute, cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute0 = cutlass::epilogue::fusion::Sm90EVT<Compute0, WScale, Accum>;

  using Compute1 = cutlass::epilogue::fusion::
      Sm90Compute<cutlass::multiplies, ElementOutput, ElementCompute, cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute1 = cutlass::epilogue::fusion::Sm90EVT<Compute1, XScale, EVTCompute0>;

  // With bias
  using ComputeWithBias = cutlass::epilogue::fusion::
      Sm90Compute<cutlass::multiply_add, ElementOutput, ElementCompute, cutlass::FloatRoundStyle::round_to_nearest>;
  using EVTComputeWithBias = cutlass::epilogue::fusion::Sm90EVT<ComputeWithBias, XScale, EVTCompute0, Bias>;

  using EpilogueEVT = typename cutlass::platform::conditional<WithBias, EVTComputeWithBias, EVTCompute1>::type;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      ArchTag,
      OperatorClass,
      TileShape,
      ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator,
      ElementCompute,
      ElementOutput,
      cutlass::layout::RowMajor,
      AlignmentC,
      ElementOutput,
      cutlass::layout::RowMajor,
      AlignmentOutput,
      EpilogueScheduleType,
      EpilogueEVT>::CollectiveOp;

  using Stages = cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
      sizeof(typename CollectiveEpilogue::SharedStorage))>;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      ArchTag,
      OperatorClass,
      ElementInputA,
      cutlass::layout::RowMajor,
      AlignmentA,
      ElementInputB,
      cutlass::layout::ColumnMajor,
      AlignmentB,
      ElementAccumulator,
      TileShape,
      ClusterShape,
      Stages,
      MainloopScheduleType>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>,  // Indicates ProblemShape
      CollectiveMainloop,
      CollectiveEpilogue,
      TileSchedulerType>;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  Gemm gemm_op;

  int m = mat_a.size(0);
  int k = mat_a.size(1);
  int n = mat_b.size(1);

  auto a_ptr = static_cast<ElementInputA*>(mat_a.data_ptr());
  auto b_ptr = static_cast<ElementInputB*>(mat_b.data_ptr());
  auto o_ptr = static_cast<ElementOutput*>(out.data_ptr());

  auto a_s_ptr = static_cast<ElementCompute*>(scales_a.data_ptr());
  auto b_s_ptr = static_cast<ElementCompute*>(scales_b.data_ptr());

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using StrideD = typename Gemm::GemmKernel::StrideD;

  StrideA stride_a = cutlass::make_cute_packed_stride(StrideA{}, make_shape(m, k, 1));
  StrideB stride_b = cutlass::make_cute_packed_stride(StrideB{}, make_shape(n, k, 1));
  StrideC stride_c;
  StrideD stride_d = cutlass::make_cute_packed_stride(StrideD{}, make_shape(m, n, 1));

  typename Gemm::Arguments args = {
      cutlass::gemm::GemmUniversalMode::kGemm,
      {m, n, k, 1},
      {a_ptr, stride_a, b_ptr, stride_b},
      {{},  // epilogue.thread
       nullptr,
       stride_c,
       o_ptr,
       stride_d}};

  if constexpr (WithBias) {
    ElementOutput* bias_ptr = static_cast<ElementOutput*>(bias->data_ptr());
    args.epilogue.thread = {
        {a_s_ptr},
        {{b_s_ptr}, {}, {}},
        {bias_ptr},
        {},
    };
  } else {
    args.epilogue.thread = {
        {a_s_ptr},
        {{b_s_ptr}, {}, {}},
        {},
    };
  }

  auto workspace = torch::empty(
      gemm_op.get_workspace_size(args), torch::TensorOptions().dtype(torch::kUInt8).device(mat_a.device()));

  auto stream = at::cuda::getCurrentCUDAStream(mat_a.get_device());

  auto can_implement = gemm_op.can_implement(args);
  TORCH_CHECK(
      can_implement == cutlass::Status::kSuccess,
      "gemm cannot implement, error: ",
      cutlassGetStatusString(can_implement));

  auto status = gemm_op(args, workspace.data_ptr(), stream);
  TORCH_CHECK(status == cutlass::Status::kSuccess, "gemm executioin failed, error: ", cutlassGetStatusString(status));
}

template <typename ElementOutput, typename TileShape, typename ClusterShape, typename MainloopScheduleType>
void sm90_dispatch_bias(
    torch::Tensor& out,
    const torch::Tensor& mat_a,
    const torch::Tensor& mat_b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  if (bias) {
    cutlass_int8_scaled_mm_sm90<ElementOutput, TileShape, ClusterShape, MainloopScheduleType, true>(
        out, mat_a, mat_b, scales_a, scales_b, bias);
  } else {
    cutlass_int8_scaled_mm_sm90<ElementOutput, TileShape, ClusterShape, MainloopScheduleType, false>(
        out, mat_a, mat_b, scales_a, scales_b, bias);
  }
}

template <typename ElementOutput>
void sm90_dispatch_shape(
    torch::Tensor& out,
    const torch::Tensor& mat_a,
    const torch::Tensor& mat_b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  int m = mat_a.size(0);
  int n = mat_b.size(1);
  if (m <= 32) {
    if (n < 8192) {
      return sm90_dispatch_bias<
          ElementOutput,
          Shape<_64, _64, _128>,
          Shape<_1, _8, _1>,
          cutlass::gemm::KernelTmaWarpSpecialized>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      return sm90_dispatch_bias<
          ElementOutput,
          Shape<_64, _128, _128>,
          Shape<_1, _8, _1>,
          cutlass::gemm::KernelTmaWarpSpecialized>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
  } else if (m <= 64) {
    if (n < 8192) {
      return sm90_dispatch_bias<
          ElementOutput,
          Shape<_64, _64, _128>,
          Shape<_1, _4, _1>,
          cutlass::gemm::KernelTmaWarpSpecialized>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      return sm90_dispatch_bias<
          ElementOutput,
          Shape<_64, _64, _256>,
          Shape<_1, _1, _1>,
          cutlass::gemm::KernelTmaWarpSpecialized>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
  } else if (m <= 128) {
    if (n <= 4096) {
      return sm90_dispatch_bias<
          ElementOutput,
          Shape<_64, _64, _128>,
          Shape<_2, _1, _1>,
          cutlass::gemm::KernelTmaWarpSpecialized>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      return sm90_dispatch_bias<
          ElementOutput,
          Shape<_64, _128, _128>,
          Shape<_2, _1, _1>,
          cutlass::gemm::KernelTmaWarpSpecialized>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
  } else {
    return sm90_dispatch_bias<
        ElementOutput,
        Shape<_128, _128, _128>,
        Shape<_2, _1, _1>,
        cutlass::gemm::KernelTmaWarpSpecializedPingpong>(out, mat_a, mat_b, scales_a, scales_b, bias);
  }
}

torch::Tensor int8_scaled_mm(
    const torch::Tensor& mat_a,
    const torch::Tensor& mat_b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const torch::Dtype& out_dtype,
    const c10::optional<torch::Tensor>& bias) {
  TORCH_CHECK(mat_a.is_cuda(), "mat_a must be a CUDA tensor");
  TORCH_CHECK(mat_b.is_cuda(), "mat_b must be a CUDA tensor");
  TORCH_CHECK(mat_a.dim() == 2, "mat_a must be a 2D tensor");
  TORCH_CHECK(mat_b.dim() == 2, "mat_b must be a 2D tensor");
  TORCH_CHECK(mat_a.stride(1) == 1, "mat_a must be a row major tensor");
  TORCH_CHECK(mat_b.stride(0) == 1, "mat_a must be a column major tensor");
  TORCH_CHECK(mat_a.size(1) == mat_b.size(0), "mat_a and mat_b shapes cannot be multiplied");
  TORCH_CHECK(mat_a.size(1) % 16 == 0, "mat_a.size(1) must be multiple of 16 for memory alignment");
  TORCH_CHECK(mat_b.size(0) % 16 == 0, "mat_b.size(0) must be multiple of 16 for memory alignment");
  TORCH_CHECK(mat_b.size(1) % 8 == 0, "mat_b.size(1) must be multiple of 8 for memory alignment");  // out.stride(0)
  TORCH_CHECK(mat_a.scalar_type() == torch::kInt8, "mat_a must be Int8");
  TORCH_CHECK(mat_b.scalar_type() == torch::kInt8, "mat_b must be Int8");
  TORCH_CHECK(out_dtype == torch::kHalf || out_dtype == torch::kBFloat16, "out_dtype must be Half or BFloat16");

  TORCH_CHECK(scales_a.numel() == mat_a.size(0), "size of scales_a is not matched");
  TORCH_CHECK(scales_b.numel() == mat_b.size(1), "size of scales_b is not matched");
  TORCH_CHECK(scales_a.is_contiguous(), "scales_a must be contiguous");
  TORCH_CHECK(scales_b.is_contiguous(), "scales_b msut be contiguous");
  TORCH_CHECK(scales_a.scalar_type() == torch::kFloat32, "scales_a must be Float32");
  TORCH_CHECK(scales_b.scalar_type() == torch::kFloat32, "scales_b must be Float32");

  if (bias) {
    TORCH_CHECK(bias->numel() == mat_b.size(1), "size of bias is not matched");
    TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
    TORCH_CHECK(bias->dtype() == out_dtype, "bias dtype must match output dtype");
  }

  torch::Tensor out = torch::empty({mat_a.size(0), mat_b.size(1)}, mat_a.options().dtype(out_dtype));

  auto sm_version = getSMVersion();

  if (sm_version >= 75 && sm_version < 80) {
    TORCH_CHECK(out_dtype == torch::kHalf, "out_dtype must be Half for SM75");
    sm75_dispatch_shape<cutlass::half_t, cutlass::arch::Sm75, cutlass::gemm::GemmShape<8, 8, 16>>(
        out, mat_a, mat_b, scales_a, scales_b, bias);
  } else if (sm_version >= 80 && sm_version < 90) {
    // sm86/sm89 has a much smaller shared memory size (100K) than sm80 (160K)
    if (sm_version == 86 || sm_version == 89) {
      if (out_dtype == torch::kBFloat16) {
        sm89_dispatch_shape<cutlass::bfloat16_t, cutlass::arch::Sm80, cutlass::gemm::GemmShape<16, 8, 32>>(
            out, mat_a, mat_b, scales_a, scales_b, bias);
      } else {
        sm89_dispatch_shape<cutlass::half_t, cutlass::arch::Sm80, cutlass::gemm::GemmShape<16, 8, 32>>(
            out, mat_a, mat_b, scales_a, scales_b, bias);
      }
    } else {
      if (out_dtype == torch::kBFloat16) {
        sm80_dispatch_shape<cutlass::bfloat16_t, cutlass::arch::Sm80, cutlass::gemm::GemmShape<16, 8, 32>>(
            out, mat_a, mat_b, scales_a, scales_b, bias);
      } else {
        sm80_dispatch_shape<cutlass::half_t, cutlass::arch::Sm80, cutlass::gemm::GemmShape<16, 8, 32>>(
            out, mat_a, mat_b, scales_a, scales_b, bias);
      }
    }
  } else if (sm_version == 90) {
#if defined CUDA_VERSION && CUDA_VERSION >= 12000
    // cutlass 3.x
    if (out_dtype == torch::kBFloat16) {
      sm90_dispatch_shape<cutlass::bfloat16_t>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      sm90_dispatch_shape<cutlass::half_t>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
#else
    // fallback to cutlass 2.x
    if (out_dtype == torch::kBFloat16) {
      sm80_dispatch_shape<cutlass::bfloat16_t, cutlass::arch::Sm80, cutlass::gemm::GemmShape<16, 8, 32>>(
          out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      sm80_dispatch_shape<cutlass::half_t, cutlass::arch::Sm80, cutlass::gemm::GemmShape<16, 8, 32>>(
          out, mat_a, mat_b, scales_a, scales_b, bias);
    }
#endif
  } else {
    TORCH_CHECK_NOT_IMPLEMENTED(false, "No implemented int8_scaled_mm for current compute capability.");
  }

  return out;
}
