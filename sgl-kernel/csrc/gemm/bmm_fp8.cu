/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/driver_types.h>

#include <flashinfer/gemm/bmm_fp8.cuh>

#include "pytorch_extension_utils.h"

void bmm_fp8(
    at::Tensor A,
    at::Tensor B,
    at::Tensor D,
    at::Tensor A_scale,
    at::Tensor B_scale,
    at::Tensor workspace_buffer,
    int64_t cublas_handle) {
  TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
  TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
  TORCH_CHECK(D.is_cuda(), "D must be a CUDA tensor");
  TORCH_CHECK(A.dim() == 3, "Expected 3D tensor for A");
  TORCH_CHECK(B.dim() == 3, "Expected 3D tensor for B");
  TORCH_CHECK(D.dim() == 3, "Expected 3D tensor for D");
  TORCH_CHECK(A.size(0) == B.size(0) && A.size(0) == D.size(0), "Batch sizes must match");
  TORCH_CHECK(A.size(2) == B.size(1), "Incompatible matrix sizes");
  TORCH_CHECK(A.size(1) == D.size(1) && B.size(2) == D.size(2), "Result tensor has incorrect shape");

  // PyTorch is row major by default. cuBLASLt is column major by default.
  // We need row major D as expected.
  // A ^ T * B = D, so D ^ T = B ^ T * A
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(B.scalar_type(), b_type, [&] {
    return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(A.scalar_type(), a_type, [&] {
      return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(D.scalar_type(), d_type, [&] {
        auto batch_size = A.size(0);
        auto m = A.size(1);
        auto k = A.size(2);
        auto n = B.size(2);

        auto lt_handle = reinterpret_cast<hipblasLtHandle_t>(cublas_handle);
        const c10::cuda::OptionalCUDAGuard device_guard(A.device());
        auto stream = at::cuda::getCurrentCUDAStream();

        auto status = flashinfer::bmm_fp8::bmm_fp8_internal_cublaslt(
            workspace_buffer.data_ptr(),
            workspace_buffer.numel(),
            static_cast<b_type*>(B.data_ptr()),
            static_cast<a_type*>(A.data_ptr()),
            static_cast<d_type*>(D.data_ptr()),
            batch_size,
            n,
            m,
            k,
            static_cast<float*>(B_scale.data_ptr()),
            static_cast<float*>(A_scale.data_ptr()),
            lt_handle,
            stream);
        TORCH_CHECK(
            status == HIPBLAS_STATUS_SUCCESS, "bmm_fp8_internal_cublaslt failed: ", cublasGetStatusString(status));
        return true;
      });
    });
  });
}
