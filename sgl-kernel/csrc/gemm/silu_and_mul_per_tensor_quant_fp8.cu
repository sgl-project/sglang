#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <c10/util/Float8_e4m3fn.h>

#include <cmath>
#include <cub/block/block_reduce.cuh>
#include <flashinfer/vec_dtypes.cuh>

#include "utils.h"

__device__ __forceinline__ float silu(const float& val) {
  return val / (1.0f + __expf(-val));
}

template <typename T>
__global__ void
silu_and_mul_per_tensor_absmax_kernel(T* __restrict__ input, const T* __restrict__ input_2, float* __restrict__ output_s, const int64_t num_elements, const int hidden_dim) {
  float max_value = 0.0f;
  unsigned int tid = threadIdx.x;
  unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = blockDim.x * gridDim.x;

  constexpr uint32_t vec_size = 16 / sizeof(T);
  using vec_t = flashinfer::vec_t<T, vec_size>;

  const int32_t num_vec_elems = num_elements / vec_size;

  for (int32_t i = gid; i < num_vec_elems; i += grid_size) {
    vec_t input_vec;
    vec_t input_vec_2;
    input_vec.cast_load(input + i * vec_size);
    input_vec_2.cast_load(input + i * vec_size + hidden_dim);

#pragma unroll
    for (uint32_t j = 0; j < vec_size; ++j) {
      input_vec[j] = silu(static_cast<float>(input_vec[j])) * static_cast<float>(input_vec_2[j]);
      float val = static_cast<float>(input_vec[j]);
      max_value = fmaxf(max_value, fabsf(val));
    }
    input_vec.cast_store(input + i * vec_size);
  }

  const int32_t remaining_start = num_vec_elems * vec_size;
  for (int32_t idx = remaining_start + gid; idx < num_elements; idx += grid_size) {
    float val = silu(static_cast<float>(input[idx])) * static_cast<float>(input_2[idx]);
    max_value = fmaxf(max_value, fabsf(static_cast<float>(val)));
    input[idx] = static_cast<T>(val);
  }

  max_value = blockReduceMax(max_value);

  if (tid == 0) {
    atomicMaxFloat(output_s, max_value / FP8_E4M3_MAX);
  }
}

template <typename T, typename DST_DTYPE>
__global__ void per_tensor_quant_fp8_kernel(
    const T* __restrict__ input,
    DST_DTYPE* __restrict__ output,
    const float* __restrict__ scale,
    const int64_t num_elements) {
  const int gid = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = blockDim.x * gridDim.x;
  const float scale_val = 1.0f / (*scale);

  // We want to store 128 bits of data at a time. 16 = 128 / 8 bits
  // Load is already vectorized, so 16 elements work for T.
  const uint32_t VEC_SIZE = 16;
  using vec_t = flashinfer::vec_t<T, VEC_SIZE>;

  const int32_t num_vec_elems = num_elements / VEC_SIZE;

  for (int32_t i = gid; i < num_vec_elems; i += grid_size) {
    vec_t input_vec;
    input_vec.cast_load(input + i * VEC_SIZE);

    DST_DTYPE output_arr[VEC_SIZE];
#pragma unroll
    for (uint32_t j = 0; j < VEC_SIZE; ++j) {
      float val = fmax(fmin(static_cast<float>(input_vec[j]) * scale_val, FP8_E4M3_MAX), -FP8_E4M3_MAX);
#ifndef USE_ROCM
      output_arr[j] = static_cast<DST_DTYPE>(val);
#else
      output_arr[j] = c10::Float8_e4m3fnuz(
          __hip_cvt_float_to_fp8(val, fp8::fp8_type::__default_saturation, fp8::fp8_type::__default_interpret),
          c10::Float8_e4m3fnuz::from_bits());
#endif
    }
    *(uint4*)(output + i * VEC_SIZE) = *(uint4*)output_arr;
  }

  const int32_t remaining_start = num_vec_elems * VEC_SIZE;
  for (int32_t idx = remaining_start + gid; idx < num_elements; idx += grid_size) {
    float val = fmax(-FP8_E4M3_MAX, fmin(static_cast<float>(input[idx]) * scale_val, FP8_E4M3_MAX));
#ifndef USE_ROCM
    output[idx] = static_cast<DST_DTYPE>(val);
#else
    output[idx] = c10::Float8_e4m3fnuz(
        __hip_cvt_float_to_fp8(val, fp8::fp8_type::__default_saturation, fp8::fp8_type::__default_interpret),
        c10::Float8_e4m3fnuz::from_bits());
#endif
  }
}

void sgl_silu_and_mul_per_tensor_quant_fp8(torch::Tensor input_gate, torch::Tensor input_up, torch::Tensor output_q, torch::Tensor output_s, bool is_static) {
  CHECK_INPUT(input_gate);
  CHECK_INPUT(input_up);
  CHECK_INPUT(output_q);
  CHECK_INPUT(output_s);
  TORCH_CHECK(is_static == false, "Static mode is not supported for silu_and_mul_per_tensor_quant_fp8");

  const int block_size = 256;
  const int num_elements = input.numel();
  const int num_blocks = min((num_elements + block_size - 1) / block_size, 1024);
  const int hidden_dim = input.size(-1);

  dim3 grid(num_blocks);
  dim3 block(block_size);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), scalar_t, [&] {

    silu_and_mul_per_tensor_absmax_kernel<scalar_t><<<grid, block, 0, stream>>>(
        static_cast<scalar_t*>(input_gate.data_ptr()), static_cast<scalar_t*>(input_up.data_ptr()) static_cast<float*>(output_s.data_ptr()), num_elements, hidden_dim);

    silu_and_mul_per_tensor_quant_fp8_kernel<scalar_t, __hip_fp8_e4m3_fnuz><<<grid, block, 0, stream>>>(
        static_cast<scalar_t*>(input_gate.data_ptr()),
        static_cast<__hip_fp8_e4m3_fnuz*>(output_q.data_ptr()),
        static_cast<float*>(output_s.data_ptr()),
        num_elements, hidden_dim);
    return true;
  });
}
