#include "hip/hip_runtime.h"
// Implemented by Haotian Tang and Shang Yang.
// @article{lin2024qserve,
//   title={QServe: W4A8KV4 Quantization and System Co-design for Efficient LLM Serving},
//   author={Lin*, Yujun and Tang*, Haotian and Yang*, Shang and Zhang, Zhekai and Xiao, Guangxuan and Gan, Chuang and
//   Han, Song}, journal={arXiv preprint arXiv:2405.04532}, year={2024}
// }
// @article{yang2025lserve,
//   title={LServe: Efficient Long-sequence LLM Serving with Unified Sparse Attention},
//   author={Yang*, Shang and Guo*, Junxian and Tang, Haotian and Hu, Qinghao and Xiao, Guangxuan and Tang, Jiaming and
//   Lin, Yujun and Liu, Zhijian and Lu, Yao and Han, Song}, year={2025}
// }

// Adapted from https://github.com/mit-han-lab/omniserve/blob/main/kernels/csrc/qgemm/w4a8_per_chn/gemm_cuda.cu

#include <ATen/cuda/HIPContext.h>
#include <hip/hip_fp16.h>
#include <cuda_pipeline_primitives.h>
#include <torch/all.h>

#include "utils.h"

#define OP_M 16
#define OP_N 8
#define OP_K 32
#define INTRIN_M 16
#define INTRIN_N 16
#define INTRIN_K 32
#define WARP_SIZE 32
#define SMEM_PAD_A 0
#define SMEM_PAD_B 0
#define PACK_SIZE 16
#if (__CUDACC_VER_MAJOR__ >= 11) && (__CUDACC_VER_MINOR__ >= 4)
#define L2_CACHEHINT(size) ".L2::" #size "B"
#else
#define L2_CACHEHINT(size)
#endif

#define KERNEL_LAUNCH_CODE                                                                                   \
  constexpr int NUM_WARPS = (CTA_M / WARP_M) * (CTA_N / WARP_N) * (CTA_K / WARP_K);                          \
  constexpr int SCALES_SMEM_SIZE = (G >= CTA_K) ? (CTA_N * STAGES * 2) : (CTA_N * (CTA_K / G) * STAGES * 2); \
  constexpr int kSmemByteSize =                                                                              \
      ((CTA_M * (CTA_K + SMEM_PAD_A) + CTA_N * (CTA_K + SMEM_PAD_B) / 2) * STAGES + SCALES_SMEM_SIZE) *      \
      sizeof(int8_t);                                                                                        \
  if (kSmemByteSize >= 99 * 1024) {                                                                          \
    printf(                                                                                                  \
        "This kernel requires %d Bytes of shared memory, which exceeds "                                     \
        "device limit.\n",                                                                                   \
        kSmemByteSize);                                                                                      \
    return;                                                                                                  \
  }                                                                                                          \
  int num_blocks_m = (num_out_feats + CTA_M - 1) / CTA_M;                                                    \
  int num_blocks_n = num_out_channels / CTA_N / 1;                                                           \
  const int log_tile = get_log_tile<8>((num_out_feats + CTA_M - 1) / CTA_M);                                 \
  const int tile_shift = 1 << log_tile;                                                                      \
  dim3 num_blocks(num_blocks_n* tile_shift, (num_blocks_m + tile_shift - 1) / tile_shift);                   \
  dim3 threads_per_block(WARP_SIZE, NUM_WARPS);                                                              \
  auto kernel_func = dense_kernel0<CTA_M, CTA_N, CTA_K, WARP_M, WARP_N, WARP_K, STAGES, G>;                  \
  hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_func), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemByteSize);             \
  kernel_func<<<num_blocks, threads_per_block, kSmemByteSize, stream>>>(                                     \
      in_feats, kernel, wscales, ascales, w_szs, a_ssums, out_feats, num_in_feats, num_out_channels, num_in_channels);

template <int N>
__inline__ __host__ __device__ int get_log_tile(int n) {
  if (N >= 8 && n >= 6)
    return 3;
  else if (N >= 4 && n >= 3)
    return 2;
  else if (N >= 2 && n >= 2)
    return 1;
  else
    return 0;
}

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
__inline__ __device__ uint2 get_block_idx_mapping(int blockIdx_x, int blockIdx_y, int log_tile) {
  return make_uint2((blockIdx_x >> log_tile), (blockIdx_y << log_tile) + ((blockIdx_x) & ((1 << (log_tile)) - 1)));
}

__inline__ __device__ uint32_t cast_smem_ptr_to_uint(void const* const ptr) {
  uint32_t smem_int_ptr;

  asm("{.reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 %0, "
      "smem_ptr; }\n"
      : "=r"(smem_int_ptr)
      : "l"(ptr));

  return smem_int_ptr;
}

__inline__ __device__ void ldmatrix_m8n8_x4_b16(int8_t* shared_warp, int ax0_0, uint32_t addr) {
  __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(((unsigned*)(shared_warp + (ax0_0 * 16)))[0]),
        "=r"(((unsigned*)(shared_warp + (ax0_0 * 16)))[1]),
        "=r"(((unsigned*)(shared_warp + (ax0_0 * 16)))[2]),
        "=r"(((unsigned*)(shared_warp + (ax0_0 * 16)))[3])
      : "r"(addr));
}

__inline__ __device__ void ldmatrix_m8n8_x4_trans_b16(int8_t* shared_warp, int ax0_0, uint32_t addr) {
  __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16"
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(((unsigned*)(shared_warp + (ax0_0 * 16)))[0]),
        "=r"(((unsigned*)(shared_warp + (ax0_0 * 16)))[1]),
        "=r"(((unsigned*)(shared_warp + (ax0_0 * 16)))[2]),
        "=r"(((unsigned*)(shared_warp + (ax0_0 * 16)))[3])
      : "r"(addr));
}

// function from lmdeploy
__inline__ __device__ void cp_async_cg_A(uint32_t smem_int_ptr, const uint4* __restrict__ src, bool mask) {
  const int cp_size = 16;
  asm volatile("{"
                "  .reg .pred p;"
                "  setp.ne.b32 p, %0, 0;"
                "  @p cp.async.cg.shared.global" L2_CACHEHINT(128) " [%1], [%2], %3;"
                "}" ::"r"((int)mask),
                "r"(smem_int_ptr),
                "l"(src),
                "n"(cp_size));
}

__device__ __inline__ void mma_m16n8k32(void* C_warp, void* A_shared_warp, void* B_shared_warp) {
  __asm__ __volatile__(
      "mma.sync.aligned.m16n8k32.row.col.s32.s8.s8.s32"
      "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};"
      : "=r"(((int*)C_warp)[0]), "=r"(((int*)C_warp)[1]), "=r"(((int*)C_warp)[2]), "=r"(((int*)C_warp)[3])
      : "r"(((unsigned*)A_shared_warp)[0]),
        "r"(((unsigned*)A_shared_warp)[1]),
        "r"(((unsigned*)A_shared_warp)[2]),
        "r"(((unsigned*)A_shared_warp)[3]),
        "r"(((unsigned*)B_shared_warp)[0]),
        "r"(((unsigned*)B_shared_warp)[1]),
        "r"(((int*)C_warp)[0]),
        "r"(((int*)C_warp)[1]),
        "r"(((int*)C_warp)[2]),
        "r"(((int*)C_warp)[3]));
}

template <int CTA_M, int CTA_N, int CTA_K, int CTA_SIZE, int SHARED_K_ITERS, int STAGES>
__device__ __inline__ void global_to_share_one_stage_A(
    int8_t* src,
    int8_t* dst,
    int global_ncols,
    int cta_offset_m,
    int cta_offset_n,
    int global_iter_k,
    int shared_iter_k,
    bool mask,
    bool* preds) {
  constexpr int total_global_iters = (CTA_M * CTA_K) / PACK_SIZE / CTA_SIZE;
  constexpr int partial_global_iters = total_global_iters / SHARED_K_ITERS;
  constexpr int cta_step_m_or_n = (CTA_SIZE * PACK_SIZE) / CTA_K;
  constexpr int warp_step_m_or_n = (WARP_SIZE * PACK_SIZE) / CTA_K;
  constexpr int threads_per_row = CTA_K / PACK_SIZE;
  constexpr int kSmemCol = CTA_K + SMEM_PAD_A;
  int8_t* dst_hoisted = dst;
  int8_t* src_hoisted = src + global_iter_k * CTA_K;

  if (mask) {
#pragma unroll
    for (int _global_iter = 0; _global_iter < partial_global_iters; ++_global_iter) {
      int global_iter = shared_iter_k * partial_global_iters + _global_iter;

      void* dst_ptr = (void*)(dst_hoisted + global_iter * cta_step_m_or_n * kSmemCol);
      uint4* src_ptr = (uint4*)(src_hoisted + global_iter * cta_step_m_or_n * global_ncols);
      // *dst_ptr = *src_ptr;
      if constexpr (STAGES > 1) {
        uint32_t addr = cast_smem_ptr_to_uint(dst_ptr);
        cp_async_cg_A(addr, src_ptr, preds[global_iter]);
      } else {
        if (preds[global_iter]) *(uint4*)dst_ptr = *src_ptr;
      }
    }
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int CTA_SIZE, int SHARED_K_ITERS, int STAGES>
__device__ __inline__ void global_to_share_one_stage_B(
    int8_t* src,
    int8_t* dst,
    int global_ncols,
    int cta_offset_m,
    int cta_offset_n,
    int global_iter_k,
    int shared_iter_k,
    bool mask) {
  constexpr int total_global_iters = (CTA_N * CTA_K) / 32 / CTA_SIZE;
  constexpr int NUM_WARPS = CTA_SIZE / WARP_SIZE;
  constexpr int warps_per_row = CTA_K / 32;
  constexpr int cta_step_m_or_n = NUM_WARPS / warps_per_row;
  constexpr int kSmemCol = CTA_K;
  int8_t* dst_hoisted = dst;
  int8_t* src_hoisted = src + global_iter_k * CTA_K * PACK_SIZE;

#pragma unroll
  for (int global_iter = 0; global_iter < total_global_iters; ++global_iter) {
    void* dst_ptr = (void*)(dst_hoisted + global_iter * cta_step_m_or_n * kSmemCol * PACK_SIZE);
    uint4* src_ptr = (uint4*)(src_hoisted + global_iter * cta_step_m_or_n * global_ncols * PACK_SIZE);
    if constexpr (STAGES > 1) {
      uint32_t addr = cast_smem_ptr_to_uint(dst_ptr);
      cp_async_cg_A(addr, src_ptr, mask);
    } else {
      if (mask) *(uint4*)dst_ptr = *src_ptr;
    }
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int CTA_SIZE, int STAGES, int G>
__device__ __inline__ void global_to_share_one_stage_zeros(
    int8_t* src,
    int8_t* dst,
    int global_ncols,
    int cta_offset_m,
    int cta_offset_n,
    int global_iter_k,
    int shared_iter_k,
    bool mask) {
  constexpr int threads_needed = CTA_N / PACK_SIZE / 1;
  constexpr int threads_used = threads_needed < CTA_SIZE ? threads_needed : CTA_SIZE;
  constexpr int total_global_iters = CTA_N / PACK_SIZE / threads_used;
  constexpr int threads_per_row = CTA_N / PACK_SIZE;
  constexpr int kSmemCol = CTA_N;
  bool local_mask = mask & (threadIdx.y * WARP_SIZE + threadIdx.x < threads_used);
  int g_idx = global_iter_k * CTA_K / G;

  void* dst_ptr = (void*)(dst + (threadIdx.x % threads_per_row) * PACK_SIZE);
  uint4* src_ptr = (uint4*)(src + g_idx * global_ncols + cta_offset_n + (threadIdx.x % threads_per_row) * PACK_SIZE);
  if (STAGES > 1) {
    uint32_t addr = cast_smem_ptr_to_uint(dst_ptr);
    cp_async_cg_A(addr, src_ptr, local_mask);
  } else {
    if (local_mask) {
      *(uint4*)dst_ptr = *src_ptr;
    }
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int CTA_SIZE, int STAGES>
__device__ __inline__ void
share_to_reg_one_stage_A(int8_t* src, int8_t* dst, int warp_offset_m, int warp_offset_n, int k_0_1, int shared_iters) {
  constexpr int kSmemCol = CTA_K + SMEM_PAD_A;
  int ld_col = (k_0_1 * INTRIN_K + (threadIdx.x / 16) * 16) / PACK_SIZE;

  for (int shared_iter = 0; shared_iter < shared_iters; ++shared_iter) {
    int ld_row = warp_offset_m + shared_iter * INTRIN_M + (threadIdx.x % 16);
    int ld_col_swizzled = ld_col ^ (ld_row / 2) & 3;
    void* addr_ptr = (void*)(src + ld_row * kSmemCol + ld_col_swizzled * PACK_SIZE);
    uint32_t addr = cast_smem_ptr_to_uint(addr_ptr);
    ldmatrix_m8n8_x4_b16(dst, shared_iter, addr);
  }
}

template <int WARP_K, int CTA_N, int CTA_K, int CTA_SIZE, int STAGES, int G>
__device__ __inline__ void share_to_reg_one_stage_B(
    int8_t* src,
    int8_t* dst,
    int8_t* zeros,
    int8_t* scales_i8,
    int warp_offset_m,
    int warp_offset_n,
    int k_0_0,
    int k_0_1,
    int shared_iters) {
  constexpr int kSmemCol = CTA_K + SMEM_PAD_B;
#pragma unroll
  for (int shared_iter = 0; shared_iter < shared_iters; ++shared_iter) {
    uint4 loaded =
        *((uint4*)(src) + warp_offset_n / 32 * kSmemCol + shared_iter * 32 / 32 * kSmemCol + k_0_1 * INTRIN_K +
          threadIdx.x);

    auto ptr = (uint32_t*)dst + shared_iter * 8;
    ptr[0] = loaded.x & 0x0F0F0F0F;
    ptr[4] = (loaded.x & 0xF0F0F0F0) >> 4;
    ptr[2] = loaded.y & 0x0F0F0F0F;
    ptr[6] = (loaded.y & 0xF0F0F0F0) >> 4;
    ptr[1] = loaded.z & 0x0F0F0F0F;
    ptr[5] = (loaded.z & 0xF0F0F0F0) >> 4;
    ptr[3] = loaded.w & 0x0F0F0F0F;
    ptr[7] = (loaded.w & 0xF0F0F0F0) >> 4;
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int WARP_M, int WARP_N, int WARP_K, int STAGES, int G>
__global__ void dense_kernel0(
    int8_t* __restrict__ A,
    int8_t* __restrict__ B,
    half2* __restrict__ wscales,
    half* __restrict__ ascales,
    half2* __restrict__ w_szs,
    half* __restrict__ a_ssums,
    half* __restrict__ C,
    int M,
    int64_t N,
    int64_t K) {
  constexpr int SPLITK = 1;
  constexpr int NUM_WARPS_MN = CTA_M / WARP_M * CTA_N / WARP_N;
  constexpr int NUM_WARPS = NUM_WARPS_MN * CTA_K / WARP_K;
  constexpr int CTA_SIZE = NUM_WARPS * WARP_SIZE;
  constexpr int CTA_SIZE_MN = NUM_WARPS_MN * WARP_SIZE;
  constexpr int SLICES = CTA_K / WARP_K;
  int num_blocks_n = (N + CTA_N - 1) / CTA_N;
  int num_blocks_m = (M + CTA_M - 1) / CTA_M;

  int blockIdx_n = blockIdx.x;
  int blockIdx_m = blockIdx.y;
  const int log_tile = get_log_tile<8>((M + CTA_M - 1) / CTA_M);
  const uint2 block_idx_mapping = get_block_idx_mapping(blockIdx_n, blockIdx_m, log_tile);
  blockIdx_n = block_idx_mapping.x;
  blockIdx_m = block_idx_mapping.y;

  int C_warp[CTA_M * CTA_N / CTA_SIZE_MN];
  constexpr int kSmemPadKA = CTA_K + SMEM_PAD_A;
  constexpr int kSmemPadKB = CTA_K + SMEM_PAD_B;
  constexpr int kSmemSizeAPerStage = CTA_M * kSmemPadKA;
  constexpr int kSmemSizeBPerStage = CTA_N * kSmemPadKB / 2;
  constexpr int kSmemSizeA = kSmemSizeAPerStage * STAGES;
  constexpr int kSmemSizeB = kSmemSizeBPerStage * STAGES;

  constexpr int scales_load_interval = G >= CTA_K ? G / CTA_K : 1;
  constexpr int scales_per_load = G < CTA_K ? CTA_K / G : 1;
  constexpr int kSmemSizeScales = CTA_N * STAGES;

  extern __shared__ int8_t mem_shared[];
  int8_t* A_shared = mem_shared;

  int8_t* B_shared = mem_shared + kSmemSizeA;
  int8_t* zeros_shared = mem_shared + kSmemSizeA + kSmemSizeB;
  int8_t* scales_i8_shared = mem_shared + kSmemSizeA + kSmemSizeB + kSmemSizeScales;

  int8_t A_shared_warp_[2][WARP_M * WARP_K / WARP_SIZE];
  int8_t B_shared_warp_[2][WARP_N * WARP_K / WARP_SIZE];
  constexpr int A_total_global_iters = (CTA_M * CTA_K) / PACK_SIZE / CTA_SIZE;
  constexpr int B_total_global_iters = (CTA_N * CTA_K) / PACK_SIZE / CTA_SIZE;
  constexpr int A_src_step_m = (CTA_SIZE * PACK_SIZE) / CTA_K;
  constexpr int A_warp_step_m = (WARP_SIZE * PACK_SIZE) / CTA_K;
  constexpr int A_threads_per_row = CTA_K / PACK_SIZE;

  constexpr int B_warps_per_row = CTA_K / 32;
  constexpr int B_src_step_n = NUM_WARPS / B_warps_per_row;

  int cta_offset_m = blockIdx_m * CTA_M;
  int cta_offset_n = blockIdx_n * CTA_N;
  int warp_mn = threadIdx.y % NUM_WARPS_MN;
  int slice_id = threadIdx.y / NUM_WARPS_MN;
  int warp_offset_m = (warp_mn % (CTA_M / WARP_M)) * WARP_M;
  int warp_offset_n = (warp_mn / (CTA_M / WARP_M)) * WARP_N;
  int warp_offset_k = slice_id * WARP_K;

  for (int i = 0; i < CTA_M * CTA_N / CTA_SIZE_MN; i++)
    C_warp[i] = 0;

  int gemm_iters = (K + CTA_K - 1) / CTA_K;
  int k_0_0_ld = 0;
  int k_0_0 = 0;
  constexpr int prologue_stages = STAGES == 1 ? 1 : STAGES - 1;
  int A_hoisted_row = threadIdx.y * A_warp_step_m + (threadIdx.x / A_threads_per_row);
  int A_hoisted_col = (threadIdx.x % A_threads_per_row);
  int A_hoisted_col_swizzled = A_hoisted_col ^ (A_hoisted_row / 2) & 3;

  int8_t* A_shared_hoisted = A_shared + A_hoisted_row * kSmemPadKA + A_hoisted_col_swizzled * PACK_SIZE;
  int8_t* B_shared_hoisted = B_shared + (threadIdx.y % B_warps_per_row) * 32 * PACK_SIZE +
                             (threadIdx.y / B_warps_per_row) * kSmemPadKB * PACK_SIZE + threadIdx.x * PACK_SIZE;
  int8_t* A_hoisted = A + cta_offset_m * K + A_hoisted_row * K + A_hoisted_col * PACK_SIZE;
  int8_t* B_hoisted = B + cta_offset_n / 32 * K * PACK_SIZE + (threadIdx.y % B_warps_per_row) * 32 * PACK_SIZE +
                      (threadIdx.y / B_warps_per_row) * K * PACK_SIZE + threadIdx.x * PACK_SIZE;

  bool A_g2s_preds[A_total_global_iters];
#pragma unroll
  for (int i = 0; i < A_total_global_iters; i++) {
    A_g2s_preds[i] = (cta_offset_m + A_hoisted_row + i * A_src_step_m) < M;
  }

  int* C_shared = reinterpret_cast<int*>(mem_shared);

#pragma unroll
  for (k_0_0_ld = 0; k_0_0_ld < prologue_stages; ++k_0_0_ld) {
    global_to_share_one_stage_A<CTA_M, CTA_N, CTA_K, CTA_SIZE, 1, STAGES>(
        A_hoisted,
        A_shared_hoisted + k_0_0_ld * kSmemSizeAPerStage,
        K,
        cta_offset_m,
        cta_offset_n,
        k_0_0_ld,
        0,
        true,
        A_g2s_preds);
    global_to_share_one_stage_B<CTA_M, CTA_N, CTA_K, CTA_SIZE, 1, STAGES>(
        B_hoisted, B_shared_hoisted + k_0_0_ld * kSmemSizeBPerStage, K, cta_offset_m, cta_offset_n, k_0_0_ld, 0, true);

    if constexpr (STAGES > 1) __pipeline_commit();
  }
  if constexpr (STAGES > 1) __pipeline_wait_prior(STAGES - 2);
  __syncthreads();

  share_to_reg_one_stage_A<CTA_M, CTA_N, CTA_K, CTA_SIZE, STAGES>(
      A_shared + warp_offset_k, A_shared_warp_[0], warp_offset_m, warp_offset_n, 0, WARP_M / INTRIN_M);
  share_to_reg_one_stage_B<CTA_M, CTA_N, CTA_K, CTA_SIZE, STAGES, G>(
      B_shared + warp_offset_k * PACK_SIZE,
      B_shared_warp_[0],
      zeros_shared,
      scales_i8_shared,
      warp_offset_m,
      warp_offset_n,
      0,
      0,
      WARP_N / 32);
  constexpr int SHARED_K_ITERS = WARP_K / INTRIN_K;

  for (; k_0_0 < gemm_iters; ++k_0_0, ++k_0_0_ld) {
    int ld_stage = k_0_0_ld % STAGES;
    int compute_stage = k_0_0 % STAGES;
    int8_t* A_shared_this_compute_stage;
    int8_t* B_shared_this_compute_stage;
    int8_t* zeros_shared_this_compute_stage;
    int8_t* scales_i8_shared_this_compute_stage;

    for (int iter_k = 0; iter_k < SHARED_K_ITERS; ++iter_k) {
      A_shared_this_compute_stage = A_shared + compute_stage * kSmemSizeAPerStage + warp_offset_k;
      B_shared_this_compute_stage = B_shared + compute_stage * kSmemSizeBPerStage + warp_offset_k * PACK_SIZE;
      zeros_shared_this_compute_stage = zeros_shared + (compute_stage)*CTA_N;
      scales_i8_shared_this_compute_stage = scales_i8_shared + (compute_stage)*CTA_N;

      share_to_reg_one_stage_A<CTA_M, CTA_N, CTA_K, CTA_SIZE, STAGES>(
          A_shared_this_compute_stage,
          A_shared_warp_[(iter_k + 1) % 2],
          warp_offset_m,
          warp_offset_n,
          (iter_k + 1) % SHARED_K_ITERS,
          WARP_M / INTRIN_M);
      share_to_reg_one_stage_B<CTA_M, CTA_N, CTA_K, CTA_SIZE, STAGES, G>(
          B_shared_this_compute_stage,
          B_shared_warp_[(iter_k + 1) % 2],
          zeros_shared_this_compute_stage,
          scales_i8_shared_this_compute_stage,
          warp_offset_m,
          warp_offset_n,
          k_0_0 + (iter_k == SHARED_K_ITERS - 1),
          (iter_k + 1) % SHARED_K_ITERS,
          WARP_N / 32);
      int8_t* A_shared_warp = A_shared_warp_[iter_k % 2];
      int8_t* B_shared_warp = B_shared_warp_[iter_k % 2];

      for (int j_0_4 = 0; j_0_4 < WARP_N / INTRIN_N; ++j_0_4) {
        for (int i_0_3 = 0; i_0_3 < WARP_M / INTRIN_M; ++i_0_3) {
          mma_m16n8k32(
              (void*)(C_warp + i_0_3 * WARP_N / INTRIN_N * 8 + j_0_4 * 8),
              (void*)(A_shared_warp + i_0_3 * 16),
              (void*)(B_shared_warp + j_0_4 * 16));
          mma_m16n8k32(
              (void*)(C_warp + i_0_3 * WARP_N / INTRIN_N * 8 + j_0_4 * 8 + 4),
              (void*)(A_shared_warp + i_0_3 * 16),
              (void*)(B_shared_warp + j_0_4 * 16 + 8));
        }
      }

      if (iter_k < SHARED_K_ITERS - 1) {
        if constexpr (STAGES == 1) __syncthreads();
        global_to_share_one_stage_A<CTA_M, CTA_N, CTA_K, CTA_SIZE, WARP_K / INTRIN_K, STAGES>(
            A_hoisted,
            A_shared_hoisted + ld_stage * kSmemSizeAPerStage,
            K,
            cta_offset_m,
            cta_offset_n,
            k_0_0_ld,
            iter_k,
            k_0_0_ld < gemm_iters,
            A_g2s_preds);
        global_to_share_one_stage_B<CTA_M, CTA_N, CTA_K, CTA_SIZE, WARP_K / INTRIN_K, STAGES>(
            B_hoisted,
            B_shared_hoisted + ld_stage * kSmemSizeBPerStage,
            K,
            cta_offset_m,
            cta_offset_n,
            k_0_0_ld,
            iter_k,
            k_0_0_ld < gemm_iters);
      }

      if (iter_k == SHARED_K_ITERS - 2) {
        if constexpr (STAGES == 1 && SHARED_K_ITERS > 2) {
          __syncthreads();
        }
        global_to_share_one_stage_A<CTA_M, CTA_N, CTA_K, CTA_SIZE, WARP_K / INTRIN_K, STAGES>(
            A_hoisted,
            A_shared_hoisted + ld_stage * kSmemSizeAPerStage,
            K,
            cta_offset_m,
            cta_offset_n,
            k_0_0_ld,
            iter_k + 1,
            k_0_0_ld < gemm_iters,
            A_g2s_preds);
        global_to_share_one_stage_B<CTA_M, CTA_N, CTA_K, CTA_SIZE, WARP_K / INTRIN_K, STAGES>(
            B_hoisted,
            B_shared_hoisted + ld_stage * kSmemSizeBPerStage,
            K,
            cta_offset_m,
            cta_offset_n,
            k_0_0_ld,
            iter_k + 1,
            k_0_0_ld < gemm_iters);
        if constexpr (STAGES > 1) {
          __pipeline_commit();
          __pipeline_wait_prior(STAGES - 2);
        }
        compute_stage = (k_0_0 + 1) % STAGES;
        __syncthreads();
      }
    }
  }

  __pipeline_commit();
  __pipeline_wait_prior(0);
  __syncthreads();

  if constexpr (SLICES > 1) {
#pragma unroll
    for (int z = 0; z < SLICES; ++z) {
      if (slice_id == z) {
#pragma unroll
        for (int ax0_0_1 = 0; ax0_0_1 < WARP_M / INTRIN_M; ++ax0_0_1) {
#pragma unroll
          for (int ax1_0_1 = 0; ax1_0_1 < WARP_N / INTRIN_N; ++ax1_0_1) {
#pragma unroll
            for (int local_id = 0; local_id < OP_M * 16 / WARP_SIZE; ++local_id) {
              if (z > 0) {
                C_warp[ax0_0_1 * WARP_N / INTRIN_N * 8 + ax1_0_1 * 8 + local_id] += C_shared
                    [warp_offset_m * CTA_N + ax0_0_1 * OP_M * CTA_N + warp_offset_n + ax1_0_1 * 16 +
                     ((local_id % 4) / 2 * 8 + (threadIdx.x / 4)) * CTA_N + (local_id / 4) * 8 + (local_id % 2) +
                     (threadIdx.x % 4) * 2];
              }
              C_shared
                  [warp_offset_m * CTA_N + ax0_0_1 * OP_M * CTA_N + warp_offset_n + ax1_0_1 * 16 +
                   ((local_id % 4) / 2 * 8 + (threadIdx.x / 4)) * CTA_N + (local_id / 4) * 8 + (local_id % 2) +
                   (threadIdx.x % 4) * 2] = C_warp[ax0_0_1 * WARP_N / INTRIN_N * 8 + ax1_0_1 * 8 + local_id];
            };
          }
        }
      }
      __syncthreads();
    }
    if (slice_id == 0) {
#pragma unroll
      for (int ax0_0_1 = 0; ax0_0_1 < WARP_M / INTRIN_M; ++ax0_0_1) {
#pragma unroll
        for (int ax1_0_1 = 0; ax1_0_1 < WARP_N / INTRIN_N; ++ax1_0_1) {
#pragma unroll
          for (int local_id = 0; local_id < OP_M * 16 / WARP_SIZE; ++local_id) {
            C_warp[ax0_0_1 * WARP_N / INTRIN_N * 8 + ax1_0_1 * 8 + local_id] = C_shared
                [warp_offset_m * CTA_N + ax0_0_1 * OP_M * CTA_N + warp_offset_n + ax1_0_1 * 16 +
                 ((local_id % 4) / 2 * 8 + (threadIdx.x / 4)) * CTA_N + (local_id / 4) * 8 + (local_id % 2) +
                 (threadIdx.x % 4) * 2];
          };
        }
      }
    }
  }

  int row_wb_thd = cta_offset_m + warp_offset_m + (threadIdx.x / 4);
  int col_wb_thd = cta_offset_n + warp_offset_n + (threadIdx.x % 4) * 2;
  if (slice_id == 0) {
    for (int ax0_0_1 = 0; ax0_0_1 < WARP_M / INTRIN_M; ++ax0_0_1) {
      int row_wb_1 = row_wb_thd + ax0_0_1 * OP_M;
      for (int ax1_0_1 = 0; ax1_0_1 < WARP_N / INTRIN_N; ++ax1_0_1) {
        int col_wb_1 = col_wb_thd + ax1_0_1 * 16;
        int* C_warp_local = C_warp + ax0_0_1 * WARP_N / INTRIN_N * 8 + ax1_0_1 * 8;
        for (int local_id = 0; local_id < OP_M * 16 / WARP_SIZE; local_id += 2) {
          int row_wb = row_wb_1 + (local_id % 4) / 2 * 8;
          if (row_wb < M) {
            int col_wb = col_wb_1 + (local_id / 4) * 8 + (local_id % 2);
            float2 wscale = __half22float2(*(wscales + col_wb / 2));
            float2 w_sz = __half22float2(*(w_szs + col_wb / 2));
            float ascale = __half2float(ascales[row_wb]);
            float a_ssum = __half2float(a_ssums[row_wb]);
            float2 psums =
                make_float2(__int2float_rn(C_warp_local[local_id]), __int2float_rn(C_warp_local[local_id + 1]));
            psums.x = psums.x * wscale.x * ascale - w_sz.x * a_ssum;
            psums.y = psums.y * wscale.y * ascale - w_sz.y * a_ssum;
            *reinterpret_cast<half2*>(C + row_wb * N + col_wb) = __float22half2_rn(psums);
          }
        };
      }
    }
  }
}
#else
template <int CTA_M, int CTA_N, int CTA_K, int WARP_M, int WARP_N, int WARP_K, int STAGES, int G>
__global__ void dense_kernel0(
    int8_t* __restrict__ A,
    int8_t* __restrict__ B,
    half2* __restrict__ wscales,
    half* __restrict__ ascales,
    half2* __restrict__ w_szs,
    half* __restrict__ a_ssums,
    half* __restrict__ C,
    int M,
    int64_t N,
    int64_t K) {
  // Not implemented for SM < 800
  assert(false);
  return;
}
#endif

void qserve_w4a8_per_chn_gemm(
    const torch::Tensor& _in_feats,
    const torch::Tensor& _kernel,
    const torch::Tensor& _wscales,
    const torch::Tensor& _ascales,
    const torch::Tensor& _w_szs,
    const torch::Tensor& _a_ssums,
    torch::Tensor& _out_feats) {
  // Check input tensor
  TORCH_CHECK(_in_feats.is_cuda(), "_in_feats must be a CUDA tensor");
  TORCH_CHECK(_in_feats.dim() == 2, "_in_feats must be a 2D tensor");
  TORCH_CHECK(_in_feats.is_contiguous(), "_in_feats must be contiguous");
  TORCH_CHECK(_in_feats.scalar_type() == torch::kInt8, "_in_feats must be int8");
  // Check kernel tensor
  TORCH_CHECK(_kernel.is_cuda(), "_kernel must be a CUDA tensor");
  TORCH_CHECK(_kernel.dim() == 2, "_kernel must be a 2D tensor");
  TORCH_CHECK(_kernel.is_contiguous(), "_kernel must be contiguous");
  TORCH_CHECK(_kernel.scalar_type() == torch::kInt8, "_kernel must be int8");
  // Check output tensor
  TORCH_CHECK(_out_feats.is_cuda(), "_out_feats must be a CUDA tensor");
  TORCH_CHECK(_out_feats.is_contiguous(), "_out_feats must be contiguous");
  TORCH_CHECK(_out_feats.scalar_type() == torch::kHalf, "_out_feats must be half");

  int num_in_feats = _in_feats.size(0);
  int num_in_channels = _in_feats.size(1);
  int num_out_feats = _out_feats.size(-2);
  int num_out_channels = _out_feats.size(-1);

  // Check matmul shape
  TORCH_CHECK(num_out_channels == _kernel.size(0), "num_out_channels must be equal to _kernel.size(0)");
  TORCH_CHECK(num_in_feats == num_out_feats, "num_in_feats must be equal to num_out_feats");

  // Check _ascales
  TORCH_CHECK(_ascales.is_cuda(), "_ascales must be a CUDA tensor");
  TORCH_CHECK(_ascales.is_contiguous(), "_ascales must be contiguous");
  TORCH_CHECK(_ascales.scalar_type() == torch::kHalf, "_ascales must be half");
  TORCH_CHECK(_ascales.numel() == num_in_feats, "_ascales must have num_in_feats elements");

  // Check _wscales
  TORCH_CHECK(_wscales.is_cuda(), "_wscales must be a CUDA tensor");
  TORCH_CHECK(_wscales.is_contiguous(), "_wscales must be contiguous");
  TORCH_CHECK(_wscales.scalar_type() == torch::kHalf, "_wscales must be half");
  TORCH_CHECK(_wscales.numel() == num_out_channels, "_wscales must have num_out_channels elements");

  // Check _w_szs
  TORCH_CHECK(_w_szs.is_cuda(), "_w_szs must be a CUDA tensor");
  TORCH_CHECK(_w_szs.is_contiguous(), "_w_szs must be contiguous");
  TORCH_CHECK(_w_szs.scalar_type() == torch::kHalf, "_w_szs must be half");
  TORCH_CHECK(_w_szs.numel() == num_out_channels, "_w_szs must have num_out_channels elements");

  // Check _a_ssums
  TORCH_CHECK(_a_ssums.is_cuda(), "_a_ssums must be a CUDA tensor");
  TORCH_CHECK(_a_ssums.is_contiguous(), "_a_ssums must be contiguous");
  TORCH_CHECK(_a_ssums.scalar_type() == torch::kHalf, "_a_ssums must be half");
  TORCH_CHECK(_a_ssums.numel() == num_in_feats, "_a_ssums must have num_in_feats elements");

  auto in_feats = reinterpret_cast<int8_t*>(_in_feats.data_ptr<int8_t>());
  auto kernel = reinterpret_cast<int8_t*>(_kernel.data_ptr<int8_t>());
  auto w_szs = reinterpret_cast<half2*>(_w_szs.data_ptr());
  auto a_ssums = reinterpret_cast<half*>(_a_ssums.data_ptr());
  auto wscales = reinterpret_cast<half2*>(_wscales.data_ptr());
  auto ascales = reinterpret_cast<half*>(_ascales.data_ptr());
  auto out_feats = reinterpret_cast<half*>(_out_feats.data_ptr<at::Half>());
  auto stream = at::cuda::getCurrentCUDAStream(_in_feats.get_device());

  auto sm_version = getSMVersion();
  if (sm_version >= 80) {
    constexpr int G = 128;

    if (num_out_feats > 256) {
      constexpr int CTA_M = 128;
      constexpr int CTA_N = 128;
      constexpr int CTA_K = 64;
      constexpr int WARP_M = 64;
      constexpr int WARP_N = 32;
      constexpr int WARP_K = 64;
      constexpr int STAGES = 3;
      KERNEL_LAUNCH_CODE
    } else if (num_out_feats >= 128) {
      constexpr int CTA_M = 64;
      constexpr int CTA_N = 64;
      constexpr int CTA_K = 64;
      constexpr int WARP_M = 32;
      constexpr int WARP_N = 32;
      constexpr int WARP_K = 64;
      constexpr int STAGES = 4;
      KERNEL_LAUNCH_CODE
    } else {
      constexpr int CTA_M = 32;
      constexpr int CTA_N = 64;
      constexpr int CTA_K = 128;
      constexpr int WARP_M = 32;
      constexpr int WARP_N = 32;
      constexpr int WARP_K = 64;
      constexpr int STAGES = 3;
      KERNEL_LAUNCH_CODE
    }
  } else {
    TORCH_CHECK_NOT_IMPLEMENTED(
        false, "No implemented qserve_w4a8_per_chn_gemm for current compute capability: ", sm_version);
  }
  return;
}
