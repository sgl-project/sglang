#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <torch/all.h>

#include "nvfp4_quant.cuh"
#include "utils.h"

// Quantizes the provided PackedVec into the uint32_t output
template <class Type, bool UE8M0_SF = false>
__device__ uint32_t cvt_warp_fp16_to_fp4(PackedVec<Type>& vec, float SFScaleVal, uint8_t* SFout) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  // Get absolute maximum values among the local 8 values.
  auto localMax = __habs2(vec.elts[0]);

// Local maximum value.
#pragma unroll
  for (int i = 1; i < CVT_FP4_ELTS_PER_THREAD / 2; i++) {
    localMax = __hmax2(localMax, __habs2(vec.elts[i]));
  }

  // Get the absolute maximum among all 16 values (two threads).
  localMax = __hmax2(__shfl_xor_sync(uint32_t(-1), localMax, 1), localMax);
  // Get the final absolute maximum values.
  float vecMax = float(__hmax(localMax.x, localMax.y));

  // Get the SF (max value of the vector / max value of e2m1).
  // maximum value of e2m1 = 6.0.
  // TODO: use half as compute data type.
  float SFValue = SFScaleVal * (vecMax * reciprocal_approximate_ftz(6.0f));
  // 8 bits representation of the SF.
  uint8_t fp8SFVal;
  // Write the SF to global memory (STG.8).
  if constexpr (UE8M0_SF) {
    // Extract the 8 exponent bits from float32.
    // float 32bits = 1 sign bit + 8 exponent bits + 23 mantissa bits.
    uint32_t tmp = reinterpret_cast<uint32_t&>(SFValue) >> 23;
    fp8SFVal = tmp & 0xff;
    // Convert back to fp32.
    reinterpret_cast<uint32_t&>(SFValue) = tmp << 23;
  } else {
    // Here SFValue is always positive, so E4M3 is the same as UE4M3.
    __hip_fp8_e4m3_fnuz tmp = __hip_fp8_e4m3_fnuz(SFValue);
    reinterpret_cast<__hip_fp8_e4m3_fnuz&>(fp8SFVal) = tmp;
    // Convert back to fp32.
    SFValue = float(tmp);
  }
  // Get the output scale.
  // Recipe: final_scale = reciprocal(fp32(fp8(SFValue * SFScaleVal))) *
  //                       reciprocal(SFScaleVal))
  float outputScale =
      SFValue != 0 ? reciprocal_approximate_ftz(SFValue * reciprocal_approximate_ftz(SFScaleVal)) : 0.0f;

  if (SFout) {
    // Write the SF to global memory (STG.8).
    *SFout = fp8SFVal;
  }

  // Convert the input to float.
  float2 fp2Vals[CVT_FP4_ELTS_PER_THREAD / 2];

#pragma unroll
  for (int i = 0; i < CVT_FP4_ELTS_PER_THREAD / 2; i++) {
    if constexpr (std::is_same_v<Type, half>) {
      fp2Vals[i] = __half22float2(vec.elts[i]);
    } else {
      fp2Vals[i] = __bfloat1622float2(vec.elts[i]);
    }
    fp2Vals[i].x *= outputScale;
    fp2Vals[i].y *= outputScale;
  }

  // Convert to e2m1 values.
  uint32_t e2m1Vec = fp32_vec_to_e2m1(fp2Vals);

  // Write the e2m1 values to global memory.
  return e2m1Vec;
#else
  return 0;
#endif
}

__device__ __forceinline__ float silu(const float& val) {
  return val / (1.0f + __expf(-val));
}

template <class Type>
inline __device__ void silu_and_mul(PackedVec<Type>& x_vec, const PackedVec<Type>& y_vec) {
  float2 x[CVT_FP4_ELTS_PER_THREAD / 2];
  float2 y[CVT_FP4_ELTS_PER_THREAD / 2];

#pragma unroll
  for (int i = 0; i < CVT_FP4_ELTS_PER_THREAD / 2; i++) {
    if constexpr (std::is_same_v<Type, half>) {
      x[i] = __half22float2(x_vec.elts[i]);
      y[i] = __half22float2(y_vec.elts[i]);
      x[i].x = silu(x[i].x) * y[i].x;
      x[i].y = silu(x[i].y) * y[i].y;
      x_vec.elts[i] = __float22half2_rn(x[i]);
    } else {
      x[i] = __bfloat1622float2(x_vec.elts[i]);
      y[i] = __bfloat1622float2(y_vec.elts[i]);
      x[i].x = silu(x[i].x) * y[i].x;
      x[i].y = silu(x[i].y) * y[i].y;
      x_vec.elts[i] = __float22bfloat162_rn(x[i]);
    }
  }
}

// Use UE4M3 by default.
template <class Type, bool UE8M0_SF = false, bool SMALL_NUM_EXPERTS = false>
__global__ void
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
__launch_bounds__(512, 4) cvt_fp16_to_fp4(
#else
cvt_fp16_to_fp4(
#endif
    int32_t numRows,
    int32_t numCols,
    Type const* in,
    float const* SFScale,
    uint32_t* out,
    uint32_t* SFout,
    uint32_t* input_offset_by_experts,
    uint32_t* output_scale_offset_by_experts,
    int32_t* mask,
    int n_experts,
    bool low_latency) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  using PackedVec = PackedVec<Type>;
  static constexpr int CVT_FP4_NUM_THREADS_PER_SF = (CVT_FP4_SF_VEC_SIZE / CVT_FP4_ELTS_PER_THREAD);
  static_assert(sizeof(PackedVec) == sizeof(Type) * CVT_FP4_ELTS_PER_THREAD, "Vec size is not matched.");

  // Input tensor row/col loops.
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int colsPerRow = numCols / CVT_FP4_ELTS_PER_THREAD;
  // TODO(kaixih@nvidia): For now, we assume mask is used together with
  // silu_and_mal. Maybe we want a more general behavior of mask later. In the
  // silu case, the input last dim doubles.
  bool use_mask = mask != nullptr;
  int actualColsPerRow = use_mask ? colsPerRow * 2 : colsPerRow;

  // Each global thread processes one element
  for (int globalIdx = tid; globalIdx < numRows * colsPerRow; globalIdx += gridDim.x * blockDim.x) {
    // Calculate which row and column this global thread should process
    int rowIdx = globalIdx / colsPerRow;
    int colIdx = globalIdx % colsPerRow;

    // Find index within the experts using different strategies based on expert
    // count
    int rowIdx_in_expert = 0;
    int expert_idx = 0;

    if constexpr (SMALL_NUM_EXPERTS) {
      for (int i = 0; i < n_experts; i++) {
        uint32_t current_offset = __ldca(&input_offset_by_experts[i]);
        uint32_t next_offset = __ldca(&input_offset_by_experts[i + 1]);
        if (rowIdx >= current_offset && rowIdx < next_offset) {
          rowIdx_in_expert = rowIdx - current_offset;
          expert_idx = i;
          break;
        }
      }
    } else {
      // Load input offsets into registers first, then do the computation.
      // Local array size set to 17 because of register limit.
      uint32_t local_offsets[17];
      for (int chunk_start = 0; chunk_start < n_experts; chunk_start += 16) {
        *reinterpret_cast<int4*>(local_offsets) =
            __ldca(reinterpret_cast<const int4*>(&input_offset_by_experts[chunk_start]));
        *reinterpret_cast<int4*>(local_offsets + 4) =
            __ldca(reinterpret_cast<const int4*>(&input_offset_by_experts[chunk_start + 4]));
        *reinterpret_cast<int4*>(local_offsets + 8) =
            __ldca(reinterpret_cast<const int4*>(&input_offset_by_experts[chunk_start + 8]));
        *reinterpret_cast<int4*>(local_offsets + 12) =
            __ldca(reinterpret_cast<const int4*>(&input_offset_by_experts[chunk_start + 12]));
        local_offsets[16] = __ldca(&input_offset_by_experts[chunk_start + 16]);

// Check against the 16 loaded offsets
#pragma unroll
        for (int i = 0; i < 16; i++) {
          if (rowIdx >= local_offsets[i] && rowIdx < local_offsets[i + 1]) {
            rowIdx_in_expert = rowIdx - local_offsets[i];
            expert_idx = chunk_start + i;
            break;
          }
        }
      }
    }

    // Early exit when using masks.
    if (use_mask && rowIdx_in_expert >= mask[expert_idx]) {
      continue;
    }

    int64_t inOffset = rowIdx * actualColsPerRow + colIdx;
    PackedVec in_vec = reinterpret_cast<PackedVec const*>(in)[inOffset];
    if (use_mask) {
      PackedVec in_vec_mul = reinterpret_cast<PackedVec const*>(in)[inOffset + colsPerRow];
      silu_and_mul(in_vec, in_vec_mul);
    }

    // Get the output tensor offset.
    // Same as inOffset because 8 elements are packed into one uint32_t.
    int64_t outOffset = rowIdx * colsPerRow + colIdx;
    auto& out_pos = out[outOffset];

    // Get the global scaling factor, which will be applied to the SF.
    // Note SFScale is the same as next GEMM's alpha, which is
    // (448.f / (Alpha_A / 6.f)).
    float const SFScaleVal = SFScale == nullptr ? 1.0f : SFScale[expert_idx];

    int factor = CVT_FP4_SF_VEC_SIZE * 4;
    // The actual output_scales dim is computed from the padded numCols.
    int32_t numCols_padded = (numCols + factor - 1) / factor * factor;
    int numCols_SFout = numCols_padded / CVT_FP4_SF_VEC_SIZE / 4;
    uint32_t* SFout_in_expert = SFout + output_scale_offset_by_experts[expert_idx] * numCols_SFout;

    auto sf_out = cvt_quant_to_fp4_get_sf_out_offset<uint32_t, CVT_FP4_NUM_THREADS_PER_SF>(
        rowIdx_in_expert, colIdx, numCols, SFout_in_expert);

    out_pos = cvt_warp_fp16_to_fp4<Type, UE8M0_SF>(in_vec, SFScaleVal, sf_out);
  }
#endif
}

// Use UE4M3 by default.
template <class Type, bool UE8M0_SF = false>
__global__ void
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
__launch_bounds__(512, 4) cvt_fp16_to_fp4_expert(
#else
cvt_fp16_to_fp4_expert(
#endif
    int32_t numRows,
    int32_t numCols,
    Type const* in,
    float const* SFScale,
    uint32_t* out,
    uint32_t* SFout,
    int32_t* mask,
    bool use_silu_and_mul,
    int n_experts) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  using PackedVec = PackedVec<Type>;
  static constexpr int CVT_FP4_NUM_THREADS_PER_SF = (CVT_FP4_SF_VEC_SIZE / CVT_FP4_ELTS_PER_THREAD);
  static_assert(sizeof(PackedVec) == sizeof(Type) * CVT_FP4_ELTS_PER_THREAD, "Vec size is not matched.");

  // Input tensor row/col loops.
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = (gridDim.x * blockDim.x) / n_experts;
  int remainder = (gridDim.x * blockDim.x) % n_experts;
  int expert_idx;
  int tid_in_expert;
  int actual_stride;
  if (remainder > 0) {
    int bound = remainder * (stride + 1);
    if (tid < bound) {
      expert_idx = tid / (stride + 1);
      tid_in_expert = tid % (stride + 1);
      actual_stride = stride + 1;
    } else {
      expert_idx = remainder + (tid - bound) / stride;
      tid_in_expert = (tid - bound) % stride;
      actual_stride = stride;
    }
  } else {
    expert_idx = tid / stride;
    tid_in_expert = tid % stride;
    actual_stride = stride;
  }
  int m = numRows / n_experts;
  int padded_m = (m + (128 - 1)) / 128 * 128;

  int colsPerRow = numCols / CVT_FP4_ELTS_PER_THREAD;
  // TODO(kaixih@nvidia): For now, we assume mask is used together with
  // silu_and_mal. Maybe we want a more general behavior of mask later. In the
  // silu case, the input last dim doubles.
  bool use_mask = mask != nullptr;
  int actualColsPerRow = use_silu_and_mul ? colsPerRow * 2 : colsPerRow;

  // Each global thread processes one element
  for (int globalIdx = tid_in_expert + expert_idx * m * colsPerRow; globalIdx < (expert_idx + 1) * m * colsPerRow;
       globalIdx += actual_stride) {
    // Calculate which row and column this global thread should process
    int rowIdx = globalIdx / colsPerRow;
    int colIdx = globalIdx % colsPerRow;

    // Find index within the experts
    int rowIdx_in_expert = rowIdx - expert_idx * m;

    // Early exit when using masks.
    if (use_mask && rowIdx_in_expert >= mask[expert_idx]) {
      break;
    }

    int64_t inOffset = rowIdx * actualColsPerRow + colIdx;
    PackedVec in_vec = reinterpret_cast<PackedVec const*>(in)[inOffset];
    if (use_silu_and_mul) {
      PackedVec in_vec_mul = reinterpret_cast<PackedVec const*>(in)[inOffset + colsPerRow];
      silu_and_mul(in_vec, in_vec_mul);
    }

    // Get the output tensor offset.
    // Same as inOffset because 8 elements are packed into one uint32_t.
    int64_t outOffset = rowIdx * colsPerRow + colIdx;
    auto& out_pos = out[outOffset];

    // Get the global scaling factor, which will be applied to the SF.
    // Note SFScale is the same as next GEMM's alpha, which is
    // (448.f / (Alpha_A / 6.f)).
    float const SFScaleVal = SFScale == nullptr ? 1.0f : SFScale[expert_idx];

    int factor = CVT_FP4_SF_VEC_SIZE * 4;
    // The actual output_scales dim is computed from the padded numCols.
    int32_t numCols_padded = (numCols + factor - 1) / factor * factor;
    int numCols_SFout = numCols_padded / CVT_FP4_SF_VEC_SIZE / 4;
    uint32_t* SFout_in_expert = SFout + expert_idx * padded_m * numCols_SFout;

    auto sf_out = cvt_quant_to_fp4_get_sf_out_offset<uint32_t, CVT_FP4_NUM_THREADS_PER_SF>(
        rowIdx_in_expert, colIdx, numCols, SFout_in_expert);

    out_pos = cvt_warp_fp16_to_fp4<Type, UE8M0_SF>(in_vec, SFScaleVal, sf_out);
  }
#endif
}

// Kernel for LARGE_M_TOPK = true (large m_topk optimized version)
template <class Type, bool UE8M0_SF = false, bool SMALL_NUM_EXPERTS = false>
__global__ void
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
__launch_bounds__(1024, 4) cvt_fp16_to_fp4(
#else
cvt_fp16_to_fp4(
#endif
    int32_t numRows,
    int32_t numCols,
    Type const* in,
    float const* SFScale,
    uint32_t* out,
    uint32_t* SFout,
    uint32_t* input_offset_by_experts,
    uint32_t* output_scale_offset_by_experts,
    int32_t* mask,
    int n_experts) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  using PackedVec = PackedVec<Type>;
  static constexpr int CVT_FP4_NUM_THREADS_PER_SF = (CVT_FP4_SF_VEC_SIZE / CVT_FP4_ELTS_PER_THREAD);
  static_assert(sizeof(PackedVec) == sizeof(Type) * CVT_FP4_ELTS_PER_THREAD, "Vec size is not matched.");
  extern __shared__ uint32_t shared_input_offsets[];

  // Load input offsets into shared memory.
  // If n_experts is larger than 4, use vectorized int4 to save instructions.
  // If n_experts is smaller than 4, read directly.
  if constexpr (SMALL_NUM_EXPERTS) {
    for (int i = threadIdx.x; i < n_experts + 1; i += blockDim.x) {
      shared_input_offsets[i] = input_offset_by_experts[i];
    }
  } else {
    for (int i = threadIdx.x * 4; i < n_experts; i += blockDim.x * 4) {
      *reinterpret_cast<int4*>(&shared_input_offsets[i]) = *reinterpret_cast<const int4*>(&input_offset_by_experts[i]);
    }
    if (threadIdx.x == 0) {
      shared_input_offsets[n_experts] = input_offset_by_experts[n_experts];
    }
  }

  __syncthreads();

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int colsPerRow = numCols / CVT_FP4_ELTS_PER_THREAD;
  bool use_mask = mask != nullptr;
  int actualColsPerRow = use_mask ? colsPerRow * 2 : colsPerRow;

  // Each global thread processes one element
  for (int globalIdx = tid; globalIdx < numRows * colsPerRow; globalIdx += gridDim.x * blockDim.x) {
    // Calculate which row and column this global thread should process
    int rowIdx = globalIdx / colsPerRow;
    int colIdx = globalIdx % colsPerRow;

    // Find expert using binary search for better performance with large m_topk
    int rowIdx_in_expert = 0;
    int expert_idx = 0;

    // Binary search through experts using shared memory
    int left = 0, right = n_experts - 1;
    while (left <= right) {
      int mid = (left + right) / 2;
      // Get offsets: shared_input_offsets[i] corresponds to
      // input_offset_by_experts[i]
      uint32_t mid_offset = shared_input_offsets[mid];
      uint32_t next_offset = shared_input_offsets[mid + 1];

      if (rowIdx >= mid_offset && rowIdx < next_offset) {
        rowIdx_in_expert = rowIdx - mid_offset;
        expert_idx = mid;
        break;
      } else if (rowIdx < mid_offset) {
        right = mid - 1;
      } else {
        left = mid + 1;
      }
    }

    if (use_mask && rowIdx_in_expert >= mask[expert_idx]) {
      continue;
    }

    int64_t inOffset = rowIdx * actualColsPerRow + colIdx;

    PackedVec in_vec = reinterpret_cast<PackedVec const*>(in)[inOffset];
    if (use_mask) {
      PackedVec in_vec_mul = reinterpret_cast<PackedVec const*>(in)[inOffset + colsPerRow];
      silu_and_mul(in_vec, in_vec_mul);
    }

    int64_t outOffset = rowIdx * colsPerRow + colIdx;
    auto& out_pos = out[outOffset];

    float const SFScaleVal = SFScale == nullptr ? 1.0f : SFScale[expert_idx];

    int factor = CVT_FP4_SF_VEC_SIZE * 4;
    int32_t numCols_padded = (numCols + factor - 1) / factor * factor;
    int numCols_SFout = numCols_padded / CVT_FP4_SF_VEC_SIZE / 4;
    uint32_t* SFout_in_expert = SFout + output_scale_offset_by_experts[expert_idx] * numCols_SFout;

    auto sf_out = cvt_quant_to_fp4_get_sf_out_offset<uint32_t, CVT_FP4_NUM_THREADS_PER_SF>(
        rowIdx_in_expert, colIdx, numCols, SFout_in_expert);

    out_pos = cvt_warp_fp16_to_fp4<Type, UE8M0_SF>(in_vec, SFScaleVal, sf_out);
  }
#endif
}

template <typename T>
void quant_impl(
    void* output,
    void* output_scale,
    void* input,
    void* input_global_scale,
    void* input_offset_by_experts,
    void* output_scale_offset_by_experts,
    void* mask,
    bool use_silu_and_mul,
    int m_topk,
    int k,
    int n_experts,
    hipStream_t stream) {
  // TODO: this multiProcessorCount should be cached.
  int device;
  hipGetDevice(&device);
  int multiProcessorCount;
  hipDeviceGetAttribute(&multiProcessorCount, hipDeviceAttributeMultiprocessorCount, device);

  // Grid, Block size.
  // Each thread converts 8 values.
  int const workSizePerRow = k / ELTS_PER_THREAD;
  int const totalWorkSize = m_topk * workSizePerRow;
  dim3 block(std::min(workSizePerRow, 512));
  // Get number of blocks per SM (assume we can fully utilize the SM).
  int const numBlocksPerSM = 2048 / block.x;
  dim3 grid(std::min(static_cast<int>((totalWorkSize + block.x - 1) / block.x), multiProcessorCount * numBlocksPerSM));
  while (grid.x <= multiProcessorCount && block.x > 64) {
    grid.x *= 2;
    block.x = (block.x + 1) / 2;
  }

  // TODO(kaixih@nvidia): Should relax this to allow any grid size.
  if (mask != nullptr) {
    grid.x = (grid.x + n_experts - 1) / n_experts * n_experts;
    cvt_fp16_to_fp4_expert<T, false><<<grid, block, 0, stream>>>(
        m_topk,
        k,
        reinterpret_cast<T*>(input),
        reinterpret_cast<float*>(input_global_scale),
        reinterpret_cast<uint32_t*>(output),
        reinterpret_cast<uint32_t*>(output_scale),
        reinterpret_cast<int32_t*>(mask),
        use_silu_and_mul,
        n_experts);
    return;
  }

  int const blockRepeat = (totalWorkSize + block.x * grid.x - 1) / (block.x * grid.x);
  if (blockRepeat > 1) {
    size_t shared_mem_size = (n_experts + 1) * sizeof(uint32_t);
    if (n_experts >= 4) {
      cvt_fp16_to_fp4<T, false, false><<<grid, block, shared_mem_size, stream>>>(
          m_topk,
          k,
          reinterpret_cast<T*>(input),
          reinterpret_cast<float*>(input_global_scale),
          reinterpret_cast<uint32_t*>(output),
          reinterpret_cast<uint32_t*>(output_scale),
          reinterpret_cast<uint32_t*>(input_offset_by_experts),
          reinterpret_cast<uint32_t*>(output_scale_offset_by_experts),
          reinterpret_cast<int32_t*>(mask),
          n_experts);
    } else {
      cvt_fp16_to_fp4<T, false, true><<<grid, block, shared_mem_size, stream>>>(
          m_topk,
          k,
          reinterpret_cast<T*>(input),
          reinterpret_cast<float*>(input_global_scale),
          reinterpret_cast<uint32_t*>(output),
          reinterpret_cast<uint32_t*>(output_scale),
          reinterpret_cast<uint32_t*>(input_offset_by_experts),
          reinterpret_cast<uint32_t*>(output_scale_offset_by_experts),
          reinterpret_cast<int32_t*>(mask),
          n_experts);
    }
  } else {
    if (n_experts >= 16) {
      cvt_fp16_to_fp4<T, false, false><<<grid, block, 0, stream>>>(
          m_topk,
          k,
          reinterpret_cast<T*>(input),
          reinterpret_cast<float*>(input_global_scale),
          reinterpret_cast<uint32_t*>(output),
          reinterpret_cast<uint32_t*>(output_scale),
          reinterpret_cast<uint32_t*>(input_offset_by_experts),
          reinterpret_cast<uint32_t*>(output_scale_offset_by_experts),
          reinterpret_cast<int32_t*>(mask),
          n_experts,
          /* bool low_latency */ true);
    } else {
      cvt_fp16_to_fp4<T, false, true><<<grid, block, 0, stream>>>(
          m_topk,
          k,
          reinterpret_cast<T*>(input),
          reinterpret_cast<float*>(input_global_scale),
          reinterpret_cast<uint32_t*>(output),
          reinterpret_cast<uint32_t*>(output_scale),
          reinterpret_cast<uint32_t*>(input_offset_by_experts),
          reinterpret_cast<uint32_t*>(output_scale_offset_by_experts),
          reinterpret_cast<int32_t*>(mask),
          n_experts,
          /* bool low_latency */ true);
    }
  }
}

// Avoid redefinition warnings
#undef CHECK_CONTIGUOUS
#undef CHECK_TH_CUDA
#undef CHECK_INPUT

/*Quantization entry for fp4 experts quantization*/
#define CHECK_TH_CUDA(x, m) TORCH_CHECK(x.is_cuda(), m, "must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x, m) TORCH_CHECK(x.is_contiguous(), m, "must be contiguous")
#define CHECK_INPUT(x, m) \
  CHECK_TH_CUDA(x, m);    \
  CHECK_CONTIGUOUS(x, m);

// constexpr auto FP8 = at::ScalarType::Float8_e4m3fn;
constexpr auto HALF = at::ScalarType::Half;
constexpr auto BF16 = at::ScalarType::BFloat16;
constexpr auto FLOAT = at::ScalarType::Float;
constexpr auto INT = at::ScalarType::Int;
constexpr auto UINT8 = at::ScalarType::Byte;

void scaled_fp4_experts_quant_sm100a(
    torch::Tensor& output,
    torch::Tensor& output_scale,
    torch::Tensor const& input,
    torch::Tensor const& input_global_scale,
    torch::Tensor const& input_offset_by_experts,
    torch::Tensor const& output_scale_offset_by_experts) {
  auto sm_version = getSMVersion();
  TORCH_CHECK(sm_version == 100 || sm_version == 103, "fp4_quant is only supported on sm100a/sm103a");

  CHECK_INPUT(output, "output must be a CUDA tensor");
  CHECK_INPUT(output_scale, "output_scale must be a CUDA tensor");
  CHECK_INPUT(input, "input must be a CUDA tensor");
  CHECK_INPUT(input_global_scale, "input_global_scale must be a CUDA tensor");
  CHECK_INPUT(input_offset_by_experts, "input_offset_by_experts must be a CUDA tensor");
  CHECK_INPUT(output_scale_offset_by_experts, "output_scale_offset_by_experts must be a CUDA tensor");

  TORCH_CHECK(output.dim() == 2);
  TORCH_CHECK(output_scale.dim() == 2);
  TORCH_CHECK(input.dim() == 2);
  TORCH_CHECK(input_global_scale.dim() == 1);
  TORCH_CHECK(input_offset_by_experts.dim() == 1);
  TORCH_CHECK(output_scale_offset_by_experts.dim() == 1);

  TORCH_CHECK(input.scalar_type() == HALF || input.scalar_type() == BF16);
  TORCH_CHECK(input_global_scale.scalar_type() == FLOAT);
  TORCH_CHECK(input_offset_by_experts.scalar_type() == INT);
  TORCH_CHECK(output_scale_offset_by_experts.scalar_type() == INT);
  // output is uint8 (two nvfp4 values are packed into one uint8)
  // output_scale is int32 (four fp8 values are packed into one int32)
  TORCH_CHECK(output.scalar_type() == UINT8);
  TORCH_CHECK(output_scale.scalar_type() == INT);

  const int BLOCK_SIZE = 16;
  auto m_topk = input.size(0);
  auto k = input.size(1);
  TORCH_CHECK(k % BLOCK_SIZE == 0, "k must be a multiple of 16");
  auto n_experts = input_global_scale.size(0);
  TORCH_CHECK(input_offset_by_experts.size(0) == n_experts + 1);
  TORCH_CHECK(output_scale_offset_by_experts.size(0) == n_experts + 1);
  TORCH_CHECK(output.size(0) == m_topk);
  TORCH_CHECK(output.size(1) == k / 2);
  int scales_k = k / BLOCK_SIZE;
  // 4 means the swizzle requirement by nvidia nvfp4.
  int padded_k = (scales_k + (4 - 1)) / 4 * 4;
  // 4 means 4 fp8 values are packed into one int32
  TORCH_CHECK(output_scale.size(1) * 4 == padded_k);

  auto in_dtype = input.dtype();
  at::cuda::CUDAGuard device_guard{(char)input.get_device()};
  const hipStream_t stream = at::cuda::getCurrentCUDAStream(input.get_device());
  if (in_dtype == at::ScalarType::Half) {
    quant_impl<half>(
        output.data_ptr(),
        output_scale.data_ptr(),
        input.data_ptr(),
        input_global_scale.data_ptr(),
        input_offset_by_experts.data_ptr(),
        output_scale_offset_by_experts.data_ptr(),
        nullptr,  // mask
        false,    // use_silu_and_mul
        m_topk,
        k,
        n_experts,
        stream);
  } else if (in_dtype == at::ScalarType::BFloat16) {
    quant_impl<__hip_bfloat16>(
        output.data_ptr(),
        output_scale.data_ptr(),
        input.data_ptr(),
        input_global_scale.data_ptr(),
        input_offset_by_experts.data_ptr(),
        output_scale_offset_by_experts.data_ptr(),
        nullptr,  // mask
        false,    // use_silu_and_mul
        m_topk,
        k,
        n_experts,
        stream);
  } else {
    TORCH_CHECK(false, "Expected input data type to be half or bfloat16");
  }
}

void silu_and_mul_scaled_fp4_experts_quant_sm100a(
    torch::Tensor& output,
    torch::Tensor& output_scale,
    torch::Tensor const& input,
    torch::Tensor const& input_global_scale,
    torch::Tensor const& mask,
    bool use_silu_and_mul) {
  auto sm_version = getSMVersion();
  TORCH_CHECK(sm_version == 100 || sm_version == 103, "fp4_quant is only supported on sm100a/sm103a");

  CHECK_INPUT(output, "output must be a CUDA tensor");
  CHECK_INPUT(output_scale, "output_scale must be a CUDA tensor");
  CHECK_INPUT(input, "input must be a CUDA tensor");
  CHECK_INPUT(input_global_scale, "input_global_scale must be a CUDA tensor");
  CHECK_INPUT(mask, "mask must be a CUDA tensor");

  TORCH_CHECK(output.dim() == 2);
  TORCH_CHECK(output_scale.dim() == 2);
  TORCH_CHECK(input.dim() == 2);
  TORCH_CHECK(input_global_scale.dim() == 1);

  TORCH_CHECK(input.scalar_type() == HALF || input.scalar_type() == BF16);
  TORCH_CHECK(input_global_scale.scalar_type() == FLOAT);
  TORCH_CHECK(mask.scalar_type() == INT);
  // output is uint8 (two nvfp4 values are packed into one uint8)
  // output_scale is int32 (four fp8 values are packed into one int32)
  TORCH_CHECK(output.scalar_type() == UINT8);
  TORCH_CHECK(output_scale.scalar_type() == INT);

  const int BLOCK_SIZE = 16;
  auto m_topk = input.size(0);
  auto k_by_2 = input.size(1);
  auto k = k_by_2;
  if (use_silu_and_mul) {
    TORCH_CHECK(k_by_2 % 2 == 0, "k must be a multiple of 2");
    k = k_by_2 / 2;
  }
  auto n_experts = input_global_scale.size(0);
  TORCH_CHECK(mask.size(0) == n_experts);
  TORCH_CHECK(output.size(0) == m_topk);
  TORCH_CHECK(output.size(1) == k / 2);
  int scales_k = k / BLOCK_SIZE;
  // 4 means the swizzle requirement by nvidia nvfp4.
  int padded_k = (scales_k + (4 - 1)) / 4 * 4;
  // 4 means 4 fp8 values are packed into one int32
  TORCH_CHECK(output_scale.size(1) * 4 == padded_k);

  auto in_dtype = input.dtype();
  at::cuda::CUDAGuard device_guard{(char)input.get_device()};
  const hipStream_t stream = at::cuda::getCurrentCUDAStream(input.get_device());
  if (in_dtype == at::ScalarType::Half) {
    quant_impl<half>(
        output.data_ptr(),
        output_scale.data_ptr(),
        input.data_ptr(),
        input_global_scale.data_ptr(),
        nullptr,  // input_offset_by_experts
        nullptr,  // output_scale_offset_by_experts
        mask.data_ptr(),
        use_silu_and_mul,
        m_topk,
        k,
        n_experts,
        stream);
  } else if (in_dtype == at::ScalarType::BFloat16) {
    quant_impl<__hip_bfloat16>(
        output.data_ptr(),
        output_scale.data_ptr(),
        input.data_ptr(),
        input_global_scale.data_ptr(),
        nullptr,  // input_offset_by_experts
        nullptr,  // output_scale_offset_by_experts
        mask.data_ptr(),
        use_silu_and_mul,
        m_topk,
        k,
        n_experts,
        stream);
  } else {
    TORCH_CHECK(false, "Expected input data type to be half or bfloat16");
  }
}
