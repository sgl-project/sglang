/* Copyright 2025 SGLang Team. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

// Adapted from
// https://github.com/NVIDIA/TensorRT-LLM/blob/v0.16.0/cpp/tensorrt_llm/kernels/cutlass_kernels/fp8_rowwise_gemm/fp8_rowwise_gemm_template.h
// https://github.com/NVIDIA/TensorRT-LLM/blob/v0.16.0/cpp/tensorrt_llm/kernels/cutlass_kernels/fp8_rowwise_gemm/fp8_rowwise_gemm_kernel_template_sm89.h
// https://github.com/NVIDIA/TensorRT-LLM/blob/v0.16.0/cpp/tensorrt_llm/kernels/cutlass_kernels/fp8_rowwise_gemm/fp8_rowwise_gemm_kernel_template_sm90.h

#include <ATen/cuda/HIPContext.h>
#include <cudaTypedefs.h>
#include <cutlass/arch/arch.h>
#include <cutlass/arch/memory.h>
#include <cutlass/arch/mma.h>
#include <cutlass/array.h>
#include <cutlass/cutlass.h>
#include <cutlass/epilogue/thread/activation.h>
#include <cutlass/epilogue/thread/linear_combination.h>
#include <cutlass/epilogue/threadblock/default_thread_map_tensor_op.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/gemm/device/gemm_universal_adapter.h>
#include <cutlass/gemm/gemm.h>
#include <cutlass/gemm/kernel/default_gemm_universal_with_visitor.h>
#include <cutlass/gemm/thread/mma.h>
#include <cutlass/layout/matrix.h>
#include <cutlass/matrix_coord.h>
#include <cutlass/numeric_types.h>
#include <cutlass/tensor_ref.h>
#include <torch/all.h>

#include <cute/tensor.hpp>
#include <cutlass/epilogue/collective/collective_builder.hpp>
#include <cutlass/epilogue/collective/default_epilogue.hpp>
#include <cutlass/epilogue/threadblock/fusion/visitors.hpp>
#include <cutlass/gemm/collective/collective_builder.hpp>
#include <cutlass/gemm/dispatch_policy.hpp>
#include <cutlass/gemm/kernel/gemm_universal.hpp>
#include <cutlass/util/packed_stride.hpp>

#include "utils.h"

using namespace cute;

#if defined CUDA_VERSION && CUDA_VERSION >= 12040
template <
    typename ElementType,
    typename OutElementType,
    typename AccumElementType,
    typename CtaShape,
    typename WarpShape,
    int Stages,
    bool WithBias,
    typename FP8MathOperator = cutlass::arch::OpMultiplyAdd,
    template <typename...> typename EpilogueVisitor = cutlass::epilogue::threadblock::Sm80EVT,
    typename ThreadblockSwizzle = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>>
struct DeviceGemmFp8RowwiseSm89 {
  static_assert(std::is_same_v<ElementType, cutlass::float_e4m3_t>, "ElementType must be FP8(e4m3)");

  using ElementA = ElementType;
  using LayoutA = cutlass::layout::RowMajor;
  static constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementA>::value;

  using ElementB = ElementType;
  using LayoutB = cutlass::layout::ColumnMajor;
  static constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementB>::value;

  using ElementC = OutElementType;
  using LayoutC = cutlass::layout::RowMajor;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;

  using ElementOutput = OutElementType;
  using LayoutOutput = cutlass::layout::RowMajor;
  static constexpr int AlignmentOutput = 128 / cutlass::sizeof_bits<ElementOutput>::value;

  using ElementAccumulator = AccumElementType;
  using ElementComputeEpilogue = float;
  using ArchTag = cutlass::arch::Sm89;
  using OperatorClass = cutlass::arch::OpClassTensorOp;

  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 32>;
  // Number of epilogue stages in EVT
  static constexpr int EVTEpilogueStages = 1;

  using OutputTileThreadMap = cutlass::epilogue::threadblock::
      OutputTileThreadLayout<CtaShape, WarpShape, ElementC, AlignmentC, EVTEpilogueStages>;

  // Definition of EVT
  using accSrc = cutlass::epilogue::threadblock::VisitorAccFetch;

  using ComputeBScale = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::multiplies,
      ElementComputeEpilogue,
      ElementComputeEpilogue,
      cutlass::FloatRoundStyle::round_to_nearest>;
  using bScaleSrc = cutlass::epilogue::threadblock::
      VisitorRowBroadcast<OutputTileThreadMap, ElementComputeEpilogue, Stride<_0, _1, _0>>;
  using EpilogueBScale = cutlass::epilogue::threadblock::Sm80EVT<ComputeBScale, accSrc, bScaleSrc>;

  using ComputeAScale = cutlass::epilogue::threadblock::
      VisitorCompute<cutlass::multiplies, ElementC, ElementComputeEpilogue, cutlass::FloatRoundStyle::round_to_nearest>;
  using aScaleSrc = cutlass::epilogue::threadblock::
      VisitorColBroadcast<OutputTileThreadMap, ElementComputeEpilogue, Stride<_1, _0, _0>>;
  using EpilogueAScale = cutlass::epilogue::threadblock::Sm80EVT<ComputeAScale, EpilogueBScale, aScaleSrc>;

  // With bias
  using biasSrc =
      cutlass::epilogue::threadblock::VisitorRowBroadcast<OutputTileThreadMap, ElementOutput, Stride<_0, _1, _0>>;
  using ComputeAScaleWithBias = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::multiply_add,
      ElementC,
      ElementComputeEpilogue,
      cutlass::FloatRoundStyle::round_to_nearest>;
  using EpilogueAScaleWithBias =
      cutlass::epilogue::threadblock::Sm80EVT<ComputeAScaleWithBias, EpilogueBScale, aScaleSrc, biasSrc>;

  using dTar = cutlass::epilogue::threadblock::VisitorAuxStore<
      OutputTileThreadMap,
      ElementC,
      cutlass::FloatRoundStyle::round_to_nearest,
      Stride<int64_t, _1, _0>>;
  using EpilogueStore = typename cutlass::platform::conditional<
      WithBias,
      cutlass::epilogue::threadblock::Sm80EVT<dTar, EpilogueAScaleWithBias>,
      cutlass::epilogue::threadblock::Sm80EVT<dTar, EpilogueAScale>>::type;

  using EpilogueOp = EpilogueStore;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmWithVisitor<
      ElementA,
      LayoutA,
      cutlass::ComplexTransform::kNone,
      AlignmentA,
      ElementB,
      LayoutB,
      cutlass::ComplexTransform::kNone,
      AlignmentB,
      ElementC,
      LayoutC,
      AlignmentC,
      ElementAccumulator,
      ElementComputeEpilogue,
      OperatorClass,
      ArchTag,
      CtaShape,
      WarpShape,
      InstructionShape,
      EpilogueOp,
      ThreadblockSwizzle,
      Stages,
      FP8MathOperator,
      EVTEpilogueStages>::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
};

template <typename Gemm, bool WithBias>
typename Gemm::Arguments prepare_sm89_fp8_args(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  using ElementT = typename Gemm::ElementA;
  using ElementOutput = typename Gemm::ElementD;
  using ElementComputeEpilogue = float;

  int32_t m = a.size(0);
  int32_t n = b.size(1);
  int32_t k = a.size(1);

  int64_t lda = a.stride(0);
  int64_t ldb = b.stride(1);
  int64_t ldc = out.stride(0);

  ElementT const* ptr_a = reinterpret_cast<ElementT const*>(a.data_ptr());
  ElementT const* ptr_b = reinterpret_cast<ElementT const*>(b.data_ptr());
  ElementOutput const* ptr_bias = nullptr;
  if constexpr (WithBias) {
    TORCH_CHECK(bias.has_value())
    ptr_bias = reinterpret_cast<ElementOutput const*>(bias.value().data_ptr());
  }
  ElementOutput* ptr_d = reinterpret_cast<ElementOutput*>(out.data_ptr());
  ElementComputeEpilogue const* ptr_scales_a = reinterpret_cast<ElementComputeEpilogue const*>(scales_a.data_ptr());
  ElementComputeEpilogue const* ptr_scales_b = reinterpret_cast<ElementComputeEpilogue const*>(scales_b.data_ptr());

  typename Gemm::Arguments args(
      cutlass::gemm::GemmUniversalMode::kGemm,  // Mode
      {m, n, k},                                // Problem size
      1,                                        // Split-k factor
      {},                                       // Epilogue args
      ptr_a,                                    // a pointer
      ptr_b,                                    // b pointer
      nullptr,                                  // c pointer (unused)
      nullptr,                                  // d pointer (unused)
      m * k,                                    // batch stride a (unused)
      n * k,                                    // batch stride b (unused)
      m * n,                                    // batch stride c (unused)
      m * n,                                    // batch stride d (unused)
      lda,                                      // stride a
      ldb,                                      // stride b
      ldc,                                      // stride c (unused)
      ldc);                                     // stride d (unused)
  if constexpr (WithBias) {
    args.epilogue = {
        {
            {
                {},  // Accumulator
                {ptr_scales_b, ElementComputeEpilogue(0), {_0{}, _1{}, _0{}}},
                {}  // Multiplies
            },
            {ptr_scales_a, ElementComputeEpilogue(0), {_1{}, _0{}, _0{}}},
            {ptr_bias, ElementOutput(0), {_0{}, _1{}, _0{}}},
            {}  // Multiplies
        },
        {ptr_d, {n, _1{}, _0{}}}};
  } else {
    args.epilogue = {
        {
            {
                {},  // Accumulator
                {ptr_scales_b, ElementComputeEpilogue(0), {_0{}, _1{}, _0{}}},
                {}  // Multiplies
            },
            {ptr_scales_a, ElementComputeEpilogue(0), {_1{}, _0{}, _0{}}},
            {}  // Multiplies
        },
        {ptr_d, {n, _1{}, _0{}}}};
  }

  return args;
}

template <typename Gemm, bool WithBias>
void launch_sm89_fp8_scaled_mm(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  auto args = prepare_sm89_fp8_args<Gemm, WithBias>(out, a, b, scales_a, scales_b, bias);
  Gemm gemm_op;

  size_t workspace_size = gemm_op.get_workspace_size(args);
  auto const workspace_options = torch::TensorOptions().dtype(torch::kUInt8).device(a.device());
  auto workspace = torch::empty(workspace_size, workspace_options);
  auto stream = at::cuda::getCurrentCUDAStream(a.get_device());

  auto can_implement = gemm_op.can_implement(args);
  TORCH_CHECK(can_implement == cutlass::Status::kSuccess)

  auto status = gemm_op(args, workspace.data_ptr(), stream);
  TORCH_CHECK(status == cutlass::Status::kSuccess)
}

template <typename OutType, typename CtaShape, typename WarpShape, int Stages>
void sm89_fp8_dispatch_bias(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  using ElementInput = cutlass::float_e4m3_t;
  using ElementOutput = OutType;
  using AccumElementType = float;
  if (bias) {
    using Gemm = typename DeviceGemmFp8RowwiseSm89<
        ElementInput,
        ElementOutput,
        AccumElementType,
        CtaShape,
        WarpShape,
        Stages,
        true>::Gemm;
    return launch_sm89_fp8_scaled_mm<Gemm, true>(out, a, b, scales_a, scales_b, bias);
  } else {
    using Gemm = typename DeviceGemmFp8RowwiseSm89<
        ElementInput,
        ElementOutput,
        AccumElementType,
        CtaShape,
        WarpShape,
        Stages,
        false>::Gemm;
    return launch_sm89_fp8_scaled_mm<Gemm, false>(out, a, b, scales_a, scales_b, bias);
  }
}

template <typename OutType>
void sm89_fp8_dispatch_shape(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  uint32_t const m = a.size(0);
  uint32_t const n = out.size(1);

  if (m == 1) {
    if (n <= 8192) {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<16, 64, 128>,
          cutlass::gemm::GemmShape<16, 64, 64>,
          7>(out, a, b, scales_a, scales_b, bias);
    } else {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<32, 64, 128>,
          cutlass::gemm::GemmShape<16, 64, 64>,
          5>(out, a, b, scales_a, scales_b, bias);
    }
  } else if (m <= 16) {
    // M in (1, 16]
    if (n <= 8192) {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<16, 64, 128>,
          cutlass::gemm::GemmShape<16, 64, 64>,
          4>(out, a, b, scales_a, scales_b, bias);
    } else if (n <= 16384) {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<32, 64, 128>,
          cutlass::gemm::GemmShape<16, 64, 64>,
          5>(out, a, b, scales_a, scales_b, bias);
    } else {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<16, 64, 128>,
          cutlass::gemm::GemmShape<16, 64, 64>,
          7>(out, a, b, scales_a, scales_b, bias);
    }
  } else if (m <= 64) {
    // M in (16, 64]
    if (n <= 16384) {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<32, 64, 128>,
          cutlass::gemm::GemmShape<16, 64, 64>,
          7>(out, a, b, scales_a, scales_b, bias);
    } else {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<16, 64, 128>,
          cutlass::gemm::GemmShape<16, 64, 64>,
          7>(out, a, b, scales_a, scales_b, bias);
    }
  } else if (m <= 128) {
    // M in (64, 128]
    if (n <= 8192) {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<64, 64, 128>,
          cutlass::gemm::GemmShape<32, 64, 64>,
          4>(out, a, b, scales_a, scales_b, bias);
    } else if (n <= 16384) {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<64, 64, 128>,
          cutlass::gemm::GemmShape<32, 64, 64>,
          5>(out, a, b, scales_a, scales_b, bias);
    } else {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<32, 64, 128>,
          cutlass::gemm::GemmShape<16, 64, 64>,
          5>(out, a, b, scales_a, scales_b, bias);
    }
  } else if (m <= 256) {
    // M in (128, 256]
    if (n <= 8192) {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<128, 64, 64>,
          cutlass::gemm::GemmShape<64, 32, 64>,
          5>(out, a, b, scales_a, scales_b, bias);
    } else if (n <= 16384) {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<64, 128, 64>,
          cutlass::gemm::GemmShape<64, 32, 64>,
          7>(out, a, b, scales_a, scales_b, bias);
    } else {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<128, 64, 128>,
          cutlass::gemm::GemmShape<64, 32, 128>,
          4>(out, a, b, scales_a, scales_b, bias);
    }
  } else if (m <= 512) {
    // M in (256, 512)
    if (n <= 16384) {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<128, 128, 64>,
          cutlass::gemm::GemmShape<64, 32, 64>,
          2>(out, a, b, scales_a, scales_b, bias);
    } else {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<128, 128, 64>,
          cutlass::gemm::GemmShape<64, 32, 64>,
          4>(out, a, b, scales_a, scales_b, bias);
    }
  } else {
    // M in (512, inf)
    if (n <= 8192) {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<128, 128, 64>,
          cutlass::gemm::GemmShape<64, 32, 64>,
          3>(out, a, b, scales_a, scales_b, bias);
    } else {
      return sm89_fp8_dispatch_bias<
          OutType,
          cutlass::gemm::GemmShape<128, 128, 64>,
          cutlass::gemm::GemmShape<64, 32, 64>,
          2>(out, a, b, scales_a, scales_b, bias);
    }
  }
}
#endif

#if defined CUDA_VERSION && CUDA_VERSION >= 12000
template <
    typename ElementType,
    typename OutElementType,
    typename AccumElementType,
    typename CTAShape,
    typename ClusterShape,
    typename MainloopScheduleType,
    typename EpilogueScheduleType,
    typename TileSchedulerType = void,
    bool WithBias = false>
struct DeviceGemmFp8RowwiseSm90 {
  static_assert(std::is_same_v<ElementType, cutlass::float_e4m3_t>, "ElementType must be FP8(e4m3)");

  // A matrix configuration
  using ElementA = ElementType;               // Element type for A matrix operand
  using LayoutA = cutlass::layout::RowMajor;  // Layout type for A matrix operand
  static constexpr int AlignmentA =
      128 / cutlass::sizeof_bits<ElementA>::value;  // Memory access granularity/alignment of A
                                                    // matrix in units of elements (up to 16 bytes)

  // B matrix configuration
  using ElementB = ElementType;                  // Element type for B matrix operand
  using LayoutB = cutlass::layout::ColumnMajor;  // Layout type for B matrix operand
  static constexpr int AlignmentB =
      128 / cutlass::sizeof_bits<ElementB>::value;  // Memory access granularity/alignment of B
                                                    // matrix in units of elements (up to 16 bytes)

  // C/D matrix configuration
  using ElementC = void;                      // Element type for C matrix operands
  using LayoutC = cutlass::layout::RowMajor;  // Layout type for C matrix operands
  static constexpr int AlignmentC =
      128 / cutlass::sizeof_bits<OutElementType>::value;  // Memory access granularity/alignment of C matrices in
                                                          // units of elements (up to 16 bytes)

  // Output matrix configuration
  using ElementOutput = OutElementType;            // Element type for output matrix operands
  using LayoutOutput = cutlass::layout::RowMajor;  // Layout type for output matrix operands
  static constexpr int AlignmentOutput = 128 / cutlass::sizeof_bits<ElementOutput>::value;

  // // Auxiliary matrix configuration and other fusion types
  // using ElementBias = float;

  // Multiply-accumulate blocking/pipelining details
  using ElementAccumulator = AccumElementType;  // Element type for internal accumulation
  using ElementCompute = float;                 // Element type for compute
  using ElementComputeEpilogue = float;
  using ArchTag = cutlass::arch::Sm90;  // Tag indicating the minimum SM that supports the intended feature
  using OperatorClass = cutlass::arch::OpClassTensorOp;  // Operator class tag
  using TileShape = CTAShape;                            // Threadblock-level tile size

  static constexpr bool PONG = false;
  static constexpr bool FAST_ACCUM = true;
  static constexpr bool USE_BIAS = false;

  using StageCountType = cutlass::gemm::collective::StageCountAuto;      // Stage count maximized
                                                                         // based on the tile size
  using KernelSchedule = cutlass::gemm::collective::KernelScheduleAuto;  // Kernel to launch based on the default
                                                                         // setting in the Collective Builder
  // Implement rowwise scaling epilogue.
  using XScale = cutlass::epilogue::fusion::Sm90ColBroadcast<
      0,
      TileShape,
      ElementComputeEpilogue,
      ElementComputeEpilogue,
      cute::Stride<cute::Int<1>, cute::Int<0>, cute::Int<0>>>;

  using WScale = cutlass::epilogue::fusion::Sm90RowBroadcast<
      0,
      TileShape,
      ElementComputeEpilogue,
      ElementComputeEpilogue,
      cute::Stride<cute::Int<0>, cute::Int<1>, cute::Int<0>>>;

  using Bias = cutlass::epilogue::fusion::Sm90RowBroadcast<
      0,
      TileShape,
      ElementOutput,
      ElementOutput,
      cute::Stride<cute::Int<0>, cute::Int<1>, cute::Int<0>>>;

  using Accum = cutlass::epilogue::fusion::Sm90AccFetch;

  using Compute0 = cutlass::epilogue::fusion::Sm90Compute<
      cutlass::multiplies,
      ElementComputeEpilogue,  // First stage output type.
      ElementComputeEpilogue,  // First stage input types.
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute0 = cutlass::epilogue::fusion::Sm90EVT<Compute0, WScale, Accum>;

  using Compute1 = cutlass::epilogue::fusion::Sm90Compute<
      cutlass::multiplies,
      ElementOutput,
      ElementComputeEpilogue,  // Second stage input types.
      cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute1 = cutlass::epilogue::fusion::Sm90EVT<Compute1, XScale, EVTCompute0>;

  // With bias
  using ComputeWithBias = cutlass::epilogue::fusion::Sm90Compute<
      cutlass::multiply_add,
      ElementOutput,
      ElementComputeEpilogue,
      cutlass::FloatRoundStyle::round_to_nearest>;
  using EVTComputeWithBias = cutlass::epilogue::fusion::Sm90EVT<ComputeWithBias, XScale, EVTCompute0, Bias>;

  using EpilogueEVT = typename cutlass::platform::conditional<WithBias, EVTComputeWithBias, EVTCompute1>::type;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90,
      cutlass::arch::OpClassTensorOp,
      TileShape,
      ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator,
      ElementComputeEpilogue,
      ElementC,
      LayoutC,
      AlignmentC,
      ElementOutput,
      LayoutOutput,
      AlignmentOutput,
      cutlass::epilogue::TmaWarpSpecialized,
      EpilogueEVT>::CollectiveOp;

  using DefaultSchedule = cutlass::gemm::KernelTmaWarpSpecialized;
  using PongSchedule = cutlass::gemm::KernelTmaWarpSpecializedPingpong;
  using FastDefaultSchedule = cutlass::gemm::KernelTmaWarpSpecializedFP8FastAccum;
  using FastPongSchedule = cutlass::gemm::KernelTmaWarpSpecializedPingpongFP8FastAccum;

  using SlowAccum = DefaultSchedule;
  using FastAccum = FastPongSchedule;  // Default apply Pingpong

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      ArchTag,
      OperatorClass,
      ElementA,
      LayoutA,
      AlignmentA,
      ElementB,
      LayoutB,
      AlignmentB,
      ElementAccumulator,
      TileShape,
      ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
          sizeof(typename CollectiveEpilogue::SharedStorage))>,
      MainloopScheduleType>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>,  // Indicates ProblemShape
      CollectiveMainloop,
      CollectiveEpilogue,
      TileSchedulerType>;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
};

template <typename Gemm, bool WithBias>
typename Gemm::Arguments prepare_sm90_fp8_args(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  using ElementT = typename Gemm::ElementA;
  using ElementOutput = typename Gemm::ElementD;
  using ElementComputeEpilogue = float;
  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using StrideD = typename Gemm::GemmKernel::StrideD;

  int32_t m = a.size(0);
  int32_t n = b.size(1);
  int32_t k = a.size(1);
  ElementT const* ptr_a = reinterpret_cast<ElementT const*>(a.data_ptr());
  ElementT const* ptr_b = reinterpret_cast<ElementT const*>(b.data_ptr());
  ElementOutput const* ptr_bias = nullptr;
  if constexpr (WithBias) {
    TORCH_CHECK(bias.has_value())
    ptr_bias = reinterpret_cast<ElementOutput const*>(bias.value().data_ptr());
  }
  ElementOutput* ptr_d = reinterpret_cast<ElementOutput*>(out.data_ptr());
  ElementComputeEpilogue const* ptr_scales_a = reinterpret_cast<ElementComputeEpilogue const*>(scales_a.data_ptr());
  ElementComputeEpilogue const* ptr_scales_b = reinterpret_cast<ElementComputeEpilogue const*>(scales_b.data_ptr());

  StrideA stride_a = cutlass::make_cute_packed_stride(StrideA{}, make_shape(m, k, 1));
  StrideB stride_b = cutlass::make_cute_packed_stride(StrideB{}, make_shape(n, k, 1));
  StrideC stride_c;
  StrideD stride_d = cutlass::make_cute_packed_stride(StrideD{}, make_shape(m, n, 1));
  typename Gemm::Arguments args = {
      cutlass::gemm::GemmUniversalMode::kGemm,
      {m, n, k, 1},
      {ptr_a, stride_a, ptr_b, stride_b},
      {{},  // epilogue.thread
       nullptr,
       stride_c,
       ptr_d,
       stride_d}};
  if constexpr (WithBias) {
    args.epilogue.thread = {
        {ptr_scales_a},
        {
            {ptr_scales_b},
            {},  // Accumulator
            {}   // Multiplies
        },
        {ptr_bias},
        {},  // Multiplies
    };
  } else {
    args.epilogue.thread = {
        {ptr_scales_a},
        {
            {ptr_scales_b},
            {},  // Accumulator
            {}   // Multiplies
        },
        {},  // Multiplies
    };
  }

  return args;
}

template <typename Gemm, bool WithBias>
void launch_sm90_fp8_scaled_mm(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  auto args = prepare_sm90_fp8_args<Gemm, WithBias>(out, a, b, scales_a, scales_b, bias);
  Gemm gemm_op;

  size_t workspace_size = gemm_op.get_workspace_size(args);
  auto const workspace_options = torch::TensorOptions().dtype(torch::kUInt8).device(a.device());
  auto workspace = torch::empty(workspace_size, workspace_options);
  auto stream = at::cuda::getCurrentCUDAStream(a.get_device());

  auto can_implement = gemm_op.can_implement(args);
  TORCH_CHECK(can_implement == cutlass::Status::kSuccess)

  auto status = gemm_op.run(args, workspace.data_ptr(), stream);

  TORCH_CHECK(status == cutlass::Status::kSuccess)
}

template <
    typename OutType,
    typename CTAShape,
    typename ClusterShape,
    typename MainloopScheduleType,
    typename TileSchedulerType>
void sm90_fp8_dispatch_bias(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias,
    bool fast_accum = true,
    bool use_persistent = false) {
  using ElementInput = cutlass::float_e4m3_t;
  using ElementOutput = OutType;
  using AccumElementType = float;
  using EpilogueScheduleType = cutlass::epilogue::TmaWarpSpecialized;

  if (bias) {
    using Gemm = typename DeviceGemmFp8RowwiseSm90<
        ElementInput,
        ElementOutput,
        AccumElementType,
        CTAShape,
        ClusterShape,
        MainloopScheduleType,
        EpilogueScheduleType,
        TileSchedulerType,
        true>::Gemm;
    return launch_sm90_fp8_scaled_mm<Gemm, true>(out, a, b, scales_a, scales_b, bias);
  } else {
    using Gemm = typename DeviceGemmFp8RowwiseSm90<
        ElementInput,
        ElementOutput,
        AccumElementType,
        CTAShape,
        ClusterShape,
        MainloopScheduleType,
        EpilogueScheduleType,
        TileSchedulerType,
        false>::Gemm;
    return launch_sm90_fp8_scaled_mm<Gemm, false>(out, a, b, scales_a, scales_b, bias);
  }
}

template <typename OutType>
void sm90_fp8_dispatch_shape(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  uint32_t const m = a.size(0);
  using FastPingpongScheduler = cutlass::gemm::KernelTmaWarpSpecializedPingpongFP8FastAccum;
  using FastBasicScheduler = cutlass::gemm::KernelTmaWarpSpecializedFP8FastAccum;
  using PersistentTileScheduler = cutlass::gemm::PersistentScheduler;
  using BasicTileScheduler = void;
  if (m <= 1) {
    return sm90_fp8_dispatch_bias<
        OutType,
        Shape<_64, _64, _128>,
        Shape<_1, _8, _1>,
        FastBasicScheduler,
        BasicTileScheduler>(out, a, b, scales_a, scales_b, bias);
  }
  if (m <= 64) {
    // m in [1, 64]
    return sm90_fp8_dispatch_bias<
        OutType,
        Shape<_64, _64, _128>,
        Shape<_1, _4, _1>,
        FastPingpongScheduler,
        PersistentTileScheduler>(out, a, b, scales_a, scales_b, bias);
  } else if (m <= 256) {
    // m in (64, 256]
    return sm90_fp8_dispatch_bias<
        OutType,
        Shape<_64, _64, _128>,
        Shape<_1, _1, _1>,
        FastPingpongScheduler,
        PersistentTileScheduler>(out, a, b, scales_a, scales_b, bias);
  } else if (m <= 1024) {
    // m in (256, 1024]
    return sm90_fp8_dispatch_bias<
        OutType,
        Shape<_128, _128, _128>,
        Shape<_1, _1, _1>,
        FastPingpongScheduler,
        PersistentTileScheduler>(out, a, b, scales_a, scales_b, bias);
  } else {
    // m in (1024, inf)
    return sm90_fp8_dispatch_bias<
        OutType,
        Shape<_128, _128, _128>,
        Shape<_2, _1, _1>,
        FastPingpongScheduler,
        PersistentTileScheduler>(out, a, b, scales_a, scales_b, bias);
  }
}
#endif

#if defined CUDA_VERSION && CUDA_VERSION >= 12080
template <
    typename ElementType,
    typename OutElementType,
    typename AccumElementType,
    typename CTAShape,
    typename ClusterShape,
    typename MainloopScheduleType,
    typename EpilogueScheduleType,
    typename TileSchedulerType = void,
    bool WithBias = false>
struct DeviceGemmFp8RowwiseSm100 {
  static_assert(std::is_same_v<ElementType, cutlass::float_e4m3_t>, "ElementType must be FP8(e4m3)");
  using TileShape = CTAShape;
  using Accum = cutlass::epilogue::fusion::Sm90AccFetch;

  using ElementComputeEpilogue = float;
  using ScaleA = cutlass::epilogue::fusion::Sm90ColBroadcast<
      0,
      TileShape,
      ElementComputeEpilogue,
      ElementComputeEpilogue,
      cute::Stride<cute::Int<1>, cute::Int<0>, cute::Int<0>>>;

  using ScaleB = cutlass::epilogue::fusion::Sm90RowBroadcast<
      0,
      TileShape,
      ElementComputeEpilogue,
      ElementComputeEpilogue,
      cute::Stride<cute::Int<0>, cute::Int<1>, cute::Int<0>>>;

  using Bias = cutlass::epilogue::fusion::Sm90RowBroadcast<
      0,
      TileShape,
      OutElementType,
      OutElementType,
      cute::Stride<cute::Int<0>, cute::Int<1>, cute::Int<0>>>;

  using Compute0 = cutlass::epilogue::fusion::
      Sm90Compute<cutlass::multiplies, float, float, cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute0 = cutlass::epilogue::fusion::Sm90EVT<Compute0, ScaleB, Accum>;

  using LayoutA = cutlass::layout::RowMajor;
  static constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementType>::value;

  using LayoutB = cutlass::layout::ColumnMajor;
  static constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementType>::value;

  using ElementC = void;
  using LayoutC = cutlass::layout::RowMajor;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<OutElementType>::value;

  using LayoutD = cutlass::layout::RowMajor;
  static constexpr int AlignmentD = AlignmentC;

  using Compute1MulAdd = cutlass::epilogue::fusion::
      Sm90Compute<cutlass::multiply_add, OutElementType, float, cutlass::FloatRoundStyle::round_to_nearest>;
  using Compute1Mul = cutlass::epilogue::fusion::
      Sm90Compute<cutlass::multiplies, OutElementType, float, cutlass::FloatRoundStyle::round_to_nearest>;

  using EVTCompute = typename std::conditional_t<
      WithBias,
      cutlass::epilogue::fusion::Sm90EVT<Compute1MulAdd, ScaleA, EVTCompute0, Bias>,
      cutlass::epilogue::fusion::Sm90EVT<Compute1Mul, ScaleA, EVTCompute0>>;
  using ArgumentType = typename EVTCompute::Arguments;
  // MMA type
  using ElementAccumulator = AccumElementType;

  // Epilogue types
  using ElementCompute = float;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100,
      cutlass::arch::OpClassTensorOp,
      TileShape,
      ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator,
      ElementCompute,
      ElementC,
      LayoutC,
      AlignmentC,
      OutElementType,
      LayoutD,
      AlignmentD,
      EpilogueScheduleType,
      EVTCompute>::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100,
      cutlass::arch::OpClassTensorOp,
      ElementType,
      LayoutA,
      AlignmentA,
      ElementType,
      LayoutB,
      AlignmentB,
      ElementAccumulator,
      TileShape,
      ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
          sizeof(typename CollectiveEpilogue::SharedStorage))>,
      MainloopScheduleType>::CollectiveOp;
  using GemmKernel =
      cutlass::gemm::kernel::GemmUniversal<Shape<int, int, int, int>, CollectiveMainloop, CollectiveEpilogue, void>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  template <typename Descriptor, typename T>
  static auto args_from_tensor(torch::Tensor const& tensor) {
    using Arguments = typename Descriptor::Arguments;
    auto* data_ptr = static_cast<T*>(tensor.data_ptr());
    static_assert(
        std::is_same_v<Descriptor, ScaleA> || std::is_same_v<Descriptor, ScaleB> || std::is_same_v<Descriptor, Bias>);
    return Arguments{data_ptr};
  }

 public:
  static ArgumentType prepare_args(
      torch::Tensor const& a_scales,
      torch::Tensor const& b_scales,
      std::optional<torch::Tensor> const& bias = std::nullopt) {
    auto a_args = args_from_tensor<ScaleA, float>(a_scales);
    auto b_args = args_from_tensor<ScaleB, float>(b_scales);

    typename EVTCompute0::Arguments evt0_args{b_args, {}, {}};

    if constexpr (WithBias) {
      auto bias_args = args_from_tensor<Bias, OutElementType>(bias.value());
      return ArgumentType{a_args, evt0_args, bias_args, {}};
    } else {
      return ArgumentType{a_args, evt0_args, {}};
    }
  }
};

template <typename GemmType, bool WithBias>
typename GemmType::Gemm::Arguments prepare_sm100_fp8_args(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  using Gemm = typename GemmType::Gemm;
  using ElementT = typename Gemm::ElementA;
  using ElementC = typename Gemm::ElementC;
  using ElementOutput = typename Gemm::ElementD;
  using ElementComputeEpilogue = float;
  using GemmKernel = typename Gemm::GemmKernel;

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using StrideD = StrideC;
  using StrideAux = StrideC;

  int32_t m = a.size(0);
  int32_t n = b.size(1);
  int32_t k = a.size(1);

  ElementT const* ptr_a = reinterpret_cast<ElementT const*>(a.data_ptr());
  ElementT const* ptr_b = reinterpret_cast<ElementT const*>(b.data_ptr());

  StrideA stride_a = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(m, k, 1));
  StrideB stride_b = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(n, k, 1));
  StrideC stride_c = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(m, n, 1));
  StrideD stride_d = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(m, n, 1));
  StrideAux aux_stride = stride_d;

  typename GemmKernel::MainloopArguments mainloop_args{ptr_a, stride_a, ptr_b, stride_b};

  typename GemmKernel::ProblemShape prob_shape = {m, n, k, 1};
  cutlass::KernelHardwareInfo hw_info;
  typename GemmKernel::TileSchedulerArguments scheduler = {};

  auto ptr_c = static_cast<ElementOutput*>(out.data_ptr());

  auto prepare_epilogue_args = [&](const c10::optional<torch::Tensor>& bias = c10::nullopt) {
    if constexpr (WithBias) {
      TORCH_CHECK(bias.has_value(), "Bias tensor is required but not provided.");
      return typename GemmKernel::EpilogueArguments{
          GemmType::prepare_args(scales_a, scales_b, bias.value()), ptr_c, stride_c, ptr_c, stride_d};
    } else {
      return typename GemmKernel::EpilogueArguments{
          GemmType::prepare_args(scales_a, scales_b), ptr_c, stride_c, ptr_c, stride_d};
    }
  };

  typename GemmKernel::Arguments args{
      cutlass::gemm::GemmUniversalMode::kGemm,
      prob_shape,
      mainloop_args,
      prepare_epilogue_args(bias),
      hw_info,
      scheduler};
  return args;
}

template <typename Gemm, bool WithBias>
void launch_sm100_fp8_scaled_mm(
    torch::Tensor& out,
    torch::Tensor const& a,
    torch::Tensor const& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  auto args = prepare_sm100_fp8_args<Gemm, WithBias>(out, a, b, scales_a, scales_b, bias);

  typename Gemm::Gemm gemm_op;
  size_t workspace_size = gemm_op.get_workspace_size(args);
  auto const workspace_options = torch::TensorOptions().dtype(torch::kUInt8).device(a.device());
  auto workspace = torch::empty(workspace_size, workspace_options);
  auto stream = at::cuda::getCurrentCUDAStream(a.get_device());
  auto can_implement = gemm_op.can_implement(args);
  TORCH_CHECK(can_implement == cutlass::Status::kSuccess)
  auto status = gemm_op.run(args, workspace.data_ptr(), stream);
  TORCH_CHECK(status == cutlass::Status::kSuccess)
}

template <typename OutType>
void sm100_fp8_dispatch_bias(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  using CTAShape = Shape<_256, _128, _64>;
  using ClusterShape = Shape<_2, _2, _1>;
  using MainloopScheduleType = cutlass::gemm::collective::KernelScheduleAuto;
  using EpilogueScheduleType = cutlass::epilogue::collective::EpilogueScheduleAuto;
  using TileSchedulerType = void;

  using ElementInput = cutlass::float_e4m3_t;
  using ElementOutput = OutType;
  using AccumElementType = float;

  if (bias) {
    using Gemm = DeviceGemmFp8RowwiseSm100<
        ElementInput,
        ElementOutput,
        AccumElementType,
        CTAShape,
        ClusterShape,
        MainloopScheduleType,
        EpilogueScheduleType,
        TileSchedulerType,
        true>;
    return launch_sm100_fp8_scaled_mm<Gemm, true>(out, a, b, scales_a, scales_b, bias);
  } else {
    using Gemm = DeviceGemmFp8RowwiseSm100<
        ElementInput,
        ElementOutput,
        AccumElementType,
        CTAShape,
        ClusterShape,
        MainloopScheduleType,
        EpilogueScheduleType,
        TileSchedulerType,
        false>;
    return launch_sm100_fp8_scaled_mm<Gemm, false>(out, a, b, scales_a, scales_b, bias);
  }
}

template <typename OutType>
void sm100_fp8_dispatch_shape(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const c10::optional<torch::Tensor>& bias) {
  return sm100_fp8_dispatch_bias<OutType>(out, a, b, scales_a, scales_b, bias);
}
#endif

torch::Tensor fp8_scaled_mm(
    const torch::Tensor& mat_a,
    const torch::Tensor& mat_b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const torch::Dtype& out_dtype,
    const c10::optional<torch::Tensor>& bias) {
  TORCH_CHECK(mat_a.is_cuda(), "mat_a must be a CUDA tensor");
  TORCH_CHECK(mat_b.is_cuda(), "mat_b must be a CUDA tensor");
  TORCH_CHECK(mat_a.dim() == 2, "mat_a must be a 2D tensor");
  TORCH_CHECK(mat_b.dim() == 2, "mat_b must be a 2D tensor");
  TORCH_CHECK(mat_a.stride(1) == 1, "mat_a must be a row major tensor");
  TORCH_CHECK(mat_b.stride(0) == 1, "mat_a must be a column major tensor");
  TORCH_CHECK(mat_a.size(1) == mat_b.size(0), "mat_a and mat_b shapes cannot be multiplied");

  TORCH_CHECK(
      (mat_a.size(1) * mat_a.element_size()) % 16 == 0, "mat_a must be multiple of 16 bytes for memory alignment");
  TORCH_CHECK(
      (mat_b.size(0) * mat_b.element_size()) % 16 == 0, "mat_b must be multiple of 16 bytes for memory alignment");
  TORCH_CHECK(mat_a.scalar_type() == torch::kFloat8_e4m3fn, "mat_a must be Float8_e4m3fn");
  TORCH_CHECK(mat_b.scalar_type() == torch::kFloat8_e4m3fn, "mat_b must be Float8_e4m3fn");
  TORCH_CHECK(out_dtype == torch::kHalf || out_dtype == torch::kBFloat16, "out_dtype must be Half or BFloat16");

  TORCH_CHECK(scales_a.numel() == mat_a.size(0), "size of scales_a is not matched");
  TORCH_CHECK(scales_b.numel() == mat_b.size(1), "size of scales_b is not matched");
  TORCH_CHECK(scales_a.is_contiguous(), "scales_a must be contiguous");
  TORCH_CHECK(scales_b.is_contiguous(), "scales_b msut be contiguous");
  TORCH_CHECK(scales_a.scalar_type() == torch::kFloat32, "scales_a must be Float32");
  TORCH_CHECK(scales_b.scalar_type() == torch::kFloat32, "scales_b must be Float32");

  if (bias) {
    TORCH_CHECK(bias->numel() == mat_b.size(1), "size of bias is not matched");
    TORCH_CHECK(bias->is_contiguous(), "bias must be contiguous");
    TORCH_CHECK(bias->dtype() == out_dtype, "bias dtype must match output dtype");
  }

  torch::Tensor out = torch::empty({mat_a.size(0), mat_b.size(1)}, mat_a.options().dtype(out_dtype));
  TORCH_CHECK((out.size(1) * out.element_size()) % 16 == 0, "out must be multiple of 16 bytes for memory alignment");

  auto sm_version = getSMVersion();

#if defined CUDA_VERSION && CUDA_VERSION >= 12080
  if (sm_version >= 100) {
    if (out_dtype == torch::kBFloat16) {
      sm100_fp8_dispatch_shape<cutlass::bfloat16_t>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      sm100_fp8_dispatch_shape<cutlass::half_t>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
    return out;
  }
#endif

#if defined CUDA_VERSION && CUDA_VERSION >= 12000
  if (sm_version >= 90) {
    if (out_dtype == torch::kBFloat16) {
      sm90_fp8_dispatch_shape<cutlass::bfloat16_t>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      sm90_fp8_dispatch_shape<cutlass::half_t>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
    return out;
  }
#endif

#if defined CUDA_VERSION && CUDA_VERSION >= 12040
  if (sm_version == 89) {
    if (out_dtype == torch::kBFloat16) {
      sm89_fp8_dispatch_shape<cutlass::bfloat16_t>(out, mat_a, mat_b, scales_a, scales_b, bias);
    } else {
      sm89_fp8_dispatch_shape<cutlass::half_t>(out, mat_a, mat_b, scales_a, scales_b, bias);
    }
    return out;
  }
#endif

  TORCH_CHECK_NOT_IMPLEMENTED(false, "No implemented fp8_scaled_mm for current compute capability: ", sm_version);
}
