#include "hip/hip_runtime.h"
// Adapted from
// https://github.com/vllm-project/vllm/blob/eb59b5a6cba6727d3727c0372258db9002f687c1/csrc/quantization/awq/gemm_kernels.cu#L350
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_fp16.h>
#include <torch/all.h>
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
#include <hip/hip_bf16.h>
#endif

template <int lut>
__device__ inline int lop3(int a, int b, int c) {
  int res;
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n" : "=r"(res) : "r"(a), "r"(b), "r"(c), "n"(lut));
  return res;
}

__device__ uint4 dequantize_s4_to_fp16x2(uint32_t const& source) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 750
  uint4 result;

  uint32_t* h = reinterpret_cast<uint32_t*>(&result);
  uint32_t const i4s = reinterpret_cast<uint32_t const&>(source);

  // First, we extract the i4s and construct an intermediate fp16 number.
  static constexpr uint32_t immLut = (0xf0 & 0xcc) | 0xaa;
  static constexpr uint32_t BOTTOM_MASK = 0x000f000f;
  static constexpr uint32_t TOP_MASK = 0x00f000f0;
  static constexpr uint32_t I4s_TO_F16s_MAGIC_NUM = 0x64006400;

  // Note that the entire sequence only requires 1 shift instruction. This is
  // thanks to the register packing format and the fact that we force our
  // integers to be unsigned, and account for this in the fp16 subtractions. In
  // addition, I exploit the fact that sub and fma have the same throughput in
  // order to convert elt_23 and elt_67 to fp16 without having to shift them to
  // the bottom bits before hand.

  // Shift right by 8 to now consider elt_45 and elt_67. Issue first to hide RAW
  // dependency if we issue immediately before required.
  const uint32_t top_i4s = i4s >> 8;
  // Extract elt_01 - (i4s & 0x000f000f) | 0x64006400
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[0])
               : "r"(i4s), "n"(BOTTOM_MASK), "n"(I4s_TO_F16s_MAGIC_NUM), "n"(immLut));
  // Extract elt_23 (i4s & 0x00f000f0) | 0x64006400
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[1])
               : "r"(i4s), "n"(TOP_MASK), "n"(I4s_TO_F16s_MAGIC_NUM), "n"(immLut));
  // Extract elt_45 (top_i4s & 0x000f000f) | 0x64006400
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[2])
               : "r"(top_i4s), "n"(BOTTOM_MASK), "n"(I4s_TO_F16s_MAGIC_NUM), "n"(immLut));
  // Extract elt_67 (top_i4s & 0x00f000f0) | 0x64006400
  asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
               : "=r"(h[3])
               : "r"(top_i4s), "n"(TOP_MASK), "n"(I4s_TO_F16s_MAGIC_NUM), "n"(immLut));

  // This is the half2 {1024, 1024} represented as an integer.
  static constexpr uint32_t FP16_TOP_MAGIC_NUM = 0x64006400;
  // This is the half2 {1 / 16, 1 / 16} represented as an integer.
  static constexpr uint32_t ONE_SIXTEENTH = 0x2c002c00;
  // This is the half2 {-64, -64} represented as an integer.
  static constexpr uint32_t NEG_64 = 0xd400d400;

  // Finally, we construct the output numbers.
  // Convert elt_01
  asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h[0]) : "r"(h[0]), "r"(FP16_TOP_MAGIC_NUM));
  // Convert elt_23
  asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(h[1]) : "r"(h[1]), "r"(ONE_SIXTEENTH), "r"(NEG_64));
  // Convert elt_45
  asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h[2]) : "r"(h[2]), "r"(FP16_TOP_MAGIC_NUM));
  // Convert elt_67
  asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n" : "=r"(h[3]) : "r"(h[3]), "r"(ONE_SIXTEENTH), "r"(NEG_64));

  return result;
#else
  assert(false);
  return {};
#endif
}

__device__ uint4 dequantize_s4_to_bf16x2(uint32_t const& source) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
  uint4 result;
  uint32_t* h = reinterpret_cast<uint32_t*>(&result);
  uint32_t const i4s = source;

  // Define masks and constants
  static constexpr uint32_t MASK = 0x000f000f;
  static constexpr uint32_t EX = 0x43004300;
  static constexpr uint32_t MUL = 0x3F803F80;
  static constexpr uint32_t ADD = 0xC300C300;

  int lo0 = lop3<(0xf0 & 0xcc) | 0xaa>(i4s, MASK, EX);
  int hi0 = lop3<(0xf0 & 0xcc) | 0xaa>(i4s >> 4, MASK, EX);
  int lo1 = lop3<(0xf0 & 0xcc) | 0xaa>(i4s >> 8, MASK, EX);
  int hi1 = lop3<(0xf0 & 0xcc) | 0xaa>(i4s >> 12, MASK, EX);

  nv_bfloat162* res = reinterpret_cast<nv_bfloat162*>(h);
  res[0] = __hfma2(
      *reinterpret_cast<nv_bfloat162*>(&lo0),
      *reinterpret_cast<const nv_bfloat162*>(&MUL),
      *reinterpret_cast<const nv_bfloat162*>(&ADD));
  res[1] = __hfma2(
      *reinterpret_cast<nv_bfloat162*>(&hi0),
      *reinterpret_cast<const nv_bfloat162*>(&MUL),
      *reinterpret_cast<const nv_bfloat162*>(&ADD));
  res[2] = __hfma2(
      *reinterpret_cast<nv_bfloat162*>(&lo1),
      *reinterpret_cast<const nv_bfloat162*>(&MUL),
      *reinterpret_cast<const nv_bfloat162*>(&ADD));
  res[3] = __hfma2(
      *reinterpret_cast<nv_bfloat162*>(&hi1),
      *reinterpret_cast<const nv_bfloat162*>(&MUL),
      *reinterpret_cast<const nv_bfloat162*>(&ADD));

  return result;
#else
  assert(false);
  return {};
#endif
}

template <typename OutputT>
__global__ void __launch_bounds__(256) dequantize_weights(
    int* __restrict__ qweight,
    OutputT* __restrict__ scales,
    int* __restrict__ qzeros,
    OutputT* __restrict__ output,
    int group_size,
    int qweight_cols) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  int group_idx = row / group_size;
  int scale_offset = 8 * col + group_idx * qweight_cols * 8;
  uint4 loaded_scale = *(uint4*)(scales + scale_offset);

  // Handle different data types
  if constexpr (std::is_same<OutputT, half>::value) {
    // FP16 path
    uint4 zeros = dequantize_s4_to_fp16x2(qzeros[col + group_idx * qweight_cols]);
    uint4 weight_fp16 = dequantize_s4_to_fp16x2(qweight[col + row * qweight_cols]);

    // Use PTX assembly for FP16 operations
    asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.x) : "r"(weight_fp16.x), "r"(zeros.x));
    asm volatile("mul.rn.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.x) : "r"(weight_fp16.x), "r"(loaded_scale.x));
    asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.y) : "r"(weight_fp16.y), "r"(zeros.y));
    asm volatile("mul.rn.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.y) : "r"(weight_fp16.y), "r"(loaded_scale.y));
    asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.z) : "r"(weight_fp16.z), "r"(zeros.z));
    asm volatile("mul.rn.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.z) : "r"(weight_fp16.z), "r"(loaded_scale.z));
    asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.w) : "r"(weight_fp16.w), "r"(zeros.w));
    asm volatile("mul.rn.f16x2 %0, %1, %2;\n" : "=r"(weight_fp16.w) : "r"(weight_fp16.w), "r"(loaded_scale.w));

    OutputT* output_ptr = output + 8 * col + 8 * row * qweight_cols;
    *(uint4*)output_ptr = weight_fp16;
  } else if constexpr (std::is_same<OutputT, __hip_bfloat16>::value) {
    uint4 weight_raw = dequantize_s4_to_bf16x2(qweight[col + row * qweight_cols]);
    uint4 zero_raw = dequantize_s4_to_bf16x2(qzeros[col + group_idx * qweight_cols]);
    uint4 scale_raw = *reinterpret_cast<uint4*>(scales + scale_offset);

    // Vectorized processing (each uint4 contains 4 nv_bfloat162)
    nv_bfloat162* weight_vec = reinterpret_cast<nv_bfloat162*>(&weight_raw);
    nv_bfloat162* zero_vec = reinterpret_cast<nv_bfloat162*>(&zero_raw);
    nv_bfloat162* scale_vec = reinterpret_cast<nv_bfloat162*>(&scale_raw);

// Single instruction dual-channel operation
#pragma unroll
    for (int i = 0; i < 4; ++i) {  // uint4 = 4 * nv_bfloat162
      weight_vec[i] = __hmul2(__hsub2(weight_vec[i], zero_vec[i]), scale_vec[i]);
    }

    // Directly store to OutputT array (guaranteed contiguous memory)
    OutputT* output_ptr = output + 8 * col + row * qweight_cols * 8;
    static_assert(sizeof(uint4) == 8 * sizeof(OutputT), "Memory layout mismatch");
    *reinterpret_cast<uint4*>(output_ptr) = weight_raw;
  }
}

torch::Tensor awq_dequantize(torch::Tensor qweight, torch::Tensor scales, torch::Tensor qzeros) {
  int qweight_rows = qweight.size(0);
  int qweight_cols = qweight.size(1);
  int group_size = qweight_rows / scales.size(0);

  int x_num_threads = 16;
  int y_num_threads = 16;
  int x_blocks = qweight_cols / x_num_threads;
  int y_blocks = qweight_rows / y_num_threads;

  const at::cuda::OptionalCUDAGuard device_guard(device_of(qweight));

  auto output_tensor_options = torch::TensorOptions().dtype(scales.dtype()).device(scales.device());
  at::Tensor output = torch::empty({qweight_rows, qweight_cols * 8}, output_tensor_options);

  auto _qweight = reinterpret_cast<int*>(qweight.data_ptr<int>());
  auto _zeros = reinterpret_cast<int*>(qzeros.data_ptr<int>());

  dim3 num_blocks(x_blocks, y_blocks);
  dim3 threads_per_block(x_num_threads, y_num_threads);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (scales.scalar_type() == at::ScalarType::Half) {
    auto _scales = reinterpret_cast<half*>(scales.data_ptr<at::Half>());
    auto _output = reinterpret_cast<half*>(output.data_ptr<at::Half>());
    dequantize_weights<half>
        <<<num_blocks, threads_per_block, 0, stream>>>(_qweight, _scales, _zeros, _output, group_size, qweight_cols);
  } else {
    auto _scales = reinterpret_cast<__hip_bfloat16*>(scales.data_ptr<at::BFloat16>());
    auto _output = reinterpret_cast<__hip_bfloat16*>(output.data_ptr<at::BFloat16>());
    dequantize_weights<__hip_bfloat16>
        <<<num_blocks, threads_per_block, 0, stream>>>(_qweight, _scales, _zeros, _output, group_size, qweight_cols);
  }

  return output;
}
