#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <c10/util/Float8_e4m3fn.h>

#include <cmath>
#include <flashinfer/vec_dtypes.cuh>

#include "utils.h"

template <int THREADS_PER_SUBWARP>
__device__ __forceinline__ float GroupReduceMax(float val, const int tid) {
  unsigned mask = 0xffff;

  static_assert((THREADS_PER_SUBWARP == 16) or (THREADS_PER_SUBWARP == 8));

  if constexpr (THREADS_PER_SUBWARP == 16) {
    val = fmaxf(val, __shfl_xor_sync(mask, val, 8));
  }
  val = fmaxf(val, __shfl_xor_sync(mask, val, 4));
  val = fmaxf(val, __shfl_xor_sync(mask, val, 2));
  val = fmaxf(val, __shfl_xor_sync(mask, val, 1));
  return val;
}

__device__ __forceinline__ float silu(const float& val) {
  float half = 0.5f * val;
  float t = __tanhf(half);
  return half * (1.0f + t);
}

// Copied and modified from DeepEP
__forceinline__ __device__ float fast_pow2(int x) {
  // We can ensure `-126 <= x and x <= 127`
  uint32_t bits_x = (x + 127) << 23;
  return *reinterpret_cast<float*>(&bits_x);
}

// Copied and modified from DeepEP
__forceinline__ __device__ int fast_log2_ceil(float x) {
  auto bits_x = *reinterpret_cast<uint32_t*>(&x);
  auto exp_x = (bits_x >> 23) & 0xff;
  auto man_bits = bits_x & ((1 << 23) - 1);
  return exp_x - 127 + (man_bits != 0);
}

// Copied and modified from DeepEP
template <bool ROUND_SCALE, typename dtype_info>
__forceinline__ __device__ void calculate_fp8_scales(float amax, float& scale, float& scale_inv) {
  constexpr float MAX_8BIT_INV = 1.0f / dtype_info::MAX;
  if constexpr (ROUND_SCALE) {
    auto exp_scale_inv = fast_log2_ceil(amax * MAX_8BIT_INV);
    scale = fast_pow2(-exp_scale_inv);
    scale_inv = fast_pow2(exp_scale_inv);
  } else {
    scale_inv = amax * MAX_8BIT_INV;
    scale = dtype_info::MAX / amax;
  }
}

// Copied and modified from DeepEP
template <bool SCALE_UE8M0, typename OUT_DTYPE_T = std::conditional_t<SCALE_UE8M0, uint8_t, float>>
__forceinline__ __device__ OUT_DTYPE_T extract_required_scale_format(float value) {
  if constexpr (SCALE_UE8M0) {
    return static_cast<uint8_t>((*reinterpret_cast<uint32_t*>(&value)) >> 23);
  } else {
    return value;
  }
}

__device__ __forceinline__ void st_global(const int4* ptr, const int4& value) {
  asm volatile(
      "st.global.v4.s32 [%0], {%1, %2, %3, %4};" ::"l"(ptr), "r"(value.x), "r"(value.y), "r"(value.z), "r"(value.w));
}

__device__ __forceinline__ int4 ld_global_nc(const int4* ptr) {
  int4 ret;
  asm volatile("ld.global.nc.v4.s32 {%0, %1, %2, %3}, [%4];"
               : "=r"(ret.x), "=r"(ret.y), "=r"(ret.z), "=r"(ret.w)
               : "l"(ptr));
  return ret;
}

template <typename T>
struct DtypeInfo;

template <>
struct DtypeInfo<int8_t> {
  static constexpr float MIN = -128;
  static constexpr float MAX = 127;
};

template <>
struct DtypeInfo<c10::Float8_e4m3fn> {
  static constexpr float MIN = -448;
  static constexpr float MAX = 448;
};

template <bool FUSE_SILU_AND_MUL>
__device__ __forceinline__ int compute_input_group_start_offset(
    int expert_idx,
    int token_idx,
    int hidden_dim_group_idx,
    int hidden_size,
    int num_tokens_per_expert,
    int group_size) {
  return expert_idx * num_tokens_per_expert * hidden_size * (FUSE_SILU_AND_MUL ? 2 : 1) +
         token_idx * hidden_size * (FUSE_SILU_AND_MUL ? 2 : 1) + hidden_dim_group_idx * group_size;
}

constexpr float LOCAL_ABSMAX_ABS = 1e-10;
constexpr int THREADS_PER_SUBWARP = 8;
constexpr uint32_t INPUT_PRIMARY_VEC_NUM_BYTES = 32;

struct NaiveScheduler {
  static void compute_exec_config(
      int num_local_experts,
      int hidden_dim_num_groups,
      int num_groups,
      int& subwarps_per_block,
      dim3& grid,
      dim3& block) {
    subwarps_per_block = ([=]() -> int {
      if (num_groups % 16 == 0) {
        return 16;
      } else if (num_groups % 8 == 0) {
        return 8;
      } else if (num_groups % 4 == 0) {
        return 4;
      } else if (num_groups % 2 == 0) {
        return 2;
      }
      return 1;
    })();
    grid = dim3(num_groups / subwarps_per_block);
    block = dim3(subwarps_per_block * THREADS_PER_SUBWARP);
  }

  template <bool FUSE_SILU_AND_MUL, typename FUNC>
  __device__ __forceinline__ static void execute(
      const int subwarps_per_block,
      const int hidden_dim_num_groups,
      const int group_size,
      const int32_t* masked_m,
      const int num_tokens_per_expert,
      FUNC fn) {
    constexpr int expert_idx = 0;

    const int subwarp_id = threadIdx.x / THREADS_PER_SUBWARP;
    const int lane_id = threadIdx.x % THREADS_PER_SUBWARP;

    const int block_group_id = blockIdx.x * subwarps_per_block;
    const int group_id = block_group_id + subwarp_id;

    int input_group_start_offset;
    if constexpr (!FUSE_SILU_AND_MUL) {
      input_group_start_offset = group_id * group_size;
    }

    const int token_idx = group_id / hidden_dim_num_groups;
    // At the hidden_size dimension, we are handling idx-th group
    const int hidden_dim_group_idx = group_id % hidden_dim_num_groups;

    if constexpr (FUSE_SILU_AND_MUL) {
      const int hidden_size = hidden_dim_num_groups * group_size;
      input_group_start_offset = compute_input_group_start_offset<FUSE_SILU_AND_MUL>(
          expert_idx, token_idx, hidden_dim_group_idx, hidden_size, num_tokens_per_expert, group_size);
    }

    fn(expert_idx, token_idx, hidden_dim_group_idx, lane_id, input_group_start_offset);
  }
};

struct MaskedLayoutScheduler {
  // TODO can be dynamically determined (which may be good when num rank is small)
  static constexpr int TOKEN_DIM_BLOCK_NUM_PER_EXPERT = 1024;
  static constexpr int SUBWARPS_PER_BLOCK = 16;

  static void compute_exec_config(
      int num_local_experts,
      int hidden_dim_num_groups,
      int num_groups,
      int& subwarps_per_block,
      dim3& grid,
      dim3& block) {
    subwarps_per_block = SUBWARPS_PER_BLOCK;
    TORCH_CHECK(hidden_dim_num_groups % subwarps_per_block == 0);
    grid = dim3(hidden_dim_num_groups / subwarps_per_block, TOKEN_DIM_BLOCK_NUM_PER_EXPERT, num_local_experts);
    block = dim3(subwarps_per_block * THREADS_PER_SUBWARP);
  }

  template <bool FUSE_SILU_AND_MUL, typename FUNC>
  __device__ __forceinline__ static void execute(
      const int subwarps_per_block,
      const int hidden_dim_num_groups,
      const int group_size,
      const int32_t* masked_m,
      const int num_tokens_per_expert,
      FUNC fn) {
    const int subwarp_id = threadIdx.x / THREADS_PER_SUBWARP;
    const int lane_id = threadIdx.x % THREADS_PER_SUBWARP;

    const int expert_idx = blockIdx.z;
    const int token_idx_start = blockIdx.y;

    const int hidden_dim_group_idx = blockIdx.x * SUBWARPS_PER_BLOCK + subwarp_id;

    const int curr_expert_token_num = masked_m[expert_idx];

    for (int token_idx = token_idx_start; token_idx < curr_expert_token_num;
         token_idx += TOKEN_DIM_BLOCK_NUM_PER_EXPERT) {
      const int hidden_size = hidden_dim_num_groups * group_size;
      const int input_group_start_offset = compute_input_group_start_offset<FUSE_SILU_AND_MUL>(
          expert_idx, token_idx, hidden_dim_group_idx, hidden_size, num_tokens_per_expert, group_size);
      fn(expert_idx, token_idx, hidden_dim_group_idx, lane_id, input_group_start_offset);
    }
  }
};

template <
    typename SCHEDULER,
    typename T,
    typename DST_DTYPE,
    bool IS_COLUMN_MAJOR = false,
    bool SCALE_UE8M0 = false,
    bool FUSE_SILU_AND_MUL = false,
    typename scale_packed_t = std::conditional_t<SCALE_UE8M0, uint32_t, float>>
__global__ void per_token_group_quant_8bit_kernel(
    const T* __restrict__ input,
    DST_DTYPE* __restrict__ output_q,
    scale_packed_t* __restrict__ output_s,
    const int32_t* __restrict__ masked_m,
    const int group_size,
    const int subwarps_per_block,
    const int hidden_dim_num_groups,
    // TODO can this be removed?
    const int scale_expert_stride,
    const int scale_hidden_stride,
    const int num_tokens_per_expert) {
  using dst_dtype_info = DtypeInfo<DST_DTYPE>;
  using scale_element_t = std::conditional_t<SCALE_UE8M0, uint8_t, float>;
  static_assert(sizeof(scale_packed_t) % sizeof(scale_element_t) == 0);

  SCHEDULER::execute<FUSE_SILU_AND_MUL>(
      subwarps_per_block,
      hidden_dim_num_groups,
      group_size,
      masked_m,
      num_tokens_per_expert,
      [&](const int expert_idx,
          const int token_idx,
          const int hidden_dim_group_idx,
          const int lane_id,
          const int input_group_start_offset) {
        constexpr uint32_t INPUT_PRIMARY_VEC_SIZE = INPUT_PRIMARY_VEC_NUM_BYTES / sizeof(T);
        constexpr uint32_t INPUT_PRIMARY_INT4_SIZE = INPUT_PRIMARY_VEC_NUM_BYTES / sizeof(int4);

        const int offset_num_groups = expert_idx * num_tokens_per_expert * hidden_dim_num_groups +
                                      token_idx * hidden_dim_num_groups + hidden_dim_group_idx;

        int4 input_primary_int4[INPUT_PRIMARY_INT4_SIZE];
        T* input_primary_vec = reinterpret_cast<T*>(input_primary_int4);
        static_assert(sizeof(input_primary_vec[0]) * INPUT_PRIMARY_VEC_SIZE == sizeof(input_primary_int4));

        int4 input_secondary_int4[INPUT_PRIMARY_INT4_SIZE];
        T* input_secondary_vec = reinterpret_cast<T*>(input_secondary_int4);
        static_assert(sizeof(input_secondary_vec[0]) * INPUT_PRIMARY_VEC_SIZE == sizeof(input_secondary_int4));

#pragma unroll
        for (uint32_t j = 0; j < INPUT_PRIMARY_INT4_SIZE; ++j) {
          input_primary_int4[j] = ld_global_nc(
              reinterpret_cast<const int4*>(input + input_group_start_offset + lane_id * INPUT_PRIMARY_VEC_SIZE) + j);
        }
        if constexpr (FUSE_SILU_AND_MUL) {
          const int secondary_offset = hidden_dim_num_groups * group_size;
#pragma unroll
          for (uint32_t j = 0; j < INPUT_PRIMARY_INT4_SIZE; ++j) {
            input_secondary_int4[j] = ld_global_nc(
                reinterpret_cast<const int4*>(
                    input + input_group_start_offset + lane_id * INPUT_PRIMARY_VEC_SIZE + secondary_offset) +
                j);
          }
        }

        constexpr int num_elems_per_pack = static_cast<int>(sizeof(scale_packed_t) / sizeof(scale_element_t));
        scale_element_t* scale_output;
        if constexpr (IS_COLUMN_MAJOR) {
          constexpr int scale_token_stride = 1;

          const int hidden_idx_packed = hidden_dim_group_idx / num_elems_per_pack;
          const int pack_idx = hidden_dim_group_idx % num_elems_per_pack;
          scale_output = reinterpret_cast<scale_element_t*>(output_s) +
                         (expert_idx * scale_expert_stride * num_elems_per_pack +
                          hidden_idx_packed * scale_hidden_stride * num_elems_per_pack +
                          token_idx * scale_token_stride * num_elems_per_pack + pack_idx);
        } else {
          static_assert(!SCALE_UE8M0);
          scale_output = output_s + offset_num_groups;
        }

        // can speed up if too slow
        if constexpr (IS_COLUMN_MAJOR and SCALE_UE8M0) {
          const int remainder_num_groups = hidden_dim_num_groups % num_elems_per_pack;
          if ((remainder_num_groups != 0) and (hidden_dim_group_idx == hidden_dim_num_groups - 1) and
              (lane_id < num_elems_per_pack - remainder_num_groups)) {
            const int shift = 1 + lane_id;
            *(scale_output + shift) = 0;
          }
        }

        float local_absmax = LOCAL_ABSMAX_ABS;

#pragma unroll
        for (uint32_t j = 0; j < INPUT_PRIMARY_VEC_SIZE; ++j) {
          float val;
          if constexpr (FUSE_SILU_AND_MUL) {
            // TODO maybe vectorize
            T val_lowprec = static_cast<T>(silu(static_cast<float>(input_primary_vec[j]))) * input_secondary_vec[j];
            val = static_cast<float>(val_lowprec);
            input_primary_vec[j] = val_lowprec;
          } else {
            val = static_cast<float>(input_primary_vec[j]);
          }

          float abs_val = fabsf(val);
          local_absmax = fmaxf(local_absmax, abs_val);
        }

        local_absmax = GroupReduceMax<THREADS_PER_SUBWARP>(local_absmax, lane_id);

        float y_scale, y_scale_inv;
        calculate_fp8_scales<SCALE_UE8M0, dst_dtype_info>(local_absmax, y_scale, y_scale_inv);
        float2 y_scale_repeated = {y_scale, y_scale};

        if (lane_id == 0) {
          *scale_output = extract_required_scale_format<SCALE_UE8M0>(y_scale_inv);
        }

        int4 output_buf;
        static_assert(sizeof(output_buf) == INPUT_PRIMARY_VEC_SIZE * sizeof(DST_DTYPE));

        if constexpr (std::is_same_v<DST_DTYPE, c10::Float8_e4m3fn>) {
          const auto output_buf_ptr = reinterpret_cast<__hip_fp8x2_storage_t*>(&output_buf);
          static_assert(sizeof(output_buf) == INPUT_PRIMARY_VEC_SIZE / 2 * sizeof(__hip_fp8x2_storage_t));
          static_assert(INPUT_PRIMARY_VEC_SIZE % 2 == 0);

#pragma unroll
          for (uint32_t j = 0; j < INPUT_PRIMARY_VEC_SIZE; j += 2) {
            float2 inputx2 = {static_cast<float>(input_primary_vec[j]), static_cast<float>(input_primary_vec[j + 1])};
            float2 outputx2 = __fmul2_rn(inputx2, y_scale_repeated);
            output_buf_ptr[j / 2] = __hip_cvt_float2_to_fp8x2(outputx2, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
          }
        } else {
          const auto output_buf_ptr = reinterpret_cast<DST_DTYPE*>(&output_buf);

#pragma unroll
          for (uint32_t j = 0; j < INPUT_PRIMARY_VEC_SIZE; ++j) {
            float val = static_cast<float>(input_primary_vec[j]);
            float q_val = fminf(fmaxf(val * y_scale, dst_dtype_info::MIN), dst_dtype_info::MAX);
            output_buf_ptr[j] = DST_DTYPE(q_val);
          }
        }

        st_global(
            reinterpret_cast<int4*>(output_q + offset_num_groups * group_size + lane_id * INPUT_PRIMARY_VEC_SIZE),
            output_buf);
      });
}

void sgl_per_token_group_quant_8bit(
    // vanilla: (num_tokens, hidden_size)
    // fuse_silu_and_mul: (num_tokens, hidden_size * 2)
    // fuse_silu_and_mul + masked_layout: (num_experts, num_tokens-with-padding, hidden_size * 2)
    torch::Tensor input,
    torch::Tensor output_q,
    torch::Tensor output_s,
    int64_t group_size,
    double eps,
    double min_8bit,
    double max_8bit,
    bool scale_ue8m0,
    bool fuse_silu_and_mul,
    const std::optional<torch::Tensor>& masked_m) {
  CHECK_INPUT(input);
  CHECK_INPUT(output_q);
  TORCH_CHECK(input.numel() > 0);

  TORCH_CHECK(std::abs(LOCAL_ABSMAX_ABS - eps) < 1e-13);

  CHECK_EQ(input.numel() % group_size, 0);
  const int num_groups = static_cast<int>(input.numel()) / group_size / (fuse_silu_and_mul ? 2 : 1);

  const bool masked_layout = masked_m.has_value();
  TORCH_CHECK(output_s.dim() == (masked_layout ? 3 : 2));

  const int num_local_experts = masked_layout ? input.size(0) : 1;

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  auto dst_type = output_q.scalar_type();

  const bool is_column_major = output_s.stride(-2) < output_s.stride(-1);
  const int hidden_dim_num_groups = static_cast<int>(output_q.size(-1)) / group_size;
  const int num_tokens_per_expert = static_cast<int>(output_q.size(-2));
  const int scale_expert_stride = masked_layout ? static_cast<int>(output_s.stride(0)) : 0;
  const int scale_hidden_stride = static_cast<int>(output_s.stride(-1));

#define LAUNCH_KERNEL_INNER(SCHEDULER, T, DST_DTYPE, output_s_dtype, ...)                                \
  do {                                                                                                   \
    int subwarps_per_block;                                                                              \
    dim3 grid, block;                                                                                    \
    SCHEDULER::compute_exec_config(                                                                      \
        num_local_experts, hidden_dim_num_groups, num_groups, subwarps_per_block, grid, block);          \
                                                                                                         \
    per_token_group_quant_8bit_kernel<SCHEDULER, T, DST_DTYPE, __VA_ARGS__><<<grid, block, 0, stream>>>( \
        static_cast<T*>(input.data_ptr()),                                                               \
        static_cast<DST_DTYPE*>(output_q.data_ptr()),                                                    \
        static_cast<output_s_dtype*>(output_s.data_ptr()),                                               \
        static_cast<int32_t*>(masked_m.has_value() ? masked_m->data_ptr() : 0),                          \
        group_size,                                                                                      \
        subwarps_per_block,                                                                              \
        hidden_dim_num_groups,                                                                           \
        scale_expert_stride,                                                                             \
        scale_hidden_stride,                                                                             \
        num_tokens_per_expert);                                                                          \
  } while (0)

#define LAUNCH_KERNEL(T, DST_DTYPE)                                                               \
  do {                                                                                            \
    TORCH_CHECK(THREADS_PER_SUBWARP* INPUT_PRIMARY_VEC_NUM_BYTES == group_size * sizeof(T));      \
                                                                                                  \
    using dst_dtype_info = DtypeInfo<DST_DTYPE>;                                                  \
    CHECK_EQ(dst_dtype_info::MIN, min_8bit);                                                      \
    CHECK_EQ(dst_dtype_info::MAX, max_8bit);                                                      \
                                                                                                  \
    if (is_column_major) {                                                                        \
      if (scale_ue8m0) {                                                                          \
        if (fuse_silu_and_mul) {                                                                  \
          if (masked_layout) {                                                                    \
            LAUNCH_KERNEL_INNER(MaskedLayoutScheduler, T, DST_DTYPE, uint32_t, true, true, true); \
          } else {                                                                                \
            LAUNCH_KERNEL_INNER(NaiveScheduler, T, DST_DTYPE, uint32_t, true, true, true);        \
          }                                                                                       \
        } else {                                                                                  \
          LAUNCH_KERNEL_INNER(NaiveScheduler, T, DST_DTYPE, uint32_t, true, true);                \
        }                                                                                         \
      } else {                                                                                    \
        LAUNCH_KERNEL_INNER(NaiveScheduler, T, DST_DTYPE, float, true);                           \
      }                                                                                           \
    } else {                                                                                      \
      LAUNCH_KERNEL_INNER(NaiveScheduler, T, DST_DTYPE, float, false);                            \
    }                                                                                             \
  } while (0)

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), scalar_t, [&] {
    if (dst_type == at::ScalarType::Char) {
      LAUNCH_KERNEL(scalar_t, int8_t);
      return true;
    } else if (dst_type == at::ScalarType::Float8_e4m3fn) {
      LAUNCH_KERNEL(scalar_t, c10::Float8_e4m3fn);
      return true;
    }
    return false;
  });

#undef LAUNCH_KERNEL
#undef LAUNCH_KERNEL_INNER
}
