// References:
// https://docs.nvidia.com/cuda/cublas/index.html#cublasgemmgroupedbatchedex
// https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuBLAS/Extensions/GemmGroupedBatchedEx/cublas_GemmGroupedBatchedEx_example.cu
// https://github.com/zhihu/ZhiLight/blob/main/src/nn/linear/gemm_grouped.cpp

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/util/Exception.h>
#include <hipblas.h>
#include <cudaTypedefs.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/all.h>

#include <cstdio>
#include <cstdlib>
#include <string>
#include <vector>

#include "utils.h"

static void check_group_count(
    const std::vector<torch::Tensor>& inputs,
    const std::vector<torch::Tensor>& weights,
    const std::vector<torch::Tensor>& outputs) {
  TORCH_CHECK(
      ((inputs.size() == weights.size()) && (inputs.size() == outputs.size())),
      "The group count of inputs, weights and outputs should be the same.");
}

static void check_device_dtype(const torch::Dtype& dtype, const std::vector<torch::Tensor>& tensors) {
  for (const auto& t : tensors) {
    TORCH_CHECK(dtype == t.dtype(), "dtype of all the tensors should be the same");
    TORCH_CHECK(t.is_cuda(), "All tensors should be in Cuda memory");
  }
}

static std::vector<int> get_dims(const std::vector<torch::Tensor>& tensors, int dim) {
  std::vector<int> results;
  for (const auto& t : tensors) {
    TORCH_CHECK(t.dim() == 2, "Should pass in 2D matrices");
    results.push_back(t.size(dim));
  }
  return std::move(results);
}

static std::vector<int> get_strides(const std::vector<torch::Tensor>& tensors, int dim) {
  std::vector<int> results;
  for (const auto& t : tensors) {
    results.push_back(t.stride(dim));
  }
  return std::move(results);
}

static void check_equal(const std::vector<int>& a, const std::vector<int>& b, const std::string& err_msg) {
  for (int i = 0; i < a.size(); ++i) {
    TORCH_CHECK(a[i] == b[i], err_msg);
  }
}

static std::vector<void*> get_tensor_ptrs(const std::vector<torch::Tensor>& tensors) {
  std::vector<void*> ptrs;
  for (auto& t : tensors) {
    ptrs.push_back(t.data_ptr());
  }
  return std::move(ptrs);
}

static torch::Tensor create_ptr_pointer(const std::vector<void*>& ptrs, hipStream_t stream) {
  auto options = torch::TensorOptions().dtype(torch::kDouble).device(torch::kCUDA);
  torch::Tensor gpu_ptrs = torch::empty({static_cast<int>(ptrs.size())}, options);
  TORCH_CHECK(
      hipMemcpyAsync(gpu_ptrs.data_ptr(), ptrs.data(), sizeof(void*) * ptrs.size(), hipMemcpyHostToDevice, stream) ==
      HIPBLAS_STATUS_SUCCESS);
  return gpu_ptrs;
}

// We want compute input @ weight^T in row major
// This is equivalent to computing weight @ input^T in col major
// Cublas only accepts matrix in column major, so this arrangement is needed
void cublas_grouped_gemm(
    const std::vector<torch::Tensor>& inputs,   // b: (m, k) row major = (k, m) col major
    const std::vector<torch::Tensor>& weights,  // a: (n, k) row major = (n, k)^T col major
    const std::vector<torch::Tensor>& outputs,  // c: (m, n) row major = (n, m) col major
    const torch::Dtype& out_dtype,
    int64_t cublas_handle,
    int64_t hip_stream) {
  TORCH_CHECK(
      out_dtype == torch::kHalf || out_dtype == torch::kBFloat16,
      "cublas grouped_gemm can"
      "only be applied to float16 and bfloat16 dtype");

  int group_count = inputs.size();
  check_group_count(inputs, weights, outputs);
  std::vector<int> group_size(group_count, 1);

  // Make sure all tensors are on cuda and use the same dtype
  check_device_dtype(out_dtype, inputs);
  check_device_dtype(out_dtype, weights);
  check_device_dtype(out_dtype, outputs);

  // Weights should be transposed to (n, k) of column major
  std::vector<hipblasOperation_t> transa_array(group_count, HIPBLAS_OP_T);
  std::vector<hipblasOperation_t> transb_array(group_count, HIPBLAS_OP_N);

  // Get dim arrays
  std::vector<int> m_array = get_dims(weights, 0);
  std::vector<int> n_array = get_dims(inputs, 0);
  std::vector<int> k_array = get_dims(inputs, 1);

  // Make sure the dimensions in each group match
  std::vector<int> m_array1 = get_dims(outputs, 1);
  std::vector<int> n_array1 = get_dims(outputs, 0);
  std::vector<int> k_array1 = get_dims(weights, 1);
  check_equal(m_array, m_array1, "sizes don't match on m dimension");
  check_equal(n_array, n_array1, "sizes don't match on n dimension");
  check_equal(k_array, k_array1, "sizes don't match on k dimension");

  // Get leading dimensions
  std::vector<int> lda_array = get_strides(weights, 0);
  std::vector<int> ldb_array = get_strides(inputs, 0);
  std::vector<int> ldc_array = get_strides(outputs, 0);

  // Use default scaling factors
  std::vector<float> alpha_array(group_count, 1);
  std::vector<float> beta_array(group_count, 0);

  std::vector<void*> a_array = get_tensor_ptrs(weights);
  std::vector<void*> b_array = get_tensor_ptrs(inputs);
  std::vector<void*> c_array = get_tensor_ptrs(outputs);

  auto stream = reinterpret_cast<hipStream_t>(hip_stream);

  // Should allocate tensors for storage of pointers
  torch::Tensor d_a = create_ptr_pointer(a_array, stream);
  torch::Tensor d_b = create_ptr_pointer(b_array, stream);
  torch::Tensor d_c = create_ptr_pointer(c_array, stream);

#if defined CUDA_VERSION && CUDA_VERSION >= 12050
  auto handle = reinterpret_cast<hipblasHandle_t>(cublas_handle);
  hipDataType cuda_data_type = (out_dtype == torch::kHalf ? HIP_R_16F : HIP_R_16BF);

  auto status = cublasGemmGroupedBatchedEx(
      handle,
      transa_array.data(),
      transb_array.data(),
      m_array.data(),
      n_array.data(),
      k_array.data(),
      alpha_array.data(),
      (void**)d_a.data_ptr(),
      cuda_data_type,
      lda_array.data(),
      (void**)d_b.data_ptr(),
      cuda_data_type,
      ldb_array.data(),
      beta_array.data(),
      (void**)d_c.data_ptr(),
      cuda_data_type,
      ldc_array.data(),
      group_count,
      group_size.data(),
      HIPBLAS_COMPUTE_32F);
  TORCH_CHECK(status == HIPBLAS_STATUS_SUCCESS, "cublas grouped gemm failed: ", cublasGetStatusString(status));
  TORCH_CHECK(hipStreamSynchronize(stream) == hipSuccess, "Failed when stream synchronization");
  return;
#endif

  TORCH_CHECK_NOT_IMPLEMENTED(
      false, "Cublas GroupGemm is not implemented with current compute capability: ", getSMVersion());
}
