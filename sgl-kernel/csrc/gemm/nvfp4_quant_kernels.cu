#include "hip/hip_runtime.h"
/* Copyright 2025 SGLang Team. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <torch/all.h>

#include "nvfp4_quant.cuh"
#include "utils.h"

// Quantizes the provided PackedVec into the uint32_t output
template <class Type, bool UE8M0_SF = false>
__device__ uint32_t cvt_warp_fp16_to_fp4(PackedVec<Type>& vec, float SFScaleVal, uint8_t* SFout) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  // Get absolute maximum values among the local 8 values.
  auto localMax = __habs2(vec.elts[0]);

// Local maximum value.
#pragma unroll
  for (int i = 1; i < CVT_FP4_ELTS_PER_THREAD / 2; i++) {
    localMax = __hmax2(localMax, __habs2(vec.elts[i]));
  }

  // Get the absolute maximum among all 16 values (two threads).
  localMax = __hmax2(__shfl_xor_sync(uint32_t(-1), localMax, 1), localMax);
  // Get the final absolute maximum values.
  float vecMax = float(__hmax(localMax.x, localMax.y));

  // Get the SF (max value of the vector / max value of e2m1).
  // maximum value of e2m1 = 6.0.
  // TODO: use half as compute data type.
  float SFValue = SFScaleVal * (vecMax * reciprocal_approximate_ftz(6.0f));
  // 8 bits representation of the SF.
  uint8_t fp8SFVal;
  // Write the SF to global memory (STG.8).
  if constexpr (UE8M0_SF) {
    __nv_fp8_e8m0 tmp;
    tmp.__x = __nv_cvt_float_to_e8m0(SFValue, __HIP_SATFINITE, hipRoundPosInf);
    SFValue = static_cast<float>(tmp);
    fp8SFVal = tmp.__x;
  } else {
    // Here SFValue is always positive, so E4M3 is the same as UE4M3.
    __hip_fp8_e4m3_fnuz tmp = __hip_fp8_e4m3_fnuz(SFValue);
    fp8SFVal = tmp.__x;
    SFValue = static_cast<float>(tmp);
  }
  // Get the output scale.
  // Recipe: final_scale = reciprocal(fp32(fp8(SFValue * SFScaleVal))) *
  //                       reciprocal(SFScaleVal))
  float outputScale =
      SFValue != 0 ? reciprocal_approximate_ftz(SFValue * reciprocal_approximate_ftz(SFScaleVal)) : 0.0f;

  if (SFout) {
    // Write the SF to global memory (STG.8).
    *SFout = fp8SFVal;
  }

  // Convert the input to float.
  float2 fp2Vals[CVT_FP4_ELTS_PER_THREAD / 2];

#pragma unroll
  for (int i = 0; i < CVT_FP4_ELTS_PER_THREAD / 2; i++) {
    if constexpr (std::is_same_v<Type, half>) {
      fp2Vals[i] = __half22float2(vec.elts[i]);
    } else {
      fp2Vals[i] = __bfloat1622float2(vec.elts[i]);
    }
    fp2Vals[i].x *= outputScale;
    fp2Vals[i].y *= outputScale;
  }

  // Convert to e2m1 values.
  uint32_t e2m1Vec = fp32_vec_to_e2m1(fp2Vals);

  // Write the e2m1 values to global memory.
  return e2m1Vec;
#else
  return 0;
#endif
}

// Use UE4M3 by default.
template <class Type, bool UE8M0_SF = false>
__global__ void
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
__launch_bounds__(512, 4) cvt_fp16_to_fp4(
#else
cvt_fp16_to_fp4(
#endif
    int32_t numRows, int32_t numCols, Type const* in, float const* SFScale, uint32_t* out, uint32_t* SFout) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
  using PackedVec = PackedVec<Type>;
  static constexpr int CVT_FP4_NUM_THREADS_PER_SF = (CVT_FP4_SF_VEC_SIZE / CVT_FP4_ELTS_PER_THREAD);
  static_assert(sizeof(PackedVec) == sizeof(Type) * CVT_FP4_ELTS_PER_THREAD, "Vec size is not matched.");

  // Get the global scaling factor, which will be applied to the SF.
  // Note SFScale is the same as next GEMM's alpha, which is
  // (448.f / (Alpha_A / 6.f)).
  float const SFScaleVal = SFScale == nullptr ? 1.0f : SFScale[0];

  // Input tensor row/col loops.
  for (int rowIdx = blockIdx.x; rowIdx < numRows; rowIdx += gridDim.x) {
    for (int colIdx = threadIdx.x; colIdx < numCols / CVT_FP4_ELTS_PER_THREAD; colIdx += blockDim.x) {
      int64_t inOffset = rowIdx * (numCols / CVT_FP4_ELTS_PER_THREAD) + colIdx;
      PackedVec in_vec = reinterpret_cast<PackedVec const*>(in)[inOffset];
      // Get the output tensor offset.
      // Same as inOffset because 8 elements are packed into one uint32_t.
      int64_t outOffset = inOffset;
      auto& out_pos = out[outOffset];

      auto sf_out =
          cvt_quant_to_fp4_get_sf_out_offset<uint32_t, CVT_FP4_NUM_THREADS_PER_SF>(rowIdx, colIdx, numCols, SFout);

      out_pos = cvt_warp_fp16_to_fp4<Type, UE8M0_SF>(in_vec, SFScaleVal, sf_out);
    }
  }
#endif
}

template <typename T>
void invokeFP4Quantization(
    int m,
    int n,
    T const* input,
    float const* SFScale,
    int64_t* output,
    int32_t* SFOuput,
    bool useUE8M0,
    int multiProcessorCount,
    hipStream_t stream) {
  // Grid, Block size.
  // Each thread converts 8 values.
  dim3 block(std::min(int(n / ELTS_PER_THREAD), 512));
  // Get number of blocks per SM (assume we can fully utilize the SM).
  int const numBlocksPerSM = 2048 / block.x;
  dim3 grid(std::min(int(m), multiProcessorCount * numBlocksPerSM));

  // Launch the cvt kernel.
  if (useUE8M0) {
    cvt_fp16_to_fp4<T, true><<<grid, block, 0, stream>>>(
        m, n, input, SFScale, reinterpret_cast<uint32_t*>(output), reinterpret_cast<uint32_t*>(SFOuput));
  } else {
    cvt_fp16_to_fp4<T, false><<<grid, block, 0, stream>>>(
        m, n, input, SFScale, reinterpret_cast<uint32_t*>(output), reinterpret_cast<uint32_t*>(SFOuput));
  }
}

// Instantiate the function.
template void invokeFP4Quantization(
    int m,
    int n,
    half const* input,
    float const* SFScale,
    int64_t* output,
    int32_t* SFOuput,
    bool useUE8M0,
    int multiProcessorCount,
    hipStream_t stream);

template void invokeFP4Quantization(
    int m,
    int n,
    __hip_bfloat16 const* input,
    float const* SFScale,
    int64_t* output,
    int32_t* SFOuput,
    bool useUE8M0,
    int multiProcessorCount,
    hipStream_t stream);

inline int getMultiProcessorCount() {
  static int multi_processor_count = []() {
    int device_id = 0;
    int count = 0;

    // Get the current CUDA device ID
    CHECK_CUDA_SUCCESS(hipGetDevice(&device_id));

    // Get the number of multiprocessors for the current device
    CHECK_CUDA_SUCCESS(hipDeviceGetAttribute(&count, hipDeviceAttributeMultiprocessorCount, device_id));

    return count;  // Initialize the static variable
  }();

  return multi_processor_count;  // Return the cached value on subsequent calls
}

void scaled_fp4_quant_sm100a(
    torch::Tensor& output, torch::Tensor const& input, torch::Tensor& output_sf, torch::Tensor const& input_sf) {
  auto sm_version = getSMVersion();
  TORCH_CHECK(sm_version == 100 || sm_version == 103, "fp4_quant is only supported on sm100a/sm103a");

  int32_t m = input.size(0);
  int32_t n = input.size(1);

  TORCH_CHECK(n % 16 == 0, "The N dimension must be multiple of 16.");

  int multiProcessorCount = getMultiProcessorCount();

  auto input_sf_ptr = static_cast<float const*>(input_sf.data_ptr());
  auto sf_out = static_cast<int32_t*>(output_sf.data_ptr());
  auto output_ptr = static_cast<int64_t*>(output.data_ptr());
  at::cuda::CUDAGuard device_guard{(char)input.get_device()};
  const hipStream_t stream = at::cuda::getCurrentCUDAStream(input.get_device());

  // We don't support e8m0 scales at this moment.
  bool useUE8M0 = false;

  switch (input.scalar_type()) {
    case torch::kHalf: {
      auto input_ptr = reinterpret_cast<half const*>(input.data_ptr());
      invokeFP4Quantization(m, n, input_ptr, input_sf_ptr, output_ptr, sf_out, useUE8M0, multiProcessorCount, stream);
      break;
    }
    case torch::kBFloat16: {
      auto input_ptr = reinterpret_cast<__hip_bfloat16 const*>(input.data_ptr());
      invokeFP4Quantization(m, n, input_ptr, input_sf_ptr, output_ptr, sf_out, useUE8M0, multiProcessorCount, stream);
      break;
    }
    default: {
      std::cerr << "Observing: " << input.scalar_type() << " for the input datatype which is invalid";
      throw std::runtime_error("Unsupported input data type for quantize_to_fp4.");
    }
  }
}
