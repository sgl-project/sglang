#include <ATen/cuda/HIPContext.h>
#include <cudaTypedefs.h>
#include <cutlass/arch/arch.h>
#include <cutlass/arch/memory.h>
#include <cutlass/arch/mma.h>
#include <cutlass/array.h>
#include <cutlass/cutlass.h>
#include <cutlass/epilogue/thread/activation.h>
#include <cutlass/epilogue/thread/linear_combination.h>
#include <cutlass/epilogue/threadblock/default_thread_map_tensor_op.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/gemm/device/gemm_universal_adapter.h>
#include <cutlass/gemm/gemm.h>
#include <cutlass/gemm/kernel/default_gemm_universal_with_visitor.h>
#include <cutlass/gemm/thread/mma.h>
#include <cutlass/layout/matrix.h>
#include <cutlass/matrix_coord.h>
#include <cutlass/numeric_types.h>
#include <cutlass/tensor_ref.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/util/tensor_view_io.h>
#include <torch/all.h>

#include <cute/tensor.hpp>
#include <cutlass/epilogue/collective/collective_builder.hpp>
#include <cutlass/epilogue/collective/default_epilogue.hpp>
#include <cutlass/epilogue/threadblock/fusion/visitors.hpp>
#include <cutlass/gemm/collective/collective_builder.hpp>
#include <cutlass/gemm/dispatch_policy.hpp>
#include <cutlass/gemm/kernel/gemm_universal.hpp>
#include <cutlass/util/packed_stride.hpp>

#include "utils.h"

using namespace cute;

template <
    typename SchedulerType,
    typename OutType,
    typename TileShape,
    typename ClusterShape,
    typename ScaleGranularity>
void launch_sm90_fp8_blockwise_scaled_mm(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b) {
  using ElementAccumulator = float;
  using ElementCompute = float;
  using ElementBlockScale = float;

  using ElementA = cutlass::float_e4m3_t;
  using LayoutA = cutlass::layout::RowMajor;
  constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementA>::value;

  using ElementB = cutlass::float_e4m3_t;
  using LayoutB = cutlass::layout::ColumnMajor;
  constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementB>::value;

  using ElementC = void;
  using LayoutC = cutlass::layout::RowMajor;
  constexpr int AlignmentC = 128 / cutlass::sizeof_bits<OutType>::value;

  using ElementD = OutType;
  using LayoutD = cutlass::layout::RowMajor;
  constexpr int AlignmentD = AlignmentC;

  static constexpr int ScaleGranularityM = size<0>(ScaleGranularity{});
  static constexpr int ScaleGranularityN = size<1>(ScaleGranularity{});

  using ArchTag = cutlass::arch::Sm90;
  using OperatorClass = cutlass::arch::OpClassTensorOp;
  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  using EpilogueTileType = cutlass::epilogue::collective::EpilogueTileAuto;
  using StoreEpilogueCompute = typename cutlass::epilogue::fusion::Sm90EVT<cutlass::epilogue::fusion::Sm90AccFetch>;

  using KernelSchedule =
      cutlass::gemm::KernelTmaWarpSpecializedCooperativeFP8BlockScaledAccum<ScaleGranularityM, ScaleGranularityN>;
  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      ArchTag,
      OperatorClass,
      TileShape,
      ClusterShape,
      EpilogueTileType,
      ElementAccumulator,
      ElementCompute,
      ElementC,
      LayoutC,
      AlignmentC,
      ElementD,
      LayoutD,
      AlignmentD,
      EpilogueSchedule,
      StoreEpilogueCompute>::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      ArchTag,
      OperatorClass,
      ElementA,
      LayoutA,
      AlignmentA,
      ElementB,
      LayoutB,
      AlignmentB,
      ElementAccumulator,
      TileShape,
      ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
          sizeof(typename CollectiveEpilogue::SharedStorage))>,
      KernelSchedule>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>,  // Indicates ProblemShape
      CollectiveMainloop,
      CollectiveEpilogue,
      SchedulerType>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  Gemm gemm_op;

  int m = a.size(0);
  int k = a.size(1);
  int n = b.size(1);

  auto a_ptr = static_cast<ElementA*>(a.data_ptr());
  auto b_ptr = static_cast<ElementB*>(b.data_ptr());
  auto o_ptr = static_cast<ElementD*>(out.data_ptr());

  auto a_s_ptr = static_cast<ElementBlockScale*>(scales_a.data_ptr());
  auto b_s_ptr = static_cast<ElementBlockScale*>(scales_b.data_ptr());

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using StrideD = typename Gemm::GemmKernel::StrideD;

  StrideA stride_a = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(m, k, 1));
  StrideB stride_b = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(n, k, 1));
  StrideC stride_c;
  StrideD stride_d = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(m, n, 1));

  typename GemmKernel::MainloopArguments mainloop_args{a_ptr, stride_a, b_ptr, stride_b, 4, a_s_ptr, b_s_ptr};
  typename GemmKernel::EpilogueArguments epilogue_args{{}, nullptr, stride_d, o_ptr, stride_d};

  typename Gemm::Arguments args = {
      cutlass::gemm::GemmUniversalMode::kGemm,
      {m, n, k, 1},
      mainloop_args,
      epilogue_args,
  };

  size_t workspace_size = gemm_op.get_workspace_size(args);
  auto const workspace_options = torch::TensorOptions().dtype(torch::kUInt8).device(a.device());
  auto workspace = torch::empty(workspace_size, workspace_options);
  auto stream = at::cuda::getCurrentCUDAStream(a.get_device());

  auto can_implement = gemm_op.can_implement(args);
  TORCH_CHECK(can_implement == cutlass::Status::kSuccess, cutlassGetStatusString(can_implement))

  auto status = gemm_op.run(args, workspace.data_ptr(), stream);
  TORCH_CHECK(status == cutlass::Status::kSuccess, cutlassGetStatusString(status))
}

template <
    typename OutType,
    typename MmaTileShape,
    typename PerSmTileShape,
    typename EpilogueTileShape,
    typename ScalesPerTile,
    int TileSizeM_ = 128,
    class ClusterShape = Shape<_1, _1, _1>>
void launch_sm100_fp8_blockwise_scaled_mm(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b) {
  static constexpr int ScaleMsPerTile = size<0>(ScalesPerTile{});
  static constexpr int ScaleGranularityM = size<0>(MmaTileShape{}) / ScaleMsPerTile;
  static constexpr int ScaleGranularityN = size<1>(MmaTileShape{}) / size<1>(ScalesPerTile{});
  static constexpr int ScaleGranularityK = size<2>(MmaTileShape{}) / size<2>(ScalesPerTile{});

  using ElementAB = cutlass::float_e4m3_t;
  using ElementA = ElementAB;
  using ElementB = ElementAB;
  using ElementC = void;
  using ElementD = OutType;
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutD = cutlass::layout::RowMajor;
  using LayoutC = LayoutD;
  // This means both SFA and SFB are column-major.
  using ScaleConfig = cutlass::detail::Sm100BlockwiseScaleConfig<
      ScaleGranularityM,
      ScaleGranularityN,
      ScaleGranularityK,
      cute::UMMA::Major::MN,
      cute::UMMA::Major::K>;
  using LayoutSFA = decltype(ScaleConfig::deduce_layoutSFA());
  using LayoutSFB = decltype(ScaleConfig::deduce_layoutSFB());

  static constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementA>::value;
  static constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementB>::value;
  static constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;
  static constexpr int AlignmentC = AlignmentD;

  using ElementAccumulator = float;
  using ElementBlockScale = float;
  using ElementCompute = float;
  using ArchTag = cutlass::arch::Sm100;
  using OperatorClass = cutlass::arch::OpClassTensorOp;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      ArchTag,
      cutlass::arch::OpClassTensorOp,
      PerSmTileShape,
      ClusterShape,
      EpilogueTileShape,
      ElementAccumulator,
      ElementCompute,
      ElementC,
      LayoutC,
      AlignmentC,
      ElementD,
      LayoutD,
      AlignmentD,
      cutlass::epilogue::TmaWarpSpecialized1Sm>::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      ArchTag,
      OperatorClass,
      ElementA,
      cute::tuple<LayoutA, LayoutSFA>,
      AlignmentA,
      ElementB,
      cute::tuple<LayoutB, LayoutSFB>,
      AlignmentB,
      ElementAccumulator,
      MmaTileShape,
      ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
          sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::KernelTmaWarpSpecializedBlockwise1SmSm100>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>,
      CollectiveMainloop,
      CollectiveEpilogue,
      cutlass::gemm::PersistentScheduler>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  Gemm gemm_op;

  int m = a.size(0);
  int k = a.size(1);
  int n = b.size(1);

  auto a_ptr = static_cast<ElementAB*>(a.data_ptr());
  auto b_ptr = static_cast<ElementAB*>(b.data_ptr());
  auto scales_a_ptr = static_cast<float*>(scales_a.data_ptr());
  auto scales_b_ptr = static_cast<float*>(scales_b.data_ptr());
  auto c_ptr = static_cast<ElementD*>(out.data_ptr());

  using StrideA = typename GemmKernel::StrideA;
  using StrideB = typename GemmKernel::StrideB;
  using StrideD = typename GemmKernel::StrideD;
  using StrideC = typename GemmKernel::StrideD;

  StrideA a_stride = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(m, k, 1));
  StrideB b_stride = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(n, k, 1));
  StrideC c_stride = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(m, n, 1));
  LayoutSFA layout_SFA = ScaleConfig::tile_atom_to_shape_SFA(make_shape(m, n, k, 1));
  LayoutSFB layout_SFB = ScaleConfig::tile_atom_to_shape_SFB(make_shape(m, n, k, 1));

  typename GemmKernel::MainloopArguments mainloop_args{
      a_ptr, a_stride, b_ptr, b_stride, scales_a_ptr, layout_SFA, scales_b_ptr, layout_SFB};

  typename GemmKernel::EpilogueArguments epilogue_args{{}, c_ptr, c_stride, c_ptr, c_stride};
  epilogue_args.thread.alpha = 1.0f;

  typename GemmKernel::Arguments args = {
      cutlass::gemm::GemmUniversalMode::kGemm, {m, n, k, 1}, mainloop_args, epilogue_args};

  auto can_implement = gemm_op.can_implement(args);
  TORCH_CHECK(can_implement == cutlass::Status::kSuccess, cutlassGetStatusString(can_implement))

  size_t workspace_size = gemm_op.get_workspace_size(args);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  auto init_status = gemm_op.initialize(args, workspace.get());
  TORCH_CHECK(init_status == cutlass::Status::kSuccess, cutlassGetStatusString(init_status));

  auto stream = at::cuda::getCurrentCUDAStream(a.get_device());

  auto status = gemm_op.run(stream);
  TORCH_CHECK(status == cutlass::Status::kSuccess, cutlassGetStatusString(status))
}

template <typename OutType>
void sm90_fp8_blockwise_dispatch_shape(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b) {
  using TileShape = Shape<_128, _128, _128>;
  using ClusterShape = Shape<_1, _2, _1>;
  using ScaleGranularity = Shape<_1, _128, _128>;

  auto k = a.size(1);
  auto n = b.size(1);
  if (k > 3 * n) {
    launch_sm90_fp8_blockwise_scaled_mm<
        cutlass::gemm::StreamKScheduler,
        OutType,
        TileShape,
        ClusterShape,
        ScaleGranularity>(out, a, b, scales_a, scales_b);
  } else {
    launch_sm90_fp8_blockwise_scaled_mm<
        cutlass::gemm::PersistentScheduler,
        OutType,
        TileShape,
        ClusterShape,
        ScaleGranularity>(out, a, b, scales_a, scales_b);
  }
}

template <typename OutType>
void sm100_fp8_blockwise_dispatch_shape(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b) {
  if (a.size(0) <= 128) {
    using MmaTileShape = Shape<_64, _128, _128>;
    using PerSmTileShape = Shape<_64, _128, _128>;
    using EpilogueTileShape = Shape<_64, _64>;
    using ScalesPerTile = Shape<_64, _1, _1>;
    launch_sm100_fp8_blockwise_scaled_mm<OutType, MmaTileShape, PerSmTileShape, EpilogueTileShape, ScalesPerTile>(
        out, a, b, scales_a, scales_b);
  } else {
    using MmaTileShape = Shape<_128, _128, _128>;
    using PerSmTileShape = Shape<_128, _128, _128>;
    using EpilogueTileShape = Shape<_128, _64>;
    using ScalesPerTile = Shape<_128, _1, _1>;
    launch_sm100_fp8_blockwise_scaled_mm<OutType, MmaTileShape, PerSmTileShape, EpilogueTileShape, ScalesPerTile>(
        out, a, b, scales_a, scales_b);
  }
}

torch::Tensor fp8_blockwise_scaled_mm(
    const torch::Tensor& mat_a,
    const torch::Tensor& mat_b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const torch::Dtype& out_dtype) {
  TORCH_CHECK(mat_a.is_cuda(), "mat_a must be a CUDA tensor");
  TORCH_CHECK(mat_b.is_cuda(), "mat_b must be a CUDA tensor");
  TORCH_CHECK(mat_a.dim() == 2, "mat_a must be a 2D tensor");
  TORCH_CHECK(mat_b.dim() == 2, "mat_b must be a 2D tensor");
  TORCH_CHECK(mat_a.stride(1) == 1, "mat_a must be a row major tensor");
  TORCH_CHECK(mat_b.stride(0) == 1, "mat_a must be a column major tensor");
  TORCH_CHECK(mat_a.size(1) == mat_b.size(0), "mat_a and mat_b shapes cannot be multiplied");

  TORCH_CHECK(
      (mat_a.size(1) * mat_a.element_size()) % 16 == 0, "mat_a must be multiple of 16 bytes for memory alignment");
  TORCH_CHECK(
      (mat_b.size(0) * mat_b.element_size()) % 16 == 0, "mat_b must be multiple of 16 bytes for memory alignment");
  TORCH_CHECK(mat_a.scalar_type() == torch::kFloat8_e4m3fn, "mat_a must be Float8_e4m3fn");
  TORCH_CHECK(mat_b.scalar_type() == torch::kFloat8_e4m3fn, "mat_b must be Float8_e4m3fn");
  TORCH_CHECK(out_dtype == torch::kHalf || out_dtype == torch::kBFloat16, "out_dtype must be Half or BFloat16");

  auto is_contiguous_vector = [](const torch::Tensor& t) {
    auto t_sizes = t.sizes();
    return t.is_contiguous() &&
           (t.dim() == 1 || (t.dim() == 2 && *std::min_element(t_sizes.begin(), t_sizes.end()) == 1));
  };

  TORCH_CHECK(mat_a.size(0) == scales_a.size(0), "size of scales_a is not matched");
  TORCH_CHECK(mat_a.size(1) / 128 == scales_a.size(1), "size of scales_a is not matched");
  TORCH_CHECK(scales_a.stride(0) == 1 || is_contiguous_vector(scales_a), "scales_a must be M major");
  TORCH_CHECK(mat_b.size(0) / 128 == scales_b.size(0), "size of scales_b is not matched");
  TORCH_CHECK(mat_b.size(1) / 128 == scales_b.size(1), "size of scales_b is not matched");
  TORCH_CHECK(scales_b.stride(0) == 1 || is_contiguous_vector(scales_b), "scales_b must be K major");
  TORCH_CHECK(scales_a.scalar_type() == torch::kFloat32, "scales_a must be Float32");
  TORCH_CHECK(scales_b.scalar_type() == torch::kFloat32, "scales_b must be Float32");

  torch::Tensor out = torch::empty({mat_a.size(0), mat_b.size(1)}, mat_a.options().dtype(out_dtype));
  TORCH_CHECK((out.size(1) * out.element_size()) % 16 == 0, "out must be multiple of 16 bytes for memory alignment");

  auto sm_version = getSMVersion();

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
#if defined CUDA_VERSION && CUDA_VERSION >= 12000
  if (sm_version == 90) {
    torch::Tensor scales_b_contiguous = scales_b.contiguous();
    if (out_dtype == torch::kBFloat16) {
      sm90_fp8_blockwise_dispatch_shape<cutlass::bfloat16_t>(out, mat_a, mat_b, scales_a, scales_b_contiguous);
    } else {
      sm90_fp8_blockwise_dispatch_shape<cutlass::half_t>(out, mat_a, mat_b, scales_a, scales_b_contiguous);
    }
    return out;
  }
#endif
#endif

#if defined(CUTLASS_ARCH_MMA_SM100A_SUPPORTED) || defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)
#if defined CUDA_VERSION && CUDA_VERSION >= 12080
  if (sm_version == 100) {
    int64_t original_rows = mat_a.size(0);

    torch::Tensor mat_a_padded = pad_tensor(mat_a, /*alignment=*/4);
    torch::Tensor scales_a_padded = pad_tensor(scales_a, /*alignment=*/4, /*col_major=*/true);
    torch::Tensor out_padded = torch::empty({mat_a_padded.size(0), mat_b.size(1)}, out.options());

    if (out_dtype == torch::kBFloat16) {
      sm100_fp8_blockwise_dispatch_shape<cutlass::bfloat16_t>(
          out_padded, mat_a_padded, mat_b, scales_a_padded, scales_b);
    } else {
      sm100_fp8_blockwise_dispatch_shape<cutlass::half_t>(out_padded, mat_a_padded, mat_b, scales_a_padded, scales_b);
    }
    return out_padded.slice(0, 0, original_rows);
  }
#endif
#endif
  TORCH_CHECK_NOT_IMPLEMENTED(
      false, "No implemented fp8_blockwise_scaled_mm for current compute capability: ", sm_version);
}
