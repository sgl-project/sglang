#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>

#include <cmath>
#include <cub/block/block_reduce.cuh>
#include <flashinfer/vec_dtypes.cuh>

#include "utils.h"

template <typename T>
__global__ void per_token_quant_fp8_kernel(
    const T* __restrict__ input,
    FP8_TYPE* __restrict__ output_q,
    float* __restrict__ output_s,
    const int64_t hidden_dim,
    const int64_t num_tokens) {
  const int token_idx = blockIdx.x;
  if (token_idx >= num_tokens) return;

  const int tid = threadIdx.x;
  const int block_dim = blockDim.x;

  const T* token_input = input + token_idx * hidden_dim;
  FP8_TYPE* token_output = output_q + token_idx * hidden_dim;

  float max_value = 0.0f;

  // We want to store 128 bits of data at a time. 16 = 128 / 8 bits
  // Load is already vectorized, so 16 elements work for T.
  const uint32_t VEC_SIZE = 16;
  using vec_t = flashinfer::vec_t<T, VEC_SIZE>;
  const int32_t num_vec_elems = hidden_dim / VEC_SIZE;

  // Find max using vectorized loads
  for (int32_t i = tid; i < num_vec_elems; i += block_dim) {
    vec_t input_vec;
    input_vec.cast_load(token_input + i * VEC_SIZE);

#pragma unroll
    for (uint32_t j = 0; j < VEC_SIZE; ++j) {
      float val = static_cast<float>(input_vec[j]);
      max_value = fmaxf(max_value, fabsf(val));
    }
  }

  max_value = blockReduceMax(max_value);

  __shared__ float scale;
  if (tid == 0) {
    scale = max_value / FP8_E4M3_MAX;
    output_s[token_idx] = scale;
  }
  __syncthreads();

  const float scale_inv = 1.0f / scale;

  // Quantize using vectorized loads
  for (int32_t i = tid; i < num_vec_elems; i += block_dim) {
    vec_t input_vec;
    input_vec.cast_load(token_input + i * VEC_SIZE);

    FP8_TYPE output_arr[VEC_SIZE];
#pragma unroll
    for (uint32_t j = 0; j < VEC_SIZE; ++j) {
      float val = fmaxf(fminf(static_cast<float>(input_vec[j]) * scale_inv, FP8_E4M3_MAX), -FP8_E4M3_MAX);
#ifndef USE_ROCM
      output_arr[j] = static_cast<FP8_TYPE>(val);
#else
      output_arr[j] = c10::Float8_e4m3fnuz(
          __hip_cvt_float_to_fp8(val, fp8::fp8_type::__default_saturation, fp8::fp8_type::__default_interpret),
          c10::Float8_e4m3fnuz::from_bits());
#endif
    }

    *(uint4*)(token_output + i * VEC_SIZE) = *(uint4*)output_arr;
  }
}

void sgl_per_token_quant_fp8(torch::Tensor input, torch::Tensor output_q, torch::Tensor output_s) {
  CHECK_INPUT(input);
  CHECK_INPUT(output_q);
  CHECK_INPUT(output_s);

  const auto input_sizes = input.sizes();
  const int64_t num_tokens = input_sizes[0];
  const int64_t hidden_dim = input_sizes[1];

  TORCH_CHECK(hidden_dim % 16 == 0, "Hidden dimension must be divisible by 16, but got ", hidden_dim);

  const int block_size = 256;
  const int num_blocks = num_tokens;

  dim3 grid(num_blocks);
  dim3 block(block_size);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), scalar_t, [&] {
    per_token_quant_fp8_kernel<scalar_t><<<grid, block, 0, stream>>>(
        static_cast<scalar_t*>(input.data_ptr()),
        static_cast<FP8_TYPE*>(output_q.data_ptr()),
        static_cast<float*>(output_s.data_ptr()),
        hidden_dim,
        num_tokens);
    return true;
  });
}
