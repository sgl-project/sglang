#include "hip/hip_runtime.h"
// clang-format off
#include <tuple>
#include <cmath>
#include <ATen/cuda/HIPContext.h>
#include <cute/algorithm/functional.hpp>
#include <cute/algorithm/gemm.hpp>
#include <cute/arch/cluster_sm90.hpp>
#include <cute/tensor.hpp>
#include <flashinfer/vec_dtypes.cuh>
// clang-format on

#include "utils.h"

using namespace cute;

static constexpr int kWarpSize = 32;

// ---------------------------------------------------------------------------
// 1. SmallWarp kernel — warp‑local, no shared memory
//    • One warp handles one token.
//    • Eight tokens per 256‑thread CTA.
// ---------------------------------------------------------------------------
template <typename T, int kTokensPerCTA = 8, int kVecSize = 16>
__global__ void per_token_quant_fp8_kernel(
    const T* __restrict__ input,
    FP8_TYPE* __restrict__ output_q,
    float* __restrict__ output_s,
    const int64_t hidden_dim,
    const int64_t num_tokens) {
  const int warp_id = threadIdx.x / kWarpSize;        // 0‑7  (8 warps)
  const int lane_id = threadIdx.x & (kWarpSize - 1);  // 0‑31
  const int token = blockIdx.x * kTokensPerCTA + warp_id;
  if (token >= num_tokens) return;

  // Global tensors for this token
  auto gmem_in = make_tensor(input + token * hidden_dim, make_shape(hidden_dim));
  auto gmem_out = make_tensor(output_q + token * hidden_dim, make_shape(hidden_dim));
  auto gmem_s = make_tensor(output_s + token, make_shape(1));

  //
  // Pass-1: compute max across whole token
  //
  float max_value = 0.f;
  using vec_t = flashinfer::vec_t<T, kVecSize>;
  const int32_t num_vec_elems = hidden_dim / kVecSize;

  for (int32_t i = lane_id; i < num_vec_elems; i += kWarpSize) {
    vec_t input_vec;
    input_vec.cast_load(gmem_in.data() + i * kVecSize);

#pragma unroll
    for (uint32_t j = 0; j < kVecSize; ++j) {
      max_value = fmaxf(max_value, fabsf(static_cast<float>(input_vec[j])));
    }
  }

  float warp_max = warpReduceMax(max_value);

  __shared__ float scale;
  scale = warp_max / FP8_E4M3_MAX;
  // Broadcast scale
  if (lane_id == 0) {
    gmem_s(0) = scale;
  }
  float scale_inv = (scale == 0.f) ? 0.f : 1.0f / scale;

  //
  // Pass-2: quantise and write back
  //
  for (int i = lane_id; i < num_vec_elems; i += kWarpSize) {
    vec_t input_vec;
    input_vec.cast_load(gmem_in.data() + i * kVecSize);
    FP8_TYPE output_arr[kVecSize];
#pragma unroll
    for (uint32_t j = 0; j < kVecSize; ++j) {
      float val = static_cast<float>(input_vec[j]) * scale_inv;
      val = fmaxf(fminf(val, FP8_E4M3_MAX), -FP8_E4M3_MAX);

#ifndef USE_ROCM
      output_arr[j] = static_cast<FP8_TYPE>(val);
#else
      output_arr[j] = c10::Float8_e4m3fnuz(
          __hip_cvt_float_to_fp8(val, fp8::fp8_type::__default_saturation, fp8::fp8_type::__default_interpret),
          c10::Float8_e4m3fnuz::from_bits());
#endif
    }
    *(uint4*)(gmem_out.data() + i * kVecSize) = *(uint4*)output_arr;
  }
}

void sgl_per_token_quant_fp8(torch::Tensor input, torch::Tensor output_q, torch::Tensor output_s) {
  CHECK_INPUT(input);
  CHECK_INPUT(output_q);
  CHECK_INPUT(output_s);

  const auto input_sizes = input.sizes();
  const int64_t num_tokens = input_sizes[0];
  const int64_t hidden_dim = input_sizes[1];

  TORCH_CHECK(hidden_dim % 16 == 0, "Hidden dimension must be divisible by 16, but got ", hidden_dim);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), scalar_t, [&] {
    constexpr int TOKENS_PER_CTA = 8;
    constexpr int THREADS = TOKENS_PER_CTA * WARP_SIZE;  // 256
    dim3 grid((num_tokens + TOKENS_PER_CTA - 1) / TOKENS_PER_CTA);
    dim3 block(THREADS);

    per_token_quant_fp8_kernel<scalar_t, TOKENS_PER_CTA, 16><<<grid, block, 0, stream>>>(
        static_cast<const scalar_t*>(input.data_ptr()),
        static_cast<FP8_TYPE*>(output_q.data_ptr()),
        static_cast<float*>(output_s.data_ptr()),
        hidden_dim,
        num_tokens);
    return true;
  });
}
