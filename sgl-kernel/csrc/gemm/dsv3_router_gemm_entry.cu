#include "hip/hip_runtime.h"
/*
 * Adapted from
 * https://github.com/NVIDIA/TensorRT-LLM/blob/main/cpp/tensorrt_llm/kernels/dsv3MinLatencyKernels/dsv3RouterGemm.cu
 * https://github.com/NVIDIA/TensorRT-LLM/blob/main/cpp/tensorrt_llm/thop/dsv3RouterGemmOp.cpp
 *
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include "hip/hip_bf16.h"
#include "hip/hip_runtime.h"
#include "utils.h"

template <typename T, int kNumTokens, int kNumExperts, int kHiddenDim>
void invokeRouterGemmFloatOutput(float* output, T const* mat_a, T const* mat_b, hipStream_t stream);

template <typename T, int kNumTokens, int kNumExperts, int kHiddenDim>
void invokeRouterGemmBf16Output(__hip_bfloat16* output, T const* mat_a, T const* mat_b, hipStream_t stream);

template <int kBegin, int kEnd, int kNumExperts, int kHiddenDim>
struct LoopUnroller {
  static void unroll_float_output(
      int num_tokens, float* output, __hip_bfloat16 const* input, __hip_bfloat16 const* weights, hipStream_t stream) {
    if (num_tokens == kBegin) {
      invokeRouterGemmFloatOutput<__hip_bfloat16, kBegin, kNumExperts, kHiddenDim>(output, input, weights, stream);
    } else {
      LoopUnroller<kBegin + 1, kEnd, kNumExperts, kHiddenDim>::unroll_float_output(
          num_tokens, output, input, weights, stream);
    }
  }

  static void unroll_bf16_output(
      int num_tokens,
      __hip_bfloat16* output,
      __hip_bfloat16 const* input,
      __hip_bfloat16 const* weights,
      hipStream_t stream) {
    if (num_tokens == kBegin) {
      invokeRouterGemmBf16Output<__hip_bfloat16, kBegin, kNumExperts, kHiddenDim>(output, input, weights, stream);
    } else {
      LoopUnroller<kBegin + 1, kEnd, kNumExperts, kHiddenDim>::unroll_bf16_output(
          num_tokens, output, input, weights, stream);
    }
  }
};

template <int kEnd, int kNumExperts, int kHiddenDim>
struct LoopUnroller<kEnd, kEnd, kNumExperts, kHiddenDim> {
  static void unroll_float_output(
      int num_tokens, float* output, __hip_bfloat16 const* input, __hip_bfloat16 const* weights, hipStream_t stream) {
    if (num_tokens == kEnd) {
      invokeRouterGemmFloatOutput<__hip_bfloat16, kEnd, kNumExperts, kHiddenDim>(output, input, weights, stream);
    } else {
      throw std::invalid_argument("Invalid num_tokens, only supports 1 to 16");
    }
  }

  static void unroll_bf16_output(
      int num_tokens,
      __hip_bfloat16* output,
      __hip_bfloat16 const* input,
      __hip_bfloat16 const* weights,
      hipStream_t stream) {
    if (num_tokens == kEnd) {
      invokeRouterGemmBf16Output<__hip_bfloat16, kEnd, kNumExperts, kHiddenDim>(output, input, weights, stream);
    } else {
      throw std::invalid_argument("Invalid num_tokens, only supports 1 to 16");
    }
  }
};

void dsv3_router_gemm(
    torch::Tensor& output,       // [num_tokens, num_experts]
    const torch::Tensor& mat_a,  // [num_tokens, hidden_dim]
    const torch::Tensor& mat_b   // [num_experts, hidden_dim]
) {
  TORCH_CHECK(output.dim() == 2 && mat_a.dim() == 2 && mat_b.dim() == 2);

  const int num_tokens = mat_a.size(0);
  constexpr int num_experts = 256;
  constexpr int hidden_dim = 7168;

  TORCH_CHECK(mat_a.size(1) == mat_b.size(1), "mat_a and mat_b must have the same hidden_dim");
  TORCH_CHECK(mat_a.size(1) == hidden_dim, "currently hidden_dim only supports 7168");
  TORCH_CHECK(mat_b.size(0) == num_experts, "currently num_experts only supports 256");
  TORCH_CHECK(
      num_tokens >= 1 && num_tokens <= 16, "currently num_tokens must be less than or equal to 16 for router_gemm");
  TORCH_CHECK(mat_a.dtype() == torch::kBFloat16, "mat_a must be bf16");
  TORCH_CHECK(mat_b.dtype() == torch::kBFloat16, "mat_b must be bf16");
  TORCH_CHECK(
      output.dtype() == torch::kFloat32 || output.dtype() == torch::kBFloat16, "output must be float32 or bf16");

  auto const sm = getSMVersion();
  TORCH_CHECK(sm >= 90, "required CUDA ARCH >= SM_90");

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (output.dtype() == torch::kFloat32) {
    LoopUnroller<1, 16, num_experts, hidden_dim>::unroll_float_output(
        num_tokens,
        reinterpret_cast<float*>(output.mutable_data_ptr()),
        reinterpret_cast<__hip_bfloat16 const*>(mat_a.data_ptr()),
        reinterpret_cast<__hip_bfloat16 const*>(mat_b.data_ptr()),
        stream);
  } else if (output.dtype() == torch::kBFloat16) {
    LoopUnroller<1, 16, num_experts, hidden_dim>::unroll_bf16_output(
        num_tokens,
        reinterpret_cast<__hip_bfloat16*>(output.mutable_data_ptr()),
        reinterpret_cast<__hip_bfloat16 const*>(mat_a.data_ptr()),
        reinterpret_cast<__hip_bfloat16 const*>(mat_b.data_ptr()),
        stream);
  }
}
