#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025 by SGLang team.
 * Copyright (c) 2025 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/all.h>

#include <cmath>
#include <flashinfer/vec_dtypes.cuh>

#include "utils.h"

template <typename scalar_t, bool IS_NEOX>
inline __device__ void apply_token_rotary_embedding(
    scalar_t* __restrict__ arr,
    const scalar_t* __restrict__ cos_ptr,
    const scalar_t* __restrict__ sin_ptr,
    int rot_offset,
    int embed_dim) {
  int x_index, y_index;

  if (IS_NEOX) {
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;

    scalar_t cos_val = SGLANG_LDG(cos_ptr + rot_offset);
    scalar_t sin_val = SGLANG_LDG(sin_ptr + rot_offset);

    const scalar_t x = arr[x_index];
    const scalar_t y = arr[y_index];
    arr[x_index] = x * cos_val - y * sin_val;
    arr[y_index] = y * cos_val + x * sin_val;

  } else {
    // GPT-J style / LLaMA style, matching the Python if cos/sin are [..., head_size]
    x_index = rot_offset;              // first half
    y_index = rot_offset + embed_dim;  // second half

    const scalar_t cos_val_x = SGLANG_LDG(cos_ptr + rot_offset);
    const scalar_t sin_val_x = SGLANG_LDG(sin_ptr + rot_offset);
    const scalar_t cos_val_y = SGLANG_LDG(cos_ptr + rot_offset + embed_dim);
    const scalar_t sin_val_y = SGLANG_LDG(sin_ptr + rot_offset + embed_dim);

    const scalar_t x = arr[x_index];
    const scalar_t y = arr[y_index];
    arr[x_index] = x * cos_val_x - y * sin_val_x;
    arr[y_index] = y * cos_val_y + x * sin_val_y;
  }
}

template <typename scalar_t, bool IS_NEOX>
inline __device__ void apply_rotary_embedding(
    scalar_t* __restrict__ query,                        // [num_heads, head_size]
    scalar_t* __restrict__ key,                          // [num_kv_heads, head_size]
    const scalar_t* __restrict__ current_token_cos_ptr,  // [rot_dim]
    const scalar_t* __restrict__ current_token_sin_ptr,  // [rot_dim]
    const int head_size,
    const int num_heads,
    const int num_kv_heads,
    const int rot_dim,
    const int64_t head_stride_query,
    const int64_t head_stride_key) {
  const int embed_dim_for_rotation = rot_dim / 2;

  const int nq_pairs = num_heads * embed_dim_for_rotation;
  for (int i = threadIdx.x; i < nq_pairs; i += blockDim.x) {
    const int head_idx = i / embed_dim_for_rotation;
    const int rot_offset = i % embed_dim_for_rotation;

    scalar_t* query_for_token_head = query + head_idx * (int)head_stride_query;

    apply_token_rotary_embedding<scalar_t, IS_NEOX>(
        query_for_token_head, current_token_cos_ptr, current_token_sin_ptr, rot_offset, embed_dim_for_rotation);
  }

  if (key != nullptr) {
    const int nk_pairs = num_kv_heads * embed_dim_for_rotation;
    for (int i = threadIdx.x; i < nk_pairs; i += blockDim.x) {
      const int head_idx = i / embed_dim_for_rotation;
      const int rot_offset = i % embed_dim_for_rotation;

      scalar_t* key_for_token_head = key + head_idx * (int)head_stride_key;

      apply_token_rotary_embedding<scalar_t, IS_NEOX>(
          key_for_token_head, current_token_cos_ptr, current_token_sin_ptr, rot_offset, embed_dim_for_rotation);
    }
  }
}

template <typename scalar_t, bool IS_NEOX>
__global__ void rotary_embedding_kernel(
    const scalar_t* __restrict__ cos_data,  // [num_tokens, rot_dim_arg]
    const scalar_t* __restrict__ sin_data,  // [num_tokens, rot_dim_arg]
    scalar_t* __restrict__ query_total,
    scalar_t* __restrict__ key_total,
    const int rot_dim_arg,
    const int64_t query_token_stride,
    const int64_t key_token_stride,
    const int64_t head_stride_query,
    const int64_t head_stride_key,
    const int num_heads,
    const int num_kv_heads,
    const int head_size) {
  const int token_idx = blockIdx.x;
  const scalar_t* current_token_cos_ptr = cos_data + token_idx * rot_dim_arg;
  const scalar_t* current_token_sin_ptr = sin_data + token_idx * rot_dim_arg;

  scalar_t* query_for_token = query_total + token_idx * (int)query_token_stride;
  scalar_t* key_for_token = (key_total != nullptr) ? (key_total + token_idx * (int)key_token_stride) : nullptr;

  apply_rotary_embedding<scalar_t, IS_NEOX>(
      query_for_token,
      key_for_token,
      current_token_cos_ptr,
      current_token_sin_ptr,
      head_size,
      num_heads,
      num_kv_heads,
      rot_dim_arg,
      head_stride_query,
      head_stride_key);
}

void rotary_embedding(
    at::Tensor& cos,
    at::Tensor& sin,
    at::Tensor& query,
    const std::optional<at::Tensor>& key,
    int64_t head_size,
    bool is_neox) {
  TORCH_CHECK(
      query.dim() == 2 || query.dim() == 3,
      "query must be in  shape [num_tokens, hidden_size] or [num_tokens, num_heads, head_size]");
  if (key.has_value()) {
    TORCH_CHECK(
        key->dim() == 2 || key->dim() == 3,
        "key must be in  shape [num_tokens, hidden_size] or [num_tokens, num_kv_heads, head_size]");
  }

  int64_t num_tokens = query.size(0);

  TORCH_CHECK(cos.dim() == 2, "cos must be in shape [num_tokens, D_cos]");
  TORCH_CHECK(sin.dim() == 2, "sin must be in  shape [num_tokens, D_sin]");
  TORCH_CHECK(cos.size(0) == num_tokens, "cos num_tokens mismatch with query");
  TORCH_CHECK(sin.size(0) == num_tokens, "sin num_tokens mismatch with query");
  TORCH_CHECK(cos.size(1) == sin.size(1), "cos and sin D_cos/D_sin mismatch");

  TORCH_CHECK(cos.scalar_type() == query.scalar_type(), "cos dtype mismatch");
  TORCH_CHECK(sin.scalar_type() == query.scalar_type(), "sin dtype mismatch");
  TORCH_CHECK(cos.is_cuda() && sin.is_cuda() && query.is_cuda(), "All tensors must be on CUDA");
  if (key.has_value()) {
    TORCH_CHECK(key->is_cuda(), "Key tensor must be on CUDA if provided");
    TORCH_CHECK(key->scalar_type() == query.scalar_type(), "Key dtype mismatch");
  }

  int query_hidden_size_calculated;
  if (query.dim() == 2) {
    query_hidden_size_calculated = (int)query.size(1);
  } else {
    query_hidden_size_calculated = (int)query.size(1) * (int)query.size(2);
    TORCH_CHECK(query.size(2) == head_size, "Query head_size mismatch in 3D tensor");
  }
  TORCH_CHECK(query_hidden_size_calculated % head_size == 0, "query_hidden_size not divisible by head_size");
  int num_heads = (int)query_hidden_size_calculated / (int)head_size;

  int key_hidden_size_calculated = 0;
  int num_kv_heads = num_heads;
  if (key.has_value()) {
    TORCH_CHECK((int)key->size(0) == num_tokens, "Key num_tokens mismatch");
    if (key->dim() == 2) {
      key_hidden_size_calculated = (int)key->size(1);
    } else {
      key_hidden_size_calculated = (int)key->size(1) * (int)key->size(2);
      TORCH_CHECK((int)key->size(2) == head_size, "Key head_size mismatch in 3D tensor");
    }
    TORCH_CHECK(key_hidden_size_calculated % head_size == 0, "key_hidden_size not divisible by head_size");
    num_kv_heads = key_hidden_size_calculated / (int)head_size;
  }
  TORCH_CHECK(num_heads % num_kv_heads == 0, "num_heads must be divisible by num_kv_heads");

  int rot_dim_from_cache = (int)cos.size(1);

  int64_t query_token_stride = query_hidden_size_calculated;
  int64_t key_token_stride = key.has_value() ? key_hidden_size_calculated : 0;

  int64_t head_stride_query;
  if (query.dim() == 3 && query.size(1) == num_heads && query.size(2) == head_size) {
    head_stride_query = query.stride(1);
  } else {
    head_stride_query = head_size;
  }

  int64_t head_stride_key = head_size;
  if (key.has_value()) {
    if (key->dim() == 3 && key->size(1) == num_kv_heads && key->size(2) == head_size) {
      head_stride_key = key->stride(1);
    } else {
      head_stride_key = head_size;
    }
  }

  dim3 grid((int)num_tokens);

  int embed_dim_for_block_calc = rot_dim_from_cache / 2;
  int max_pairs_to_rotate_per_token =
      std::max(num_heads * embed_dim_for_block_calc, num_kv_heads * embed_dim_for_block_calc);
  dim3 block(std::min<int64_t>(max_pairs_to_rotate_per_token, 512L));

  if (block.x == 0 && num_tokens > 0) block.x = 1;

  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  SGLANG_DISPATCH_FLOATING_TYPES(query.scalar_type(), "rotary_embedding", [&] {
    if (is_neox) {
      rotary_embedding_kernel<scalar_t, true><<<grid, block, 0, stream>>>(
          cos.data_ptr<scalar_t>(),
          sin.data_ptr<scalar_t>(),
          query.data_ptr<scalar_t>(),
          key.has_value() ? key->data_ptr<scalar_t>() : nullptr,
          rot_dim_from_cache,
          query_token_stride,
          key_token_stride,
          head_stride_query,
          head_stride_key,
          num_heads,
          num_kv_heads,
          (int)head_size);
    } else {
      rotary_embedding_kernel<scalar_t, false><<<grid, block, 0, stream>>>(
          cos.data_ptr<scalar_t>(),
          sin.data_ptr<scalar_t>(),
          query.data_ptr<scalar_t>(),
          key.has_value() ? key->data_ptr<scalar_t>() : nullptr,
          rot_dim_from_cache,
          query_token_stride,
          key_token_stride,
          head_stride_query,
          head_stride_key,
          num_heads,
          num_kv_heads,
          (int)head_size);
    }
  });
}
