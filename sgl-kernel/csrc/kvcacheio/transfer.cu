#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAException.h>
#include <c10/util/irange.h>

#include <cstdint>

#include "pytorch_extension_utils.h"

__device__ __forceinline__ void
transfer_item_warp(int32_t lane_id, const void* src_addr, void* dst_addr, int64_t item_size_bytes) {
  // todo, different chunk size
  int total_chunks = item_size_bytes / 8;
  const int64_t* src_8 = reinterpret_cast<const int64_t*>(src_addr);
  int64_t* dst_8 = reinterpret_cast<int64_t*>(dst_addr);
#pragma unroll
  for (int j = lane_id; j < total_chunks; j += 32) {
    const int64_t* src_addr_lane = &src_8[j];
    int64_t* dst_addr_lane = &dst_8[j];
    int64_t temp_val;
    asm volatile("ld.global.nc.b64 %0, [%1];" : "=l"(temp_val) : "l"(src_addr_lane) : "memory");
    asm volatile("st.global.cg.b64 [%0], %1;" ::"l"(dst_addr_lane), "l"(temp_val) : "memory");
  }
}

template <typename T>
__device__ __forceinline__ T* get_global_offset_lf(
    T* base,
    const uintptr_t* __restrict__ /*unused*/,
    int64_t layer_id,
    int64_t layer_dim,
    int64_t page_id,
    int64_t item_size_bytes) {
  // layer first
  return base + layer_id * layer_dim + page_id * item_size_bytes;
}

template <typename T>
__device__ __forceinline__ T* get_global_offset_pf(
    T* base,
    const uintptr_t* __restrict__ /*unused*/,
    int64_t layer_id,
    int64_t page_dim,
    int64_t page_id,
    int64_t item_size_bytes) {
  // page first
  return base + page_id * page_dim + layer_id * item_size_bytes;
}

// get offset from layer base table when layers are not contiguous
template <typename T>
__device__ __forceinline__ T* get_global_offset_lf_tbl(
    T* /*unused*/,
    const uintptr_t* __restrict__ layer_base_tbl,
    int64_t layer_id,
    int64_t /*unused*/,
    int64_t page_id,
    int64_t item_size_bytes) {
  return reinterpret_cast<T*>(layer_base_tbl[layer_id]) + page_id * item_size_bytes;
}

template <auto SrcOffsetFn, auto DstOffsetFn, bool IsMLA>
__global__ void transfer_kernel_impl(
    const void* __restrict__ src_k,
    void* __restrict__ dst_k,
    const void* __restrict__ src_v,
    void* __restrict__ dst_v,
    const int64_t* __restrict__ src_indices,
    const int64_t* __restrict__ dst_indices,
    int64_t start_layer_id,
    int64_t num_layers_to_process,
    int64_t num_items,
    int64_t items_per_warp,
    int64_t item_size_bytes,
    int64_t src_layout_dim,
    int64_t dst_layout_dim,
    const uintptr_t* __restrict__ src_k_layer_tbl,
    const uintptr_t* __restrict__ dst_k_layer_tbl,
    const uintptr_t* __restrict__ src_v_layer_tbl,
    const uintptr_t* __restrict__ dst_v_layer_tbl) {
  int32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  int32_t lane_id = tid % 32;
  int32_t warp_id = tid / 32;

  for (int i = 0; i < items_per_warp; ++i) {
    int64_t item_id = warp_id * items_per_warp + i;
    if (item_id >= num_items) {
      break;
    }
    const int64_t src_page_id = src_indices[item_id];
    const int64_t dst_page_id = dst_indices[item_id];

    // Loop over layers if necessary
    for (int64_t layer_id = start_layer_id; layer_id < start_layer_id + num_layers_to_process; ++layer_id) {
      const char* src_ptr = SrcOffsetFn(
          static_cast<const char*>(src_k), src_k_layer_tbl, layer_id, src_layout_dim, src_page_id, item_size_bytes);
      char* dst_ptr = DstOffsetFn(
          static_cast<char*>(dst_k), dst_k_layer_tbl, layer_id, dst_layout_dim, dst_page_id, item_size_bytes);
      transfer_item_warp(lane_id, src_ptr, dst_ptr, item_size_bytes);

      if constexpr (!IsMLA) {
        const char* src_v_ptr = SrcOffsetFn(
            static_cast<const char*>(src_v), src_v_layer_tbl, layer_id, src_layout_dim, src_page_id, item_size_bytes);
        char* dst_v_ptr = DstOffsetFn(
            static_cast<char*>(dst_v), dst_v_layer_tbl, layer_id, dst_layout_dim, dst_page_id, item_size_bytes);
        transfer_item_warp(lane_id, src_v_ptr, dst_v_ptr, item_size_bytes);
      }
    }
  }
}

template <auto SrcOffsetFn, auto DstOffsetFn, bool IsMLA>
void transfer_kv_launcher(
    const at::Tensor& src_k,
    at::Tensor& dst_k,
    const at::Tensor& src_v,
    at::Tensor& dst_v,
    const at::Tensor& src_indices,
    const at::Tensor& dst_indices,
    int64_t start_layer_id,
    int64_t num_layers_to_process,
    int64_t item_size,
    int64_t src_layout_dim,
    int64_t dst_layout_dim,
    const at::Tensor& src_k_layers,
    const at::Tensor& dst_k_layers,
    const at::Tensor& src_v_layers,
    const at::Tensor& dst_v_layers,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  TORCH_CHECK(src_indices.is_cuda(), "Source indices must be a CUDA tensor");
  TORCH_CHECK(dst_indices.is_cuda(), "Destination indices must be a CUDA tensor");
  TORCH_CHECK(src_indices.scalar_type() == at::kLong, "Source indices must be of type long");
  TORCH_CHECK(dst_indices.scalar_type() == at::kLong, "Destination indices must be of type long");
  TORCH_CHECK(src_indices.numel() == dst_indices.numel(), "Source and destination indices must have the same length");
  TORCH_CHECK(item_size % 8 == 0, "Item byte size must be divisible by 8");

  auto div_up = [](int64_t x, int64_t y) { return (x + y - 1) / y; };
  const int64_t num_items = src_indices.numel();
  const int64_t items_per_warp = div_up(num_items, block_quota * num_warps_per_block);
  const int32_t num_blocks = div_up(num_items, items_per_warp * num_warps_per_block);
  dim3 grid_dim(num_blocks, 1, 1);
  const int32_t threads_per_block = num_warps_per_block * 32;

  const void* src_k_ptr = src_k.defined() ? src_k.data_ptr() : nullptr;
  void* dst_k_ptr = dst_k.defined() ? dst_k.data_ptr() : nullptr;
  const void* src_v_ptr = IsMLA || !src_v.defined() ? nullptr : src_v.data_ptr();
  void* dst_v_ptr = IsMLA || !dst_v.defined() ? nullptr : dst_v.data_ptr();
  const uintptr_t* src_k_tbl_ptr = src_k_layers.defined() ? src_k_layers.data_ptr<uintptr_t>() : nullptr;
  const uintptr_t* dst_k_tbl_ptr = dst_k_layers.defined() ? dst_k_layers.data_ptr<uintptr_t>() : nullptr;
  const uintptr_t* src_v_tbl_ptr = IsMLA || !src_v_layers.defined() ? nullptr : src_v_layers.data_ptr<uintptr_t>();
  const uintptr_t* dst_v_tbl_ptr = IsMLA || !dst_v_layers.defined() ? nullptr : dst_v_layers.data_ptr<uintptr_t>();

  hipStream_t torch_current_stream = at::cuda::getCurrentCUDAStream();
  transfer_kernel_impl<SrcOffsetFn, DstOffsetFn, IsMLA><<<grid_dim, threads_per_block, 0, torch_current_stream>>>(
      src_k_ptr,
      dst_k_ptr,
      src_v_ptr,
      dst_v_ptr,
      src_indices.data_ptr<int64_t>(),
      dst_indices.data_ptr<int64_t>(),
      start_layer_id,
      num_layers_to_process,
      num_items,
      items_per_warp,
      item_size,
      src_layout_dim,
      dst_layout_dim,
      src_k_tbl_ptr,
      dst_k_tbl_ptr,
      src_v_tbl_ptr,
      dst_v_tbl_ptr);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void transfer_kv_per_layer(
    const at::Tensor src_k,
    at::Tensor dst_k,
    const at::Tensor src_v,
    at::Tensor dst_v,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  at::Tensor empty;
  transfer_kv_launcher<get_global_offset_lf<const char>, get_global_offset_lf<char>, false>(
      src_k,
      dst_k,
      src_v,
      dst_v,
      src_indices,
      dst_indices,
      0,
      1,
      item_size,
      0,
      0,
      empty,
      empty,
      empty,
      empty,
      block_quota,
      num_warps_per_block);
}

void transfer_kv_per_layer_pf_lf(
    const at::Tensor src_k,
    at::Tensor dst_k,
    const at::Tensor src_v,
    at::Tensor dst_v,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t src_layout_dim,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  at::Tensor empty;
  transfer_kv_launcher<get_global_offset_pf<const char>, get_global_offset_lf<char>, false>(
      src_k,
      dst_k,
      src_v,
      dst_v,
      src_indices,
      dst_indices,
      0,
      1,
      item_size,
      src_layout_dim,
      0,
      empty,
      empty,
      empty,
      empty,
      block_quota,
      num_warps_per_block);
}

void transfer_kv_all_layer(
    const at::Tensor src_k_layers,
    const at::Tensor dst_k_layers,
    const at::Tensor src_v_layers,
    const at::Tensor dst_v_layers,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t num_layers,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  TORCH_CHECK(num_layers == src_k_layers.size(0), "Number of layers in source k tensor does not match num_layers");
  at::Tensor empty;
  transfer_kv_launcher<get_global_offset_lf_tbl<const char>, get_global_offset_lf_tbl<char>, false>(
      empty,
      empty,
      empty,
      empty,
      src_indices,
      dst_indices,
      0,
      num_layers,
      item_size,
      0,
      0,
      src_k_layers,
      dst_k_layers,
      src_v_layers,
      dst_v_layers,
      block_quota,
      num_warps_per_block);
}

void transfer_kv_all_layer_lf_pf(
    const at::Tensor src_k_layers,
    at::Tensor dst_k,
    const at::Tensor src_v_layers,
    at::Tensor dst_v,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t dst_layout_dim,
    int64_t num_layers,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  TORCH_CHECK(num_layers == src_k_layers.size(0), "Number of layers in source k tensor does not match num_layers");
  at::Tensor empty;
  transfer_kv_launcher<get_global_offset_lf_tbl<const char>, get_global_offset_pf<char>, false>(
      empty,
      dst_k,
      empty,
      dst_v,
      src_indices,
      dst_indices,
      0,
      num_layers,
      item_size,
      0,
      dst_layout_dim,
      src_k_layers,
      empty,
      src_v_layers,
      empty,
      block_quota,
      num_warps_per_block);
}

void transfer_kv_per_layer_mla(
    const at::Tensor src,
    at::Tensor dst,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  at::Tensor empty;
  transfer_kv_launcher<get_global_offset_lf<const char>, get_global_offset_lf<char>, true>(
      src,
      dst,
      empty,
      empty,
      src_indices,
      dst_indices,
      0,
      1,
      item_size,
      0,
      0,
      empty,
      empty,
      empty,
      empty,
      block_quota,
      num_warps_per_block);
}

void transfer_kv_per_layer_mla_pf_lf(
    const at::Tensor src,
    at::Tensor dst,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t src_layout_dim,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  at::Tensor empty;
  transfer_kv_launcher<get_global_offset_pf<const char>, get_global_offset_lf<char>, true>(
      src,
      dst,
      empty,
      empty,
      src_indices,
      dst_indices,
      0,
      1,
      item_size,
      src_layout_dim,
      0,
      empty,
      empty,
      empty,
      empty,
      block_quota,
      num_warps_per_block);
}

void transfer_kv_all_layer_mla(
    const at::Tensor src_layers,
    const at::Tensor dst_layers,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t num_layers,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  TORCH_CHECK(num_layers == src_layers.size(0), "Number of layers in source tensor does not match num_layers");
  at::Tensor empty;
  transfer_kv_launcher<get_global_offset_lf_tbl<const char>, get_global_offset_lf_tbl<char>, true>(
      empty,
      empty,
      empty,
      empty,
      src_indices,
      dst_indices,
      0,
      num_layers,
      item_size,
      0,
      0,
      src_layers,
      dst_layers,
      empty,
      empty,
      block_quota,
      num_warps_per_block);
}

void transfer_kv_all_layer_mla_lf_pf(
    const at::Tensor src_layers,
    at::Tensor dst,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t dst_layout_dim,
    int64_t num_layers,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  TORCH_CHECK(num_layers == src_layers.size(0), "Number of layers in source tensor does not match num_layers");
  at::Tensor empty;
  transfer_kv_launcher<get_global_offset_lf_tbl<const char>, get_global_offset_pf<char>, true>(
      empty,
      dst,
      empty,
      empty,
      src_indices,
      dst_indices,
      0,
      num_layers,
      item_size,
      0,
      dst_layout_dim,
      src_layers,
      empty,
      empty,
      empty,
      block_quota,
      num_warps_per_block);
}

inline void transfer_page_direct(
    const at::Tensor& src_buffer,
    at::Tensor& dst_buffer,
    int64_t src_page_index,
    int64_t dst_page_index,
    int64_t page_size) {
  dst_buffer.slice(0, dst_page_index, dst_page_index + page_size)
      .copy_(
          src_buffer.slice(0, src_page_index, src_page_index + page_size),
          /* non_blocking= */ true);
}

void transfer_kv_direct(
    const std::vector<at::Tensor>& src_layers,
    std::vector<at::Tensor> dst_layers,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t page_size) {
  TORCH_CHECK(
      src_layers.size() == dst_layers.size(), "Source and destination layers must have the same number of layers");
  TORCH_CHECK(src_indices.numel() == dst_indices.numel(), "Source and destination indices must have the same length");
  TORCH_CHECK(page_size > 0, "Page size must be positive");
  TORCH_CHECK(src_indices.numel() % page_size == 0, "Source indices size must be divisible by page size");

  auto src_indices_cpu = src_indices.cpu();
  auto dst_indices_cpu = dst_indices.cpu();

  const int64_t num_pages = src_indices_cpu.size(0) / page_size;
  const int64_t num_layers = src_layers.size();

  for (int64_t i = 0; i < num_pages; ++i) {
    auto src_index = src_indices_cpu[i * page_size].item<int64_t>();
    auto dst_index = dst_indices_cpu[i * page_size].item<int64_t>();

    for (int64_t j = 0; j < num_layers; ++j) {
      transfer_page_direct(src_layers[j], dst_layers[j], src_index, dst_index, page_size);
    }
  }
}
