#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAException.h>
#include <c10/util/irange.h>

#include <cstdint>

#include "pytorch_extension_utils.h"

__device__ __forceinline__ void
transfer_item_warp(int32_t lane_id, const void* src_addr, void* dst_addr, int64_t item_size_bytes) {
  // todo, different chunk size
  int total_chunks = item_size_bytes / 8;
  const int64_t* src_8 = reinterpret_cast<const int64_t*>(src_addr);
  int64_t* dst_8 = reinterpret_cast<int64_t*>(dst_addr);
#pragma unroll
  for (int j = lane_id; j < total_chunks; j += 32) {
    const int64_t* src_addr_lane = &src_8[j];
    int64_t* dst_addr_lane = &dst_8[j];
    int64_t temp_val;
    asm volatile("ld.global.nc.b64 %0, [%1];" : "=l"(temp_val) : "l"(src_addr_lane) : "memory");
    asm volatile("st.global.cg.b64 [%0], %1;" ::"l"(dst_addr_lane), "l"(temp_val) : "memory");
  }
}

// todo, structs for different memory layout
__device__ __forceinline__ int64_t
get_global_offset_lf(int64_t layer_id, int64_t layer_dim, int64_t page_id, int64_t item_size_bytes) {
  // layer first
  return layer_id * layer_dim + page_id * item_size_bytes;
}

__device__ __forceinline__ int64_t
get_global_offset_pf(int64_t layer_id, int64_t page_dim, int64_t page_id, int64_t item_size_bytes) {
  // page first
  return page_id * page_dim + layer_id * item_size_bytes;
}

template <auto SrcOffsetFn, auto DstOffsetFn, bool IsMLA>
__global__ void transfer_kernel_impl(
    const void* __restrict__ src_k,
    void* __restrict__ dst_k,
    const void* __restrict__ src_v,
    void* __restrict__ dst_v,
    const int64_t* __restrict__ src_indices,
    const int64_t* __restrict__ dst_indices,
    int64_t start_layer_id,
    int64_t num_layers_to_process,
    int64_t num_items,
    int64_t items_per_warp,
    int64_t item_size_bytes,
    int64_t src_layout_dim,
    int64_t dst_layout_dim) {
  int32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  int32_t lane_id = tid % 32;
  int32_t warp_id = tid / 32;

  for (int i = 0; i < items_per_warp; ++i) {
    int32_t item_id = warp_id * items_per_warp + i;
    if (item_id >= num_items) {
      return;
    }
    const int64_t src_page_id = src_indices[item_id];
    const int64_t dst_page_id = dst_indices[item_id];

    // Loop over layers if necessary
    for (int64_t layer_id = start_layer_id; layer_id < start_layer_id + num_layers_to_process; ++layer_id) {
      // Calculate offsets using the provided function pointers
      const int64_t src_offset = SrcOffsetFn(layer_id, src_layout_dim, src_page_id, item_size_bytes);
      const int64_t dst_offset = DstOffsetFn(layer_id, dst_layout_dim, dst_page_id, item_size_bytes);

      if constexpr (IsMLA) {
        transfer_item_warp(
            lane_id,
            static_cast<const char*>(src_k) + src_offset,
            static_cast<char*>(dst_k) + dst_offset,
            item_size_bytes);
      } else {
        transfer_item_warp(
            lane_id,
            static_cast<const char*>(src_k) + src_offset,
            static_cast<char*>(dst_k) + dst_offset,
            item_size_bytes);
        transfer_item_warp(
            lane_id,
            static_cast<const char*>(src_v) + src_offset,
            static_cast<char*>(dst_v) + dst_offset,
            item_size_bytes);
      }
    }
  }
}

template <auto SrcOffsetFn, auto DstOffsetFn, bool IsMLA>
void transfer_kv_launcher(
    const at::Tensor& src_k,
    at::Tensor& dst_k,
    const at::Tensor& src_v,
    at::Tensor& dst_v,
    const at::Tensor& src_indices,
    const at::Tensor& dst_indices,
    int64_t start_layer_id,
    int64_t num_layers_to_process,
    int64_t item_size,
    int64_t src_layout_dim,
    int64_t dst_layout_dim,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  TORCH_CHECK(src_k.scalar_type() == dst_k.scalar_type(), "Source and destination keys must have the same type");
  TORCH_CHECK(src_indices.is_cuda(), "Source indices must be a CUDA tensor");
  TORCH_CHECK(dst_indices.is_cuda(), "Destination indices must be a CUDA tensor");
  TORCH_CHECK(src_indices.scalar_type() == at::kLong, "Source indices must be of type long");
  TORCH_CHECK(dst_indices.scalar_type() == at::kLong, "Destination indices must be of type long");
  TORCH_CHECK(src_indices.numel() == dst_indices.numel(), "Source and destination indices must have the same length");

  if (!IsMLA) {
    TORCH_CHECK(src_v.scalar_type() == dst_v.scalar_type(), "Source and destination values must have the same type");
  }

  int dtype_size = src_k.element_size();
  TORCH_CHECK((item_size * dtype_size) % 8 == 0, "Item byte size must be divisible by 8");

  auto div_up = [](int32_t x, int32_t y) { return (x + y - 1) / y; };
  const int64_t num_items = src_indices.numel();
  const int64_t items_per_warp = div_up(num_items, block_quota * num_warps_per_block);
  const int32_t num_blocks = div_up(num_items, items_per_warp * num_warps_per_block);
  dim3 grid_dim(num_blocks, 1, 1);
  const int32_t threads_per_block = num_warps_per_block * 32;

  hipStream_t torch_current_stream = at::cuda::getCurrentCUDAStream();
  transfer_kernel_impl<SrcOffsetFn, DstOffsetFn, IsMLA><<<grid_dim, threads_per_block, 0, torch_current_stream>>>(
      src_k.data_ptr(),
      dst_k.data_ptr(),
      (IsMLA ? nullptr : src_v.data_ptr()),
      (IsMLA ? nullptr : dst_v.data_ptr()),
      src_indices.data_ptr<int64_t>(),
      dst_indices.data_ptr<int64_t>(),
      start_layer_id,
      num_layers_to_process,
      num_items,
      items_per_warp,
      item_size * dtype_size,
      src_layout_dim * dtype_size,
      dst_layout_dim * dtype_size);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void transfer_kv_per_layer(
    const at::Tensor src_k,
    at::Tensor dst_k,
    const at::Tensor src_v,
    at::Tensor dst_v,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  transfer_kv_launcher<get_global_offset_lf, get_global_offset_lf, false>(
      src_k, dst_k, src_v, dst_v, src_indices, dst_indices, 0, 1, item_size, 0, 0, block_quota, num_warps_per_block);
}

void transfer_kv_all_layer(
    const at::Tensor src_k,
    at::Tensor dst_k,
    const at::Tensor src_v,
    at::Tensor dst_v,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t num_layers,
    int64_t src_layer_offset,
    int64_t dst_layer_offset,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  transfer_kv_launcher<get_global_offset_lf, get_global_offset_lf, false>(
      src_k,
      dst_k,
      src_v,
      dst_v,
      src_indices,
      dst_indices,
      0,
      num_layers,
      item_size,
      src_layer_offset,
      dst_layer_offset,
      block_quota,
      num_warps_per_block);
}

void transfer_kv_per_layer_mla(
    const at::Tensor src,
    at::Tensor dst,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  at::Tensor empty_tensor = at::Tensor();
  transfer_kv_launcher<get_global_offset_lf, get_global_offset_lf, true>(
      src,
      dst,
      empty_tensor,
      empty_tensor,
      src_indices,
      dst_indices,
      0,
      1,
      item_size,
      0,
      0,
      block_quota,
      num_warps_per_block);
}

void transfer_kv_all_layer_mla(
    const at::Tensor src,
    at::Tensor dst,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t num_layers,
    int64_t src_layer_offset,
    int64_t dst_layer_offset,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  at::Tensor empty_tensor = at::Tensor();
  transfer_kv_launcher<get_global_offset_lf, get_global_offset_lf, true>(
      src,
      dst,
      empty_tensor,
      empty_tensor,
      src_indices,
      dst_indices,
      0,
      num_layers,
      item_size,
      src_layer_offset,
      dst_layer_offset,
      block_quota,
      num_warps_per_block);
}

inline void transfer_page_direct(
    const at::Tensor src_buffer,
    at::Tensor dst_buffer,
    int64_t src_page_index,
    int64_t dst_page_index,
    int64_t page_size) {
  dst_buffer.slice(0, dst_page_index, dst_page_index + page_size)
      .copy_(
          src_buffer.slice(0, src_page_index, src_page_index + page_size),
          /* non_blocking= */ true);
}

template <bool IsMLA, bool AllLayers>
inline void transfer_kv_direct_impl(
    const at::Tensor& src_k,
    at::Tensor& dst_k,
    const at::Tensor& src_v_opt,  // Only used when IsMLA is false (for src_v)
    at::Tensor& dst_v_opt,        // Only used when IsMLA is false (for dst_v)
    const at::Tensor& src_indices,
    const at::Tensor& dst_indices,
    int64_t page_size,
    int64_t num_layers = 1) {
  TORCH_CHECK(src_indices.numel() == dst_indices.numel(), "Source and destination indices must have the same length");
  TORCH_CHECK(page_size > 0, "Page size must be positive");
  TORCH_CHECK(src_indices.numel() % page_size == 0, "Source indices size must be divisible by page size");

  auto src_indices_cpu = src_indices.cpu();
  auto dst_indices_cpu = dst_indices.cpu();

  const int64_t num_pages = src_indices_cpu.size(0) / page_size;

  for (const auto i : c10::irange(num_pages)) {
    auto s_index = src_indices_cpu[i * page_size].item<int64_t>();
    auto d_index = dst_indices_cpu[i * page_size].item<int64_t>();

    if constexpr (AllLayers) {
      for (const auto j : c10::irange(num_layers)) {
        if constexpr (IsMLA) {
          transfer_page_direct(src_k.select(0, j), dst_k.select(0, j), s_index, d_index, page_size);
        } else {
          transfer_page_direct(src_k.select(0, j), dst_k.select(0, j), s_index, d_index, page_size);
          transfer_page_direct(src_v_opt.select(0, j), dst_v_opt.select(0, j), s_index, d_index, page_size);
        }
      }
    } else {  // Per-layer
      if constexpr (IsMLA) {
        transfer_page_direct(src_k, dst_k, s_index, d_index, page_size);
      } else {
        transfer_page_direct(src_k, dst_k, s_index, d_index, page_size);
        transfer_page_direct(src_v_opt, dst_v_opt, s_index, d_index, page_size);
      }
    }
  }
}

void transfer_kv_per_layer_direct(
    const at::Tensor src_k,
    at::Tensor dst_k,
    const at::Tensor src_v,
    at::Tensor dst_v,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t page_size) {
  transfer_kv_direct_impl<false, false>(src_k, dst_k, src_v, dst_v, src_indices, dst_indices, page_size);
}

void transfer_kv_all_layer_direct(
    const at::Tensor src_k,
    at::Tensor dst_k,
    const at::Tensor src_v,
    at::Tensor dst_v,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t page_size,
    int64_t num_layers) {
  transfer_kv_direct_impl<false, true>(src_k, dst_k, src_v, dst_v, src_indices, dst_indices, page_size, num_layers);
}

void transfer_kv_per_layer_mla_direct(
    const at::Tensor src,
    at::Tensor dst,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t page_size) {
  at::Tensor empty_tensor = at::Tensor();

  transfer_kv_direct_impl<true, false>(src, dst, empty_tensor, empty_tensor, src_indices, dst_indices, page_size);
}

void transfer_kv_all_layer_mla_direct(
    const at::Tensor src,
    at::Tensor dst,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t page_size,
    int64_t num_layers) {
  at::Tensor empty_tensor = at::Tensor();
  transfer_kv_direct_impl<true, true>(
      src, dst, empty_tensor, empty_tensor, src_indices, dst_indices, page_size, num_layers);
}
