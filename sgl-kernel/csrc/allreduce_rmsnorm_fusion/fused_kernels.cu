#include "hip/hip_runtime.h"
#include "type_convert.cuh"
#include "dispatch_utils.h"

#include <torch/hip/hip_runtime.h>
#include <c10/cuda/CUDAGuard.h>

#ifndef USE_ROCM
#include <hipcub/hipcub.hpp>
#else
#include <hipcub/hipcub.hpp>
#endif

#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <cmath>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900) && CUDART_VERSION >= 12010
#define NVCC_SUPPORTS_MULTICAST 1
#endif

#include <ATen/ATen.h>
#if !defined(USE_ROCM)
#include <hip/hip_bf16.h>
#endif

#include "multimem_utils.cuh"
#include <cassert>


//copy from https://github.com/microsoft/tokenweave/blob/main/csrc/tokenweave_fused_kernels.cu


/* 
* ********************************************************* *
* FUSED RS + RESIDUAL ADD + RMS NORM + AG CTA-BASED KERNEL  *
* Function specialization in the case of BF16/FP16 tensors. *
* ********************************************************* *
*/
namespace vllm
{
template <typename scalar_t, int width>
__global__ std::enable_if_t<(width > 0) && _typeConvert<scalar_t>::exists>
fused_rs_ln_ag_cta_kernel(
    scalar_t *__restrict__ input,        // [..., hidden_size]
    scalar_t *__restrict__ mcptr,        // [..., hidden_size] multimem_ptr
    scalar_t *__restrict__ residual,     // [..., hidden_size]
    const scalar_t *__restrict__ weight, // [hidden_size]
    uint32_t **signal_pads,
    size_t rank,
    size_t world_size,
    const float epsilon,
    const int num_tokens,
    const int hidden_size)
{

  // Check vectorization assumptions
  static_assert(std::is_pod_v<_f16Vec<scalar_t, width>>);
  static_assert(sizeof(_f16Vec<scalar_t, width>) == sizeof(scalar_t) * width);

  const int vec_hidden_size = hidden_size / width;
  using vec_t = _f16Vec<scalar_t, width>;

  // Type-punned vector pointers
  auto *__restrict__ input_v = reinterpret_cast<vec_t *>(input);
  auto *__restrict__ residual_v = reinterpret_cast<vec_t *>(residual);
  auto *__restrict__ weight_v = reinterpret_cast<const vec_t *>(weight);
  int tokens_per_iter = (num_tokens + gridDim.x - 1) / gridDim.x;

  sync_remote_blocks<MemOpSem::Relaxed>(signal_pads, rank, world_size);
  __syncthreads();

  #pragma unroll
  for (int iter = 0; iter < tokens_per_iter; iter++)
  {
    int token_id = blockIdx.x + iter * gridDim.x;
    if (token_id >= num_tokens)
      continue;
    float variance[1] = {0.0f};
    const int tid = threadIdx.x;
    const int bdimx = blockDim.x;

    __shared__ float s_variance;
    int offset = token_id * vec_hidden_size;
    int offset_scalar = token_id * hidden_size;
    auto input_o = input_v + offset;
    auto residual_o = residual_v + offset;

    for (int idx = tid; idx < vec_hidden_size; idx += bdimx)
    {
      auto mtemp = multimem_ld_reduce_add<16>(mcptr + offset_scalar + idx * width);
      vec_t temp = *(reinterpret_cast<vec_t *>(&mtemp));
      temp += residual_o[idx];
      variance[0] += temp.sum_squares(); // FP32 accumulation
      residual_o[idx] = temp;
      // multimem_st<16>(residual_mcptr + offset_scalar + idx * width, 
      //                *(reinterpret_cast<Vec<16> *>(&temp)));
    }

    blockReduceSum<float, 1>(variance);
    if (threadIdx.x == 0)
    {
      s_variance = rsqrtf(variance[0] / hidden_size + epsilon);
    }
    __syncthreads();

    // Second pass: normalize and apply weight
    for (int idx = tid; idx < vec_hidden_size; idx += bdimx)
    {
      vec_t shared_weight = weight_v[idx];
      vec_t temp = residual_o[idx];
      temp *= s_variance;
      temp *= shared_weight;
      multimem_st<16>(mcptr + offset_scalar + idx * width, *(reinterpret_cast<Vec<16> *>(&temp)));
    }
  }
  __syncthreads();
  sync_remote_blocks<MemOpSem::AcqRel>(signal_pads, rank, world_size);
}

/* 
* ********************************************************* *
* FUSED RS + RESIDUAL ADD + RMS NORM + AG CTA-BASED KERNEL  *
* GENERIC NOT SUPPORTED                                     *
* ********************************************************* *
*/
template <typename scalar_t, int width>
__global__ std::enable_if_t<(width == 0) || !_typeConvert<scalar_t>::exists>
fused_rs_ln_ag_cta_kernel(
    scalar_t *__restrict__ input,        // [..., hidden_size]
    scalar_t *__restrict__ mcptr,        // [..., hidden_size] multimem_ptr
    scalar_t *__restrict__ residual,     // [..., hidden_size]
    const scalar_t *__restrict__ weight, // [hidden_size]
    uint32_t **signal_pads,
    size_t rank,
    size_t world_size,
    const float epsilon,
    const int num_tokens,
    const int hidden_size)
{
  /* Not supported */
  assert(false && "TokenWeave currently only supports bf16/fp16 with width 8.");
}
} //namespace vllm

/* 
* ******************************************************************* *
* Fused ReduceScatter plus Fused(Residual, RMSNorm) plus AllGather    *
* ******************************************************************* *
*/
#define LAUNCH_FUSED_RS_LN_AG_CTA(width)                                                                                                   \
  VLLM_BF16_DISPATCH_FLOATING_TYPES(                                                                                                       \
      input.scalar_type(), "fused_rs_ln_ag_cta_kernel", [&] { vllm::fused_rs_ln_ag_cta_kernel<scalar_t, width>                             \
                                                                  <<<grid, block, 0, stream>>>(input.data_ptr<scalar_t>(),                 \
                                                                                               reinterpret_cast<scalar_t *>(mcptr),        \
                                                                                               residual.data_ptr<scalar_t>(),              \
                                                                                               weight.data_ptr<scalar_t>(),                \
                                                                                               reinterpret_cast<uint32_t **>(signal_pads), \
                                                                                               static_cast<size_t>(rank),                  \
                                                                                               static_cast<size_t>(world_size),            \
                                                                                               epsilon, num_tokens, hidden_size); });
void fused_rs_ln_ag_cta(torch::Tensor &input,    // [..., hidden_size]
                        torch::Tensor &residual, // [..., hidden_size]
                        torch::Tensor &weight,   // [hidden_size]
                        int64_t mcptr,           // [..., hidden_size] multimem_ptr
                        int64_t signal_pads,     // [..., hidden_size] signal pads
                        int64_t rank,
                        int64_t world_size,
                        int64_t MAX_CTAS,
                        double epsilon)
{
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(MAX_CTAS);                                          // full coverage
  dim3 block(std::min(1024, (hidden_size / 8 + 31) / 32 * 32)); // match kernel assumptions
  /* This kernel is memory-latency bound in many scenarios.
     When num_tokens is large, a smaller block size allows
     for increased block occupancy on CUs and better latency
     hiding on global mem ops. */
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  /*If the tensor types are FP16/BF16, try to use the optimized kernel
    with packed + vectorized ops.
    Max optimization is achieved with a width-8 vector of FP16/BF16s
    since we can load at most 128 bits at once in a global memory op.
    However, this requires each tensor's data to be aligned to 16
    bytes.
   */
  auto inp_ptr = reinterpret_cast<std::uintptr_t>(input.data_ptr());
  auto res_ptr = reinterpret_cast<std::uintptr_t>(residual.data_ptr());
  auto wt_ptr = reinterpret_cast<std::uintptr_t>(weight.data_ptr());
  bool ptrs_are_aligned =
      inp_ptr % 16 == 0 && res_ptr % 16 == 0 && wt_ptr % 16 == 0;
  if (ptrs_are_aligned && hidden_size % 8 == 0)
  {
    LAUNCH_FUSED_RS_LN_AG_CTA(8);
  }
  else
  {
    TORCH_CHECK(false, "Input, residual, and weight tensors must be 16-byte aligned and hidden_size must be divisible by 8 for optimized kernel.");
  }
}