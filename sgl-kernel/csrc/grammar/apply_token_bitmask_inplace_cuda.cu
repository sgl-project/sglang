#include "hip/hip_runtime.h"
// Adapted from
// https://github.com/mlc-ai/xgrammar/blob/v0.1.18/python/xgrammar/kernels/apply_token_bitmask_inplace_cuda.cu

/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// clang-format off
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
// clang-format on

#if defined CUDA_VERSION && CUDA_VERSION >= 12040

#ifndef HIPRT_INF_FP16
#define HIPRT_INF_FP16 __ushort_as_half((unsigned short)0x7C00U)
#endif

#ifndef CUDART_INF_BF16
#define CUDART_INF_BF16 __ushort_as_bfloat16((unsigned short)0x7F80U)
#endif

constexpr int32_t BITS_PER_BLOCK = 32;
constexpr int32_t THREADS_PER_THREAD_BLOCK = 256;

template <typename T>
__device__ T NegativeInfinity() {
  return -INFINITY;
}

template <>
__device__ __half NegativeInfinity<__half>() {
  return -HIPRT_INF_FP16;
}

template <>
__device__ __hip_bfloat16 NegativeInfinity<__hip_bfloat16>() {
  return -CUDART_INF_BF16;
}

template <typename T, typename PackedT>
__device__ PackedT PackedNegativeInfinity() {
  constexpr int kAlignment = sizeof(PackedT) / sizeof(T);
  T packed[kAlignment];
#pragma unroll
  for (int i = 0; i < kAlignment; i++) {
    packed[i] = NegativeInfinity<T>();
  }
  return *reinterpret_cast<PackedT*>(packed);
}

template <typename T, typename PackedT, int32_t kBitsPerThread>
__global__ void __launch_bounds__(THREADS_PER_THREAD_BLOCK) LogitsBitmaskKernel(
    T* __restrict__ logits,
    const int32_t* __restrict__ bitmask,
    const int32_t* __restrict__ indices,
    int32_t vocab_size,
    int32_t logits_stride,
    int32_t bitmask_stride) {
  constexpr int kAlignment = sizeof(PackedT) / sizeof(T);
  constexpr uint32_t kPackedMask = (1 << kAlignment) - 1;

  const int batch_idx = (indices == nullptr) ? blockIdx.y : indices[blockIdx.y];

  const int block_offset = blockIdx.x * THREADS_PER_THREAD_BLOCK * kBitsPerThread;
  T* logits_gmem_ptr = logits + batch_idx * logits_stride + block_offset;
  const int32_t* bitmask_gmem_ptr = bitmask + batch_idx * bitmask_stride + block_offset / BITS_PER_BLOCK;
  const int bitmask_inner_idx = threadIdx.x % (BITS_PER_BLOCK / kAlignment);
  T logits_reg[kAlignment];

#pragma unroll
  for (int offset = threadIdx.x * kAlignment; offset < THREADS_PER_THREAD_BLOCK * kBitsPerThread;
       offset += THREADS_PER_THREAD_BLOCK * kAlignment) {
    if (block_offset + offset >= vocab_size) {
      break;
    }

    const uint32_t bitmask_val =
        (~bitmask_gmem_ptr[offset / BITS_PER_BLOCK] >> (bitmask_inner_idx * kAlignment)) & kPackedMask;

    if (bitmask_val == 0) {
      continue;
    }

    if (bitmask_val == kPackedMask) {
      *reinterpret_cast<PackedT*>(logits_gmem_ptr + offset) = PackedNegativeInfinity<T, PackedT>();
      continue;
    }

    *reinterpret_cast<PackedT*>(logits_reg) = *reinterpret_cast<PackedT*>(logits_gmem_ptr + offset);
#pragma unroll
    for (int i = 0; i < kAlignment; i++) {
      if (((bitmask_val >> i) & 1)) {
        logits_reg[i] = NegativeInfinity<T>();
      }
    }
    *reinterpret_cast<PackedT*>(logits_gmem_ptr + offset) = *reinterpret_cast<PackedT*>(logits_reg);
  }
}

template <typename T, typename = std::enable_if_t<std::is_integral<T>::value>>
constexpr auto CeilDiv(T numerator, T denominator) {
  return (numerator + denominator - 1) / denominator;
}

template <typename T, typename PackedT>
void ApplyTokenBitmaskInplaceDispatchToBitsPerThread(
    T* __restrict__ logits,
    const int32_t* __restrict__ bitmask,
    const int32_t* __restrict__ indices,
    int32_t vocab_size,
    int32_t logits_stride,
    int32_t bitmask_stride,
    int32_t num_rows) {
  constexpr int kAlignment = sizeof(PackedT) / sizeof(T);
  const int32_t num_blocks_per_row = CeilDiv(2048 / THREADS_PER_THREAD_BLOCK * 128, num_rows);
  const int32_t num_bits_per_thread = CeilDiv(vocab_size, THREADS_PER_THREAD_BLOCK * num_blocks_per_row);

  const dim3 block(THREADS_PER_THREAD_BLOCK);
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

  if (num_bits_per_thread <= 4 && kAlignment <= 4) {
    const dim3 grid(CeilDiv(vocab_size, THREADS_PER_THREAD_BLOCK * 4), num_rows);
    LogitsBitmaskKernel<T, PackedT, 4>
        <<<grid, block, 0, stream>>>(logits, bitmask, indices, vocab_size, logits_stride, bitmask_stride);
  } else if (num_bits_per_thread <= 8 && kAlignment <= 8) {
    const dim3 grid(CeilDiv(vocab_size, THREADS_PER_THREAD_BLOCK * 8), num_rows);
    LogitsBitmaskKernel<T, PackedT, 8>
        <<<grid, block, 0, stream>>>(logits, bitmask, indices, vocab_size, logits_stride, bitmask_stride);
  } else if (num_bits_per_thread <= 16 && kAlignment <= 16) {
    const dim3 grid(CeilDiv(vocab_size, THREADS_PER_THREAD_BLOCK * 16), num_rows);
    LogitsBitmaskKernel<T, PackedT, 16>
        <<<grid, block, 0, stream>>>(logits, bitmask, indices, vocab_size, logits_stride, bitmask_stride);
  } else {
    const dim3 grid(CeilDiv(vocab_size, THREADS_PER_THREAD_BLOCK * 32), num_rows);
    LogitsBitmaskKernel<T, PackedT, 32>
        <<<grid, block, 0, stream>>>(logits, bitmask, indices, vocab_size, logits_stride, bitmask_stride);
  }
}

template <typename T>
void ApplyTokenBitmaskInplaceDispatchToPackedT(
    T* __restrict__ logits,
    const int32_t* __restrict__ bitmask,
    const int32_t* __restrict__ indices,
    int32_t vocab_size,
    int32_t logits_stride,
    int32_t bitmask_stride,
    int32_t num_rows) {
  if (logits_stride % (sizeof(float4) / sizeof(T)) == 0) {
    ApplyTokenBitmaskInplaceDispatchToBitsPerThread<T, float4>(
        logits, bitmask, indices, vocab_size, logits_stride, bitmask_stride, num_rows);
  } else {
    ApplyTokenBitmaskInplaceDispatchToBitsPerThread<T, T>(
        logits, bitmask, indices, vocab_size, logits_stride, bitmask_stride, num_rows);
  }
}

void ApplyTokenBitmaskInplace(at::Tensor logits, at::Tensor bitmask, at::optional<at::Tensor> indices = at::nullopt) {
  TORCH_CHECK(logits.is_cuda(), "logits must be a CUDA tensor.");
  TORCH_CHECK(logits.is_contiguous(), "logits must be contiguous.");
  TORCH_CHECK(logits.dim() == 1 || logits.dim() == 2, "logits must be a 1D or 2D tensor.");
  std::pair<int32_t, int32_t> logits_shape =
      logits.dim() == 2 ? std::make_pair(static_cast<int32_t>(logits.size(0)), static_cast<int32_t>(logits.size(1)))
                        : std::make_pair(1, static_cast<int32_t>(logits.size(0)));

  TORCH_CHECK(bitmask.is_cuda(), "bitmask must be a CUDA tensor.");
  TORCH_CHECK(bitmask.is_contiguous(), "bitmask must be contiguous.");
  TORCH_CHECK(bitmask.dim() == 1 || bitmask.dim() == 2, "bitmask must be a 1D or 2D tensor.");
  std::pair<int32_t, int32_t> bitmask_shape =
      bitmask.dim() == 2 ? std::make_pair(static_cast<int32_t>(bitmask.size(0)), static_cast<int32_t>(bitmask.size(1)))
                         : std::make_pair(1, static_cast<int32_t>(bitmask.size(0)));

  TORCH_CHECK(bitmask.dtype() == torch::kInt32, "bitmask must be of type int32.");

  TORCH_CHECK(
      (logits_shape.second + BITS_PER_BLOCK - 1) / BITS_PER_BLOCK >= bitmask_shape.second,
      "The provided logits's vocab size should be no less than the bitmask's vocab size "
      "(converted from bitmask size). But got vocab size ",
      logits_shape.second,
      " vs bitmask size ",
      bitmask_shape.second);

  int vocab_size = std::min(logits_shape.second, bitmask_shape.second * BITS_PER_BLOCK);

  int32_t num_rows = logits_shape.first;
  int32_t* indices_ptr = nullptr;
  if (indices) {
    TORCH_CHECK(indices->is_cuda(), "indices must be a CUDA tensor.");
    TORCH_CHECK(indices->is_contiguous(), "indices must be contiguous.");
    TORCH_CHECK(indices->dim() == 1, "indices must be a 1D tensor.");
    TORCH_CHECK(indices->dtype() == torch::kInt32, "indices must be of type int32.");
    num_rows = indices->size(0);
    indices_ptr = indices->data_ptr<int32_t>();
  } else {
    TORCH_CHECK(logits_shape.first == bitmask_shape.first, "logits and bitmask must have the same batch size.");
  }

  switch (logits.scalar_type()) {
    case torch::kFloat32: {
      ApplyTokenBitmaskInplaceDispatchToPackedT(
          logits.data_ptr<float>(),
          bitmask.data_ptr<int32_t>(),
          indices_ptr,
          vocab_size,
          logits_shape.second,
          bitmask_shape.second,
          num_rows);
      break;
    }
    case torch::kFloat16: {
      ApplyTokenBitmaskInplaceDispatchToPackedT(
          reinterpret_cast<__half*>(logits.data_ptr<torch::Half>()),
          bitmask.data_ptr<int32_t>(),
          indices_ptr,
          vocab_size,
          logits_shape.second,
          bitmask_shape.second,
          num_rows);
      break;
    }
    case torch::kBFloat16: {
      ApplyTokenBitmaskInplaceDispatchToPackedT(
          reinterpret_cast<__hip_bfloat16*>(logits.data_ptr<torch::BFloat16>()),
          bitmask.data_ptr<int32_t>(),
          indices_ptr,
          vocab_size,
          logits_shape.second,
          bitmask_shape.second,
          num_rows);
      break;
    }
    default:
      TORCH_CHECK(false, "logits dtype must be float, half or bfloat16.");
      break;
  }
}
#endif
