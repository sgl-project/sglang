#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#ifndef USE_ROCM
#include "pytorch_extension_utils.h"
#else
#include "pytorch_extension_utils_rocm.h"
#endif

void reconstruct_indices_from_tree_mask(
    at::Tensor tree_mask,
    at::Tensor verified_seq_len,
    at::Tensor positions,
    at::Tensor retrive_index,
    at::Tensor retrive_next_token,
    at::Tensor retrive_next_sibling,
    int64_t batch_size,
    int64_t draft_token_num) {
  dim3 grid(batch_size);
  dim3 block(draft_token_num);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  reconstructIndicesFromTreeMask<<<grid, block, 0, stream>>>(
      static_cast<bool*>(tree_mask.data_ptr()),
      static_cast<int64_t*>(verified_seq_len.data_ptr()),
      static_cast<int64_t*>(positions.data_ptr()),
      static_cast<int64_t*>(retrive_index.data_ptr()),
      static_cast<int64_t*>(retrive_next_token.data_ptr()),
      static_cast<int64_t*>(retrive_next_sibling.data_ptr()),
      int(batch_size),
      int(draft_token_num));
}
