#include <torch/all.h>

#include <cstdlib>
#include <iomanip>
#include <iostream>

#include "cuda_utils.h"
#include "greenctx_stream.h"

static std::vector<int64_t> create_greenctx_stream_fallback(CUgreenCtx gctx[2]) {
  hipStream_t streamA, streamB;
  hipCtx_t ctx;

  CUDA_DRV(cuCtxFromGreenCtx(&ctx, gctx[0]));
  CUDA_DRV(hipCtxPushCurrent(ctx));
  CUDA_DRV(hipStreamCreateWithFlags(&streamA, hipStreamNonBlocking));
  CUDA_DRV(hipCtxPopCurrent(nullptr));

  CUDA_DRV(cuCtxFromGreenCtx(&ctx, gctx[1]));
  CUDA_DRV(hipCtxPushCurrent(ctx));
  CUDA_DRV(hipStreamCreateWithFlags(&streamB, hipStreamNonBlocking));
  CUDA_DRV(hipCtxPopCurrent(nullptr));

  return {(int64_t)streamA, (int64_t)streamB};
}

typedef hipError_t(CUDAAPI* PFN_cuGreenCtxStreamCreate)(hipStream_t*, CUgreenCtx, unsigned int, int);

static std::vector<int64_t> create_greenctx_stream_direct_dynamic(CUgreenCtx gctx[2]) {
  static PFN_cuGreenCtxStreamCreate pfn = nullptr;
  static std::once_flag pfn_probed_flag;

  // detect compatibility in runtime
  std::call_once(pfn_probed_flag, []() {
    hipGetProcAddress("cuGreenCtxStreamCreate", reinterpret_cast<void**>(&pfn), 0, 0, nullptr);
  });

  if (!pfn) {  // fallback if not compatible
    return create_greenctx_stream_fallback(gctx);
  }

  hipStream_t streamA, streamB;
  CUDA_DRV(pfn(&streamA, gctx[0], hipStreamNonBlocking, 0));
  CUDA_DRV(pfn(&streamB, gctx[1], hipStreamNonBlocking, 0));

  return {(int64_t)streamA, (int64_t)streamB};
}

inline void destroy_green_context(int64_t h) {
  if (h) CUDA_DRV(cuGreenCtxDestroy(reinterpret_cast<CUgreenCtx>(h)));
}

std::vector<int64_t> create_greenctx_stream_by_value(int64_t smA, int64_t smB, int64_t device) {
  TORCH_CHECK(CUDA_VERSION >= 12040, "Green Contexts feature requires CUDA Toolkit 12.4 or newer.");

  CUgreenCtx gctx[3];
  CUdevResourceDesc desc[3];
  CUdevResource input;
  CUdevResource resources[4];
  if (smA <= 0 || smB <= 0) {
    TORCH_CHECK(false, "SM counts must be positive");
  }

  CUDA_DRV(cuDeviceGetDevResource((hipDevice_t)device, &input, CU_DEV_RESOURCE_TYPE_SM));

  const unsigned minCount = smA + smB;
  const unsigned minCountA = smA;
  TORCH_CHECK(minCount <= input.sm.smCount, "Not enough SMs available for the requested configuration");

  unsigned nbGroups = 1;
  CUDA_DRV(cuDevSmResourceSplitByCount(&resources[2], &nbGroups, &input, &resources[3], 0, minCount));
  CUDA_DRV(cuDevResourceGenerateDesc(&desc[2], &resources[2], 1));
  CUDA_DRV(cuGreenCtxCreate(&gctx[2], desc[2], (hipDevice_t)device, CU_GREEN_CTX_DEFAULT_STREAM));
  CUDA_DRV(cuGreenCtxGetDevResource(gctx[2], &input, CU_DEV_RESOURCE_TYPE_SM));
  nbGroups = 1;
  CUDA_DRV(cuDevSmResourceSplitByCount(&resources[0], &nbGroups, &input, &resources[1], 0, minCountA));
  CUDA_DRV(cuDevResourceGenerateDesc(&desc[0], &resources[0], 1));
  CUDA_DRV(cuGreenCtxCreate(&gctx[0], desc[0], (hipDevice_t)device, CU_GREEN_CTX_DEFAULT_STREAM));
  CUDA_DRV(cuDevResourceGenerateDesc(&desc[1], &resources[1], 1));
  CUDA_DRV(cuGreenCtxCreate(&gctx[1], desc[1], (hipDevice_t)device, CU_GREEN_CTX_DEFAULT_STREAM));

  const int smCountA = resources[0].sm.smCount;
  const int smCountB = resources[1].sm.smCount;

  std::vector<int64_t> streams = create_greenctx_stream_direct_dynamic(gctx);

  CUDA_DRV(cuGreenCtxDestroy(gctx[2]));

  std::vector<int64_t> vec = {
      streams[0],  // streamA
      streams[1],  // streamB
      (int64_t)smCountA,
      (int64_t)smCountB};

  return vec;
}
