#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <flashinfer/activation.cuh>

#include "pytorch_extension_utils.h"

using namespace flashinfer;

__device__ __forceinline__ float silu(const float& val) {
  return val / (1.0f + __expf(-val));
}

__device__ __forceinline__ float gelu(const float& val) {
  constexpr float kAlpha = M_SQRT1_2;
  return val * 0.5f * (1.0f + ::erf(val * kAlpha));
}

__device__ __forceinline__ float gelu_tanh(const float& val) {
  const float cdf = 0.5f * (1.0f + math::tanh((0.7978845608028654f * (val + 0.044715f * val * val * val))));
  return val * cdf;
}

void silu_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream) {
  int d = input.size(-1) / 2;
  int64_t num_tokens = input.numel() / input.size(-1);
  dim3 grid(num_tokens);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    uint32_t vec_size = 16 / sizeof(c_type);
    dim3 block(std::min(d / vec_size, 1024U));
    flashinfer::activation::act_and_mul_kernel<c_type, silu>
        <<<grid, block, 0, stream>>>(static_cast<c_type*>(out.data_ptr()), static_cast<c_type*>(input.data_ptr()), d);

    return true;
  });
}

void gelu_tanh_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream) {
  int d = input.size(-1) / 2;
  int64_t num_tokens = input.numel() / input.size(-1);
  dim3 grid(num_tokens);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    uint32_t vec_size = 16 / sizeof(c_type);
    dim3 block(std::min(d / vec_size, 1024U));
    flashinfer::activation::act_and_mul_kernel<c_type, gelu_tanh>
        <<<grid, block, 0, stream>>>(static_cast<c_type*>(out.data_ptr()), static_cast<c_type*>(input.data_ptr()), d);

    return true;
  });
}

void gelu_and_mul(at::Tensor& out, at::Tensor& input, int64_t hip_stream) {
  int d = input.size(-1) / 2;
  int64_t num_tokens = input.numel() / input.size(-1);
  dim3 grid(num_tokens);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input.scalar_type(), c_type, [&] {
    uint32_t vec_size = 16 / sizeof(c_type);
    dim3 block(std::min(d / vec_size, 1024U));
    flashinfer::activation::act_and_mul_kernel<c_type, gelu>
        <<<grid, block, 0, stream>>>(static_cast<c_type*>(out.data_ptr()), static_cast<c_type*>(input.data_ptr()), d);

    return true;
  });
}
