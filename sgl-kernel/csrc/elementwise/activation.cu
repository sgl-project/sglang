#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>

#ifndef USE_ROCM

#include <flashinfer/activation.cuh>

#include "utils.h"

#else
#include "hip_act_and_mul.cuh"
#endif

// Adapted from flashinfer activation
// https://github.com/flashinfer-ai/flashinfer/blob/4e8eb1879f9c3ba6d75511e5893183bf8f289a62/csrc/activation.cu#L44

namespace detail {

template <typename T>
__device__ __forceinline__ float to_f32(const T& x) {
#if USE_ROCM
  return castToFloat(x);
#else
  return static_cast<float>(x);
#endif
}

template <typename T>
__device__ __forceinline__ T from_f32(float f32) {
#if USE_ROCM
  return castFromFloat<T>(f32);
#else
  return static_cast<T>(f32);
#endif
}

}  // namespace detail

template <typename T>
__device__ __forceinline__ T silu(const T& x) {
  float f32_val = detail::to_f32(x);
  return detail::from_f32<T>(f32_val / (1.0f + expf(-f32_val)));
}

template <typename T>
__device__ __forceinline__ T gelu(const T& x) {
  constexpr float kAlpha = M_SQRT1_2;
  float f32_val = detail::to_f32(x);
  return detail::from_f32<T>(f32_val * (0.5f * (1.0f + erf(f32_val * kAlpha))));
}

// gelu_quick(x) = x * torch.sigmoid(1.702 * x)
template <typename T>
__device__ __forceinline__ T gelu_quick_act(const T& x) {
  float f32_val = detail::to_f32(x);
  return detail::from_f32<T>(f32_val / (1.0f + expf(-f32_val * 1.702f)));
}

template <typename T>
__device__ __forceinline__ T gelu_tanh(const T& x) {
  constexpr float kAlpha = 0.044715f;
  constexpr float kBeta = 0.7978845608028654f;
  float f32_val = detail::to_f32(x);
  const float cdf = 0.5f * (1.0f + tanhf((kBeta * (f32_val + kAlpha * f32_val * f32_val * f32_val))));
  return detail::from_f32<T>(f32_val * cdf);
}

void silu_and_mul(at::Tensor& out, at::Tensor& input) {
  int d = input.size(-1) / 2;
  int64_t num_tokens = input.numel() / input.size(-1);
  dim3 grid(num_tokens);

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), c_type, [&] {
    uint32_t vec_size = 16 / sizeof(c_type);
    dim3 block(std::min(d / vec_size, 1024U));
#if USE_ROCM
    sgl_hip::activation::act_and_mul_kernel<c_type, silu>
        <<<grid, block, 0, stream>>>(static_cast<c_type*>(out.data_ptr()), static_cast<c_type*>(input.data_ptr()), d);
#else
    flashinfer::activation::act_and_mul_kernel<c_type, silu>
        <<<grid, block, 0, stream>>>(static_cast<c_type*>(out.data_ptr()), static_cast<c_type*>(input.data_ptr()), d);
#endif
    return true;
  });
}

void gelu_tanh_and_mul(at::Tensor& out, at::Tensor& input) {
  int d = input.size(-1) / 2;
  int64_t num_tokens = input.numel() / input.size(-1);
  dim3 grid(num_tokens);

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), c_type, [&] {
    uint32_t vec_size = 16 / sizeof(c_type);
    dim3 block(std::min(d / vec_size, 1024U));
#if USE_ROCM
    sgl_hip::activation::act_and_mul_kernel<c_type, gelu_tanh>
        <<<grid, block, 0, stream>>>(static_cast<c_type*>(out.data_ptr()), static_cast<c_type*>(input.data_ptr()), d);
#else
    flashinfer::activation::act_and_mul_kernel<c_type, gelu_tanh>
        <<<grid, block, 0, stream>>>(static_cast<c_type*>(out.data_ptr()), static_cast<c_type*>(input.data_ptr()), d);
#endif
    return true;
  });
}

void gelu_and_mul(at::Tensor& out, at::Tensor& input) {
  int d = input.size(-1) / 2;
  int64_t num_tokens = input.numel() / input.size(-1);
  dim3 grid(num_tokens);

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), c_type, [&] {
    uint32_t vec_size = 16 / sizeof(c_type);
    dim3 block(std::min(d / vec_size, 1024U));
#if USE_ROCM
    sgl_hip::activation::act_and_mul_kernel<c_type, gelu>
        <<<grid, block, 0, stream>>>(static_cast<c_type*>(out.data_ptr()), static_cast<c_type*>(input.data_ptr()), d);
#else
    flashinfer::activation::act_and_mul_kernel<c_type, gelu>
        <<<grid, block, 0, stream>>>(static_cast<c_type*>(out.data_ptr()), static_cast<c_type*>(input.data_ptr()), d);
#endif

    return true;
  });
}

#if USE_ROCM
void gelu_quick(at::Tensor& out, const at::Tensor& input) {
  int d = input.size(-1);
  int64_t num_tokens = input.numel() / input.size(-1);
  dim3 grid(num_tokens);

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), c_type, [&] {
    uint32_t vec_size = 16 / sizeof(c_type);
    dim3 block(std::min(d / vec_size, 1024U));
    sgl_hip::activation::act_only_kernel<c_type, gelu_quick_act>
        <<<grid, block, 0, stream>>>(static_cast<c_type*>(out.data_ptr()), static_cast<c_type*>(input.data_ptr()), d);

    return true;
  });
}
#endif
