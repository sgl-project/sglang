#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
// #include <flashinfer/pos_enc.cuh>

#include "pos_enc.cuh"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

void apply_rope_pos_ids_cos_sin_cache(
    at::Tensor q,
    at::Tensor k,
    at::Tensor q_rope,
    at::Tensor k_rope,
    at::Tensor cos_sin_cache,
    at::Tensor pos_ids,
    bool interleave,
    int64_t hip_stream,
    const std::optional<at::Tensor>& v,
    const std::optional<at::Tensor>& k_buffer,
    const std::optional<at::Tensor>& v_buffer,
    const std::optional<at::Tensor>& kv_cache_loc) {
  CHECK_LAST_DIM_CONTIGUOUS(q);
  CHECK_LAST_DIM_CONTIGUOUS(k);
  if (save_kv_cache) {
    CHECK_LAST_DIM_CONTIGUOUS(v);
    CHECK_LAST_DIM_CONTIGUOUS(k_buffer);
    CHECK_LAST_DIM_CONTIGUOUS(v_buffer);
    CHECK_DIM(3, k_buffer);   // k_buffer: (nnz, H_K, D)
    CHECK_DIM(3, v_buffer);   // v_buffer: (nnz, H_V, D)
    CHECK_DIM(3, v);          // v: (nnz, H_V, D)
    CHECK_DIM(1, cache_loc);  // v: (n)
    CHECK_INPUT(cache_loc);
    size_t k_buffer_stride_n = k_buffer.stride(0);
    size_t k_buffer_stride_h = k_buffer.stride(1);
    size_t v_buffer_stride_n = v_buffer.stride(0);
    size_t v_buffer_stride_h = v_buffer.stride(1);
    size_t v_stride_n = v.stride(0);
    size_t v_stride_h = v.stride(1);
    auto v_ptr = static_cast<c_type*>(v.data_ptr());
    auto k_buffer_ptr = static_cast<c_type*>(k_buffer.data_ptr());
    auto v_buffer_ptr = static_cast<c_type*>(v_buffer.data_ptr());
    auto cache_loc_ptr = static_cast<int64_t*>(cache_loc.data_ptr());
  } else {
    size_t k_buffer_stride_n = 0;
    size_t k_buffer_stride_h = 0;
    size_t v_buffer_stride_n = 0;
    size_t v_buffer_stride_h = 0;
    size_t v_stride_n = 0;
    size_t v_stride_h = 0;
    auto v_ptr = nullptr;
    auto k_buffer_ptr = nullptr;
    auto v_buffer_ptr = nullptr;
    auto cache_loc_ptr = nullptr;
  }

  CHECK_INPUT(cos_sin_cache);
  CHECK_INPUT(pos_ids);
  auto device = q.device();
  CHECK_EQ(k.device(), device);
  CHECK_EQ(cos_sin_cache.device(), device);
  CHECK_EQ(pos_ids.device(), device);
  CHECK_DIM(3, q);  // q: (nnz, H_Q, D)
  CHECK_DIM(3, k);  // k: (nnz, H_K, D)

  // cos_sin_cache: (max_seq_len, R)
  // First half of R is cos, second half is sin
  CHECK_DIM(2, cos_sin_cache);
  CHECK_EQ(q.size(0), k.size(0));
  CHECK_EQ(q.size(2), k.size(2));
  unsigned int rotary_dim = cos_sin_cache.size(1);
  unsigned int num_qo_heads = q.size(1);
  unsigned int num_kv_heads = k.size(1);
  unsigned int head_dim = q.size(2);
  unsigned int nnz = q.size(0);
  size_t q_stride_n = q.stride(0);
  size_t q_stride_h = q.stride(1);
  size_t k_stride_n = k.stride(0);
  size_t k_stride_h = k.stride(1);

  size_t q_rope_stride_n = q_rope.stride(0);
  size_t q_rope_stride_h = q_rope.stride(1);
  size_t k_rope_stride_n = k_rope.stride(0);
  size_t k_rope_stride_h = k_rope.stride(1);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(q.scalar_type(), c_type, [&] {
    hipError_t status = BatchQKApplyRotaryPosIdsCosSinCacheWithSetKVBuffer(
        static_cast<c_type*>(q.data_ptr()),
        static_cast<c_type*>(k.data_ptr()),
        v_ptr,
        static_cast<c_type*>(q_rope.data_ptr()),
        static_cast<c_type*>(k_rope.data_ptr()),
        k_buffer_ptr,
        v_buffer_ptr,
        static_cast<float*>(cos_sin_cache.data_ptr()),
        static_cast<int64_t*>(pos_ids.data_ptr()),
        nnz,
        num_qo_heads,
        num_kv_heads,
        rotary_dim,
        head_dim,
        q_stride_n,
        q_stride_h,
        k_stride_n,
        k_stride_h,
        v_stride_n,
        v_stride_h,
        q_rope_stride_n,
        q_rope_stride_h,
        k_rope_stride_n,
        k_rope_stride_h,
        k_buffer_stride_n,
        k_buffer_stride_h,
        v_buffer_stride_n,
        v_buffer_stride_h,
        cache_loc_ptr,
        interleave,
        stream,
        k_scale,
        v_scale,
        save_kv_cache);
    TORCH_CHECK(
        status == hipSuccess,
        "BatchQKApplyRotaryPosIdsCosSinCacheWithSetKVBuffer failed with error code " +
            std::string(hipGetErrorString(status)));
    return true;
  });
}
