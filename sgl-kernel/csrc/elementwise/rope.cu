#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "pos_enc.cuh"
#include "pytorch_extension_utils.h"

using namespace flashinfer;

void apply_rope_pos_ids_cos_sin_cache(
    at::Tensor q,
    at::Tensor k,
    at::Tensor q_rope,
    at::Tensor k_rope,
    at::Tensor cos_sin_cache,
    at::Tensor pos_ids,
    bool interleave,
    int64_t hip_stream,
    const std::optional<at::Tensor>& v,
    const std::optional<at::Tensor>& k_buffer,
    const std::optional<at::Tensor>& v_buffer,
    const std::optional<at::Tensor>& kv_cache_loc) {
  CHECK_LAST_DIM_CONTIGUOUS(q);
  CHECK_LAST_DIM_CONTIGUOUS(k);

  const bool save_kv_cache = v.has_value();
  if (save_kv_cache) {
    TORCH_CHECK(v.has_value());
    TORCH_CHECK(k_buffer.has_value());
    TORCH_CHECK(v_buffer.has_value());
    TORCH_CHECK(kv_cache_loc.has_value());
    CHECK_LAST_DIM_CONTIGUOUS(v.value());
    CHECK_LAST_DIM_CONTIGUOUS(k_buffer.value());
    CHECK_LAST_DIM_CONTIGUOUS(v_buffer.value());
    CHECK_DIM(3, k_buffer.value());      // k_buffer: (nnz, H_K, D)
    CHECK_DIM(3, v_buffer.value());      // v_buffer: (nnz, H_V, D)
    CHECK_DIM(3, v.value());             // v: (nnz, H_V, D)
    CHECK_DIM(1, kv_cache_loc.value());  // v: (n)
    CHECK_INPUT(kv_cache_loc.value());
  }
  size_t k_buffer_stride_n = save_kv_cache ? k_buffer->stride(0) : 0;
  size_t k_buffer_stride_h = save_kv_cache ? k_buffer->stride(1) : 0;
  size_t v_buffer_stride_n = save_kv_cache ? v_buffer->stride(0) : 0;
  size_t v_buffer_stride_h = save_kv_cache ? v_buffer->stride(1) : 0;
  size_t v_stride_n = save_kv_cache ? v->stride(0) : 0;
  size_t v_stride_h = save_kv_cache ? v->stride(1) : 0;
  auto kv_cache_loc_ptr = save_kv_cache ? static_cast<int64_t*>(kv_cache_loc->data_ptr()) : nullptr;

  CHECK_INPUT(cos_sin_cache);
  CHECK_INPUT(pos_ids);
  auto device = q.device();
  CHECK_EQ(k.device(), device);
  CHECK_EQ(cos_sin_cache.device(), device);
  CHECK_EQ(pos_ids.device(), device);
  CHECK_DIM(3, q);  // q: (nnz, H_Q, D)
  CHECK_DIM(3, k);  // k: (nnz, H_K, D)

  // cos_sin_cache: (max_seq_len, R)
  // First half of R is cos, second half is sin
  CHECK_DIM(2, cos_sin_cache);
  CHECK_EQ(q.size(0), k.size(0));
  CHECK_EQ(q.size(2), k.size(2));
  unsigned int rotary_dim = cos_sin_cache.size(1);
  unsigned int num_qo_heads = q.size(1);
  unsigned int num_kv_heads = k.size(1);
  unsigned int head_dim = q.size(2);
  unsigned int nnz = q.size(0);
  size_t q_stride_n = q.stride(0);
  size_t q_stride_h = q.stride(1);
  size_t k_stride_n = k.stride(0);
  size_t k_stride_h = k.stride(1);

  size_t q_rope_stride_n = q_rope.stride(0);
  size_t q_rope_stride_h = q_rope.stride(1);
  size_t k_rope_stride_n = k_rope.stride(0);
  size_t k_rope_stride_h = k_rope.stride(1);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(q.scalar_type(), c_type, [&] {
    // TODO temporarily only use `BatchQKApplyRotaryPosIdsCosSinCacheEnhanced` when save_kv_cache
    // to avoid changing original code path; but this branch is feature-complete and should switch to this later
    if (save_kv_cache) {
      hipError_t status = BatchQKApplyRotaryPosIdsCosSinCacheEnhanced(
          static_cast<c_type*>(q.data_ptr()),
          static_cast<c_type*>(k.data_ptr()),
          save_kv_cache ? static_cast<c_type*>(v->data_ptr()) : nullptr,
          static_cast<c_type*>(q_rope.data_ptr()),
          static_cast<c_type*>(k_rope.data_ptr()),
          save_kv_cache ? static_cast<c_type*>(k_buffer->data_ptr()) : nullptr,
          save_kv_cache ? static_cast<c_type*>(v_buffer->data_ptr()) : nullptr,
          static_cast<float*>(cos_sin_cache.data_ptr()),
          static_cast<int64_t*>(pos_ids.data_ptr()),
          nnz,
          num_qo_heads,
          num_kv_heads,
          rotary_dim,
          head_dim,
          q_stride_n,
          q_stride_h,
          k_stride_n,
          k_stride_h,
          v_stride_n,
          v_stride_h,
          q_rope_stride_n,
          q_rope_stride_h,
          k_rope_stride_n,
          k_rope_stride_h,
          k_buffer_stride_n,
          k_buffer_stride_h,
          v_buffer_stride_n,
          v_buffer_stride_h,
          kv_cache_loc_ptr,
          interleave,
          save_kv_cache,
          stream);
      TORCH_CHECK(
          status == hipSuccess,
          "BatchQKApplyRotaryPosIdsCosSinCacheEnhanced failed with error code " +
              std::string(hipGetErrorString(status)));
    } else {
      hipError_t status = BatchQKApplyRotaryPosIdsCosSinCache(
          static_cast<c_type*>(q.data_ptr()),
          static_cast<c_type*>(k.data_ptr()),
          static_cast<c_type*>(q_rope.data_ptr()),
          static_cast<c_type*>(k_rope.data_ptr()),
          static_cast<float*>(cos_sin_cache.data_ptr()),
          static_cast<int64_t*>(pos_ids.data_ptr()),
          nnz,
          num_qo_heads,
          num_kv_heads,
          rotary_dim,
          head_dim,
          q_stride_n,
          q_stride_h,
          k_stride_n,
          k_stride_h,
          q_rope_stride_n,
          q_rope_stride_h,
          k_rope_stride_n,
          k_rope_stride_h,
          interleave,
          stream);
      TORCH_CHECK(
          status == hipSuccess,
          "BatchQKApplyRotaryPosIdsCosSinCache failed with error code " + std::string(hipGetErrorString(status)));
    }
    return true;
  });
}
