#include "hip/hip_runtime.h"
#include "pytorch_extension_utils.h"

template <typename T>
struct ConvertToFP8 {
  static __device__ __hip_fp8_storage_t convert_to_fp8(T value) {
    return 0;
  }
};

template <>
struct ConvertToFP8<__hip_bfloat16> {
  static __device__ __hip_fp8_storage_t convert_to_fp8(__hip_bfloat16 value) {
    return __hip_cvt_bfloat16raw_to_fp8(value, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
  }
};

template <>
struct ConvertToFP8<half> {
  static __device__ __hip_fp8_storage_t convert_to_fp8(half value) {
    return __hip_cvt_halfraw_to_fp8(value, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
  }
};

template <typename T>
struct ConvertFromFloat {
  static __device__ T convert_from_float(float value) {
    return 0;
  }
};

template <>
struct ConvertFromFloat<__hip_bfloat16> {
  static __device__ __hip_bfloat16 convert_from_float(float value) {
    return __float2bfloat16(value);
  }
};

template <>
struct ConvertFromFloat<half> {
  static __device__ half convert_from_float(float value) {
    return __float2half(value);
  }
};

template <typename T>
__global__ void fused_downcast_kernel(
    const T* cache_k,
    const T* cache_v,
    const float* k_scale,
    const float* v_scale,
    __hip_fp8_storage_t* output_k,
    __hip_fp8_storage_t* output_v,
    const int input_sl,
    const int head,
    const int dim,
    const T max_fp8,
    const T min_fp8,
    const int64_t mult,
    const int64_t offset,
    const int64_t* loc) {
  // TODO: change name
  int token_idx = blockIdx.x;
  int thread_idx = threadIdx.x;
  int total_threads = blockDim.x;

  T k_scale_val = ConvertFromFloat<T>::convert_from_float(k_scale[0]);
  T v_scale_val = ConvertFromFloat<T>::convert_from_float(v_scale[0]);

  T k_scale_inv = static_cast<T>(1.f) / k_scale_val;
  T v_scale_inv = static_cast<T>(1.f) / v_scale_val;

  auto clamp = [&](T val) { return val > max_fp8 ? max_fp8 : (min_fp8 > val ? min_fp8 : val); };

  if (token_idx < input_sl) {
    int out_seq_idx = loc[token_idx];

#pragma unroll
    for (int i = thread_idx; i < head * dim; i += total_threads) {
      int in_idx = token_idx * head * dim + i;
      int out_idx = (out_seq_idx * mult + offset) * head * dim + i;

      T k_val = cache_k[in_idx] * k_scale_inv;
      k_val = clamp(k_val);
      output_k[out_idx] = ConvertToFP8<T>::convert_to_fp8(k_val);

      T v_val = cache_v[in_idx] * v_scale_inv;
      v_val = clamp(v_val);
      output_v[out_idx] = ConvertToFP8<T>::convert_to_fp8(v_val);
    }
  }
}

template <typename T>
void downcast_fp8_impl(
    at::Tensor& k,
    at::Tensor& v,
    at::Tensor& k_out,
    at::Tensor& v_out,
    at::Tensor& k_scale,
    at::Tensor& v_scale,
    at::Tensor& loc,
    int64_t mult,
    int64_t offset,
    hipStream_t stream) {
  CHECK_INPUT(k);
  CHECK_INPUT(v);
  CHECK_INPUT(k_out);
  CHECK_INPUT(v_out);
  CHECK_INPUT(k_scale);
  CHECK_INPUT(v_scale);
  CHECK_INPUT(loc);

  int64_t input_sl = k.size(0);
  int64_t head = k.size(1);
  int64_t dim = k.size(2);

  dim3 grid(input_sl * head);
  int vec_size = 8;
  dim3 block(std::min(int(dim) / vec_size, 1024));

  const T max_fp8 = static_cast<T>(448.0f);
  const T min_fp8 = static_cast<T>(-448.0f);

  fused_downcast_kernel<T><<<grid, block, 0, stream>>>(
      static_cast<const T*>(k.data_ptr()),
      static_cast<const T*>(v.data_ptr()),
      static_cast<const float*>(k_scale.data_ptr()),
      static_cast<const float*>(v_scale.data_ptr()),
      static_cast<__hip_fp8_storage_t*>(k_out.data_ptr()),
      static_cast<__hip_fp8_storage_t*>(v_out.data_ptr()),
      input_sl,
      head,
      dim,
      max_fp8,
      min_fp8,
      mult,
      offset,
      static_cast<const int64_t*>(loc.data_ptr()));

  hipError_t status = hipGetLastError();
  TORCH_CHECK(status == hipSuccess, "Kernel launch failed: " + std::string(hipGetErrorString(status)));
}

void downcast_fp8(
    at::Tensor& k,
    at::Tensor& v,
    at::Tensor& k_out,
    at::Tensor& v_out,
    at::Tensor& k_scale,
    at::Tensor& v_scale,
    at::Tensor& loc,
    int64_t mult,
    int64_t offset,
    int64_t hip_stream) {
  CHECK_INPUT(k);
  CHECK_INPUT(v);
  CHECK_INPUT(k_out);
  CHECK_INPUT(v_out);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  switch (k.scalar_type()) {
    case at::ScalarType::BFloat16:
      downcast_fp8_impl<__hip_bfloat16>(k, v, k_out, v_out, k_scale, v_scale, loc, mult, offset, stream);
      break;
    case at::ScalarType::Half:
      downcast_fp8_impl<__half>(k, v, k_out, v_out, k_scale, v_scale, loc, mult, offset, stream);
      break;
    default:
      TORCH_CHECK(false, "Unsupported input type for downcast_fp8. Expected bfloat16 or float16.");
  }
}
