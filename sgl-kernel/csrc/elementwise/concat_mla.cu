#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDADataType.h>
#include <hip/hip_runtime.h>

#include "pytorch_extension_utils.h"

constexpr int NUM_LOCAL_HEADS = 128;
constexpr int QK_NOPE_HEAD_DIM = 128;
constexpr int QK_ROPE_HEAD_DIM = 64;
constexpr int K_HEAD_DIM = QK_NOPE_HEAD_DIM + QK_ROPE_HEAD_DIM;

constexpr int HEAD_CHUNK_SIZE = 16;
constexpr int NUM_HEAD_CHUNKS = NUM_LOCAL_HEADS / HEAD_CHUNK_SIZE;

__forceinline__ __device__ int get_lane_id() {
  int lane_id;
  asm("mov.s32 %0, %laneid;" : "=r"(lane_id));
  return lane_id;
}

int ceil_div(int a, int b) {
  return (a + b - 1) / b;
}

__global__ void concat_mla_k_kernel(
    hip_bfloat16* k,
    hip_bfloat16* k_nope,
    hip_bfloat16* k_rope,
    const int num_tokens,
    const int k_stride_0,
    const int k_stride_1,
    const int k_nope_stride_0,
    const int k_nope_stride_1,
    const int k_rope_stride_0) {
  const int flat_warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  const int token_id = flat_warp_id / NUM_HEAD_CHUNKS;
  const int head_chunk_id = flat_warp_id % NUM_HEAD_CHUNKS;
  const int lane_id = get_lane_id();

  if (token_id >= num_tokens) {
    return;
  }

  using KNopeBufType = int2;
  static_assert(sizeof(KNopeBufType) == QK_NOPE_HEAD_DIM * sizeof(k[0]) / 32);
  KNopeBufType k_nope_buf[HEAD_CHUNK_SIZE];

  using KRopeBufType = int;
  static_assert(sizeof(KRopeBufType) == QK_ROPE_HEAD_DIM * sizeof(k[0]) / 32);
  KRopeBufType k_rope_buf;

  {
    const int* base_addr = reinterpret_cast<int*>(k_rope + token_id * k_rope_stride_0);
    k_rope_buf = *(base_addr + lane_id);
  }

#pragma unroll
  for (int i = 0; i < HEAD_CHUNK_SIZE; ++i) {
    const int head_id = head_chunk_id * HEAD_CHUNK_SIZE + i;
    const int2* base_addr = reinterpret_cast<int2*>(k_nope + token_id * k_nope_stride_0 + head_id * k_nope_stride_1);
    k_nope_buf[i] = *(base_addr + lane_id);
  }

#pragma unroll
  for (int i = 0; i < HEAD_CHUNK_SIZE; ++i) {
    const int head_id = head_chunk_id * HEAD_CHUNK_SIZE + i;

    {
      int2* base_addr = reinterpret_cast<int2*>(k + token_id * k_stride_0 + head_id * k_stride_1);
      *(base_addr + lane_id) = k_nope_buf[i];
    }
    {
      int* base_addr = reinterpret_cast<int*>(k + token_id * k_stride_0 + head_id * k_stride_1 + QK_NOPE_HEAD_DIM);
      *(base_addr + lane_id) = k_rope_buf;
    }
  }
}

inline void check_tensor(const at::Tensor& t, int64_t shape0, int64_t shape1, int64_t shape2, c10::ScalarType dtype) {
  TORCH_CHECK_EQ(t.dim(), 3);
  TORCH_CHECK_EQ(t.size(0), shape0);
  TORCH_CHECK_EQ(t.size(1), shape1);
  TORCH_CHECK_EQ(t.size(2), shape2);
  TORCH_CHECK_EQ(t.dtype(), dtype);
  TORCH_CHECK(t.device().is_cuda());
  TORCH_CHECK_EQ(((int64_t)t.data_ptr()) % 16, 0);  // alignment
}

void concat_mla_k(at::Tensor k, at::Tensor k_nope, at::Tensor k_rope) {
  const int num_tokens = k.size(0);

  check_tensor(k, num_tokens, NUM_LOCAL_HEADS, K_HEAD_DIM, at::kBFloat16);
  check_tensor(k_nope, num_tokens, NUM_LOCAL_HEADS, QK_NOPE_HEAD_DIM, at::kBFloat16);
  check_tensor(k_rope, num_tokens, 1, QK_ROPE_HEAD_DIM, at::kBFloat16);
  TORCH_CHECK_EQ(k.stride(2), 1);
  TORCH_CHECK_EQ(k_nope.stride(2), 1);
  TORCH_CHECK_EQ(k_rope.stride(2), 1);

  const auto stream = at::cuda::getCurrentCUDAStream().stream();

  constexpr int num_warps_per_block = 32;
  const int grid_size = ceil_div(num_tokens * NUM_HEAD_CHUNKS, num_warps_per_block);
  const int block_size = num_warps_per_block * 32;

  concat_mla_k_kernel<<<grid_size, block_size, 0, stream>>>(
      reinterpret_cast<hip_bfloat16*>(k.data_ptr()),
      reinterpret_cast<hip_bfloat16*>(k_nope.data_ptr()),
      reinterpret_cast<hip_bfloat16*>(k_rope.data_ptr()),
      num_tokens,
      k.stride(0),
      k.stride(1),
      k_nope.stride(0),
      k_nope.stride(1),
      k_rope.stride(0));
  hipError_t err = hipGetLastError();
  TORCH_CHECK(err == hipSuccess, "CUDA kernel launch failed: ", hipGetErrorString(err));
}

// ============================== concat_mla_absorb_q ==============================

// TODO give a name prefix, also maybe refactor code above
constexpr int A_LAST_DIM = 512;
constexpr int B_LAST_DIM = 64;

__global__ void concat_mla_absorb_q_kernel(
    hip_bfloat16* a,
    hip_bfloat16* b,
    hip_bfloat16* out,
    const int num_items,
    const int dim_1,
    const int a_stride_0,
    const int a_stride_1,
    const int b_stride_0,
    const int b_stride_1,
    const int out_stride_0,
    const int out_stride_1) {
  const int flat_warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  const int lane_id = get_lane_id();

  const int idx_0 = flat_warp_id / dim_1;
  const int idx_1 = flat_warp_id % dim_1;

  if (flat_warp_id >= num_items) {
    return;
  }

  using ABufType = int4;
  constexpr int A_NUM_UNROLL = 2;
  static_assert(sizeof(ABufType) * A_NUM_UNROLL == A_LAST_DIM * sizeof(a[0]) / 32);
  ABufType a_buf[A_NUM_UNROLL];

  using BBufType = int;
  constexpr int B_NUM_UNROLL = 1;
  static_assert(sizeof(BBufType) * B_NUM_UNROLL == B_LAST_DIM * sizeof(b[0]) / 32);
  BBufType b_buf;

  {
    const BBufType* base_addr = reinterpret_cast<BBufType*>(b + idx_0 * b_stride_0 + idx_1 * b_stride_1);
    b_buf = *(base_addr + lane_id);
  }

#pragma unroll
  for (int i = 0; i < A_NUM_UNROLL; ++i) {
    const ABufType* base_addr = reinterpret_cast<ABufType*>(a + idx_0 * a_stride_0 + idx_1 * a_stride_1);
    a_buf[i] = *(base_addr + i * 32 + lane_id);
  }

  {
    BBufType* base_addr = reinterpret_cast<BBufType*>(out + idx_0 * out_stride_0 + idx_1 * out_stride_1 + A_LAST_DIM);
    *(base_addr + lane_id) = b_buf;
  }

#pragma unroll
  for (int i = 0; i < A_NUM_UNROLL; ++i) {
    ABufType* base_addr = reinterpret_cast<ABufType*>(out + idx_0 * out_stride_0 + idx_1 * out_stride_1);
    *(base_addr + i * 32 + lane_id) = a_buf[i];
  }
}

inline void check_tensor_concat_mla_absorb_q(const at::Tensor& t, int64_t shape2) {
  TORCH_CHECK_EQ(t.dim(), 3);
  TORCH_CHECK_EQ(t.size(2), shape2);
  TORCH_CHECK_EQ(t.stride(2), 1);
  TORCH_CHECK_EQ(t.dtype(), at::kBFloat16);
  TORCH_CHECK(t.device().is_cuda());
  TORCH_CHECK_EQ(((int64_t)t.data_ptr()) % 16, 0);  // alignment
}

// TODO further optimize it later
void concat_mla_absorb_q(at::Tensor a, at::Tensor b, at::Tensor out) {
  check_tensor_concat_mla_absorb_q(a, A_LAST_DIM);
  check_tensor_concat_mla_absorb_q(b, B_LAST_DIM);
  check_tensor_concat_mla_absorb_q(out, A_LAST_DIM + B_LAST_DIM);

  const auto stream = at::cuda::getCurrentCUDAStream().stream();

  TORCH_CHECK_EQ(a.size(0) * a.size(1), b.size(0) * b.size(1));
  TORCH_CHECK_EQ(a.size(1), b.size(1));
  const int num_items = a.size(0) * a.size(1);

  constexpr int num_warps_per_block = 32;
  const int grid_size = ceil_div(num_items, num_warps_per_block);
  const int block_size = num_warps_per_block * 32;

  concat_mla_absorb_q_kernel<<<grid_size, block_size, 0, stream>>>(
      reinterpret_cast<hip_bfloat16*>(a.data_ptr()),
      reinterpret_cast<hip_bfloat16*>(b.data_ptr()),
      reinterpret_cast<hip_bfloat16*>(out.data_ptr()),
      num_items,
      a.size(1),
      a.stride(0),
      a.stride(1),
      b.stride(0),
      b.stride(1),
      out.stride(0),
      out.stride(1));
  hipError_t err = hipGetLastError();
  TORCH_CHECK(err == hipSuccess, "CUDA kernel launch failed: ", hipGetErrorString(err));
}
