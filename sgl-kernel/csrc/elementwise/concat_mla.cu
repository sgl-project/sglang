#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDADataType.h>
#include <hip/hip_runtime.h>

#include "pytorch_extension_utils.h"

constexpr int NUM_LOCAL_HEADS = 128;
constexpr int QK_NOPE_HEAD_DIM = 128;
constexpr int QK_ROPE_HEAD_DIM = 64;
constexpr int K_HEAD_DIM = QK_NOPE_HEAD_DIM + QK_ROPE_HEAD_DIM;

constexpr int HEAD_CHUNK_SIZE = 16;
constexpr int NUM_HEAD_CHUNKS = NUM_LOCAL_HEADS / HEAD_CHUNK_SIZE;

__forceinline__ __device__ int get_lane_id() {
  int lane_id;
  asm("mov.s32 %0, %laneid;" : "=r"(lane_id));
  return lane_id;
}

int ceil_div(int a, int b) {
  return (a + b - 1) / b;
}

__global__ void concat_mla_k_kernel(
    hip_bfloat16* k,
    hip_bfloat16* k_nope,
    hip_bfloat16* k_rope,
    const int num_tokens,
    const int k_stride_0,
    const int k_stride_1,
    const int k_nope_stride_0,
    const int k_nope_stride_1,
    const int k_rope_stride_0) {
  const int flat_warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  const int token_id = flat_warp_id / NUM_HEAD_CHUNKS;
  const int head_chunk_id = flat_warp_id % NUM_HEAD_CHUNKS;
  const int lane_id = get_lane_id();

  if (token_id >= num_tokens) {
    return;
  }

  using KNopeBufType = int2;
  static_assert(sizeof(KNopeBufType) == QK_NOPE_HEAD_DIM * sizeof(k[0]) / 32);
  KNopeBufType k_nope_buf[HEAD_CHUNK_SIZE];

  using KRopeBufType = int;
  static_assert(sizeof(KRopeBufType) == QK_ROPE_HEAD_DIM * sizeof(k[0]) / 32);
  KRopeBufType k_rope_buf;

  {
    const int* base_addr = reinterpret_cast<int*>(k_rope + token_id * k_rope_stride_0);
    k_rope_buf = *(base_addr + lane_id);
  }

#pragma unroll
  for (int i = 0; i < HEAD_CHUNK_SIZE; ++i) {
    const int head_id = head_chunk_id * HEAD_CHUNK_SIZE + i;
    const int2* base_addr = reinterpret_cast<int2*>(k_nope + token_id * k_nope_stride_0 + head_id * k_nope_stride_1);
    k_nope_buf[i] = *(base_addr + lane_id);
  }

#pragma unroll
  for (int i = 0; i < HEAD_CHUNK_SIZE; ++i) {
    const int head_id = head_chunk_id * HEAD_CHUNK_SIZE + i;

    {
      int2* base_addr = reinterpret_cast<int2*>(k + token_id * k_stride_0 + head_id * k_stride_1);
      *(base_addr + lane_id) = k_nope_buf[i];
    }
    {
      int* base_addr = reinterpret_cast<int*>(k + token_id * k_stride_0 + head_id * k_stride_1 + QK_NOPE_HEAD_DIM);
      *(base_addr + lane_id) = k_rope_buf;
    }
  }
}

inline void check_tensor(const at::Tensor& t, int64_t shape0, int64_t shape1, int64_t shape2, c10::ScalarType dtype) {
  TORCH_CHECK_EQ(t.dim(), 3);
  TORCH_CHECK_EQ(t.size(0), shape0);
  TORCH_CHECK_EQ(t.size(1), shape1);
  TORCH_CHECK_EQ(t.size(2), shape2);
  TORCH_CHECK_EQ(t.dtype(), dtype);
  TORCH_CHECK(t.device().is_cuda());
  TORCH_CHECK_EQ(((int64_t)t.data_ptr()) % 16, 0);  // alignment
}

void concat_mla_k(at::Tensor k, at::Tensor k_nope, at::Tensor k_rope) {
  const int num_tokens = k.size(0);

  check_tensor(k, num_tokens, NUM_LOCAL_HEADS, K_HEAD_DIM, at::kBFloat16);
  check_tensor(k_nope, num_tokens, NUM_LOCAL_HEADS, QK_NOPE_HEAD_DIM, at::kBFloat16);
  check_tensor(k_rope, num_tokens, 1, QK_ROPE_HEAD_DIM, at::kBFloat16);
  TORCH_CHECK_EQ(k.stride(2), 1);
  TORCH_CHECK_EQ(k_nope.stride(2), 1);
  TORCH_CHECK_EQ(k_rope.stride(2), 1);

  const auto stream = at::cuda::getCurrentCUDAStream().stream();

  constexpr int num_warps_per_block = 32;
  const int grid_size = ceil_div(num_tokens * NUM_HEAD_CHUNKS, num_warps_per_block);
  const int block_size = num_warps_per_block * 32;

  concat_mla_k_kernel<<<grid_size, block_size, 0, stream>>>(
      reinterpret_cast<hip_bfloat16*>(k.data_ptr()),
      reinterpret_cast<hip_bfloat16*>(k_nope.data_ptr()),
      reinterpret_cast<hip_bfloat16*>(k_rope.data_ptr()),
      num_tokens,
      k.stride(0),
      k.stride(1),
      k_nope.stride(0),
      k_nope.stride(1),
      k_rope.stride(0));
  hipError_t err = hipGetLastError();
  TORCH_CHECK(err == hipSuccess, "CUDA kernel launch failed: ", hipGetErrorString(err));
}
