#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDADataType.h>
#include <hip/hip_runtime.h>

#include "pytorch_extension_utils.h"
#include "utils.cuh"

constexpr int NUM_LOCAL_HEADS = 128;
constexpr int QK_NOPE_HEAD_DIM = 128;
constexpr int QK_ROPE_HEAD_DIM = 64;
constexpr int K_HEAD_DIM = QK_NOPE_HEAD_DIM + QK_ROPE_HEAD_DIM;

constexpr int HEAD_CHUNK_SIZE = 16;
constexpr int NUM_HEAD_CHUNKS = NUM_LOCAL_HEADS / HEAD_CHUNK_SIZE;

__global__ void concat_mla_k_kernel(
    hip_bfloat16* __restrict__ k,
    const hip_bfloat16* __restrict__ k_nope,
    const hip_bfloat16* __restrict__ k_rope,
    const int num_tokens,
    const int k_stride_0,
    const int k_stride_1,
    const int k_nope_stride_0,
    const int k_nope_stride_1,
    const int k_rope_stride_0) {
  const int flat_warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  const int token_id = flat_warp_id / NUM_HEAD_CHUNKS;
  const int head_chunk_id = flat_warp_id % NUM_HEAD_CHUNKS;
  const int lane_id = get_lane_id();
  if (token_id >= num_tokens) return;

  using NopeVec = int2;  // 8B/thread，32 thread = 256B/row
  using RopeVec = int;   // 4B/thread，32 thread = 128B/row
  static_assert(sizeof(NopeVec) * 32 == QK_NOPE_HEAD_DIM * sizeof(hip_bfloat16), "nope vec mismatch");
  static_assert(sizeof(RopeVec) * 32 == QK_ROPE_HEAD_DIM * sizeof(hip_bfloat16), "rope vec mismatch");

  const int head_row0 = head_chunk_id * HEAD_CHUNK_SIZE;

  const int2* __restrict__ nope_src =
      reinterpret_cast<const int2*>(k_nope + token_id * k_nope_stride_0 + head_row0 * k_nope_stride_1) + lane_id;

  int2* __restrict__ nope_dst = reinterpret_cast<int2*>(k + token_id * k_stride_0 + head_row0 * k_stride_1) + lane_id;

  int* __restrict__ rope_dst =
      reinterpret_cast<int*>(k + token_id * k_stride_0 + head_row0 * k_stride_1 + QK_NOPE_HEAD_DIM) + lane_id;

  const int nope_src_stride_v = (k_nope_stride_1 >> 2);  // int2 covers 4 bf16
  const int nope_dst_stride_v = (k_stride_1 >> 2);
  const int rope_dst_stride_v = (k_stride_1 >> 1);  // int covers 2 bf16

  const int* rope_base = reinterpret_cast<const int*>(k_rope + token_id * k_rope_stride_0);
  const RopeVec rope_val = ld_na_global_v1(rope_base + lane_id);

  prefetch_L2(nope_src);
  NopeVec cur = ld_na_global_v2(nope_src);

#pragma unroll
  for (int i = 0; i < HEAD_CHUNK_SIZE; ++i) {
    NopeVec next;
    if (i + 1 < HEAD_CHUNK_SIZE) {
      const int2* next_src = nope_src + nope_src_stride_v;
      prefetch_L2(next_src);
      next = ld_na_global_v2(next_src);
    }

    st_na_global_v2(nope_dst, cur);
    st_na_global_v1(rope_dst, rope_val);

    nope_src += nope_src_stride_v;
    nope_dst += nope_dst_stride_v;
    rope_dst += rope_dst_stride_v;

    cur = next;
  }
}

inline void check_tensor(const at::Tensor& t, int64_t shape0, int64_t shape1, int64_t shape2, c10::ScalarType dtype) {
  TORCH_CHECK_EQ(t.dim(), 3);
  TORCH_CHECK_EQ(t.size(0), shape0);
  TORCH_CHECK_EQ(t.size(1), shape1);
  TORCH_CHECK_EQ(t.size(2), shape2);
  TORCH_CHECK_EQ(t.dtype(), dtype);
  TORCH_CHECK(t.device().is_cuda());
  TORCH_CHECK_EQ(((int64_t)t.data_ptr()) % 16, 0);  // alignment
}

void concat_mla_k(at::Tensor k, at::Tensor k_nope, at::Tensor k_rope) {
  const int num_tokens = k.size(0);

  check_tensor(k, num_tokens, NUM_LOCAL_HEADS, K_HEAD_DIM, at::kBFloat16);
  check_tensor(k_nope, num_tokens, NUM_LOCAL_HEADS, QK_NOPE_HEAD_DIM, at::kBFloat16);
  check_tensor(k_rope, num_tokens, 1, QK_ROPE_HEAD_DIM, at::kBFloat16);
  TORCH_CHECK_EQ(k.stride(2), 1);
  TORCH_CHECK_EQ(k_nope.stride(2), 1);
  TORCH_CHECK_EQ(k_rope.stride(2), 1);

  const auto stream = at::cuda::getCurrentCUDAStream().stream();

  constexpr int num_warps_per_block = 32;
  const int grid_size = ceil_div(num_tokens * NUM_HEAD_CHUNKS, num_warps_per_block);
  const int block_size = num_warps_per_block * 32;

  concat_mla_k_kernel<<<grid_size, block_size, 0, stream>>>(
      reinterpret_cast<hip_bfloat16*>(k.data_ptr()),
      reinterpret_cast<hip_bfloat16*>(k_nope.data_ptr()),
      reinterpret_cast<hip_bfloat16*>(k_rope.data_ptr()),
      num_tokens,
      k.stride(0),
      k.stride(1),
      k_nope.stride(0),
      k_nope.stride(1),
      k_rope.stride(0));
  hipError_t err = hipGetLastError();
  TORCH_CHECK(err == hipSuccess, "CUDA kernel launch failed: ", hipGetErrorString(err));
}

// ============================== concat_mla_absorb_q ==============================

// TODO give a name prefix, also maybe refactor code above
constexpr int A_LAST_DIM = 512;
constexpr int B_LAST_DIM = 64;

__global__ void concat_mla_absorb_q_kernel(
    hip_bfloat16* a,
    hip_bfloat16* b,
    hip_bfloat16* out,
    const int num_items,
    const int dim_1,
    const int a_stride_0,
    const int a_stride_1,
    const int b_stride_0,
    const int b_stride_1,
    const int out_stride_0,
    const int out_stride_1) {
  const int flat_warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  const int lane_id = get_lane_id();

  const int idx_0 = flat_warp_id / dim_1;
  const int idx_1 = flat_warp_id % dim_1;

  if (flat_warp_id >= num_items) {
    return;
  }

  using ABufType = int4;
  constexpr int A_NUM_UNROLL = 2;
  static_assert(sizeof(ABufType) * A_NUM_UNROLL == A_LAST_DIM * sizeof(a[0]) / 32);
  ABufType a_buf[A_NUM_UNROLL];

  using BBufType = int;
  constexpr int B_NUM_UNROLL = 1;
  static_assert(sizeof(BBufType) * B_NUM_UNROLL == B_LAST_DIM * sizeof(b[0]) / 32);
  BBufType b_buf;

  {
    const BBufType* base_addr = reinterpret_cast<BBufType*>(b + idx_0 * b_stride_0 + idx_1 * b_stride_1);
    b_buf = *(base_addr + lane_id);
  }

#pragma unroll
  for (int i = 0; i < A_NUM_UNROLL; ++i) {
    const ABufType* base_addr = reinterpret_cast<ABufType*>(a + idx_0 * a_stride_0 + idx_1 * a_stride_1);
    a_buf[i] = *(base_addr + i * 32 + lane_id);
  }

  {
    BBufType* base_addr = reinterpret_cast<BBufType*>(out + idx_0 * out_stride_0 + idx_1 * out_stride_1 + A_LAST_DIM);
    *(base_addr + lane_id) = b_buf;
  }

#pragma unroll
  for (int i = 0; i < A_NUM_UNROLL; ++i) {
    ABufType* base_addr = reinterpret_cast<ABufType*>(out + idx_0 * out_stride_0 + idx_1 * out_stride_1);
    *(base_addr + i * 32 + lane_id) = a_buf[i];
  }
}

inline void check_tensor_concat_mla_absorb_q(const at::Tensor& t, int64_t shape2) {
  TORCH_CHECK_EQ(t.dim(), 3);
  TORCH_CHECK_EQ(t.size(2), shape2);
  TORCH_CHECK_EQ(t.stride(2), 1);
  TORCH_CHECK_EQ(t.dtype(), at::kBFloat16);
  TORCH_CHECK(t.device().is_cuda());
  TORCH_CHECK_EQ(((int64_t)t.data_ptr()) % 16, 0);  // alignment
}

// TODO further optimize it later
void concat_mla_absorb_q(at::Tensor a, at::Tensor b, at::Tensor out) {
  check_tensor_concat_mla_absorb_q(a, A_LAST_DIM);
  check_tensor_concat_mla_absorb_q(b, B_LAST_DIM);
  check_tensor_concat_mla_absorb_q(out, A_LAST_DIM + B_LAST_DIM);

  const auto stream = at::cuda::getCurrentCUDAStream().stream();

  TORCH_CHECK_EQ(a.size(0) * a.size(1), b.size(0) * b.size(1));
  TORCH_CHECK_EQ(a.size(1), b.size(1));
  const int num_items = a.size(0) * a.size(1);

  constexpr int num_warps_per_block = 32;
  const int grid_size = ceil_div(num_items, num_warps_per_block);
  const int block_size = num_warps_per_block * 32;

  concat_mla_absorb_q_kernel<<<grid_size, block_size, 0, stream>>>(
      reinterpret_cast<hip_bfloat16*>(a.data_ptr()),
      reinterpret_cast<hip_bfloat16*>(b.data_ptr()),
      reinterpret_cast<hip_bfloat16*>(out.data_ptr()),
      num_items,
      a.size(1),
      a.stride(0),
      a.stride(1),
      b.stride(0),
      b.stride(1),
      out.stride(0),
      out.stride(1));
  hipError_t err = hipGetLastError();
  TORCH_CHECK(err == hipSuccess, "CUDA kernel launch failed: ", hipGetErrorString(err));
}
