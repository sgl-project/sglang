#include "hip/hip_runtime.h"
/* Copyright 2025 SGLang Team. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include <ATen/cuda/HIPContext.h>

#include <flashinfer/norm.cuh>

#include "utils.h"

using namespace flashinfer;

void sgl_fused_add_rmsnorm(
    torch::Tensor input, torch::Tensor residual, torch::Tensor weight, double eps, bool enable_pdl) {
  CHECK_INPUT(input);
  CHECK_INPUT(residual);
  CHECK_INPUT(weight);
  auto device = input.device();
  CHECK_EQ(residual.device(), device);
  CHECK_EQ(weight.device(), device);
  CHECK_DIM(2, input);     // input: (batch_size, hidden_size)
  CHECK_DIM(2, residual);  // residual: (batch_size, hidden_size)
  CHECK_DIM(1, weight);    // weight: (hidden_size)
  CHECK_EQ(input.size(0), residual.size(0));
  CHECK_EQ(input.size(1), residual.size(1));
  CHECK_EQ(input.size(1), weight.size(0));
  unsigned int batch_size = input.size(0);
  unsigned int hidden_size = input.size(1);

  hipStream_t torch_current_stream = at::cuda::getCurrentCUDAStream();
  // support float16, bfloat16 and float32
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FLOAT_FP16(input.scalar_type(), c_type, [&] {
    hipError_t status = norm::FusedAddRMSNorm(
        static_cast<c_type*>(input.data_ptr()),
        static_cast<c_type*>(residual.data_ptr()),
        static_cast<c_type*>(weight.data_ptr()),
        batch_size,
        hidden_size,
        input.stride(0),
        residual.stride(0),
        eps,
        enable_pdl,
        torch_current_stream);
    TORCH_CHECK(
        status == hipSuccess, "FusedAddRMSNorm failed with error code " + std::string(hipGetErrorString(status)));
    return true;
  });
}
