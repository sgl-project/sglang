#include "hip/hip_runtime.h"
/**
 * @NOTE: This file is adapted from
 * https://github.com/tile-ai/tilelang/blob/main/examples/deepseek_v32/topk_selector.py
 * We:
 * 1. adapt from tilelang to pure cuda
 * 2. optimize the performance a little
 * 3. fix the potential illegal memory access
 */
#include <ATen/core/TensorBase.h>
#include <ATen/core/TensorBody.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/macros/Macros.h>
#include <c10/util/Exception.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <cstddef>
#include <cstdint>
#include <optional>

namespace {

constexpr int TopK = 2048;
constexpr int kThreadsPerBlock = 1024;
constexpr size_t kSmem = 32 * 1024 * sizeof(uint32_t);  // 128KB

struct FastTopKParams {
  const float* __restrict__ input;  // [B, input_stride]
  int32_t* __restrict__ indices;    // [B, TopK]
  int32_t* __restrict__ lengths;    // [B]
  int64_t input_stride;
};

// when length <= TopK, we can directly write the indices
__device__ void naive_topk_cuda(const float* __restrict__ score, int32_t* __restrict__ indice, int32_t length) {
  const auto tid = threadIdx.x;
  for (int i = tid; i < TopK; i += kThreadsPerBlock) {
    indice[i] = (i < length) ? i : -1;
  }
}

// keep the first `length` entries, set others to -1
__device__ void naive_topk_transform(
    const float* __restrict__ score,
    int32_t length,
    int32_t* __restrict__ dst_page_table,
    const int32_t* __restrict__ src_page_table) {
  const auto tid = threadIdx.x;
  for (auto i = tid; i < TopK; i += kThreadsPerBlock) {
    dst_page_table[i] = (i < length) ? src_page_table[i] : -1;
  }
}

// keep the first `length` entries, set others to -1
__device__ void naive_topk_transform_ragged(
    const float* __restrict__ score, int32_t length, int32_t* __restrict__ topk_indices_ragged, int32_t offset) {
  const auto tid = threadIdx.x;
  for (auto i = tid; i < TopK; i += kThreadsPerBlock) {
    topk_indices_ragged[i] = (i < length) ? static_cast<int32_t>(i) + offset : -1;
  }
}

__device__ __forceinline__ auto convert_to_uint8(float x) -> uint8_t {
  __half h = __float2half_rn(x);
  uint16_t bits = __half_as_ushort(h);
  uint16_t key = (bits & 0x8000) ? static_cast<uint16_t>(~bits) : static_cast<uint16_t>(bits | 0x8000);
  return static_cast<uint8_t>(key >> 8);
}

__device__ __forceinline__ auto convert_to_uint32(float x) -> uint32_t {
  uint32_t bits = __float_as_uint(x);
  return (bits & 0x80000000u) ? ~bits : (bits | 0x80000000u);
}

__device__ void fast_topk_cuda_tl(const float* __restrict__ input, int* __restrict__ index, int length) {
  // An optimized topk kernel copied from tilelang kernel
  // We assume length > TopK here, or it will crash
  int topk = TopK;
  constexpr auto BLOCK_SIZE = 1024;
  constexpr auto RADIX = 256;
  constexpr auto SMEM_INPUT_SIZE = kSmem / (2 * sizeof(int));

  alignas(128) __shared__ int s_histogram_buf[2][RADIX + 128];
  alignas(128) __shared__ int s_counter;
  alignas(128) __shared__ int s_threshold_bin_id;
  alignas(128) __shared__ int s_num_input[2];

  auto& s_histogram = s_histogram_buf[0];
  // allocate for two rounds
  extern __shared__ int s_input_idx[][SMEM_INPUT_SIZE];

  const int tx = threadIdx.x;

  // stage 1: 8bit coarse histogram
  if (tx < RADIX + 1) s_histogram[tx] = 0;
  __syncthreads();

  for (int idx = tx; idx < length; idx += BLOCK_SIZE) {
    const auto bin = convert_to_uint8(input[idx]);
    ::atomicAdd(&s_histogram[bin], 1);
  }
  __syncthreads();

  const auto run_cumsum = [&] {
#pragma unroll 8
    for (int i = 0; i < 8; ++i) {
      static_assert(1 << 8 == RADIX);
      if (C10_LIKELY(tx < RADIX)) {
        const auto j = 1 << i;
        const auto k = i & 1;
        auto value = s_histogram_buf[k][tx];
        if (tx < RADIX - j) {
          value += s_histogram_buf[k][tx + j];
        }
        s_histogram_buf[k ^ 1][tx] = value;
      }
      __syncthreads();
    }
  };

  run_cumsum();
  if (tx < RADIX && s_histogram[tx] > topk && s_histogram[tx + 1] <= topk) {
    s_threshold_bin_id = tx;
    s_num_input[0] = 0;
    s_counter = 0;
  }
  __syncthreads();

  const auto threshold_bin = s_threshold_bin_id;
  topk -= s_histogram[threshold_bin + 1];

  if (topk == 0) {
    for (int idx = tx; idx < length; idx += BLOCK_SIZE) {
      const auto bin = static_cast<int>(convert_to_uint8(input[idx]));
      if (bin > threshold_bin) {
        const auto pos = ::atomicAdd(&s_counter, 1);
        index[pos] = idx;
      }
    }
    __syncthreads();
    return;
  } else {
    __syncthreads();
    if (tx < RADIX + 1) {
      s_histogram[tx] = 0;
    }
    __syncthreads();

    for (int idx = tx; idx < length; idx += BLOCK_SIZE) {
      const auto raw_input = input[idx];
      const auto bin = static_cast<int>(convert_to_uint8(raw_input));
      if (bin > threshold_bin) {
        const auto pos = ::atomicAdd(&s_counter, 1);
        index[pos] = idx;
      } else if (bin == threshold_bin) {
        const auto pos = ::atomicAdd(&s_num_input[0], 1);
        /// NOTE: (dark) fuse the histogram computation here
        if (C10_LIKELY(pos < SMEM_INPUT_SIZE)) {
          s_input_idx[0][pos] = idx;
          const auto bin = convert_to_uint32(raw_input);
          const auto sub_bin = (bin >> 24) & 0xFF;
          ::atomicAdd(&s_histogram[sub_bin], 1);
        }
      }
    }
    __syncthreads();
  }

  // stage 2: refine with 8bit radix passes
#pragma unroll 4
  for (int round = 0; round < 4; ++round) {
    __shared__ int s_last_remain;
    const auto r_idx = round % 2;

    // clip here to prevent overflow
    const auto _raw_num_input = s_num_input[r_idx];
    const auto num_input = (_raw_num_input < int(SMEM_INPUT_SIZE)) ? _raw_num_input : int(SMEM_INPUT_SIZE);

    run_cumsum();
    if (tx < RADIX && s_histogram[tx] > topk && s_histogram[tx + 1] <= topk) {
      s_threshold_bin_id = tx;
      s_num_input[r_idx ^ 1] = 0;
      s_last_remain = topk - s_histogram[tx + 1];
    }
    __syncthreads();

    const auto threshold_bin = s_threshold_bin_id;
    topk -= s_histogram[threshold_bin + 1];

    if (topk == 0) {
      for (int i = tx; i < num_input; i += BLOCK_SIZE) {
        const auto idx = s_input_idx[r_idx][i];
        const auto offset = 24 - round * 8;
        const auto bin = (convert_to_uint32(input[idx]) >> offset) & 0xFF;
        if (bin > threshold_bin) {
          const auto pos = ::atomicAdd(&s_counter, 1);
          index[pos] = idx;
        }
      }
      __syncthreads();
      break;
    } else {
      __syncthreads();
      if (tx < RADIX + 1) {
        s_histogram[tx] = 0;
      }
      __syncthreads();
      for (int i = tx; i < num_input; i += BLOCK_SIZE) {
        const auto idx = s_input_idx[r_idx][i];
        const auto raw_input = input[idx];
        const auto offset = 24 - round * 8;
        const auto bin = (convert_to_uint32(raw_input) >> offset) & 0xFF;
        if (bin > threshold_bin) {
          const auto pos = ::atomicAdd(&s_counter, 1);
          index[pos] = idx;
        } else if (bin == threshold_bin) {
          if (round == 3) {
            const auto pos = ::atomicAdd(&s_last_remain, -1);
            if (pos > 0) {
              index[TopK - pos] = idx;
            }
          } else {
            const auto pos = ::atomicAdd(&s_num_input[r_idx ^ 1], 1);
            if (C10_LIKELY(pos < SMEM_INPUT_SIZE)) {
              /// NOTE: (dark) fuse the histogram computation here
              s_input_idx[r_idx ^ 1][pos] = idx;
              const auto bin = convert_to_uint32(raw_input);
              const auto sub_bin = (bin >> (offset - 8)) & 0xFF;
              ::atomicAdd(&s_histogram[sub_bin], 1);
            }
          }
        }
      }
      __syncthreads();
    }
  }
}

__global__ __launch_bounds__(kThreadsPerBlock)  // topk
    void topk_kernel(const FastTopKParams params) {
  const auto& [input, indices, lengths, input_stride] = params;
  const auto bid = static_cast<uint64_t>(blockIdx.x);
  const auto length = lengths[bid];
  const auto indice = indices + bid * TopK;
  const auto score = input + bid * input_stride;
  if (length <= TopK) {
    return naive_topk_cuda(score, indice, length);
  } else {
    return fast_topk_cuda_tl(score, indice, length);
  }
}

__global__ __launch_bounds__(kThreadsPerBlock)  // decode
    void topk_transform_decode_kernel(
        const FastTopKParams params,
        int32_t* __restrict__ dst_page_table,
        const int32_t* __restrict__ src_page_table,
        const int64_t src_stride) {
  const auto& [input, _, lengths, input_stride] = params;
  const auto bid = static_cast<uint64_t>(blockIdx.x);
  const auto tid = threadIdx.x;
  const auto length = lengths[bid];
  const auto src_page_entry = src_page_table + bid * src_stride;
  const auto dst_page_entry = dst_page_table + bid * TopK;
  const auto score = input + bid * input_stride;
  if (length <= TopK) {
    return naive_topk_transform(score, length, dst_page_entry, src_page_entry);
  } else {
    __shared__ int s_indices[TopK];
    fast_topk_cuda_tl(score, s_indices, length);
    // copy src[s_indices] to dst, we manually unroll here
    static_assert(TopK % kThreadsPerBlock == 0);
    static_assert(TopK / kThreadsPerBlock == 2);
    const auto idx_0 = tid;
    const auto pos_0 = s_indices[idx_0];
    dst_page_entry[idx_0] = src_page_entry[pos_0];
    const auto idx_1 = tid + kThreadsPerBlock;
    const auto pos_1 = s_indices[idx_1];
    dst_page_entry[idx_1] = src_page_entry[pos_1];
  }
}

__global__ __launch_bounds__(kThreadsPerBlock)  // prefill
    void topk_transform_prefill_kernel(
        const FastTopKParams params,
        int32_t* __restrict__ dst_page_table,
        const int32_t* __restrict__ src_page_table,
        const int64_t src_stride,
        const int32_t* __restrict__ cu_seqlens_q,
        const int64_t prefill_bs) {
  const auto& [input, _, lengths, input_stride] = params;
  const auto bid = static_cast<uint64_t>(blockIdx.x);
  const auto tid = threadIdx.x;
  const auto length = lengths[bid];
  const auto dst_page_entry = dst_page_table + bid * TopK;
  const auto score = input + bid * input_stride;

  /// NOTE: prefill bs is usually small, we can just use a simple loop here
  /// We ensure that last cu_seqlens is equal to number of blocks launched
  __shared__ const int32_t* s_src_page_entry;
  if (C10_LIKELY(prefill_bs <= kThreadsPerBlock)) {
    if (tid < prefill_bs) {
      if (bid >= cu_seqlens_q[tid] && bid < cu_seqlens_q[tid + 1]) {
        s_src_page_entry = src_page_table + tid * src_stride;
      }
    }
  } else {
    for (int64_t i = tid; i < prefill_bs; i += kThreadsPerBlock) {
      if (bid >= cu_seqlens_q[i] && bid < cu_seqlens_q[i + 1]) {
        s_src_page_entry = src_page_table + i * src_stride;
      }
    }
  }
  __syncthreads();
  const auto src_page_entry = s_src_page_entry;

  if (length <= TopK) {
    return naive_topk_transform(score, length, dst_page_entry, src_page_entry);
  } else {
    __shared__ int s_indices[TopK];
    fast_topk_cuda_tl(score, s_indices, length);
    // copy src[s_indices] to dst, we manually unroll here
    static_assert(TopK % kThreadsPerBlock == 0);
    static_assert(TopK / kThreadsPerBlock == 2);
    const auto idx_0 = tid;
    const auto pos_0 = s_indices[idx_0];
    dst_page_entry[idx_0] = src_page_entry[pos_0];
    const auto idx_1 = tid + kThreadsPerBlock;
    const auto pos_1 = s_indices[idx_1];
    dst_page_entry[idx_1] = src_page_entry[pos_1];
  }
}

__global__ __launch_bounds__(kThreadsPerBlock)  // prefill, ragged kv
    void topk_transform_prefill_ragged_kernel(
        const FastTopKParams params,
        int32_t* __restrict__ topk_indices_ragged,
        const int32_t* __restrict__ topk_indices_offset) {
  const auto& [input, _, lengths, input_stride] = params;
  const auto bid = static_cast<uint64_t>(blockIdx.x);
  const auto tid = threadIdx.x;
  const auto length = lengths[bid];
  const auto dst_indices_entry = topk_indices_ragged + bid * TopK;
  const auto score = input + bid * input_stride;
  const auto offset = topk_indices_offset[bid];

  if (length <= TopK) {
    return naive_topk_transform_ragged(score, length, dst_indices_entry, offset);
  } else {
    __shared__ int s_indices[TopK];
    fast_topk_cuda_tl(score, s_indices, length);
    // copy src[s_indices] to dst, we manually unroll here
    static_assert(TopK % kThreadsPerBlock == 0);
    static_assert(TopK / kThreadsPerBlock == 2);
    const auto idx_0 = tid;
    const auto pos_0 = s_indices[idx_0];
    dst_indices_entry[idx_0] = pos_0 + offset;
    const auto idx_1 = tid + kThreadsPerBlock;
    const auto pos_1 = s_indices[idx_1];
    dst_indices_entry[idx_1] = pos_1 + offset;
  }
}

auto get_params(
    const at::Tensor& score,
    const at::Tensor& lengths,
    std::optional<at::Tensor> indices_opt = std::nullopt) -> FastTopKParams {
  const auto B = score.size(0);
  TORCH_CHECK(score.dim() == 2 && score.stride(1) == 1);
  TORCH_CHECK(lengths.dim() == 1 && lengths.is_contiguous());
  TORCH_CHECK(lengths.size(0) == B);
  int32_t* indices_data_ptr = nullptr;
  if (indices_opt.has_value()) {
    const auto& indices = indices_opt.value();
    TORCH_CHECK(indices.dim() == 2 && indices.is_contiguous());
    TORCH_CHECK(indices.size(0) == B);
    TORCH_CHECK(indices.size(1) == TopK);
    indices_data_ptr = indices.data_ptr<int32_t>();
  }

  return FastTopKParams{
      .input = score.data_ptr<float>(),
      .indices = indices_data_ptr,
      .lengths = lengths.data_ptr<int32_t>(),
      .input_stride = score.stride(0),
  };
}

template <auto* f, size_t max_dynamic_smem>
void setup_kernel_smem_once() {
  [[maybe_unused]]
  static const auto result =
      [] { return ::hipFuncSetAttribute(reinterpret_cast<const void*>(f), ::hipFuncAttributeMaxDynamicSharedMemorySize, max_dynamic_smem); }();
  TORCH_CHECK(result == hipSuccess, "set_up_kernel_once failed:", ::hipGetErrorString(result));
}

}  // namespace

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")

void fast_topk_interface(const at::Tensor& score, at::Tensor& indices, const at::Tensor& lengths) {
  CHECK_CUDA(score);
  CHECK_CUDA(indices);
  CHECK_CUDA(lengths);
  const auto params = get_params(score, lengths, indices);
  const auto B = score.size(0);
  const auto stream = at::cuda::getCurrentCUDAStream().stream();
  const auto grid = dim3{static_cast<uint32_t>(B)};
  const auto block = dim3{kThreadsPerBlock};
  setup_kernel_smem_once<topk_kernel, kSmem>();
  topk_kernel<<<grid, block, kSmem, stream>>>(params);
  const auto result = hipGetLastError();
  TORCH_CHECK(result == hipSuccess, "topk kernel failed:", ::hipGetErrorString(result));
}

void fast_topk_transform_interface(
    const at::Tensor& score,
    const at::Tensor& lengths,
    at::Tensor& dst_page_table,
    const at::Tensor& src_page_table,
    const at::Tensor& cu_seqlens_q) {
  CHECK_CUDA(score);
  CHECK_CUDA(lengths);
  CHECK_CUDA(dst_page_table);
  CHECK_CUDA(src_page_table);
  CHECK_CUDA(cu_seqlens_q);
  const auto params = get_params(score, lengths);
  const auto B = score.size(0);
  TORCH_CHECK(dst_page_table.dim() == 2 && dst_page_table.is_contiguous());
  TORCH_CHECK(src_page_table.dim() == 2 && src_page_table.stride(1) == 1);
  TORCH_CHECK(cu_seqlens_q.dim() == 1 && cu_seqlens_q.is_contiguous());
  const auto prefill_bs = cu_seqlens_q.size(0) - 1;
  TORCH_CHECK(dst_page_table.size(0) == B);
  TORCH_CHECK(dst_page_table.size(1) == TopK);
  TORCH_CHECK(src_page_table.size(0) == prefill_bs);
  TORCH_CHECK(prefill_bs <= B);  // prefill_bs should be smaller than expanded bs

  // launch kernel
  const auto stream = at::cuda::getCurrentCUDAStream().stream();
  const auto grid = dim3{static_cast<uint32_t>(B)};
  const auto block = dim3{kThreadsPerBlock};
  const auto src_stride = src_page_table.stride(0);

  // dispatch to decode or prefill
  const auto is_decode = (prefill_bs == B);
  if (is_decode) {
    setup_kernel_smem_once<topk_transform_decode_kernel, kSmem>();
    topk_transform_decode_kernel<<<grid, block, kSmem, stream>>>(
        params, dst_page_table.data_ptr<int32_t>(), src_page_table.data_ptr<int32_t>(), src_stride);
  } else {
    setup_kernel_smem_once<topk_transform_prefill_kernel, kSmem>();
    topk_transform_prefill_kernel<<<grid, block, kSmem, stream>>>(
        params,
        dst_page_table.data_ptr<int32_t>(),
        src_page_table.data_ptr<int32_t>(),
        src_stride,
        cu_seqlens_q.data_ptr<int32_t>(),
        prefill_bs);
  }

  const auto result = hipGetLastError();
  TORCH_CHECK(result == hipSuccess, "topk kernel failed:", ::hipGetErrorString(result));
}

void fast_topk_transform_ragged_interface(
    const at::Tensor& score,
    const at::Tensor& lengths,
    at::Tensor& topk_indices_ragged,
    const at::Tensor& topk_indices_offset) {
  CHECK_CUDA(score);
  CHECK_CUDA(lengths);
  CHECK_CUDA(topk_indices_ragged);
  CHECK_CUDA(topk_indices_offset);

  const auto params = get_params(score, lengths);
  const auto B = score.size(0);
  TORCH_CHECK(topk_indices_ragged.dim() == 2 && topk_indices_ragged.is_contiguous());
  TORCH_CHECK(topk_indices_offset.dim() == 1);

  TORCH_CHECK(topk_indices_ragged.size(0) == B);
  TORCH_CHECK(topk_indices_ragged.size(1) == TopK);
  TORCH_CHECK(topk_indices_offset.size(0) == B);

  // launch kernel
  const auto stream = at::cuda::getCurrentCUDAStream().stream();
  const auto grid = dim3{static_cast<uint32_t>(B)};
  const auto block = dim3{kThreadsPerBlock};

  setup_kernel_smem_once<topk_transform_prefill_ragged_kernel, kSmem>();
  topk_transform_prefill_ragged_kernel<<<grid, block, kSmem, stream>>>(
      params, topk_indices_ragged.data_ptr<int32_t>(), topk_indices_offset.data_ptr<int32_t>());

  const auto result = hipGetLastError();
  TORCH_CHECK(result == hipSuccess, "topk kernel failed:", ::hipGetErrorString(result));
}
