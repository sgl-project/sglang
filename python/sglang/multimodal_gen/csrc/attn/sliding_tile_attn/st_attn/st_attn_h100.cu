#include "hip/hip_runtime.h"
// # Define TORCH_COMPILE macro

#include "kittens.cuh"
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <stdio.h>

// #define CLAMP(value, min, max) ((value) < (min) ? (min) : ((value) > (max) ?
// (max) : (value)))
__device__ __forceinline__ int clamp_int(int value, int min, int max) {
  return (value < min) ? min : ((value > max) ? max : value);
}
// #define ABS(x) ((x) < 0 ? -(x) : (x))
__device__ __forceinline__ int abs_int(int value) {
  return (value < 0) ? -value : value;
}

constexpr int CONSUMER_WARPGROUPS = (3);
constexpr int PRODUCER_WARPGROUPS = (1);
constexpr int NUM_WARPGROUPS = (CONSUMER_WARPGROUPS + PRODUCER_WARPGROUPS);
constexpr int NUM_WORKERS = (NUM_WARPGROUPS * kittens::WARPGROUP_WARPS);

using namespace kittens;
namespace cg = cooperative_groups;

template <int D> struct fwd_attend_ker_tile_dims {};
template <> struct fwd_attend_ker_tile_dims<64> {
  constexpr static int tile_width = (64);
  constexpr static int qo_height = (4 * 16);
  constexpr static int kv_height = (8 * 16);
  constexpr static int stages = (4);
};
template <> struct fwd_attend_ker_tile_dims<128> {
  constexpr static int tile_width = (128);
  constexpr static int qo_height = (4 * 16);
  constexpr static int kv_height = (8 * 16);
  constexpr static int stages = (2);
};

template <int D> struct fwd_globals {
  using q_tile = st_bf<fwd_attend_ker_tile_dims<D>::qo_height,
                       fwd_attend_ker_tile_dims<D>::tile_width>;
  using k_tile = st_bf<fwd_attend_ker_tile_dims<D>::kv_height,
                       fwd_attend_ker_tile_dims<D>::tile_width>;
  using v_tile = st_bf<fwd_attend_ker_tile_dims<D>::kv_height,
                       fwd_attend_ker_tile_dims<D>::tile_width>;
  using l_col_vec = col_vec<st_fl<fwd_attend_ker_tile_dims<D>::qo_height,
                                  fwd_attend_ker_tile_dims<D>::tile_width>>;
  using o_tile = st_bf<fwd_attend_ker_tile_dims<D>::qo_height,
                       fwd_attend_ker_tile_dims<D>::tile_width>;

  using q_gl = gl<bf16, -1, -1, -1, -1, q_tile>;
  using k_gl = gl<bf16, -1, -1, -1, -1, k_tile>;
  using v_gl = gl<bf16, -1, -1, -1, -1, v_tile>;
  using l_gl = gl<float, -1, -1, -1, -1, l_col_vec>;
  using o_gl = gl<bf16, -1, -1, -1, -1, o_tile>;

  q_gl q;
  k_gl k;
  v_gl v;
  l_gl l;
  o_gl o;

  const int N;
  const int text_L;
  const int hr;
};

template <int D, bool is_causal, bool text_q, bool text_kv, int DT, int DH,
          int DW, int CT, int CH, int CW>
__global__ __launch_bounds__(
    (NUM_WORKERS)*kittens::WARP_THREADS,
    1) void fwd_attend_ker(const __grid_constant__ fwd_globals<D> g) {
  extern __shared__ int __shm[];
  tma_swizzle_allocator al((int *)&__shm[0]);
  int warpid = kittens::warpid(),
      warpgroupid = warpid / kittens::WARPGROUP_WARPS;

  using K = fwd_attend_ker_tile_dims<D>;

  using q_tile = st_bf<K::qo_height, K::tile_width>;
  using k_tile = st_bf<K::kv_height, K::tile_width>;
  using v_tile = st_bf<K::kv_height, K::tile_width>;
  using l_col_vec = col_vec<st_fl<K::qo_height, K::tile_width>>;
  using o_tile = st_bf<K::qo_height, K::tile_width>;

  q_tile(&q_smem)[CONSUMER_WARPGROUPS] =
      al.allocate<q_tile, CONSUMER_WARPGROUPS>();
  k_tile(&k_smem)[K::stages] = al.allocate<k_tile, K::stages>();
  v_tile(&v_smem)[K::stages] = al.allocate<v_tile, K::stages>();
  l_col_vec(&l_smem)[CONSUMER_WARPGROUPS] =
      al.allocate<l_col_vec, CONSUMER_WARPGROUPS>();
  auto(*o_smem) = reinterpret_cast<o_tile(*)>(q_smem);
  int img_kv_blocks;
  int kv_blocks = g.N / (K::kv_height);
  if constexpr (text_kv) {
    img_kv_blocks = kv_blocks - 3;
  } else {
    img_kv_blocks = kv_blocks;
  }
  int kv_head_idx = blockIdx.y / g.hr;
  int seq_idx;
  if constexpr (text_q) {
    seq_idx = CT * CH * CW * 6.0 + blockIdx.x * CONSUMER_WARPGROUPS;
  } else {
    seq_idx = blockIdx.x * CONSUMER_WARPGROUPS;
  }
  __shared__ kittens::semaphore qsmem_semaphore, k_smem_arrived[K::stages],
      v_smem_arrived[K::stages], compute_done[K::stages];
  if (threadIdx.x == 0) {
    init_semaphore(qsmem_semaphore, 0, 1);
    for (int j = 0; j < K::stages; j++) {
      init_semaphore(k_smem_arrived[j], 0, 1);
      init_semaphore(v_smem_arrived[j], 0, 1);
      init_semaphore(compute_done[j], CONSUMER_WARPGROUPS, 0);
    }

    tma::expect_bytes(qsmem_semaphore, sizeof(q_smem));

    for (int wg = 0; wg < CONSUMER_WARPGROUPS; wg++) {
      coord<q_tile> q_tile_idx = {blockIdx.z, blockIdx.y, (seq_idx) + wg, 0};
      tma::load_async(q_smem[wg], g.q, q_tile_idx, qsmem_semaphore);
    }

    if constexpr (text_q) {
      for (int j = 0; j < K::stages - 1; j++) {
        coord<k_tile> kv_tile_idx = {blockIdx.z, kv_head_idx, j, 0};
        tma::expect_bytes(k_smem_arrived[j], sizeof(k_tile));
        tma::load_async(k_smem[j], g.k, kv_tile_idx, k_smem_arrived[j]);
        tma::expect_bytes(v_smem_arrived[j], sizeof(v_tile));
        tma::load_async(v_smem[j], g.v, kv_tile_idx, v_smem_arrived[j]);
      }
    } else {
      int qt = seq_idx / 6 / (CH * CW);
      int qh = (seq_idx / 6) % (CH * CW) / CW;
      int qw = (seq_idx / 6) % CW;
      qt = clamp_int(qt, DT, CT - DT - 1);
      qh = clamp_int(qh, DH, CH - DH - 1);
      qw = clamp_int(qw, DW, CW - DW - 1);
      int count = 0;
      int j = 0;
      while (count < K::stages - 1) {
        int kt = j / 3 / (CH * CW);
        int kh = (j / 3) % (CH * CW) / CW;
        int kw = (j / 3) % CW;
        bool mask = (abs_int(qt - kt) <= DT) && (abs_int(qh - kh) <= DH) &&
                    (abs_int(qw - kw) <= DW);
        if (mask) {
          coord<k_tile> kv_tile_idx = {blockIdx.z, kv_head_idx, j, 0};
          tma::expect_bytes(k_smem_arrived[count], sizeof(k_tile));
          tma::load_async(k_smem[count], g.k, kv_tile_idx,
                          k_smem_arrived[count]);
          tma::expect_bytes(v_smem_arrived[count], sizeof(v_tile));
          tma::load_async(v_smem[count], g.v, kv_tile_idx,
                          v_smem_arrived[count]);
          count += 1;
        }
        j += 1;
      }
    }
  }
  __syncthreads();

  int pipe_idx = K::stages - 1;

  if (warpgroupid == NUM_WARPGROUPS - 1) {
    warpgroup::decrease_registers<32>();

    int kv_iters;
    if constexpr (is_causal) {
      kv_iters =
          (seq_idx * (K::qo_height / kittens::TILE_ROW_DIM<bf16>)) - 1 +
          (CONSUMER_WARPGROUPS * (K::qo_height / kittens::TILE_ROW_DIM<bf16>));
      kv_iters =
          ((kv_iters / (K::kv_height / kittens::TILE_ROW_DIM<bf16>)) == 0)
              ? (0)
              : ((kv_iters / (K::kv_height / kittens::TILE_ROW_DIM<bf16>)) - 1);
    } else {
      kv_iters = kv_blocks - 2;
    }

    if (warpid == NUM_WORKERS - 4) {
      if constexpr (text_q) {
        for (auto kv_idx = pipe_idx - 1; kv_idx <= kv_iters; kv_idx++) {
          coord<k_tile> kv_tile_idx = {blockIdx.z, kv_head_idx, kv_idx + 1, 0};
          tma::expect_bytes(k_smem_arrived[(kv_idx + 1) % K::stages],
                            sizeof(k_tile));
          tma::load_async(k_smem[(kv_idx + 1) % K::stages], g.k, kv_tile_idx,
                          k_smem_arrived[(kv_idx + 1) % K::stages]);
          tma::expect_bytes(v_smem_arrived[(kv_idx + 1) % K::stages],
                            sizeof(v_tile));
          tma::load_async(v_smem[(kv_idx + 1) % K::stages], g.v, kv_tile_idx,
                          v_smem_arrived[(kv_idx + 1) % K::stages]);
          kittens::wait(compute_done[(kv_idx) % K::stages],
                        (kv_idx / K::stages) % 2);
        }
      } else {
        int qt = seq_idx / 6 / (CH * CW);
        int qh = (seq_idx / 6) % (CH * CW) / CW;
        int qw = (seq_idx / 6) % CW;
        qt = clamp_int(qt, DT, CT - DT - 1);
        qh = clamp_int(qh, DH, CH - DH - 1);
        qw = clamp_int(qw, DW, CW - DW - 1);
        int k_t_min = clamp_int(qt - DT, 0, CT - 1);
        int k_t_max = clamp_int(qt + DT, 0, CT - 1);
        int k_h_min = clamp_int(qh - DH, 0, CH - 1);
        int k_h_max = clamp_int(qh + DH, 0, CH - 1);
        int k_w_min = clamp_int(qw - DW, 0, CW - 1);
        int k_w_max = clamp_int(qw + DW, 0, CW - 1);
        int count = 0;
        for (int kt = k_t_min; kt <= k_t_max; kt++) {
          for (int kh = k_h_min; kh <= k_h_max; kh++) {
            for (int kw = k_w_min; kw <= k_w_max; kw++) {
              for (int j = 0; j <= 2; j++) {
                if (count >= K::stages - 1) {
                  int index = ((kt * (CH * CW)) + (kh * CW) + kw) * 3 + j;
                  coord<k_tile> kv_tile_idx = {blockIdx.z, kv_head_idx, index,
                                               0};
                  tma::expect_bytes(k_smem_arrived[count % K::stages],
                                    sizeof(k_tile));
                  tma::load_async(k_smem[count % K::stages], g.k, kv_tile_idx,
                                  k_smem_arrived[count % K::stages]);
                  tma::expect_bytes(v_smem_arrived[count % K::stages],
                                    sizeof(v_tile));
                  tma::load_async(v_smem[count % K::stages], g.v, kv_tile_idx,
                                  v_smem_arrived[count % K::stages]);
                  kittens::wait(compute_done[(count - 1) % K::stages],
                                ((count - 1) / K::stages) % 2);
                  count += 1;
                } else {
                  count += 1;
                }
              }
            }
          }
        }
        // for text
        for (int index = img_kv_blocks; index < kv_blocks; index++) {
          coord<k_tile> kv_tile_idx = {blockIdx.z, kv_head_idx, index, 0};
          tma::expect_bytes(k_smem_arrived[count % K::stages], sizeof(k_tile));
          tma::load_async(k_smem[count % K::stages], g.k, kv_tile_idx,
                          k_smem_arrived[count % K::stages]);
          tma::expect_bytes(v_smem_arrived[count % K::stages], sizeof(v_tile));
          tma::load_async(v_smem[count % K::stages], g.v, kv_tile_idx,
                          v_smem_arrived[count % K::stages]);
          kittens::wait(compute_done[(count - 1) % K::stages],
                        ((count - 1) / K::stages) % 2);
          count += 1;
        }
      }
    }
  } else {
    warpgroup::increase_registers<160>();

    rt_fl<16, K::kv_height> att_block;
    rt_bf<16, K::kv_height> att_block_mma;
    rt_fl<16, K::tile_width> o_reg;

    col_vec<rt_fl<16, K::kv_height>> max_vec, norm_vec, max_vec_last_scaled,
        max_vec_scaled;

    neg_infty(max_vec);
    zero(norm_vec);
    zero(o_reg);

    int kv_iters;
    if constexpr (is_causal) {
      kv_iters = (seq_idx * 4) - 1 + (CONSUMER_WARPGROUPS * 4);
      kv_iters = (kv_iters / 8);
    } else if constexpr (text_q) {
      // the last three kv blocks are for text, we process them separately
      kv_iters = img_kv_blocks - 1;
    } else {
      kv_iters = clamp_int(DT * 2 + 1, 1, CT) * clamp_int(DH * 2 + 1, 1, CH) *
                     clamp_int(DW * 2 + 1, 1, CW) * 3 -
                 1;
    }

    kittens::wait(qsmem_semaphore, 0);
    for (auto kv_idx = 0; kv_idx <= kv_iters; kv_idx++) {

      kittens::wait(k_smem_arrived[(kv_idx) % K::stages],
                    (kv_idx / K::stages) % 2);
      warpgroup::mm_ABt(att_block, q_smem[warpgroupid],
                        k_smem[(kv_idx) % K::stages]);

      copy(max_vec_last_scaled, max_vec);
      if constexpr (D == 64) {
        mul(max_vec_last_scaled, max_vec_last_scaled, 1.44269504089f * 0.125f);
      } else {
        mul(max_vec_last_scaled, max_vec_last_scaled,
            1.44269504089f * 0.08838834764f);
      }

      warpgroup::mma_async_wait();

      row_max(max_vec, att_block, max_vec);

      if constexpr (D == 64) {
        mul(att_block, att_block, 1.44269504089f * 0.125f);
        mul(max_vec_scaled, max_vec, 1.44269504089f * 0.125f);
      } else {
        mul(att_block, att_block, 1.44269504089f * 0.08838834764f);
        mul(max_vec_scaled, max_vec, 1.44269504089f * 0.08838834764f);
      }

      sub_row(att_block, att_block, max_vec_scaled);
      exp2(att_block, att_block);
      sub(max_vec_last_scaled, max_vec_last_scaled, max_vec_scaled);
      exp2(max_vec_last_scaled, max_vec_last_scaled);
      mul(norm_vec, norm_vec, max_vec_last_scaled);
      row_sum(norm_vec, att_block, norm_vec);
      add(att_block, att_block, 0.f);
      copy(att_block_mma, att_block);
      mul_row(o_reg, o_reg, max_vec_last_scaled);

      kittens::wait(v_smem_arrived[(kv_idx) % K::stages],
                    (kv_idx / K::stages) % 2);

      warpgroup::mma_AB(o_reg, att_block_mma, v_smem[(kv_idx) % K::stages]);
      warpgroup::mma_async_wait();

      if (warpgroup::laneid() == 0)
        arrive(compute_done[(kv_idx) % K::stages], 1);
    }
    // the last three kv blocks are for text, we process them separately
    if constexpr (text_kv) {
      for (auto kv_idx = kv_iters + 1; kv_idx <= kv_iters + 3; kv_idx++) {

        kittens::wait(k_smem_arrived[(kv_idx) % K::stages],
                      (kv_idx / K::stages) % 2);
        warpgroup::mm_ABt(att_block, q_smem[warpgroupid],
                          k_smem[(kv_idx) % K::stages]);

        copy(max_vec_last_scaled, max_vec);
        if constexpr (D == 64) {
          mul(max_vec_last_scaled, max_vec_last_scaled,
              1.44269504089f * 0.125f);
        } else {
          mul(max_vec_last_scaled, max_vec_last_scaled,
              1.44269504089f * 0.08838834764f);
        }

        warpgroup::mma_async_wait();
        // apply non-pad mask
        int offset = g.text_L - (kv_idx - (kv_iters + 1)) * K::kv_height;
        // printf("k_idx_start: %d, k_idx_end: %d, text_end: %d, offset: %d\n",
        // k_idx_start, k_idx_end, text_end, offset);
        right_fill(att_block, att_block, offset,
                   base_types::constants<float>::neg_infty());

        row_max(max_vec, att_block, max_vec);

        if constexpr (D == 64) {
          mul(att_block, att_block, 1.44269504089f * 0.125f);
          mul(max_vec_scaled, max_vec, 1.44269504089f * 0.125f);
        } else {
          mul(att_block, att_block, 1.44269504089f * 0.08838834764f);
          mul(max_vec_scaled, max_vec, 1.44269504089f * 0.08838834764f);
        }

        sub_row(att_block, att_block, max_vec_scaled);
        exp2(att_block, att_block);
        sub(max_vec_last_scaled, max_vec_last_scaled, max_vec_scaled);
        exp2(max_vec_last_scaled, max_vec_last_scaled);
        mul(norm_vec, norm_vec, max_vec_last_scaled);
        row_sum(norm_vec, att_block, norm_vec);
        add(att_block, att_block, 0.f);
        copy(att_block_mma, att_block);
        mul_row(o_reg, o_reg, max_vec_last_scaled);

        kittens::wait(v_smem_arrived[(kv_idx) % K::stages],
                      (kv_idx / K::stages) % 2);

        warpgroup::mma_AB(o_reg, att_block_mma, v_smem[(kv_idx) % K::stages]);
        warpgroup::mma_async_wait();

        if (warpgroup::laneid() == 0)
          arrive(compute_done[(kv_idx) % K::stages], 1);
      }
    }

    div_row(o_reg, o_reg, norm_vec);
    warpgroup::store(o_smem[warpgroupid], o_reg);
    warpgroup::sync(warpgroupid + 4);

    if (warpid % 4 == 0) {
      coord<o_tile> o_tile_idx = {blockIdx.z, blockIdx.y,
                                  (seq_idx) + warpgroupid, 0};
      tma::store_async(g.o, o_smem[warpgroupid], o_tile_idx);
    }

    mul(max_vec_scaled, max_vec_scaled, 0.69314718056f);
    log(norm_vec, norm_vec);
    add(norm_vec, norm_vec, max_vec_scaled);

    if constexpr (D == 64) {
      mul(norm_vec, norm_vec, -8.0f);
    } else {
      mul(norm_vec, norm_vec, -11.313708499f);
    }

    warpgroup::store(l_smem[warpgroupid], norm_vec);
    warpgroup::sync(warpgroupid + 4);

    if (warpid % 4 == 0) {
      coord<l_col_vec> tile_idx = {blockIdx.z, blockIdx.y, 0,
                                   (seq_idx) + warpgroupid};
      tma::store_async(g.l, l_smem[warpgroupid], tile_idx);
    }
    tma::store_async_wait();
  }
}

#include "pyutils/torch_helpers.cuh"
#include <ATen/cuda/HIPContext.h>
#include <iostream>

torch::Tensor sta_forward(torch::Tensor q, torch::Tensor k, torch::Tensor v,
                          torch::Tensor o, int kernel_t_size, int kernel_h_size,
                          int kernel_w_size, int text_length, bool process_text,
                          bool has_text, int kernel_aspect_ratio_flag) {
  CHECK_INPUT(q);
  CHECK_INPUT(k);
  CHECK_INPUT(v);

  auto batch = q.size(0);
  auto seq_len = q.size(2);
  auto head_dim = q.size(3);
  auto qo_heads = q.size(1);
  auto kv_heads = k.size(1);

  // check to see that these dimensions match for all inputs
  TORCH_CHECK(q.size(0) == batch,
              "Q batch dimension - idx 0 - must match for all inputs");
  TORCH_CHECK(k.size(0) == batch,
              "K batch dimension - idx 0 - must match for all inputs");
  TORCH_CHECK(v.size(0) == batch,
              "V batch dimension - idx 0 - must match for all inputs");

  TORCH_CHECK(
      q.size(2) == seq_len,
      "Q sequence length dimension - idx 2 - must match for all inputs");
  TORCH_CHECK(
      k.size(2) == seq_len,
      "K sequence length dimension - idx 2 - must match for all inputs");
  TORCH_CHECK(
      v.size(2) == seq_len,
      "V sequence length dimension - idx 2 - must match for all inputs");

  TORCH_CHECK(
      q.size(3) == head_dim,
      "Q head dimension - idx 3 - must match for all non-vector inputs");
  TORCH_CHECK(
      k.size(3) == head_dim,
      "K head dimension - idx 3 - must match for all non-vector inputs");
  TORCH_CHECK(
      v.size(3) == head_dim,
      "V head dimension - idx 3 - must match for all non-vector inputs");

  TORCH_CHECK(qo_heads >= kv_heads,
              "QO heads must be greater than or equal to KV heads");
  TORCH_CHECK(qo_heads % kv_heads == 0,
              "QO heads must be divisible by KV heads");
  TORCH_CHECK(q.size(1) == qo_heads,
              "QO head dimension - idx 1 - must match for all inputs");
  TORCH_CHECK(k.size(1) == kv_heads,
              "KV head dimension - idx 1 - must match for all inputs");
  TORCH_CHECK(v.size(1) == kv_heads,
              "KV head dimension - idx 1 - must match for all inputs");

  auto hr = qo_heads / kv_heads;

  c10::BFloat16 *q_ptr = q.data_ptr<c10::BFloat16>();
  c10::BFloat16 *k_ptr = k.data_ptr<c10::BFloat16>();
  c10::BFloat16 *v_ptr = v.data_ptr<c10::BFloat16>();

  bf16 *d_q = reinterpret_cast<bf16 *>(q_ptr);
  bf16 *d_k = reinterpret_cast<bf16 *>(k_ptr);
  bf16 *d_v = reinterpret_cast<bf16 *>(v_ptr);

  torch::Tensor l_vec = torch::empty(
      {static_cast<const uint>(batch), static_cast<const uint>(qo_heads),
       static_cast<const uint>(seq_len), static_cast<const uint>(1)},
      torch::TensorOptions()
          .dtype(torch::kFloat)
          .device(q.device())
          .memory_format(at::MemoryFormat::Contiguous));

  bf16 *o_ptr = reinterpret_cast<bf16 *>(o.data_ptr<c10::BFloat16>());
  bf16 *d_o = reinterpret_cast<bf16 *>(o_ptr);

  float *l_ptr = reinterpret_cast<float *>(l_vec.data_ptr<float>());
  float *d_l = reinterpret_cast<float *>(l_ptr);

  // cudadevicesynchronize();
  const c10::cuda::OptionalCUDAGuard device_guard(q.device());
  const hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

  if (head_dim == 128) {
    using q_tile = st_bf<fwd_attend_ker_tile_dims<128>::qo_height,
                         fwd_attend_ker_tile_dims<128>::tile_width>;
    using k_tile = st_bf<fwd_attend_ker_tile_dims<128>::kv_height,
                         fwd_attend_ker_tile_dims<128>::tile_width>;
    using v_tile = st_bf<fwd_attend_ker_tile_dims<128>::kv_height,
                         fwd_attend_ker_tile_dims<128>::tile_width>;
    using l_col_vec = col_vec<st_fl<fwd_attend_ker_tile_dims<128>::qo_height,
                                    fwd_attend_ker_tile_dims<128>::tile_width>>;
    using o_tile = st_bf<fwd_attend_ker_tile_dims<128>::qo_height,
                         fwd_attend_ker_tile_dims<128>::tile_width>;

    using q_global = gl<bf16, -1, -1, -1, -1, q_tile>;
    using k_global = gl<bf16, -1, -1, -1, -1, k_tile>;
    using v_global = gl<bf16, -1, -1, -1, -1, v_tile>;
    using l_global = gl<float, -1, -1, -1, -1, l_col_vec>;
    using o_global = gl<bf16, -1, -1, -1, -1, o_tile>;

    using globals = fwd_globals<128>;

    q_global qg_arg{d_q, static_cast<unsigned int>(batch),
                    static_cast<unsigned int>(qo_heads),
                    static_cast<unsigned int>(seq_len), 128U};
    k_global kg_arg{d_k, static_cast<unsigned int>(batch),
                    static_cast<unsigned int>(kv_heads),
                    static_cast<unsigned int>(seq_len), 128U};
    v_global vg_arg{d_v, static_cast<unsigned int>(batch),
                    static_cast<unsigned int>(kv_heads),
                    static_cast<unsigned int>(seq_len), 128U};
    l_global lg_arg{d_l, static_cast<unsigned int>(batch),
                    static_cast<unsigned int>(qo_heads), 1U,
                    static_cast<unsigned int>(seq_len)};
    o_global og_arg{d_o, static_cast<unsigned int>(batch),
                    static_cast<unsigned int>(qo_heads),
                    static_cast<unsigned int>(seq_len), 128U};

    globals g{qg_arg,
              kg_arg,
              vg_arg,
              lg_arg,
              og_arg,
              static_cast<int>(seq_len),
              static_cast<int>(text_length),
              static_cast<int>(hr)};

    constexpr int mem_size = kittens::MAX_SHARED_MEMORY;
    int threads = NUM_WORKERS * kittens::WARP_THREADS;
    if (has_text) {
      // TORCH_CHECK(seq_len % (CONSUMER_WARPGROUPS*kittens::TILE_DIM*4) == 0,
      // "sequence length must be divisible by 192");
      dim3 grid_image(
          seq_len / (CONSUMER_WARPGROUPS * kittens::TILE_ROW_DIM<bf16> * 4) - 2,
          qo_heads, batch);
      dim3 grid_text(2, qo_heads, batch);
      if (!process_text) {
        if (kernel_t_size == 3 && kernel_h_size == 3 && kernel_w_size == 3) {

          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, true, 1, 1, 1, 5, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, true, 1, 1, 1, 5, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 3 && kernel_h_size == 3 &&
                   kernel_w_size == 5) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, true, 1, 1, 2, 5, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, true, 1, 1, 2, 5, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 5 && kernel_h_size == 3 &&
                   kernel_w_size == 3) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, true, 2, 1, 1, 5, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, true, 2, 1, 1, 5, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 3 && kernel_h_size == 5 &&
                   kernel_w_size == 5) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, true, 1, 2, 2, 5, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, true, 1, 2, 2, 5, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 5 && kernel_h_size == 6 &&
                   kernel_w_size == 1) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, true, 2, 3, 0, 5, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, true, 2, 3, 0, 5, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 5 && kernel_h_size == 3 &&
                   kernel_w_size == 5) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, true, 2, 1, 2, 5, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, true, 2, 1, 2, 5, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 5 && kernel_h_size == 5 &&
                   kernel_w_size == 5) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, true, 2, 2, 2, 5, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, true, 2, 2, 2, 5, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 5 && kernel_h_size == 5 &&
                   kernel_w_size == 7) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, true, 2, 2, 3, 5, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, true, 2, 2, 3, 5, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 5 && kernel_h_size == 6 &&
                   kernel_w_size == 10) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, true, 2, 3, 5, 5, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, true, 2, 3, 5, 5, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 5 && kernel_h_size == 1 &&
                   kernel_w_size == 1) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, true, 2, 0, 0, 5, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, true, 2, 0, 0, 5, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 1 && kernel_h_size == 6 &&
                   kernel_w_size == 10) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, true, 0, 3, 5, 5, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, true, 0, 3, 5, 5, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 5 && kernel_h_size == 1 &&
                   kernel_w_size == 10) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, true, 2, 0, 5, 5, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, true, 2, 0, 5, 5, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else {
          // print error
          std::cout << "Invalid kernel size" << std::endl;
          // print kernel size
          std::cout << "Kernel size: " << kernel_t_size << " " << kernel_h_size
                    << " " << kernel_w_size << std::endl;
        }
      } else {
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            fwd_attend_ker<128), false, true, true, 1, 1, 1, 5, 6, 10>,
            hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
        fwd_attend_ker<128, false, true, true, 1, 1, 1, 5, 6, 10>
            <<<grid_text, (32 * NUM_WORKERS), mem_size, stream>>>(g);
      }

    } else {
      dim3 grid_image(
          seq_len / (CONSUMER_WARPGROUPS * kittens::TILE_ROW_DIM<bf16> * 4),
          qo_heads, batch);
      if (kernel_aspect_ratio_flag == 2) {
        if (kernel_t_size == 3 && kernel_h_size == 3 && kernel_w_size == 3) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 1, 1, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 1, 1, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 3 && kernel_h_size == 3 &&
                   kernel_w_size == 6) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 1, 3, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 1, 3, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 6 && kernel_h_size == 3 &&
                   kernel_w_size == 3) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 3, 1, 1, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 3, 1, 1, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 3 && kernel_h_size == 6 &&
                   kernel_w_size == 6) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 3, 3, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 3, 3, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 3 && kernel_h_size == 6 &&
                   kernel_w_size == 3) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 3, 1, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 3, 1, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 6 && kernel_h_size == 3 &&
                   kernel_w_size == 6) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 3, 1, 3, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 3, 1, 3, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 6 && kernel_h_size == 6 &&
                   kernel_w_size == 6) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 3, 3, 3, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 3, 3, 3, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 6 && kernel_h_size == 1 &&
                   kernel_w_size == 1) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 3, 0, 0, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 3, 0, 0, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 6 && kernel_h_size == 1 &&
                   kernel_w_size == 6) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 3, 0, 3, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 3, 0, 3, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 6 && kernel_h_size == 6 &&
                   kernel_w_size == 1) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 3, 3, 0, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 3, 3, 0, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 1 && kernel_h_size == 6 &&
                   kernel_w_size == 6) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 0, 3, 3, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 0, 3, 3, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 1 && kernel_h_size == 1 &&
                   kernel_w_size == 6) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 0, 0, 3, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 0, 0, 3, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 1 && kernel_h_size == 6 &&
                   kernel_w_size == 1) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 0, 3, 0, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 0, 3, 0, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 6 && kernel_h_size == 6 &&
                   kernel_w_size == 1) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 3, 3, 0, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 3, 3, 0, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 6 && kernel_h_size == 1 &&
                   kernel_w_size == 6) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 3, 0, 3, 6, 6, 6>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 3, 0, 3, 6, 6, 6>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else {
          // print error
          std::cout << "Invalid kernel size" << std::endl;
          // print kernel size
          std::cout << "Kernel size: " << kernel_t_size << " " << kernel_h_size
                    << " " << kernel_w_size << std::endl;
        }
      } else if (kernel_aspect_ratio_flag == 3) {
        if (kernel_t_size == 3 && kernel_h_size == 3 && kernel_w_size == 3) {

          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 1, 1, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 1, 1, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 3 && kernel_h_size == 3 &&
                   kernel_w_size == 5) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 1, 2, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 1, 2, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 3 && kernel_h_size == 5 &&
                   kernel_w_size == 5) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 2, 2, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 2, 2, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 3 && kernel_h_size == 6 &&
                   kernel_w_size == 1) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 3, 0, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 3, 0, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);

        } else if (kernel_t_size == 3 && kernel_h_size == 5 &&
                   kernel_w_size == 7) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 2, 3, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 2, 3, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 3 && kernel_h_size == 5 &&
                   kernel_w_size == 9) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 2, 4, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 2, 4, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 3 && kernel_h_size == 6 &&
                   kernel_w_size == 10) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 3, 5, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 3, 5, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 3 && kernel_h_size == 6 &&
                   kernel_w_size == 3) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 3, 1, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 3, 1, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 3 && kernel_h_size == 1 &&
                   kernel_w_size == 1) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 0, 0, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 0, 0, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 1 && kernel_h_size == 6 &&
                   kernel_w_size == 10) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 0, 3, 5, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 0, 3, 5, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 1 && kernel_h_size == 5 &&
                   kernel_w_size == 10) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 0, 2, 5, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 0, 2, 5, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 1 && kernel_h_size == 6 &&
                   kernel_w_size == 7) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 0, 3, 3, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 0, 3, 3, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 1 && kernel_h_size == 5 &&
                   kernel_w_size == 7) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 0, 2, 3, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 0, 2, 3, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 1 && kernel_h_size == 5 &&
                   kernel_w_size == 9) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 0, 2, 4, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 0, 2, 4, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 3 && kernel_h_size == 1 &&
                   kernel_w_size == 10) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 0, 5, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 0, 5, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 3 && kernel_h_size == 3 &&
                   kernel_w_size == 10) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 1, 1, 5, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 1, 1, 5, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 1 && kernel_h_size == 3 &&
                   kernel_w_size == 10) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 0, 1, 5, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 0, 1, 5, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else if (kernel_t_size == 1 && kernel_h_size == 6 &&
                   kernel_w_size == 5) {
          hipFuncSetAttribute(reinterpret_cast<const void*>(
              fwd_attend_ker<128), false, false, false, 0, 3, 2, 3, 6, 10>,
              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
          fwd_attend_ker<128, false, false, false, 0, 3, 2, 3, 6, 10>
              <<<grid_image, (32 * NUM_WORKERS), mem_size, stream>>>(g);
        } else {
          // print error
          std::cout << "Invalid kernel size" << std::endl;
          // print kernel size
          std::cout << "Kernel size: " << kernel_t_size << " " << kernel_h_size
                    << " " << kernel_w_size << std::endl;
        }
      }

      else {
        std::cout << "Unsupported kernel_aspect_ratio_flag: "
                  << kernel_aspect_ratio_flag << std::endl;
      }
    }
    CHECK_CUDA_ERROR(hipGetLastError());
    // hipStreamSynchronize(stream);
  }

  return o;
  // cudadevicesynchronize();
}
